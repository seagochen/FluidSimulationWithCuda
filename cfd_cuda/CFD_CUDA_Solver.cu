#include "hip/hip_runtime.h"
#include "CFD_FuncPrototypes.h"
#include <hip/hip_runtime.h>
#include <>
#include <stdio.h>
#include <stdlib.h>
#include <SGE\SGUtils.h>

#define CUDA_Release(ptr) {if (ptr != 0) hipFree(ptr);}
#define Error_Free(ptr0, ptr1, ptr2, ptr3) {CUDA_Release(ptr0); CUDA_Release(ptr1); CUDA_Release(ptr2); CUDA_Release(ptr3);}

#define IX(i,j) ((i)+(GridSize+2)*(j))
#define SWAP(grid0,grid) {float * tmp=grid0;grid0=grid;grid=tmp;}

__global__ void add_source_kernel ( float * grid_out, float * src_in, float dt_in )
{
	int i = threadIdx.x;
	grid_out[i] += dt_in*src_in[i];
}

void set_bnd ( int GridSize, int boundary, float * grid )
{
	int i;

	for ( i=1 ; i<=GridSize ; i++ ) {
		grid[IX(0  ,i)] = boundary==1 ? -grid[IX(1,i)] : grid[IX(1,i)];
		grid[IX(GridSize+1,i)] = boundary==1 ? -grid[IX(GridSize,i)] : grid[IX(GridSize,i)];
		grid[IX(i,0  )] = boundary==2 ? -grid[IX(i,1)] : grid[IX(i,1)];
		grid[IX(i,GridSize+1)] = boundary==2 ? -grid[IX(i,GridSize)] : grid[IX(i,GridSize)];
	}
	grid[IX(0  ,0  )] = 0.5f*(grid[IX(1,0  )]+grid[IX(0  ,1)]);
	grid[IX(0  ,GridSize+1)] = 0.5f*(grid[IX(1,GridSize+1)]+grid[IX(0  ,GridSize)]);
	grid[IX(GridSize+1,0  )] = 0.5f*(grid[IX(GridSize,0  )]+grid[IX(GridSize+1,1)]);
	grid[IX(GridSize+1,GridSize+1)] = 0.5f*(grid[IX(GridSize,GridSize+1)]+grid[IX(GridSize+1,GridSize)]);
}


void lin_solve ( int GridSize, int boundary, float * grid, float * grid0, float a, float c )
{
	int i, j, k;

	for ( k=0 ; k<20 ; k++ ) 
	{
		for ( i=1 ; i<=GridSize ; i++ )
		{
			for ( j=1 ; j<=GridSize ; j++ ) 
			{
				grid[IX(i,j)] = (grid0[IX(i,j)] + a*(grid[IX(i-1,j)]+grid[IX(i+1,j)]+grid[IX(i,j-1)]+grid[IX(i,j+1)]))/c;
			}
		}
		set_bnd ( GridSize, boundary, grid );
	}
}


void diffuse ( int GridSize, int boundary, float * grid, float * grid0, float diff, float dt )
{
	float a=dt*diff*GridSize*GridSize;
	lin_solve ( GridSize, boundary, grid, grid0, a, 1+4*a );
}


void advect ( int GridSize, int boundary, float * density, float * density0, float * u, float * v, float dt )
{
	int i, j, i0, j0, i1, j1;
	float grid, y, s0, t0, s1, t1, dt0;

	dt0 = dt*GridSize;
	for ( i=1 ; i<=GridSize ; i++ ) 
	{
		for ( j=1 ; j<=GridSize ; j++ ) 
		{
			grid = i-dt0*u[IX(i,j)]; y = j-dt0*v[IX(i,j)];
			if (grid<0.5f) grid=0.5f; if (grid>GridSize+0.5f) grid=GridSize+0.5f; i0=(int)grid; i1=i0+1;
			if (y<0.5f) y=0.5f; if (y>GridSize+0.5f) y=GridSize+0.5f; j0=(int)y; j1=j0+1;
			s1 = grid-i0; s0 = 1-s1; t1 = y-j0; t0 = 1-t1;
			density[IX(i,j)] = s0*(t0*density0[IX(i0,j0)]+t1*density0[IX(i0,j1)])+
				s1*(t0*density0[IX(i1,j0)]+t1*density0[IX(i1,j1)]);
		}
	}
	set_bnd ( GridSize, boundary, density );
}


void project ( int GridSize, float * u, float * v, float * p, float * div )
{
	int i, j;

	for ( i=1 ; i<=GridSize ; i++ )
	{
		for ( j=1 ; j<=GridSize ; j++ )
		{
			div[IX(i,j)] = -0.5f*(u[IX(i+1,j)]-u[IX(i-1,j)]+v[IX(i,j+1)]-v[IX(i,j-1)])/GridSize;		
			p[IX(i,j)] = 0;
		}
	}	
	set_bnd ( GridSize, 0, div ); set_bnd ( GridSize, 0, p );

	lin_solve ( GridSize, 0, p, div, 1, 4 );

	for ( i=1 ; i<=GridSize ; i++ )
	{
		for ( j=1 ; j<=GridSize ; j++ ) 
		{
			u[IX(i,j)] -= 0.5f*GridSize*(p[IX(i+1,j)]-p[IX(i-1,j)]);
			v[IX(i,j)] -= 0.5f*GridSize*(p[IX(i,j+1)]-p[IX(i,j-1)]);
		}
	}
	set_bnd ( GridSize, 1, u ); set_bnd ( GridSize, 2, v );
}

hipError_t cuda_dens_step( int GridSize, float *grid, float *grid0, float *u, float *v, float diff, float dt )
{
	float *grid_cuda = 0;
	float *grid0_cuda= 0;
	float *u_cuda = 0;
	float *v_cuda = 0;

	int size = (GridSize+2)*(GridSize+2);

	hipError_t cuda_status;

	// Choose which GPU to run on, change this on a multi-GPU system
	cuda_status = hipSetDevice(0);
	if ( cuda_status != hipError_t::hipSuccess ) {
		ErrorMSG("hipSetDevice failed! Do you have a CUDA-capable GPU installed?");
		Error_Free(grid_cuda, grid0_cuda, u_cuda, v_cuda);
		return cuda_status;
	}

	// Allocate GPU buffers for those vectors
	if ( (cuda_status = hipMalloc((void**)&grid_cuda, size * sizeof(float))) != hipError_t::hipSuccess ) 
	{
		ErrorMSG("hipMalloc failed!");
		Error_Free(grid_cuda, grid0_cuda, u_cuda, v_cuda);
		return cuda_status;
	}

	if ( (cuda_status = hipMalloc((void**)&grid0_cuda, size * sizeof(float))) != hipError_t::hipSuccess )
	{
		ErrorMSG("hipMalloc failed!");
		Error_Free(grid_cuda, grid0_cuda, u_cuda, v_cuda);
		return cuda_status;
	}

	if ( (cuda_status = hipMalloc((void**)&u_cuda, size * sizeof(float))) != hipError_t::hipSuccess )
	{
		ErrorMSG("hipMalloc failed!");
		Error_Free(grid_cuda, grid0_cuda, u_cuda, v_cuda);
		return cuda_status;
	}

	if ( (cuda_status = hipMalloc((void**)&v_cuda, size * sizeof(float))) != hipError_t::hipSuccess )
	{
		ErrorMSG("hipMalloc failed!");
		Error_Free(grid_cuda, grid0_cuda, u_cuda, v_cuda);
		return cuda_status;
	}

	// Copy input vectors from host memory to GPU buffers
	// Do add_source_kernel first
	cuda_status = hipMemcpy(grid0_cuda, grid0, size * sizeof(float), hipMemcpyKind::hipMemcpyHostToDevice);
	if ( cuda_status != hipError_t::hipSuccess )
	{
		ErrorMSG("hipMemcpy failed!");
		Error_Free(grid_cuda, grid0_cuda, u_cuda, v_cuda);
		return cuda_status;
	}

	// Launch a kernel on GPU with one thread for each element
	add_source_kernel<<<1, size>>>(grid_cuda, grid0_cuda, dt);

	// Check for any errors launching the kernel
	cuda_status = hipGetLastError();
	if ( cuda_status != hipError_t::hipSuccess ) 
	{
		fprintf(stderr, "add_source_kernel launch failed: %s\n", hipGetErrorString(cuda_status));
		Error_Free(grid_cuda, grid0_cuda, u_cuda, v_cuda);
		return cuda_status;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns any errors encountered during the launch
	cuda_status = hipDeviceSynchronize();
	if ( cuda_status != hipError_t::hipSuccess )
	{
		fprintf(stderr, "hipDeviceSynchronize returned error code %d (%s) after launching kernel!\n",
			cuda_status, hipGetErrorString(cuda_status));
		Error_Free(grid_cuda, grid0_cuda, u_cuda, v_cuda);
		return cuda_status;
	}

	// Copy output vector from GPU buffer to host memory
	cuda_status = hipMemcpy(grid, grid_cuda, size*sizeof(float), hipMemcpyKind::hipMemcpyDeviceToHost);
	if ( cuda_status != hipError_t::hipSuccess )
	{
		ErrorMSG("hipMemcpy failed!");
		Error_Free(grid_cuda, grid0_cuda, u_cuda, v_cuda);
		return cuda_status;
	}

	SWAP ( grid0, grid ); diffuse ( GridSize, 0, grid, grid0, diff, dt );
	SWAP ( grid0, grid ); advect ( GridSize, 0, grid, grid0, u, v, dt );

	return cuda_status;
};

void add_source ( int GridSize, float * grid, float * src, float dt )
{
	int i, size=(GridSize+2)*(GridSize+2);
	for ( i=0 ; i<size ; i++ ) grid[i] += dt*src[i];
}

hipError_t cuda_vel_step( int GridSize, float * u, float * v, float * u0, float * v0, float visc, float dt )
{
	add_source ( GridSize, u, u0, dt ); add_source ( GridSize, v, v0, dt );
	SWAP ( u0, u ); diffuse ( GridSize, 1, u, u0, visc, dt );
	SWAP ( v0, v ); diffuse ( GridSize, 2, v, v0, visc, dt );
	project ( GridSize, u, v, u0, v0 );
	SWAP ( u0, u ); SWAP ( v0, v );
	advect ( GridSize, 1, u, u0, u0, v0, dt ); advect ( GridSize, 2, v, v0, u0, v0, dt );
	project ( GridSize, u, v, u0, v0 );
};

extern "C" void dens_step( int GridSize, float *grid, float *grid0, float *u, float *v, float diff, float dt )
{
	cuda_dens_step( GridSize, grid, grid0, u, v, diff, dt );
}

extern "C" void vel_step( int GridSize, float * u, float * v, float * u0, float * v0, float visc, float dt )
{
	cuda_vel_step(GridSize,  u,  v,  u0,  v0,  visc, dt);
}