/**
* <Author>        Orlando Chen
* <Email>         seagochen@gmail.com
* <First Time>    Dec 15, 2013
* <Last Time>     Mar 04, 2014
* <File Name>     FluidSimProcKernels.cu
*/

#include <time.h>
#include <iostream>
#include <utility>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <>
#include "MacroDefinition.h"
#include "FluidSimProc.h"
#include "MacroDefinition.h"
#include "FrameworkDynamic.h"
#include "Kernels.h"