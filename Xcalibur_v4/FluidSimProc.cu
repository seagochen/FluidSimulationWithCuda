/**
* <Author>        Orlando Chen
* <Email>         seagochen@gmail.com
* <First Time>    Dec 15, 2013
* <Last Time>     Mar 24, 2014
* <File Name>     FluidSimProc.cu
*/

#include <time.h>
#include <iostream>
#include <utility>
#include <hip/hip_runtime_api.h>
#include <>
#include "MacroDefinition.h"
#include "FluidSimProc.h"
#include "MacroDefinition.h"
#include "Kernels.h"

using namespace sge;

void FluidSimProc::CreateTopology( void )
{
	for ( int k = 0; k < GNODES_X; k++ )
	{
		for ( int j = 0; j < GNODES_X; j++ )
		{
			for ( int i = 0; i < GNODES_X; i++ )
			{
				/* left */
				if ( i >= 1 )
					gpu_node[cudaIndex3D( i, j, k, GNODES_X )]->ptrLeft  = gpu_node[cudaIndex3D( i-1, j, k, GNODES_X )];
				/* right */
				if ( i <= GNODES_X - 2 )
					gpu_node[cudaIndex3D( i, j, k, GNODES_X )]->ptrRight = gpu_node[cudaIndex3D( i+1, j, k, GNODES_X )];
				/* down */
				if ( j >= 1 )
					gpu_node[cudaIndex3D( i, j, k, GNODES_X )]->ptrDown  = gpu_node[cudaIndex3D( i, j-1, k, GNODES_X )];
				/* up */
				if ( j <= GNODES_X - 2 )
					gpu_node[cudaIndex3D( i, j, k, GNODES_X )]->ptrUp    = gpu_node[cudaIndex3D( i, j+1, k, GNODES_X )];
				/* back */
				if ( k >= 1 )
					gpu_node[cudaIndex3D( i, j, k, GNODES_X )]->ptrBack  = gpu_node[cudaIndex3D( i, j, k-1, GNODES_X )];
				/* front */
				if ( k <= GNODES_X - 2 )
					gpu_node[cudaIndex3D( i, j, k, GNODES_X )]->ptrFront = gpu_node[cudaIndex3D( i, j, k+1, GNODES_X )];
			}
		}
	}

	for ( int k = 0; k < HNODES_X; k++ )
	{
		for ( int j = 0; j < HNODES_X; j++ )
		{
			for ( int i = 0; i < HNODES_X; i++ )
			{
				/* left */
				if ( i >= 1 )
					host_node[cudaIndex3D( i, j, k, HNODES_X )]->ptrLeft  = host_node[cudaIndex3D( i-1, j, k, HNODES_X )];
				/* right */
				if ( i <= HNODES_X - 2 )
					host_node[cudaIndex3D( i, j, k, HNODES_X )]->ptrRight = host_node[cudaIndex3D( i+1, j, k, HNODES_X )];
				/* down */
				if ( j >= 1 )
					host_node[cudaIndex3D( i, j, k, HNODES_X )]->ptrDown  = host_node[cudaIndex3D( i, j-1, k, HNODES_X )];
				/* up */
				if ( j <= HNODES_X - 2 )
					host_node[cudaIndex3D( i, j, k, HNODES_X )]->ptrUp    = host_node[cudaIndex3D( i, j+1, k, HNODES_X )];
				/* back */
				if ( k >= 1 )
					host_node[cudaIndex3D( i, j, k, HNODES_X )]->ptrBack  = host_node[cudaIndex3D( i, j, k-1, HNODES_X )];
				/* front */
				if ( k <= HNODES_X - 2 )
					host_node[cudaIndex3D( i, j, k, HNODES_X )]->ptrFront = host_node[cudaIndex3D( i, j, k+1, HNODES_X )];

				host_node[cudaIndex3D( i, j, k, HNODES_X )]->x = i;
				host_node[cudaIndex3D( i, j, k, HNODES_X )]->y = j;
				host_node[cudaIndex3D( i, j, k, HNODES_X )]->z = k;
			}
		}
	}
};

void FluidSimProc::RefreshStatus( FLUIDSPARAM *fluid )
{
	/* waiting for all kernels end */
	if ( hipDeviceSynchronize() not_eq hipSuccess )
	{
		printf( "hipDeviceSynchronize failed\n" );
		FreeResource();
		exit( 1 );
	}

	for ( int i = 0; i < HNODES_X * HNODES_X * HNODES_X; i++ ) host_node[i]->updated = false;

	/* counting FPS */
	fluid->fps.dwFrames ++;
	fluid->fps.dwCurrentTime = GetTickCount();
	fluid->fps.dwElapsedTime = fluid->fps.dwCurrentTime - fluid->fps.dwLastUpdateTime;

	/* 1 second */
	if ( fluid->fps.dwElapsedTime >= 1000 )
	{
		fluid->fps.uFPS     = fluid->fps.dwFrames * 1000 / fluid->fps.dwElapsedTime;
		fluid->fps.dwFrames = 0;
		fluid->fps.dwLastUpdateTime = fluid->fps.dwCurrentTime;
	}

	/* updating image */
	if ( hipMemcpy( host_visual, dev_visual, m_volm_size, hipMemcpyDeviceToHost ) not_eq hipSuccess )
	{
		helper.GetCUDALastError( "host function: hipMemcpy failed", __FILE__, __LINE__ );
		FreeResource();
		exit( 1 );
	}
	fluid->volume.ptrData = host_visual;
};

void FluidSimProc::InitBoundary( void )
{
	 helper.DeviceDim3D( &blockDim, &gridDim, THREADS_X, TILE_X, GRIDS_X, GRIDS_X, GRIDS_X );

	/* zero boundary buffers */
	kernelZeroBuffers __device_func__ ( dev_obs, GRIDS_X, GRIDS_X, GRIDS_X );

	for ( int i = 0; i < host_obstacle.size(); i++ )
	{
		if ( hipMemcpy( host_obstacle[i], dev_obs, m_node_size, hipMemcpyDeviceToHost ) not_eq hipSuccess )
		{
			helper.GetCUDALastError( "hipMemcpy failed", __FILE__, __LINE__ );
			FreeResource();
			exit( 1 );
		}
	}

	kernelCopyGrids __device_func__ ( gd_obstacle, dev_obs );

	// TODO more boundary condition
};

void FluidSimProc::FluidSimSolver( FLUIDSPARAM *fluid )
{
	if ( !fluid->run ) return;
	
	helper.DeviceDim3D( &blockDim, &gridDim, THREADS_X, TILE_X, GRIDS_X, GRIDS_X, GRIDS_X );

	kernelCopyGrids __device_func__ ( dev_den, gd_density );
	kernelCopyGrids __device_func__ ( dev_u, gd_velocity_u );
	kernelCopyGrids __device_func__ ( dev_v, gd_velocity_v );
	kernelCopyGrids __device_func__ ( dev_w, gd_velocity_w );

	SolveNavierStokesEquation( DELTATIME, true );

	double rate = 1.f/(double)HNODES_X;
		
	for ( int k = 0; k < HNODES_X; k++ ) for ( int j = 0; j < HNODES_X; j++ ) for ( int i = 0; i < HNODES_X; i++ )
	{
		ptr = host_node[cudaIndex3D(i,j,k,HNODES_X)];

		kernelInterRootGrids __device_func__ ( dev_density[cudaIndex3D(i,j,k,HNODES_X)], dev_den, i, j, k, rate );
		kernelInterRootGrids __device_func__ ( dev_velocity_u[cudaIndex3D(i,j,k,HNODES_X)], dev_u, i, j, k, rate );
		kernelInterRootGrids __device_func__ ( dev_velocity_v[cudaIndex3D(i,j,k,HNODES_X)], dev_v, i, j, k, rate );
		kernelInterRootGrids __device_func__ ( dev_velocity_w[cudaIndex3D(i,j,k,HNODES_X)], dev_w, i, j, k, rate );
	}

	for ( int i = 0; i < HNODES_X * HNODES_X * HNODES_X; i++ )
	{
		ptr = host_node[i];
		kernelInterLeafGrids __device_func__ ( gd_density, dev_density[i], ptr->x, ptr->y, ptr->z, rate );
		kernelInterLeafGrids __device_func__ ( gd_velocity_u, dev_velocity_u[i], ptr->x, ptr->y, ptr->z, rate );
		kernelInterLeafGrids __device_func__ ( gd_velocity_v, dev_velocity_v[i], ptr->x, ptr->y, ptr->z, rate );
		kernelInterLeafGrids __device_func__ ( gd_velocity_w, dev_velocity_w[i], ptr->x, ptr->y, ptr->z, rate );
	}

	for ( int k = 0; k < HNODES_X; k++ ) for ( int j = 0; j < HNODES_X; j++ ) for ( int i = 0; i < HNODES_X; i++ )
	{
		kernelPickData __device_func__ ( dev_visual, dev_density[cudaIndex3D(i,j,k,HNODES_X)], i, j, k, GRIDS_X );
	}

	RefreshStatus( fluid );
};

void FluidSimProc::ClearBuffers( void )
{
	DeviceParamDim();

	_zero( gd_density );
	_zero( gd_velocity_u );
	_zero( gd_velocity_v );
	_zero( gd_velocity_w );

	for ( int i = 0; i < m_vectCompBufs.size(); i++ ) _zero( m_vectCompBufs[i] );
		 	 

	for ( int i = 0; i < NODES_X * NODES_Y * NODES_Z; i++ )
	{
		_zero( m_vectGPUDens[i] ); _zero( m_vectNewDens[i] );
		_zero( m_vectGPUVelU[i] ); _zero( m_vectNewDens[i] );
		_zero( m_vectGPUVelV[i] ); _zero( m_vectNewDens[i] );
		_zero( m_vectGPUVelW[i] ); _zero( m_vectNewDens[i] );
		_zero( m_vectGPUObst[i] ); _zero( m_vectNewDens[i] );
	}

	if ( helper.GetCUDALastError( "host function failed: ZeroBuffers", __FILE__, __LINE__ ) )
	{
		FreeResource();
		exit( 1 );
	}
};