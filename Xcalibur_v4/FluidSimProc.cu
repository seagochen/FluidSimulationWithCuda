/**
* <Author>        Orlando Chen
* <Email>         seagochen@gmail.com
* <First Time>    Dec 15, 2013
* <Last Time>     Mar 23, 2014
* <File Name>     FluidSimProc.cu
*/

#include <time.h>
#include <iostream>
#include <utility>
#include "FluidSimProc.h"
#include "MacroDefinition.h"
#include "FrameworkDynamic.h"
#include "Kernels.h"
#include "SimulationParam.h"

using std::cout;
using std::endl;

using namespace sge;

#define __device_func__ <<<gridDim, blockDim>>>

void FluidSimProc::ClearCompNodes( void )
{
	m_scHelper.DeviceParamDim( &gridDim, &blockDim, THREADS_S, TILE_X, TILE_Y, GRIDS_X, GRIDS_Y, GRIDS_Z );

	for ( int i = 0; i < m_nNodeNum; i++ )
	{
		kernelZeroBuffers __device_func__ ( m_vectGPUDens[i], GRIDS_X, GRIDS_Y, GRIDS_Z );
		kernelZeroBuffers __device_func__ ( m_vectGPUVelU[i], GRIDS_X, GRIDS_Y, GRIDS_Z );
		kernelZeroBuffers __device_func__ ( m_vectGPUVelV[i], GRIDS_X, GRIDS_Y, GRIDS_Z );
		kernelZeroBuffers __device_func__ ( m_vectGPUVelW[i], GRIDS_X, GRIDS_Y, GRIDS_Z );
		kernelZeroBuffers __device_func__ ( m_vectGPUObst[i], GRIDS_X, GRIDS_Y, GRIDS_Z );

		kernelZeroBuffers __device_func__ ( m_vectNewDens[i], GRIDS_X, GRIDS_Y, GRIDS_Z );
		kernelZeroBuffers __device_func__ ( m_vectNewVelU[i], GRIDS_X, GRIDS_Y, GRIDS_Z );
		kernelZeroBuffers __device_func__ ( m_vectNewVelV[i], GRIDS_X, GRIDS_Y, GRIDS_Z );
		kernelZeroBuffers __device_func__ ( m_vectNewVelW[i], GRIDS_X, GRIDS_Y, GRIDS_Z );
	}

	if ( m_scHelper.GetCUDALastError( "call member function ClearCompNodes failed", __FILE__, __LINE__ ) )
	{
		FreeResource();
		exit(1);
	}

	for ( int i = 0; i < m_nNodeNum; i++ )
	{
		hipMemcpy( m_vectHostDens[i], m_vectNewDens[i], sizeof(double) * m_nNodeSize, hipMemcpyDeviceToHost );
		hipMemcpy( m_vectHostVelU[i], m_vectNewVelU[i], sizeof(double) * m_nNodeSize, hipMemcpyDeviceToHost );
		hipMemcpy( m_vectHostVelV[i], m_vectNewVelV[i], sizeof(double) * m_nNodeSize, hipMemcpyDeviceToHost );
		hipMemcpy( m_vectHostVelW[i], m_vectNewVelW[i], sizeof(double) * m_nNodeSize, hipMemcpyDeviceToHost );
		hipMemcpy( m_vectHostObst[i], m_vectGPUObst[i], sizeof(double) * m_nNodeSize, hipMemcpyDeviceToHost );
	}

	if ( m_scHelper.GetCUDALastError( "call member function ClearCompNodes failed", __FILE__, __LINE__ ) )
	{
		FreeResource();
		exit(1);
	}
};

void FluidSimProc::ClearDeviceBuffers( void )
{
	m_scHelper.DeviceParamDim( &gridDim, &blockDim, THREADS_S, 22, 22, BULLET_X, BULLET_Y, BULLET_Z );

	for ( int i = 0; i < m_vectGPUBuffers.size(); i++ )
	{
		kernelZeroBuffers __device_func__ ( m_vectGPUBuffers[i], BULLET_X, BULLET_Y, BULLET_Z );
	}

	if ( m_scHelper.GetCUDALastError( "call member function ClearDeviceBuffers failed", __FILE__, __LINE__ ) )
	{
		FreeResource();
		exit(1);
	}

	m_scHelper.DeviceParamDim( &gridDim, &blockDim, THREADS_S, TPBUFFER_S );

	kernelZeroBuffers __device_func__ ( m_ptrGPUShare, TPBUFFER_S );

	if ( m_scHelper.GetCUDALastError( "call member function ClearDeviceBuffers failed", __FILE__, __LINE__ ) )
	{
		FreeResource();
		exit(1);
	}

	hipMemcpy( m_ptrHostShare, m_ptrGPUShare, sizeof(double) * TPBUFFER_S, hipMemcpyDeviceToHost );

	if ( m_scHelper.GetCUDALastError( "call member function ClearDeviceBuffers failed", __FILE__, __LINE__ ) )
	{
		FreeResource();
		exit(1);
	}
};

void FluidSimProc::ClearVisualBuffers( void )
{
	m_scHelper.DeviceParamDim( &gridDim, &blockDim, THREADS_S, TILE_X, TILE_Y, VOLUME_X, VOLUME_Y, VOLUME_Z );

	kernelZeroBuffers __device_func__ ( m_ptrGPUVisual, VOLUME_X, VOLUME_Y, VOLUME_Z );
	hipMemcpy( m_ptrHostVisual, m_ptrGPUVisual, sizeof(uchar) * m_nVolumSize, hipMemcpyDeviceToHost );

	if ( m_scHelper.GetCUDALastError( "call member function ClearVisualBuffers failed", __FILE__, __LINE__ ) )
	{
		FreeResource();
		exit(1);
	}
};

void FluidSimProc::PushCompNode( int id )
{
	m_scHelper.DeviceParamDim( &gridDim, &blockDim, THREADS_S, TILE_X, TILE_Y, GRIDS_X, GRIDS_Y, GRIDS_Z );

	kernelLoadBullet __device_func__ ( dev_den, m_vectGPUDens[id], BULLET_X, BULLET_Y, BULLET_Z, GRIDS_X, GRIDS_Y, GRIDS_Z );
	kernelLoadBullet __device_func__ ( dev_u, m_vectGPUVelU[id], BULLET_X, BULLET_Y, BULLET_Z, GRIDS_X, GRIDS_Y, GRIDS_Z );
	kernelLoadBullet __device_func__ ( dev_v, m_vectGPUVelV[id], BULLET_X, BULLET_Y, BULLET_Z, GRIDS_X, GRIDS_Y, GRIDS_Z );
	kernelLoadBullet __device_func__ ( dev_w, m_vectGPUVelW[id], BULLET_X, BULLET_Y, BULLET_Z, GRIDS_X, GRIDS_Y, GRIDS_Z );
	kernelLoadBullet __device_func__ ( dev_obs, m_vectGPUObst[id], BULLET_X, BULLET_Y, BULLET_Z, GRIDS_X, GRIDS_Y, GRIDS_Z );

	if ( m_scHelper.GetCUDALastError( "call member function PushCompNode failed", __FILE__, __LINE__ ) )
	{
		FreeResource();
		exit(1);
	}
};

void FluidSimProc::PopCompNode( int id )
{
	m_scHelper.DeviceParamDim( &gridDim, &blockDim, THREADS_S, TILE_X, TILE_Y, GRIDS_X, GRIDS_Y, GRIDS_Z );

	kernelExitBullet __device_func__ ( m_vectNewDens[id], dev_den, GRIDS_X, GRIDS_Y, GRIDS_Z, BULLET_X, BULLET_Y, BULLET_Z );
	kernelExitBullet __device_func__ ( m_vectNewVelU[id], dev_u, GRIDS_X, GRIDS_Y, GRIDS_Z, BULLET_X, BULLET_Y, BULLET_Z );
	kernelExitBullet __device_func__ ( m_vectNewVelV[id], dev_v, GRIDS_X, GRIDS_Y, GRIDS_Z, BULLET_X, BULLET_Y, BULLET_Z );
	kernelExitBullet __device_func__ ( m_vectNewVelW[id], dev_w, GRIDS_X, GRIDS_Y, GRIDS_Z, BULLET_X, BULLET_Y, BULLET_Z );

	if ( m_scHelper.GetCUDALastError( "call member function PopCompNode failed", __FILE__, __LINE__ ) )
	{
		FreeResource();
		exit(1);
	}
};

void FluidSimProc::GenVolumeImage( FLUIDSPARAM *fluid )
{
	m_scHelper.DeviceParamDim( &gridDim, &blockDim, THREADS_S, TILE_X, TILE_Y, GRIDS_X, GRIDS_Y, GRIDS_Z );

	for ( int k = 0; k < NODES_Z; k++ )
	{
		for ( int j = 0; j < NODES_Y; j++ )
		{
			for ( int i = 0; i < NODES_X; i++ )
			{
				kernelDensToVolume __device_func__
					( m_ptrGPUVisual, m_vectGPUDens[IX(i,j,k,NODES_X,NODES_Y,NODES_Z)], i, j, k,
					GRIDS_X, GRIDS_Y, GRIDS_Z, VOLUME_X, VOLUME_Y, VOLUME_Z );
			}
		}
	}

	if ( hipMemcpy( m_ptrHostVisual, m_ptrGPUVisual, sizeof(uchar) * m_nVolumSize, 
		hipMemcpyDeviceToHost ) not_eq hipSuccess )
	{
		m_scHelper.GetCUDALastError( "call member function GenVolumeImage failed", __FILE__, __LINE__ );
		FreeResource();
		exit( 1 );
	}
	
	fluid->volume.ptrData = m_ptrHostVisual;
};

void FluidSimProc::VelocitySolver( cdouble timestep )
{
	// diffuse the velocity field (per axis):
	Diffusion( dev_u0, dev_u, VISOCITY );
	Diffusion( dev_v0, dev_v, VISOCITY );
	Diffusion( dev_w0, dev_w, VISOCITY );
	
	if ( m_scHelper.GetCUDALastError( "host function failed: Diffusion", __FILE__, __LINE__ ) )
	{
		FreeResource();
		exit( 1 );
	}

	std::swap( dev_u0, dev_u );
	std::swap( dev_v0, dev_v );
	std::swap( dev_w0, dev_w );

	// stabilize it: (vx0, vy0 are whatever, being used as temporaries to store gradient field)
	Projection( dev_u, dev_v, dev_w, dev_div, dev_p );

	if ( m_scHelper.GetCUDALastError( "host function failed: Projection", __FILE__, __LINE__ ) )
	{
		FreeResource();
		exit( 1 );
	}
	
	// advect the velocity field (per axis):
	Advection( dev_u0, dev_u, timestep, dev_u, dev_v, dev_w );
	Advection( dev_v0, dev_v, timestep, dev_u, dev_v, dev_w );
	Advection( dev_w0, dev_w, timestep, dev_u, dev_v, dev_w );

	if ( m_scHelper.GetCUDALastError( "host function failed: Advection", __FILE__, __LINE__ ) )
	{
		FreeResource();
		exit( 1 );
	}

	std::swap( dev_u0, dev_u );
	std::swap( dev_v0, dev_v );
	std::swap( dev_w0, dev_w );
	
	// stabilize it: (vx0, vy0 are whatever, being used as temporaries to store gradient field)
	Projection( dev_u, dev_v, dev_w, dev_div, dev_p );
};

void FluidSimProc::DensitySolver( cdouble timestep )
{
	Diffusion( dev_den0, dev_den, DIFFUSION );
	std::swap( dev_den0, dev_den );
	Advection ( dev_den, dev_den0, timestep, dev_u, dev_v, dev_w );

	if ( m_scHelper.GetCUDALastError( "host function failed: DensitySolver", __FILE__, __LINE__ ) )
	{
		FreeResource();
		exit( 1 );
	}
};

void FluidSimProc::SourceSolver( void )
{
	m_scHelper.DeviceParamDim( &gridDim, &blockDim, THREADS_S, TILE_X, TILE_Y, GRIDS_X, GRIDS_Y, GRIDS_Z );	

	kernelAddSource __device_func__ ( dev_den, dev_u, dev_v, dev_w, dev_obs, DENSITY, VELOCITY, DELTATIME );
//	if ( decrease_times eqt 0 )
//	{
//		
//
//		if ( helper.GetCUDALastError( "device kernel: kernelPickData failed", __FILE__, __LINE__ ) )
//		{
//			FreeResource();
//			exit( 1 );
//		}
//
//		increase_times++;
//
//		if ( increase_times eqt 200 )
//		{
//			decrease_times = increase_times;
//			increase_times = 0;
//		}
//	}
//	else
//	{
//		decrease_times--;
//	}
};

void FluidSimProc::Jacobi( double *out, cdouble *in, cdouble diff, cdouble divisor )
{
	m_scHelper.DeviceParamDim( &gridDim, &blockDim, THREADS_S, TILE_X, TILE_Y, GRIDS_X, GRIDS_Y, GRIDS_Z );

	for ( int k=0; k<20; k++)
	{
		kernelJacobi<<<gridDim,blockDim>>>( out, in, diff, divisor);
	}
};

void FluidSimProc::Advection( double *out, cdouble *in, cdouble timestep, cdouble *u, cdouble *v, cdouble *w )
{
	m_scHelper.DeviceParamDim( &gridDim, &blockDim, THREADS_S, TILE_X, TILE_Y, GRIDS_X, GRIDS_Y, GRIDS_Z );

	kernelAdvection<<<gridDim,blockDim>>>( out, in, timestep, u, v, w );
};

void FluidSimProc::Diffusion( double *out, cdouble *in, cdouble diff )
{
	double rate = diff * GRIDS_X * GRIDS_Y * GRIDS_Z;
	Jacobi ( out, in, rate, 1+6*rate );
};

void FluidSimProc::Projection( double *u, double *v, double *w, double *div, double *p )
{
	m_scHelper.DeviceParamDim( &gridDim, &blockDim, THREADS_S, TILE_X, TILE_Y, GRIDS_X, GRIDS_Y, GRIDS_Z );

	// the velocity gradient
	kernelGradient<<<gridDim,blockDim>>>( div, p, u, v, w );

	// reuse the Gauss-Seidel relaxation solver to safely diffuse the velocity gradients from p to div
	Jacobi(p, div, 1.f, 6.f);

	// now subtract this gradient from our current velocity field
	kernelSubtract<<<gridDim,blockDim>>>( u, v, w, p );
};