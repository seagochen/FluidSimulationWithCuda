/**
* <Author>        Orlando Chen
* <Email>         seagochen@gmail.com
* <First Time>    Dec 15, 2013
* <Last Time>     Mar 19, 2014
* <File Name>     FluidSimProc.cu
*/

#include <time.h>
#include <iostream>
#include <utility>
#include "FluidSimProc.h"
#include "MacroDefinition.h"
#include "FrameworkDynamic.h"
#include "Kernels.h"
#include "SimulationParam.h"

using namespace sge;

#define __device_func__ <<<gridDim, blockDim>>>


void FluidSimProc::ClearCompNodes( void )
{
	m_scHelper.DeviceParamDim( &gridDim, &blockDim, THREADS_S, TILE_X, TILE_Y, GRIDS_X, GRIDS_Y, GRIDS_Z );

	for ( int i = 0; i < m_nNodeNum; i++ )
	{
		kernelZeroBuffers __device_func__ ( m_vectGPUDens[i], GRIDS_X, GRIDS_Y, GRIDS_Z );
		kernelZeroBuffers __device_func__ ( m_vectGPUVelU[i], GRIDS_X, GRIDS_Y, GRIDS_Z );
		kernelZeroBuffers __device_func__ ( m_vectGPUVelV[i], GRIDS_X, GRIDS_Y, GRIDS_Z );
		kernelZeroBuffers __device_func__ ( m_vectGPUVelW[i], GRIDS_X, GRIDS_Y, GRIDS_Z );
		kernelZeroBuffers __device_func__ ( m_vectGPUObst[i], GRIDS_X, GRIDS_Y, GRIDS_Z );

		kernelZeroBuffers __device_func__ ( m_vectNewDens[i], GRIDS_X, GRIDS_Y, GRIDS_Z );
		kernelZeroBuffers __device_func__ ( m_vectNewVelU[i], GRIDS_X, GRIDS_Y, GRIDS_Z );
		kernelZeroBuffers __device_func__ ( m_vectNewVelV[i], GRIDS_X, GRIDS_Y, GRIDS_Z );
		kernelZeroBuffers __device_func__ ( m_vectNewVelW[i], GRIDS_X, GRIDS_Y, GRIDS_Z );
	}

	if ( m_scHelper.GetCUDALastError( "call member function ClearCompNodes failed", __FILE__, __LINE__ ) )
	{
		FreeResource();
		exit(1);
	}

	for ( int i = 0; i < m_nNodeNum; i++ )
	{
		hipMemcpy( m_vectHostDens[i], m_vectNewDens[i], sizeof(double) * m_nNodeSize, hipMemcpyDeviceToHost );
		hipMemcpy( m_vectHostVelU[i], m_vectNewVelU[i], sizeof(double) * m_nNodeSize, hipMemcpyDeviceToHost );
		hipMemcpy( m_vectHostVelV[i], m_vectNewVelV[i], sizeof(double) * m_nNodeSize, hipMemcpyDeviceToHost );
		hipMemcpy( m_vectHostVelW[i], m_vectNewVelW[i], sizeof(double) * m_nNodeSize, hipMemcpyDeviceToHost );
		hipMemcpy( m_vectHostObst[i], m_vectGPUObst[i], sizeof(double) * m_nNodeSize, hipMemcpyDeviceToHost );
	}

	if ( m_scHelper.GetCUDALastError( "call member function ClearCompNodes failed", __FILE__, __LINE__ ) )
	{
		FreeResource();
		exit(1);
	}
};

void FluidSimProc::ClearDeviceBuffers( void )
{
	m_scHelper.DeviceParamDim( &gridDim, &blockDim, THREADS_S, 22, 22, BULLET_X, BULLET_Y, BULLET_Z );

	for ( int i = 0; i < m_vectGPUBuffers.size(); i++ )
	{
		kernelZeroBuffers __device_func__ ( m_vectGPUBuffers[i], BULLET_X, BULLET_Y, BULLET_Z );
	}

	if ( m_scHelper.GetCUDALastError( "call member function ClearDeviceBuffers failed", __FILE__, __LINE__ ) )
	{
		FreeResource();
		exit(1);
	}

	m_scHelper.DeviceParamDim( &gridDim, &blockDim, THREADS_S, TPBUFFER_S );

	kernelZeroBuffers __device_func__ ( m_ptrGPUShare, TPBUFFER_S );

	if ( m_scHelper.GetCUDALastError( "call member function ClearDeviceBuffers failed", __FILE__, __LINE__ ) )
	{
		FreeResource();
		exit(1);
	}

	hipMemcpy( m_ptrHostShare, m_ptrGPUShare, sizeof(double) * TPBUFFER_S, hipMemcpyDeviceToHost );

	if ( m_scHelper.GetCUDALastError( "call member function ClearDeviceBuffers failed", __FILE__, __LINE__ ) )
	{
		FreeResource();
		exit(1);
	}
};

void FluidSimProc::ClearVisualBuffers( void )
{
	m_scHelper.DeviceParamDim( &gridDim, &blockDim, THREADS_S, TILE_X, TILE_Y, VOLUME_X, VOLUME_Y, VOLUME_Z );

	kernelZeroBuffers __device_func__ ( m_ptrGPUVisual, VOLUME_X, VOLUME_Y, VOLUME_Z );
	hipMemcpy( m_ptrHostVisual, m_ptrGPUVisual, sizeof(uchar) * m_nVolumSize, hipMemcpyDeviceToHost );

	if ( m_scHelper.GetCUDALastError( "call member function ClearVisualBuffers failed", __FILE__, __LINE__ ) )
	{
		FreeResource();
		exit(1);
	}
};

