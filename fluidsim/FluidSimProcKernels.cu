#include "hip/hip_runtime.h"
/**
* <Author>        Orlando Chen
* <Email>         seagochen@gmail.com
* <First Time>    Nov 15, 2013
* <Last Time>     Feb 17, 2014
* <File Name>     FluidSimProc.cu
*/

#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <>
#include "FluidSimProc.h"
#include "CUDAInterfaces.h"

using namespace sge;

/* Ĭ�ϵĹ��캯������������ģ������Ҫ�Ŀռ䣬�Լ���ʼ����ز��� */
FluidSimProc::FluidSimProc( FLUIDSPARAM *fluid )
{
	/* initialize the parameters of fluid simulation */
	InitParams( fluid );

	/* allocate the space for fluid simulation */
	if ( !AllocateResource() )
	{
		FreeResource();
		printf(" malloc buffers for fluid simulation failed! \n");
		exit(1);
	}
	else
		printf( "allocate resource success!\n" );

	/* initialize the nodes */
	InitSimNodes();

	/* building structure order */
	BuildOrder();

	/* select and active a node for fluid simulation */
	ActiveNode( 1, 1, 0 );

	/* zero all buffers */
	ZeroBuffers();
	
	/* finally, print the state message and zero the data */
	printf( "fluid simulation ready...\n" );
};

/* ��ʼ������ģ�����ز��� */
void FluidSimProc::InitParams( FLUIDSPARAM *fluid )
{
	/* initilize the status of FPS counter */
	fluid->fps.dwCurrentTime    = 0;
	fluid->fps.dwElapsedTime    = 0;
	fluid->fps.dwFrames         = 0;
	fluid->fps.dwLastUpdateTime = 0;
	fluid->fps.uFPS             = 0;
};

/* fluid simulation processing function */
void FluidSimProc::FluidSimSolver( FLUIDSPARAM *fluid )
{
	if ( fluid->run )
	{
		for ( int i = 0; i < NODES_X; i++ )
		{
			for ( int j = 0; j < NODES_X; j++ )
			{
				for ( int k = 0; k < NODES_X; k++ )
				{
					// TODO: �ڼ�������ǰ����һֱ���ü򵥶����˵���ѯ��������������ڵ��״̬��
					// �ڵ�ǰ������£������нڵ�Ĭ��Ϊ����״̬������������ֱ�۵Ĳ��Ը����ڵ����ݴ��������
					/* ���㿪ʼ */
					SelectNode( i, j, k );
					UploadBuffers();

					AddSource();
					DensitySolver( dev_dens, dev_vel_u, dev_vel_v, dev_vel_w, dev_dens0 );
					VelocitySolver( dev_vel_u, dev_vel_v, dev_vel_w,
						dev_vel_u0, dev_vel_v0, dev_vel_w0, dev_div, dev_p );
//					hostAddSource( dev_center, SG_VELOCITY_U_FIELD );
//					hostAddSource( dev_center, SG_VELOCITY_V_FIELD );
//					hostAddSource( dev_center, SG_VELOCITY_W_FIELD );
//					hostAddSource( dev_center, SG_DENSITY_FIELD );


					/* ������� */
					DownloadBuffers();
					hostPickData( dev_visual, dev_center, &nodeIX ); 
				}
			}
		}
	}
};

__global__ void kernelAddSource( double *dens, double *u, double *v, double *w, double const *obs )
{
	GetIndex();

	if ( obs[ Index(i,j,k) ] eqt BOUND_SOURCE )
	{
		dens[ Index(i,j,k) ] = SOURCE_DENSITY;
		v[ Index(i,j,k) ] = SOURCE_VELOCITY;
	}
};

/* add source */
void FluidSimProc::AddSource( void )
{
	cudaDeviceDim3D();

	kernelAddSource cudaDevice(gridDim, blockDim)
		( dev_dens, dev_vel_u, dev_vel_v, dev_vel_w, dev_obs );
};

/* allocate resource */
bool FluidSimProc::AllocateResource( void )
{
	size_t size = GRIDS_X * GRIDS_X * GRIDS_X;

	/* allocate device buffers */
	for ( int i = 0; i < dev_buffers_num; i++ )
	{
		double *buf;
		if ( m_helper.CreateDeviceBuffers( size, 1, &buf ) not_eq SG_RUNTIME_OK )
			return false;

		dev_buffers.push_back( buf );
	}

	/* allocate host buffers */
	for ( int i = 0; i < NODES_X * NODES_X * NODES_X; i++)
	{
		double *dens, *u, *v, *w, *obs;
		if ( m_helper.CreateHostBuffers( size, 5, &dens, &u, &v, &w, &obs ) not_eq SG_RUNTIME_OK )
			return false;
		
		host_density.push_back( dens );
		host_velocity_u.push_back( u );
		host_velocity_v.push_back( v );
		host_velocity_w.push_back( w );
		host_obstacle.push_back( obs );
	}

	/* allocate visual buffers */	
	size = VOLUME_X * VOLUME_X * VOLUME_X;
	if ( m_helper.CreateVolumetricBuffers( size, &host_visual, &dev_visual ) not_eq SG_RUNTIME_OK )
		return false;

	/* push nodes to linker */
	for ( int i = 0; i < NODES_X * NODES_X * NODES_X; i++ )
	{
		LinkNode* node = (LinkNode*)malloc( sizeof(LinkNode) );
		node->ptrBack = node->ptrDown = node->ptrFront =
			node->ptrLeft = node->ptrRight = node->ptrUp = nullptr;

		if ( node eqt nullptr )
		{
			printf( "malloc node failed\n" );
			return false;
		}

		host_link.push_back( node );
	}	

	return true;
};

/* when program existed, release resource */
void FluidSimProc::FreeResource( void )
{
	/* free device L-0 buffers */
	for ( int i = 0; i < dev_buffers_num; i++ )
	{
		hipFree( dev_buffers[i] );
	}

	/* free host L-0 buffers */
	int i;
	for ( i = 0; i < host_density.size(); i++ )
		SAFE_FREE_PTR( host_density[i] );
	for ( i = 0; i < host_velocity_u.size(); i++ )
		SAFE_FREE_PTR( host_velocity_u[i] );
	for ( i = 0; i < host_velocity_v.size(); i++ )
		SAFE_FREE_PTR( host_velocity_v[i] );
	for ( i = 0; i < host_velocity_w.size(); i++ )
		SAFE_FREE_PTR( host_velocity_w[i] );
	for ( i = 0; i < host_obstacle.size(); i++ )
		SAFE_FREE_PTR( host_obstacle[i] );

	/* free L-0 visual buffers */
	SAFE_FREE_PTR( host_visual );
	hipFree( dev_visual );
};

/* zero the buffers for fluid simulation */
void FluidSimProc::ZeroBuffers( void )
{
	/* zero center node first */
	hostZeroBuffer( dev_center );

	size_t size = GRIDS_X * GRIDS_X * GRIDS_X * sizeof(double);

	for ( int i = 0; i < NODES_X * NODES_X * NODES_X; i++ )
	{
		if ( hipMemcpy( host_density[i], dev_center, size, hipMemcpyDeviceToHost ) != hipSuccess )
		{
			m_helper.CheckRuntimeErrors( "hipMemcpy failed", __FILE__, __LINE__ );
			exit( 1 );
		}
		if ( hipMemcpy( host_velocity_u[i], dev_center, size, hipMemcpyDeviceToHost ) != hipSuccess )
		{
			m_helper.CheckRuntimeErrors( "hipMemcpy failed", __FILE__, __LINE__ );
			exit( 1 );
		}
		if ( hipMemcpy( host_velocity_v[i], dev_center, size, hipMemcpyDeviceToHost ) != hipSuccess )
		{
			m_helper.CheckRuntimeErrors( "hipMemcpy failed", __FILE__, __LINE__ );
			exit( 1 );
		}
		if ( hipMemcpy( host_velocity_w[i], dev_center, size, hipMemcpyDeviceToHost ) != hipSuccess )
		{
			m_helper.CheckRuntimeErrors( "hipMemcpy failed", __FILE__, __LINE__ );
			exit( 1 );
		}

	}
};

/* choose the node and mark it as actived */
void FluidSimProc::ActiveNode( int i, int j, int k )
{
	int ix = 0;

	if ( i >= 0 and i < NODES_X and j >= 0 and j < NODES_X and k >= 0 and k < NODES_X )
	{
		ix = cudaIndex3D( i, j, k, NODES_X );
		host_link[ix]->active = true;

		/* print status */
		if ( host_link[ix]->active )			
			printf ( "node no.%d is actived!\n", ix );
		else
			printf ( "node no.%d is deactived!\n", ix );
	}	
};

/* choose the node and mark it as deactived */
void FluidSimProc::DeactiveNode( int i, int j, int k )
{
	int ix = 0;

	if ( i >= 0 and i < NODES_X and j >= 0 and j < NODES_X and k >= 0 and k < NODES_X )
	{
		ix = cudaIndex3D( i, j, k, NODES_X );
		host_link[ix]->active = false;

		/* print status */
		if ( host_link[ix]->active )			
			printf ( "node no.%d is actived!\n", ix );
		else
			printf ( "node no.%d is deactived!\n", ix );
	}	
};


void FluidSimProc::SelectNode( int i, int j, int k )
{
	if ( i >= 0 and i < NODES_X and j >= 0 and j < NODES_X and k >= 0 and k < NODES_X )
	{
		nodeIX.x = i;
		nodeIX.y = j;
		nodeIX.z = k;
	}
};

/* zero data, set the bounds */
void FluidSimProc::InitSimNodes( void )
{
	hostZeroBuffer( dev_center );

	for ( int i = 0; i < host_obstacle.size(); i++ )
	{
		if ( hipMemcpy( host_obstacle[i], dev_center, 
			sizeof(double) * GRIDS_X * GRIDS_X * GRIDS_X, hipMemcpyDeviceToHost ) not_eq hipSuccess )
		{
			m_helper.CheckRuntimeErrors( "hipMalloc failed", __FILE__, __LINE__ );
			FreeResource();
			exit(1);
		}
	}

	int half = GRIDS_X / 2;
	double *obs = host_obstacle[0];
	obs[Index(half, half, half)] = BOUND_SOURCE;
	obs[Index(half+1, half, half)] = BOUND_SOURCE;
	obs[Index(half-1, half, half)] = BOUND_SOURCE;
	obs[Index(half, half+1, half)] = BOUND_SOURCE;
	obs[Index(half, half-1, half)] = BOUND_SOURCE;
	obs[Index(half, half, half+1)] = BOUND_SOURCE;
	obs[Index(half, half, half-1)] = BOUND_SOURCE;
};

/* create simulation nodes' topological structure */
void FluidSimProc::BuildOrder( void )
{
	printf( "structure:\n" );
	for ( int i = 0; i < NODES_X; i++ )
	{
		for ( int j = 0; j < NODES_X; j++ )
		{
			for ( int k = 0; k < NODES_X; k++ )
			{
				int index = cudaIndex3D( i, j, k, NODES_X );

				if ( index >= host_link.size() or index < 0 )
				{
					printf ( "index out of range! %s, line: %d \n", __FILE__, __LINE__ );
					exit ( 1 );
				}

				/* left */
				if ( i >= 1 )
					host_link[index]->ptrLeft = host_link[index-1];
				/* right */
				if ( i <= NODES_X - 2 )
					host_link[index]->ptrRight = host_link[index+1];
				/* down */
				if ( j >= 1 )
					host_link[index]->ptrDown = host_link[index-NODES_X];
				/* up */
				if ( j <= NODES_X - 2 )
					host_link[index]->ptrUp = host_link[index+NODES_X];
				/* back */
				if ( k >= 1 )
					host_link[index]->ptrBack = host_link[index-NODES_X*NODES_X];
				/* front */
				if ( k <= NODES_X - 2 )
					host_link[index]->ptrFront = host_link[index+NODES_X*NODES_X];

				host_link[index]->n3Pos.x = i;
				host_link[index]->n3Pos.y = j;
				host_link[index]->n3Pos.z = k;

				printf ( "no: %d | offset: %d%d%d | L: %d | R: %d | U: %d | D: %d | F: %d | B: %d \n",
					index,
					host_link[index]->n3Pos.x, 
					host_link[index]->n3Pos.y, 
					host_link[index]->n3Pos.z,
					host_link[index]->ptrLeft not_eq nullptr,
					host_link[index]->ptrRight not_eq nullptr,
					host_link[index]->ptrUp not_eq nullptr,
					host_link[index]->ptrDown not_eq nullptr,
					host_link[index]->ptrFront not_eq nullptr,
					host_link[index]->ptrBack not_eq nullptr );
			}
		}
	}

	printf( "-----------------------------------------------\n" );
};

/* copy host data to CUDA device */
void FluidSimProc::UploadBuffers( void )
{
	int ix = cudaIndex3D( nodeIX.x, nodeIX.y, nodeIX.z, NODES_X );

	/* zero all buffers first */
	hostZeroBuffer( dev_dens );
	hostZeroBuffer( dev_vel_u );
	hostZeroBuffer( dev_vel_v );
	hostZeroBuffer( dev_vel_w );

	size_t size = GRIDS_X * GRIDS_X * GRIDS_X * sizeof( double );

	if ( hipMemcpy( dev_dens, host_density[ix], size, hipMemcpyHostToDevice ) not_eq hipSuccess )
	{
		m_helper.CheckRuntimeErrors( "hipMemcpy failed", __FILE__, __LINE__ );
		FreeResource();
		exit(1);
	}
	if ( hipMemcpy( dev_vel_u, host_velocity_u[ix], size, hipMemcpyHostToDevice ) not_eq hipSuccess )
	{
		m_helper.CheckRuntimeErrors( "hipMemcpy failed", __FILE__, __LINE__ );
		FreeResource();
		exit(1);
	}
	if ( hipMemcpy( dev_vel_v, host_velocity_v[ix], size, hipMemcpyHostToDevice ) not_eq hipSuccess )
	{
		m_helper.CheckRuntimeErrors( "hipMemcpy failed", __FILE__, __LINE__ );
		FreeResource();
		exit(1);
	}
	if ( hipMemcpy( dev_vel_w, host_velocity_w[ix], size, hipMemcpyHostToDevice ) not_eq hipSuccess )
	{
		m_helper.CheckRuntimeErrors( "hipMemcpy failed", __FILE__, __LINE__ );
		FreeResource();
		exit(1);
	}
	if ( hipMemcpy( dev_obs, host_obstacle[ix], size, hipMemcpyHostToDevice ) not_eq hipSuccess )
	{
		m_helper.CheckRuntimeErrors( "hipMemcpy failed", __FILE__, __LINE__ );
		FreeResource();
		exit(1);
	}

};

/* retrieve data back to host */
void FluidSimProc::DownloadBuffers( void )
{
	int ix = cudaIndex3D( nodeIX.x, nodeIX.y, nodeIX.z, NODES_X );

	size_t size = GRIDS_X * GRIDS_X * GRIDS_X * sizeof( double );

	if ( hipMemcpy( host_density[ix], dev_dens, size, hipMemcpyDeviceToHost ) not_eq hipSuccess )
	{
		m_helper.CheckRuntimeErrors( "hipMemcpy failed", __FILE__, __LINE__ );
		FreeResource();
		exit(1);
	}
	if ( hipMemcpy( host_velocity_u[ix], dev_vel_u, size, hipMemcpyDeviceToHost ) not_eq hipSuccess )
	{
		m_helper.CheckRuntimeErrors( "hipMemcpy failed", __FILE__, __LINE__ );
		FreeResource();
		exit(1);
	}
	if ( hipMemcpy( host_velocity_v[ix], dev_vel_v, size, hipMemcpyDeviceToHost ) not_eq hipSuccess )
	{
		m_helper.CheckRuntimeErrors( "hipMemcpy failed", __FILE__, __LINE__ );
		FreeResource();
		exit(1);
	}
	if ( hipMemcpy( host_velocity_w[ix], dev_vel_w, size, hipMemcpyDeviceToHost ) not_eq hipSuccess )
	{
		m_helper.CheckRuntimeErrors( "hipMemcpy failed", __FILE__, __LINE__ );
		FreeResource();
		exit(1);
	}
};

/* retrieve the density back and load into volumetric data for rendering */
void FluidSimProc::PickVolumetric( FLUIDSPARAM *fluid )
{
	if ( hipMemcpy( host_visual, dev_visual,
		sizeof(SGUCHAR) * VOLUME_X * VOLUME_X * VOLUME_X, hipMemcpyDeviceToHost ) != hipSuccess )
	{
		m_helper.CheckRuntimeErrors( "hipMemcpy failed", __FILE__, __LINE__ );
		FreeResource();
		exit(1);
	}

	fluid->volume.ptrData = host_visual;
};