/**
* <Author>        Orlando Chen
* <Email>         seagochen@gmail.com
* <First Time>    Nov 15, 2013
* <Last Time>     Feb 15, 2014
* <File Name>     FluidSimProc.cu
*/

#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <>
#include "FluidSimProc.h"
#include "CUDAInterfaces.h"

using namespace sge;

/* Ĭ�ϵĹ��캯������������ģ������Ҫ�Ŀռ䣬�Լ���ʼ����ز��� */
FluidSimProc::FluidSimProc( FLUIDSPARAM *fluid )
{
	/* initialize the parameters of fluid simulation */
	InitParams( fluid );

	/* allocate the space for fluid simulation */
	if ( !AllocateResource() )
	{
		FreeResource();
		printf(" malloc buffers for fluid simulation failed! \n");
		exit(1);
	}
	else
		printf( "allocate resource success!\n" );

	/* initialize the nodes */
	InitSimNodes();

	/* building structure order */
	BuildOrder();

	/* select and active a node for fluid simulation */
	ActiveNode( 1, 1, 0 );
	
	/* finally, print the state message and zero the data */
	printf( "fluid simulation ready...\n" );
};

/* ��ʼ������ģ�����ز��� */
void FluidSimProc::InitParams( FLUIDSPARAM *fluid )
{
	/* initilize the status of FPS counter */
	fluid->fps.dwCurrentTime    = 0;
	fluid->fps.dwElapsedTime    = 0;
	fluid->fps.dwFrames         = 0;
	fluid->fps.dwLastUpdateTime = 0;
	fluid->fps.uFPS             = 0;
};

/* fluid simulation processing function */
void FluidSimProc::FluidSimSolver( FLUIDSPARAM *fluid )
{
	if ( fluid->run )
	{
	}
};

/* allocate resource */
bool FluidSimProc::AllocateResource( void )
{
	size_t size;

	/* allocate device L-0 buffers */
	for ( int i = 0; i < dev_L0_vector_num; i++ )
	{
		SGSIMPLENODES *buf;
		if ( hipMalloc( (void**)&buf, sizeof(SGSIMPLENODES) ) != hipSuccess )
		{
			m_helper.CheckRuntimeErrors( "hipMalloc failed", __FILE__, __LINE__ );
			return false;
		}
		dev_L0_vector.push_back( buf );
	}

	/* allocate device L-1 buffer */
	if ( hipMalloc( (void**)&dev_L1_bufs, sizeof(SGCUDANODES) ) != hipSuccess )
	{
		m_helper.CheckRuntimeErrors( "hipMalloc failed", __FILE__, __LINE__ );
		return false;
	}

	/* allocate device L-2 buffers */
	size = GRIDS_X * GRIDS_X * GRIDS_X;
	for ( int i = 0; i < dev_L2_vector_num; i++ )
	{
		SGSTDGRID *buf;
		if ( hipMalloc( (void**)&buf, sizeof(SGSTDGRID) * size ) != hipSuccess)
		{
			m_helper.CheckRuntimeErrors( "hipMalloc failed", __FILE__, __LINE__ );
			return false;
		}
		dev_L2_vector.push_back( buf );
	}

	/* allocate host L-0 buffers */
	size = VOLUME_X * VOLUME_X * VOLUME_X;
	for ( int i = 0; i < NODES_X * NODES_X * NODES_X; i++)
	{
		SGHOSTNODE *buf;
		buf = (SGHOSTNODE*)malloc( sizeof(SGHOSTNODE) );
		if ( buf eqt nullptr )
		{
			printf( "malloc falied\n" );
			return false;
		}
		host_L0_vector.push_back( buf );
	}

	/* allocate visual buffers */
	host_L0_visual = (SGUCHAR*) malloc ( sizeof(SGUCHAR) * size );
	if ( host_L0_visual eqt nullptr )
	{
		printf( "malloc falied\n" );
		return false;
	}

	if ( hipMalloc( (void**)&dev_L0_visual, sizeof(SGUCHAR) * size) != hipSuccess )
	{
		m_helper.CheckRuntimeErrors( "hipMalloc failed", __FILE__, __LINE__ );
		return false;
	}
		
	return true;
};

/* when program existed, release resource */
void FluidSimProc::FreeResource( void )
{
	/* free device L-0 buffers */
	for ( int i = 0; i < dev_L0_vector_num; i++ )
	{
		hipFree( dev_L0_vector[i] );
	}

	/* free device L-1 buffers */
	hipFree( dev_L1_bufs );

	/* free device L-2 buffers */
	for ( int i = 0; i < dev_L2_vector_num; i++ )
	{
		hipFree( dev_L2_vector[i] );
	}

	/* free host L-0 buffers */
	for ( int i = 0; i < host_L0_vector.size(); i++ )
	{
		SAFE_FREE_PTR( host_L0_vector[i] );
	}

	/* free L-0 visual buffers */
	SAFE_FREE_PTR( host_L0_visual );
	hipFree( dev_L0_visual );
};

/* zero the buffers for fluid simulation */
void FluidSimProc::ZeroBuffers( void )
{
	/* zero center node first */
	hostZeroBuffer( dev_center );

	for ( int i = 0; i < NODES_X * NODES_X * NODES_X; i++ )
	{
		if ( hipMemcpy( host_L0_vector[i]->ptrGrids, dev_center, 
			sizeof(SGSTDGRID) * GRIDS_X * GRIDS_X * GRIDS_X, hipMemcpyDeviceToHost ) != hipSuccess )
		{
			m_helper.CheckRuntimeErrors( "hipMemcpy failed", __FILE__, __LINE__ );
			exit( 1 );
		}
	}

	/* update fluid simulation! */
	UploadBuffers();
};

/* choose the node and mark it as actived */
void FluidSimProc::ActiveNode( int i, int j, int k )
{
	int ix = 0;

	if ( i >= 0 and i < NODES_X and j >= 0 and j < NODES_X and k >= 0 and k < NODES_X )
	{
		/* update index */
		nodeIX.x = i;
		nodeIX.y = j;
		nodeIX.z = k;
		ix = cudaIndex3D( nodeIX.x, nodeIX.y, nodeIX.z, NODES_X );
		host_L0_vector[ix]->bActive = true;

		/* print status */
		if ( host_L0_vector[ix]->bActive )			
			printf ( "node no.%d is actived!\n", ix );
		else
			printf ( "node no.%d is deactived!\n", ix );
	}	
};

/* choose the node and mark it as deactived */
void FluidSimProc::DeactiveNode( int i, int j, int k )
{
	int ix = 0;

	if ( i >= 0 and i < NODES_X and j >= 0 and j < NODES_X and k >= 0 and k < NODES_X )
	{
		/* update index */
		nodeIX.x = i;
		nodeIX.y = j;
		nodeIX.z = k;
		ix = cudaIndex3D( nodeIX.x, nodeIX.y, nodeIX.z, NODES_X );
		host_L0_vector[ix]->bActive = false;

		/* print status */
		if ( host_L0_vector[ix]->bActive )			
			printf ( "node no.%d is actived!\n", ix );
		else
			printf ( "node no.%d is deactived!\n", ix );
	}	
};

/* retrieve the density back and load into volumetric data for rendering */
void FluidSimProc::PickVolumetric( FLUIDSPARAM *fluid )
{};

/* copy host data to CUDA device */
void FluidSimProc::UploadBuffers( void )
{};

/* retrieve data back to host */
void FluidSimProc::DownloadBuffers( void )
{};

/* zero data, set the bounds */
void FluidSimProc::InitSimNodes( void )
{};

/* create simulation nodes' topological structure */
void FluidSimProc::BuildOrder( void )
{};