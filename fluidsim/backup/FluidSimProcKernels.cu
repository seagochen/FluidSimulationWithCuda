/**
* <Author>        Orlando Chen
* <Email>         seagochen@gmail.com
* <First Time>    Nov 15, 2013
* <Last Time>     Feb 17, 2014
* <File Name>     FluidSimProc.cu
*/

#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <>
#include "FluidSimProc.h"
#include "CUDAInterfaces.h"

using namespace sge;

/* Ĭ�ϵĹ��캯������������ģ������Ҫ�Ŀռ䣬�Լ���ʼ����ز��� */
FluidSimProc::FluidSimProc( FLUIDSPARAM *fluid )
{
	/* initialize the parameters of fluid simulation */
	InitParams( fluid );

	/* allocate the space for fluid simulation */
	if ( !AllocateResource() )
	{
		FreeResource();
		printf(" malloc buffers for fluid simulation failed! \n");
		exit(1);
	}
	else
		printf( "allocate resource success!\n" );

	/* initialize the nodes */
	InitSimNodes();

	/* building structure order */
	BuildOrder();

	/* select and active a node for fluid simulation */
	ActiveNode( 1, 1, 0 );

	/* zero all buffers */
	ZeroBuffers();
	
	/* finally, print the state message and zero the data */
	printf( "fluid simulation ready...\n" );
};

/* ��ʼ������ģ�����ز��� */
void FluidSimProc::InitParams( FLUIDSPARAM *fluid )
{
	/* initilize the status of FPS counter */
	fluid->fps.dwCurrentTime    = 0;
	fluid->fps.dwElapsedTime    = 0;
	fluid->fps.dwFrames         = 0;
	fluid->fps.dwLastUpdateTime = 0;
	fluid->fps.uFPS             = 0;
};

/* fluid simulation processing function */
void FluidSimProc::FluidSimSolver( FLUIDSPARAM *fluid )
{
	if ( fluid->run )
	{
		for ( int i = 0; i < NODES_X; i++ )
		{
			for ( int j = 0; j < NODES_X; j++ )
			{
				for ( int k = 0; k < NODES_X; k++ )
				{
					// TODO: �ڼ�������ǰ����һֱ���ü򵥶����˵���ѯ��������������ڵ��״̬��
					// �ڵ�ǰ������£������нڵ�Ĭ��Ϊ����״̬������������ֱ�۵Ĳ��Ը����ڵ����ݴ��������
					/* ���㿪ʼ */
					SelectNode( i, j, k );
					UploadBuffers();

//					hostAddSource( dev_center, SG_VELOCITY_U_FIELD );
//					hostAddSource( dev_center, SG_VELOCITY_V_FIELD );
//					hostAddSource( dev_center, SG_VELOCITY_W_FIELD );
//					hostAddSource( dev_center, SG_DENSITY_FIELD );


					/* ������� */
					DownloadBuffers();
					hostPickData( dev_L0_visual, dev_center, &nodeIX ); 
				}
			}
		}
	}
};

/* allocate resource */
bool FluidSimProc::AllocateResource( void )
{
	/* allocate device L-0 buffers */
	for ( int i = 0; i < dev_L0_vector_num; i++ )
	{
		SGSIMPLENODES *buf;
		if ( m_helper.CreateCUDABuffers( &buf ) not_eq SG_RUNTIME_OK )
		{
			m_helper.CheckRuntimeErrors( "hipMalloc failed", __FILE__, __LINE__ );
			return false;
		}
		dev_L0_vector.push_back( buf );
	}

	/* allocate device L-1 buffer */
	if ( m_helper.CreateCUDABuffers( &dev_L1_bufs ) not_eq SG_RUNTIME_OK )
	{
		m_helper.CheckRuntimeErrors( "hipMalloc failed", __FILE__, __LINE__ );
		return false;
	}

	/* allocate device L-2 buffers */
	for ( int i = 0; i < dev_L2_vector_num; i++ )
	{
		SGSTDGRID *buf;
		if ( m_helper.CreateCUDABuffers( &buf ) not_eq SG_RUNTIME_OK )
		{
			m_helper.CheckRuntimeErrors( "hipMalloc failed", __FILE__, __LINE__ );
			return false;
		}
		dev_L2_vector.push_back( buf );
	}

	/* allocate host L-0 buffers */
	for ( int i = 0; i < NODES_X * NODES_X * NODES_X; i++)
	{
		SGHOSTNODE *buf;
		if ( m_helper.CreateHostBuffers( buf ) not_eq SG_RUNTIME_OK )
		{
			printf( "malloc falied\n" );
			return false;
		}
		host_L0_vector.push_back( buf );
	}

	/* allocate visual buffers */	
	if ( m_helper.CreateVolumetricBuffers( &host_L0_visual, &dev_L0_visual ) not_eq SG_RUNTIME_OK )
	{
		m_helper.CheckRuntimeErrors( "hipMalloc failed", __FILE__, __LINE__ );
		return false;
	}

	/* allocate temporary buffers */
	if ( m_helper.CreateDoubleBuffers( TPBUFFER_X, 1, &dev_L0_temps ) not_eq SG_RUNTIME_OK )
	{
		printf( "malloc falied\n" );
		return false;
	}
		
	return true;
};

/* when program existed, release resource */
void FluidSimProc::FreeResource( void )
{
	/* free device L-0 buffers */
	for ( int i = 0; i < dev_L0_vector_num; i++ )
	{
		hipFree( dev_L0_vector[i] );
	}

	/* free device L-1 buffers */
	hipFree( dev_L1_bufs );

	/* free device L-2 buffers */
	for ( int i = 0; i < dev_L2_vector_num; i++ )
	{
		hipFree( dev_L2_vector[i] );
	}

	/* free host L-0 buffers */
	for ( int i = 0; i < host_L0_vector.size(); i++ )
	{
		SAFE_FREE_PTR( host_L0_vector[i] );
	}

	/* free L-0 visual buffers */
	SAFE_FREE_PTR( host_L0_visual );
	hipFree( dev_L0_visual );
	hipFree( dev_L0_temps );
};

/* zero the buffers for fluid simulation */
void FluidSimProc::ZeroBuffers( void )
{
	/* zero center node first */
	hostZeroBuffer( dev_center );

	size_t size = GRIDS_X * GRIDS_X * GRIDS_X;

	for ( int i = 0; i < NODES_X * NODES_X * NODES_X; i++ )
	{
		if ( hipMemcpy( host_L0_vector[i]->ptrGrids, dev_center, 
			sizeof(SGSTDGRID) * size, hipMemcpyDeviceToHost ) != hipSuccess )
		{
			m_helper.CheckRuntimeErrors( "hipMemcpy failed", __FILE__, __LINE__ );
			exit( 1 );
		}
	}
};

/* choose the node and mark it as actived */
void FluidSimProc::ActiveNode( int i, int j, int k )
{
	int ix = 0;

	if ( i >= 0 and i < NODES_X and j >= 0 and j < NODES_X and k >= 0 and k < NODES_X )
	{
		ix = cudaIndex3D( i, j, k, NODES_X );
		host_L0_vector[ix]->bActive = true;

		/* print status */
		if ( host_L0_vector[ix]->bActive )			
			printf ( "node no.%d is actived!\n", ix );
		else
			printf ( "node no.%d is deactived!\n", ix );
	}	
};

/* choose the node and mark it as deactived */
void FluidSimProc::DeactiveNode( int i, int j, int k )
{
	int ix = 0;

	if ( i >= 0 and i < NODES_X and j >= 0 and j < NODES_X and k >= 0 and k < NODES_X )
	{
		ix = cudaIndex3D( i, j, k, NODES_X );
		host_L0_vector[ix]->bActive = false;

		/* print status */
		if ( host_L0_vector[ix]->bActive )			
			printf ( "node no.%d is actived!\n", ix );
		else
			printf ( "node no.%d is deactived!\n", ix );
	}	
};


void FluidSimProc::SelectNode( int i, int j, int k )
{
	if ( i >= 0 and i < NODES_X and j >= 0 and j < NODES_X and k >= 0 and k < NODES_X )
	{
		nodeIX.x = i;
		nodeIX.y = j;
		nodeIX.z = k;
	}
};

/* zero data, set the bounds */
void FluidSimProc::InitSimNodes( void )
{
	int IX = 0;

	for ( int i = 0; i < GRIDS_X; i++ )
	{
		for ( int j = 0; j < GRIDS_X; j++ )
		{
			for ( int k = 0; k < GRIDS_X; k++ )
			{
				for ( IX = 0; IX < host_L0_vector.size(); IX++ )
				{
					host_L0_vector[IX]->ptrGrids[cudaIndex3D(i,j,k,NODES_X)].obstacle = SG_BLANK;
				}
			}
		}
	}

	IX = cudaIndex3D(1,1,0,NODES_X);

	int half = GRIDS_X / 2;
//	host_L0_vector[IX]->ptrGrids[cudaIndex3D(half,half,half,GRIDS_X)].obstacle = SG_SOURCE;
//	host_L0_vector[IX]->ptrGrids[cudaIndex3D(half-1,half,0,NODES_X)].obstacle = SG_SOURCE;
//	host_L0_vector[IX]->ptrGrids[cudaIndex3D(half+1,half,0,NODES_X)].obstacle = SG_SOURCE;
//	host_L0_vector[IX]->ptrGrids[cudaIndex3D(half,half-1,0,NODES_X)].obstacle = SG_SOURCE;
//	host_L0_vector[IX]->ptrGrids[cudaIndex3D(half,half+1,0,NODES_X)].obstacle = SG_SOURCE;
};

/* create simulation nodes' topological structure */
void FluidSimProc::BuildOrder( void )
{
	printf( "structure:\n" );
	for ( int i = 0; i < NODES_X; i++ )
	{
		for ( int j = 0; j < NODES_X; j++ )
		{
			for ( int k = 0; k < NODES_X; k++ )
			{
				int index = cudaIndex3D( i, j, k, NODES_X );

				if ( index >= host_L0_vector.size() or index < 0 )
				{
					printf ( "index out of range! %s, line: %d \n", __FILE__, __LINE__ );
					exit ( 1 );
				}

				/* left */
				if ( i >= 1 )
					host_L0_vector[index]->ptrLeft = host_L0_vector[index-1];
				/* right */
				if ( i <= NODES_X - 2 )
					host_L0_vector[index]->ptrRight = host_L0_vector[index+1];
				/* down */
				if ( j >= 1 )
					host_L0_vector[index]->ptrDown = host_L0_vector[index-NODES_X];
				/* up */
				if ( j <= NODES_X - 2 )
					host_L0_vector[index]->ptrUp = host_L0_vector[index+NODES_X];
				/* back */
				if ( k >= 1 )
					host_L0_vector[index]->ptrBack = host_L0_vector[index-NODES_X*NODES_X];
				/* front */
				if ( k <= NODES_X - 2 )
					host_L0_vector[index]->ptrFront = host_L0_vector[index+NODES_X*NODES_X];

				host_L0_vector[index]->n3Pos.x = i;
				host_L0_vector[index]->n3Pos.y = j;
				host_L0_vector[index]->n3Pos.z = k;

				printf ( "no: %d | offset: %d%d%d | L: %d | R: %d | U: %d | D: %d | F: %d | B: %d \n",
					index,
					host_L0_vector[index]->n3Pos.x, 
					host_L0_vector[index]->n3Pos.y, 
					host_L0_vector[index]->n3Pos.z,
					host_L0_vector[index]->ptrLeft != NULL,
					host_L0_vector[index]->ptrRight != NULL,
					host_L0_vector[index]->ptrUp != NULL,
					host_L0_vector[index]->ptrDown != NULL,
					host_L0_vector[index]->ptrFront != NULL,
					host_L0_vector[index]->ptrBack != NULL );
			}
		}
	}

	printf( "-----------------------------------------------\n" );
};

/* copy host data to CUDA device */
void FluidSimProc::UploadBuffers( void )
{
	int index = cudaIndex3D( nodeIX.x, nodeIX.y, nodeIX.z, NODES_X );

	/* zero all buffers first */
	hostZeroBuffer( dev_center );
	hostZeroBuffer( dev_left );
	hostZeroBuffer( dev_right );
	hostZeroBuffer( dev_up );
	hostZeroBuffer( dev_down );
	hostZeroBuffer( dev_front );
	hostZeroBuffer( dev_back );
	//printf( "buffers cleared!\n" );

	SGHOSTNODE *ptr = host_L0_vector[index];
	size_t size = sizeof( SGSTDGRID ) * GRIDS_X * GRIDS_X * GRIDS_X;

	if ( ptr->ptrLeft not_eq nullptr and 
		hipMemcpy( dev_left, ptr->ptrLeft->ptrGrids, size, hipMemcpyHostToDevice ) != hipSuccess )
	{
		m_helper.CheckRuntimeErrors( "hipMemcpy failed", __FILE__, __LINE__ );
		FreeResource();
		exit(1);
	}
	if ( ptr->ptrRight not_eq nullptr and 
		hipMemcpy( dev_right, ptr->ptrRight->ptrGrids, size, hipMemcpyHostToDevice ) != hipSuccess )
	{
		m_helper.CheckRuntimeErrors( "hipMemcpy failed", __FILE__, __LINE__ );
		FreeResource();
		exit(1);
	}
	if ( ptr->ptrUp not_eq nullptr and 
		hipMemcpy( dev_up, ptr->ptrUp->ptrGrids, size, hipMemcpyHostToDevice ) != hipSuccess )
	{
		m_helper.CheckRuntimeErrors( "hipMemcpy failed", __FILE__, __LINE__ );
		FreeResource();
		exit(1);
	}
	if ( ptr->ptrDown not_eq nullptr and 
		hipMemcpy( dev_down, ptr->ptrDown->ptrGrids, size, hipMemcpyHostToDevice ) != hipSuccess )
	{
		m_helper.CheckRuntimeErrors( "hipMemcpy failed", __FILE__, __LINE__ );
		FreeResource();
		exit(1);
	}
	if ( ptr->ptrFront not_eq nullptr and 
		hipMemcpy( dev_front, ptr->ptrFront->ptrGrids, size, hipMemcpyHostToDevice ) != hipSuccess )
	{
		m_helper.CheckRuntimeErrors( "hipMemcpy failed", __FILE__, __LINE__ );
		FreeResource();
		exit(1);
	}
	if ( ptr->ptrBack not_eq nullptr and 
		hipMemcpy( dev_back, ptr->ptrBack->ptrGrids, size, hipMemcpyHostToDevice ) != hipSuccess )
	{
		m_helper.CheckRuntimeErrors( "hipMemcpy failed", __FILE__, __LINE__ );
		FreeResource();
		exit(1);
	}
	if ( hipMemcpy( dev_center, ptr->ptrGrids, size, hipMemcpyHostToDevice ) != hipSuccess )
	{
		m_helper.CheckRuntimeErrors( "hipMemcpy failed", __FILE__, __LINE__ );
		FreeResource();
		exit(1);
	}
	//printf( "data has been uploaded to CUDA device!\n" );
};

/* retrieve data back to host */
void FluidSimProc::DownloadBuffers( void )
{
	int index = cudaIndex3D( nodeIX.x, nodeIX.y, nodeIX.z, NODES_X );

	SGHOSTNODE *ptr = host_L0_vector[index];
	size_t size = sizeof( SGSTDGRID ) * GRIDS_X * GRIDS_X * GRIDS_X;

	if ( ptr->ptrLeft not_eq nullptr and 
		hipMemcpy( ptr->ptrLeft->ptrGrids, dev_left, size, hipMemcpyDeviceToHost ) != hipSuccess )
	{
		m_helper.CheckRuntimeErrors( "hipMemcpy failed", __FILE__, __LINE__ );
		FreeResource();
		exit(1);
	}
	if ( ptr->ptrRight not_eq nullptr and 
		hipMemcpy( ptr->ptrRight->ptrGrids, dev_right, size, hipMemcpyDeviceToHost ) != hipSuccess )
	{
		m_helper.CheckRuntimeErrors( "hipMemcpy failed", __FILE__, __LINE__ );
		FreeResource();
		exit(1);
	}
	if ( ptr->ptrUp not_eq nullptr and 
		hipMemcpy( ptr->ptrUp->ptrGrids, dev_up, size, hipMemcpyDeviceToHost ) != hipSuccess )
	{
		m_helper.CheckRuntimeErrors( "hipMemcpy failed", __FILE__, __LINE__ );
		FreeResource();
		exit(1);
	}
	if ( ptr->ptrDown not_eq nullptr and 
		hipMemcpy( ptr->ptrDown->ptrGrids, dev_down, size, hipMemcpyDeviceToHost ) != hipSuccess )
	{
		m_helper.CheckRuntimeErrors( "hipMemcpy failed", __FILE__, __LINE__ );
		FreeResource();
		exit(1);
	}
	if ( ptr->ptrFront not_eq nullptr and 
		hipMemcpy( ptr->ptrFront->ptrGrids, dev_front, size, hipMemcpyDeviceToHost ) != hipSuccess )
	{
		m_helper.CheckRuntimeErrors( "hipMemcpy failed", __FILE__, __LINE__ );
		FreeResource();
		exit(1);
	}
	if ( ptr->ptrBack not_eq nullptr and 
		hipMemcpy( ptr->ptrBack->ptrGrids, dev_back, size, hipMemcpyDeviceToHost ) != hipSuccess )
	{
		m_helper.CheckRuntimeErrors( "hipMemcpy failed", __FILE__, __LINE__ );
		FreeResource();
		exit(1);
	}
	if ( hipMemcpy( ptr->ptrGrids, dev_center, size, hipMemcpyDeviceToHost ) != hipSuccess )
	{
		m_helper.CheckRuntimeErrors( "hipMemcpy failed", __FILE__, __LINE__ );
		FreeResource();
		exit(1);
	}
	//printf( "data has been downloaded to host!\n" );
};

/* retrieve the density back and load into volumetric data for rendering */
void FluidSimProc::PickVolumetric( FLUIDSPARAM *fluid )
{
	if ( hipMemcpy( host_L0_visual, dev_L0_visual,
		sizeof(SGUCHAR) * VOLUME_X * VOLUME_X * VOLUME_X, hipMemcpyDeviceToHost ) != hipSuccess )
	{
		m_helper.CheckRuntimeErrors( "hipMemcpy failed", __FILE__, __LINE__ );
		FreeResource();
		exit(1);
	}

	fluid->volume.ptrData = host_L0_visual;
};