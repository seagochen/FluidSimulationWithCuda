#include "hip/hip_runtime.h"
﻿/**
*
* Copyright (C) <2013> <Orlando Chen>
* Permission is hereby granted, free of charge, to any person obtaining a copy of this software and
* associated documentation files (the "Software"), to deal in the Software without restriction, 
* including without limitation the rights to use, copy, modify, merge, publish, distribute, sublicense, 
* and/or sell copies of the Software, and to permit persons to whom the Software is furnished to do so, 
* subject to the following conditions:
*
* The above copyright notice and this permission notice shall be included in all copies or substantial
* portions of the Software.
*
* THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR IMPLIED, INCLUDING BUT 
* NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.
* IN NO EVENT SHALL THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER LIABILITY, 
* WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM, OUT OF OR IN CONNECTION WITH THE
* SOFTWARE OR THE USE OR OTHER DEALINGS IN THE SOFTWARE.
 */

/**
* <Author>      Orlando Chen
* <First>       Dec 12, 2013
* <Last>		Dec 23, 2013
* <File>        kernel.cu
*/

#include <iostream>
#include <cstdio>
#include <fstream>
#include <cstdlib>

#include <GL\glew.h>
#include <GL\freeglut.h>
#include <SGE\SGUtils.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include "fluidsim.h"
#include "bufferOp.h"
#include "myMath.h"

using namespace sge;
using namespace std;


__global__ void kernelAddSource ( double *dens, double *vel_u, double *vel_v, double *vel_w )
{
	GetIndex();

	if ( dens != NULL && j < 10 )
		dens [ Index(i,j,k) ] = INDENSITY;

	if ( vel_v != NULL && j < 10 )
		vel_v [ Index(i,j,k) ] = 1.f;
};

__global__ void kernelGridAdvection ( double *grid_out, double const *grid_in, double const *u_in, double const *v_in, double const *w_in )
{
	GetIndex();

	double u = i - u_in [ Index(i,j,k) ] * DELTA_TIME;
	double v = j - v_in [ Index(i,j,k) ] * DELTA_TIME;
	double w = k - w_in [ Index(i,j,k) ] * DELTA_TIME;

	grid_out [ Index(i,j,k) ] = trilinear ( grid_in, u, v, w );
};

void FluidSimProc::VelocitySolver ( void )
{
	cudaDeviceDim3D ();

	kernelAddSource <<<gridDim, blockDim>>> ( NULL, dev_u, dev_v, dev_w );
	if ( hipDeviceSynchronize() != hipSuccess )  goto Error;

	kernelGridAdvection <<<gridDim, blockDim>>> ( dev_den0, dev_den, dev_u, dev_v, dev_w );
	if ( hipDeviceSynchronize() != hipSuccess )  goto Error;

	kernelSwapBuffer <<<gridDim, blockDim>>> ( dev_den0, dev_den );
	if ( hipDeviceSynchronize() != hipSuccess )  goto Error;

	goto Success;

Error:
	cudaCheckErrors ("hipDeviceSynchronize failed", __FILE__, __LINE__);
	FreeResourcePtrs ();
	exit (1);

Success:
	;
};

void FluidSimProc::DensitySolver ( void )
{
	cudaDeviceDim3D ();

	kernelAddSource <<<gridDim, blockDim>>> ( dev_den0, NULL, NULL, NULL );
	if ( hipDeviceSynchronize() != hipSuccess )  goto Error;

	kernelSwapBuffer <<<gridDim, blockDim>>> ( dev_den0, dev_den );
	if ( hipDeviceSynchronize() != hipSuccess )  goto Error;

	goto Success;

Error:
	cudaCheckErrors ("hipDeviceSynchronize failed", __FILE__, __LINE__);
	FreeResourcePtrs ();
	exit (1);

Success:
	;
};

void FluidSimProc::FluidSimSolver ( fluidsim *fluid )
{
	if ( !fluid->drawing.bContinue ) return ;

	// For fluid simulation, copy the data to device
	CopyDataToDevice();

	// Fluid process
	VelocitySolver ();
	DensitySolver ();
	PickData ( fluid );

	// Synchronize the device
	if ( hipDeviceSynchronize() != hipSuccess ) goto Error;

	// After simulation process, retrieve data back to host, in order to 
	// avoid data flipping
	CopyDataToHost();

	goto Success;

Error:
	cudaCheckErrors ("hipDeviceSynchronize failed", __FILE__, __LINE__);
	FreeResourcePtrs ();
	exit (1);

Success:
	fluid->volume.ptrData = host_data;
};

void FluidSimProc::PickData ( fluidsim *fluid )
{
	cudaDeviceDim3D ();
	kernelPickData  <<<gridDim, blockDim>>> ( dev_data, dev_den );

	if ( hipMemcpy (host_data, dev_data, 
		sizeof(unsigned char) * (fluid->volume.nVolDepth * fluid->volume.nVolHeight * fluid->volume.nVolWidth), 
		hipMemcpyDeviceToHost ) != hipSuccess )
	{
		cudaCheckErrors ("hipMemcpy failed", __FILE__, __LINE__);
		FreeResourcePtrs ();
		exit (1);
	}
};