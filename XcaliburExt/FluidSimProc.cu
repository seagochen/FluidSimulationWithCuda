/**
* <Author>        Orlando Chen
* <Email>         seagochen@gmail.com
* <First Time>    Dec 15, 2013
* <Last Time>     Mar 26, 2014
* <File Name>     FluidSimProc.cu
*/

#include <iostream>
#include <utility>
#include <ctime>
#include <hip/hip_runtime_api.h>
#include <>

#include "FluidSimProc.h"
#include "MacroDefinition.h"
#include "Kernels.h"

using namespace sge;
using std::cout;
using std::endl;

FluidSimProc::FluidSimProc( FLUIDSPARAM *fluid )
{
	/* choose which GPU to run on, change this on a multi-GPU system. */
	if ( hipSetDevice ( 0 ) != hipSuccess )
	{
		m_scHelper.GetCUDALastError( "cannot set device", __FILE__, __LINE__ );
		exit(1);
	}

	/* initialize FPS */
	InitParams( fluid );

	/* allocate resources */
	AllocateResource();
	
	/* clear buffer */
	ClearBuffers();

	/* initialize boundary */
	InitBound();

	/* finally, print message */
	printf( "fluid simulation ready...\n" );
};

void FluidSimProc::InitParams( FLUIDSPARAM *fluid )
{
	fluid->fps.dwCurrentTime = 0;
	fluid->fps.dwElapsedTime = 0;
	fluid->fps.dwFrames = 0;
	fluid->fps.dwLastUpdateTime = 0;
	fluid->fps.uFPS = 0;

	srand(time(NULL));

	m_szTitle = APP_TITLE;
};

void FluidSimProc::AllocateResource( void )
{
	size_t size = GRIDS_X * GRIDS_Y * GRIDS_Z * sizeof(double);
	size_t node = NODES_X * NODES_Y * NODES_Z;
	
	if ( not m_scHelper.CreateCompNodesForHost( &m_vectHostDens, size, node ) ) goto Error;
	if ( not m_scHelper.CreateCompNodesForHost( &m_vectHostVelU, size, node ) ) goto Error;
	if ( not m_scHelper.CreateCompNodesForHost( &m_vectHostVelV, size, node ) ) goto Error;
	if ( not m_scHelper.CreateCompNodesForHost( &m_vectHostVelW, size, node ) ) goto Error;
	if ( not m_scHelper.CreateCompNodesForHost( &m_vectHostObst, size, node ) ) goto Error;

	if ( not m_scHelper.CreateCompNodesForDevice( &m_vectGPUDens, size, node ) ) goto Error;
	if ( not m_scHelper.CreateCompNodesForDevice( &m_vectGPUVelU, size, node ) ) goto Error;
	if ( not m_scHelper.CreateCompNodesForDevice( &m_vectGPUVelV, size, node ) ) goto Error;
	if ( not m_scHelper.CreateCompNodesForDevice( &m_vectGPUVelW, size, node ) ) goto Error;
	if ( not m_scHelper.CreateCompNodesForDevice( &m_vectGPUObst, size, node ) ) goto Error;

	if ( not m_scHelper.CreateCompNodesForDevice( &m_vectNewDens, size, node ) ) goto Error;
	if ( not m_scHelper.CreateCompNodesForDevice( &m_vectNewVelU, size, node ) ) goto Error;
	if ( not m_scHelper.CreateCompNodesForDevice( &m_vectNewVelV, size, node ) ) goto Error;
	if ( not m_scHelper.CreateCompNodesForDevice( &m_vectNewVelW, size, node ) ) goto Error;

	size = GRIDS_X * GRIDS_Y * GRIDS_Z * sizeof(double);
	node = TEMPND_S;

	if ( not m_scHelper.CreateCompNodesForDevice( &m_vectGlobalBufs, size, node ) ) goto Error;

	size = BULLET_X * BULLET_Y * BULLET_Z * sizeof(double);
	node = BULLET_S;

	if ( not m_scHelper.CreateCompNodesForDevice( &m_vectCompBufs, size, node ) ) goto Error;

	goto Success;

Error:
	cout << "create computation nodes failed" << endl;
		FreeResource();
		exit(1);

Success:
	
	m_scHelper.CreateDeviceBuffers( VOLUME_X * VOLUME_Y * VOLUME_Z * sizeof(SGUCHAR), 1, &m_ptrDeviceVisual );
	m_scHelper.CreateHostBuffers( VOLUME_X * VOLUME_Y * VOLUME_Z * sizeof(SGUCHAR), 1, &m_ptrHostVisual );

	if ( m_scHelper.GetCUDALastError( "call member function AllocateResource failed", __FILE__, __LINE__ ) )
	{
		FreeResource();
		exit(1);
	}
};

void FluidSimProc::FreeResource( void )
{
	for ( int i = 0; i < NODES_X * NODES_Y * NODES_Z; i++ )
	{
		m_scHelper.FreeDeviceBuffers( 9,
			&m_vectGPUDens[i], &m_vectNewDens[i],
			&m_vectGPUVelU[i], &m_vectNewVelU[i],
			&m_vectGPUVelV[i], &m_vectNewVelV[i],
			&m_vectGPUVelW[i], &m_vectNewVelW[i],
			&m_vectGPUObst[i] );

		m_scHelper.FreeHostBuffers( 5,
			&m_vectHostDens[i], &m_vectHostVelU[i],
			&m_vectHostVelV[i], &m_vectHostVelW[i], &m_vectHostObst[i] );
	}


	for ( int i = 0; i < m_vectCompBufs.size(); i++ )
		m_scHelper.FreeDeviceBuffers( 1, &m_vectCompBufs[i] );

	for ( int i = 0; i < m_vectGlobalBufs.size(); i++ )
		m_scHelper.FreeDeviceBuffers( 1, &m_vectGlobalBufs[i] );


	m_scHelper.FreeDeviceBuffers( 1, &m_ptrDeviceVisual );
	m_scHelper.FreeHostBuffers( 1, &m_ptrHostVisual );


	if ( m_scHelper.GetCUDALastError( "call member function FreeResource failed",
		__FILE__, __LINE__ ) ) exit(1);
};

void FluidSimProc::RefreshStatus( FLUIDSPARAM *fluid )
{
	/* waiting for all kernels end */
	if ( hipDeviceSynchronize() not_eq hipSuccess )
	{
		printf( "hipDeviceSynchronize failed\n" );
		FreeResource();
		exit( 1 );
	}

	/* counting FPS */
	fluid->fps.dwFrames ++;
	fluid->fps.dwCurrentTime = GetTickCount();
	fluid->fps.dwElapsedTime = fluid->fps.dwCurrentTime - fluid->fps.dwLastUpdateTime;

	/* 1 second */
	if ( fluid->fps.dwElapsedTime >= 1000 )
	{
		fluid->fps.uFPS     = fluid->fps.dwFrames * 1000 / fluid->fps.dwElapsedTime;
		fluid->fps.dwFrames = 0;
		fluid->fps.dwLastUpdateTime = fluid->fps.dwCurrentTime;
	}

	/* updating image */
	if ( hipMemcpy( m_ptrHostVisual, m_ptrDeviceVisual, 
		VOLUME_X * VOLUME_Y * VOLUME_Z * sizeof(SGUCHAR), hipMemcpyDeviceToHost ) not_eq hipSuccess )
	{
		m_scHelper.GetCUDALastError( "host function: hipMemcpy failed", __FILE__, __LINE__ );
		FreeResource();
		exit( 1 );
	}
	fluid->volume.ptrData = m_ptrHostVisual;

	if ( m_scHelper.GetCUDALastError( "call member function RefreshStatus failed", __FILE__, __LINE__ ) )
	{
		FreeResource();
		exit(1);
	}
};

void FluidSimProc::ClearBuffers( void )
{
	Dim3ParamDim();
	
	for ( int i = 0; i < m_vectCompBufs.size(); i++ ) _zero( m_vectCompBufs[i] );
		 	 

	for ( int i = 0; i < NODES_X * NODES_Y * NODES_Z; i++ )
	{
		_zero( m_vectGPUDens[i] ); _zero( m_vectNewDens[i] );
		_zero( m_vectGPUVelU[i] ); _zero( m_vectNewDens[i] );
		_zero( m_vectGPUVelV[i] ); _zero( m_vectNewDens[i] );
		_zero( m_vectGPUVelW[i] ); _zero( m_vectNewDens[i] );
		_zero( m_vectGPUObst[i] ); _zero( m_vectNewDens[i] );
	}

	if ( m_scHelper.GetCUDALastError( "host function failed: ZeroBuffers", __FILE__, __LINE__ ) )
	{
		FreeResource();
		exit( 1 );
	}
};

void FluidSimProc::InitBound( void )
{
	cint halfx = GRIDS_X / 2;
	cint halfz = GRIDS_Z / 2;

	for ( int k = 0; k < GRIDS_Z; k++ )
	{
		for ( int j = 0; j < GRIDS_Y; j++ )
		{
			for ( int i = 0; i < GRIDS_X; i++ )
			{
				for ( int n = 0; n < NODES_X * NODES_Y * NODES_Z; n++ )
					m_vectHostObst[n][ix(i,j,k,GRIDS_X,GRIDS_Y)] = MACRO_BOUNDARY_BLANK;
			}
		}
	}

	for ( int k = 0; k < GRIDS_Z; k++ )
	{
		for ( int j = 0; j < GRIDS_Y; j++ )
		{
			for ( int i = 0; i < GRIDS_X; i++ )
			{
				if ( j < 3 and 
					i >= halfx - 2 and i < halfx + 2 and 
					k >= halfz - 2 and k < halfz + 2 )
					m_vectHostObst[0][ix(i,j,k,GRIDS_X,GRIDS_Y)] = MACRO_BOUNDARY_SOURCE;
			}
		}
	}

	for ( int n = 0; n < NODES_X * NODES_Y * NODES_Z; n++ )
	{
		hipMemcpy( m_vectGPUObst[n], m_vectHostObst[n], 
			sizeof(double) * GRIDS_X * GRIDS_Y * GRIDS_Z, hipMemcpyHostToDevice );
	}

	if ( m_scHelper.GetCUDALastError( "call member function InitBound failed", __FILE__, __LINE__ ) )
	{
		FreeResource();
		exit(1);
	}
}

void FluidSimProc::FluidSimSolver( FLUIDSPARAM *fluid )
{
	if ( !fluid->run ) return;

	m_scHelper.DeviceParamDim( &gridDim, &blockDim, THREADS_S, 22, 22, BULLET_X, BULLET_Y, BULLET_Z );
	for ( int i = 0; i < m_vectCompBufs.size(); i++ )
	{
		kernelZeroBuffers __device_func__ ( m_vectCompBufs[i], BULLET_X, BULLET_Y, BULLET_Z );
	}
	
	m_scHelper.DeviceParamDim( &gridDim, &blockDim, THREADS_S, TILE_X, TILE_Y, GRIDS_X, GRIDS_Y, GRIDS_Z );
	kernelLoadBullet __device_func__ ( dev_den, m_vectGPUDens[0], BULLET_X, BULLET_Y, BULLET_Z, GRIDS_X, GRIDS_Y, GRIDS_Z );
	kernelLoadBullet __device_func__ ( dev_u, m_vectGPUVelU[0], BULLET_X, BULLET_Y, BULLET_Z, GRIDS_X, GRIDS_Y, GRIDS_Z );
	kernelLoadBullet __device_func__ ( dev_v, m_vectGPUVelV[0], BULLET_X, BULLET_Y, BULLET_Z, GRIDS_X, GRIDS_Y, GRIDS_Z );
	kernelLoadBullet __device_func__ ( dev_w, m_vectGPUVelW[0], BULLET_X, BULLET_Y, BULLET_Z, GRIDS_X, GRIDS_Y, GRIDS_Z );
	kernelLoadBullet __device_func__ ( dev_obs, m_vectGPUObst[0], BULLET_X, BULLET_Y, BULLET_Z, GRIDS_X, GRIDS_Y, GRIDS_Z );

	SolveNavierStokesEquation( DELTATIME, true );

	kernelExitBullet __device_func__ ( m_vectGPUDens[0], dev_den, GRIDS_X, GRIDS_Y, GRIDS_Z, BULLET_X, BULLET_Y, BULLET_Z );
	kernelExitBullet __device_func__ ( m_vectGPUVelU[0], dev_u, GRIDS_X, GRIDS_Y, GRIDS_Z, BULLET_X, BULLET_Y, BULLET_Z );
	kernelExitBullet __device_func__ ( m_vectGPUVelV[0], dev_v, GRIDS_X, GRIDS_Y, GRIDS_Z, BULLET_X, BULLET_Y, BULLET_Z );
	kernelExitBullet __device_func__ ( m_vectGPUVelW[0], dev_w, GRIDS_X, GRIDS_Y, GRIDS_Z, BULLET_X, BULLET_Y, BULLET_Z );

	kernelPickData __device_func__ ( m_ptrDeviceVisual, m_vectGPUDens[0], 0, 0, 0, GRIDS_X, GRIDS_Y, GRIDS_Z );

	if ( m_scHelper.GetCUDALastError( "call member function FluidSimSolver failed", __FILE__, __LINE__ ) )
	{
		FreeResource();
		exit(1);
	}

	RefreshStatus( fluid );
};