/**
* <Author>        Orlando Chen
* <Email>         seagochen@gmail.com
* <First Time>    Dec 15, 2013
* <Last Time>     Mar 04, 2014
* <File Name>     FluidSimProcKernels.cu
*/

#include <iostream>
#include <utility>
#include "MacroDefinition.h"
#include "FrameworkDynamic.h"
#include "Kernels.h"

using namespace sge;

FluidSimProc::FluidSimProc ( FLUIDSPARAM *fluid )
{
	/* initialize FPS */
	InitParams( fluid );

	/* allocate resources */
	if ( !AllocateResource ( fluid ) ) { FreeResource (); exit (1); }

	/* build order */
	CreateTopology();
	
	/* clear buffer */
	ZeroBuffers();

	/* set boundary */
	InitBoundary();

	/* finally, print message */
	printf( "fluid simulation ready...\n" );
};

ptrStr FluidSimProc::GetTitleBar( void )
{
	return &m_sz_title; 
};

void FluidSimProc::InitParams( FLUIDSPARAM *fluid )
{
	fluid->fps.dwCurrentTime = 0;
	fluid->fps.dwElapsedTime = 0;
	fluid->fps.dwFrames = 0;
	fluid->fps.dwLastUpdateTime = 0;
	fluid->fps.uFPS = 0;

	m_node_size = GRIDS_X * GRIDS_X * GRIDS_X * sizeof(double);
	m_volm_size = VOLUME_X * VOLUME_X * VOLUME_X * sizeof(SGUCHAR);

	increase_times = decrease_times = 0;

	m_sz_title = "Excalibur OTL 2.10.00, large-scale. ------------ FPS: %d ";
};

void FluidSimProc::CreateTopology( void )
{
	for ( int k = 0; k < NODES_X; k++ )
	{
		for ( int j = 0; j < NODES_X; j++ )
		{
			for ( int i = 0; i < NODES_X; i++ )
			{
				/* left */
				if ( i >= 1 )
					gpu_node[cudaIndex3D( i, j, k, NODES_X )]->ptrLeft  = gpu_node[cudaIndex3D( i-1, j, k, NODES_X )];
				/* right */
				if ( i <= NODES_X - 2 )
					gpu_node[cudaIndex3D( i, j, k, NODES_X )]->ptrRight = gpu_node[cudaIndex3D( i+1, j, k, NODES_X )];
				/* down */
				if ( j >= 1 )
					gpu_node[cudaIndex3D( i, j, k, NODES_X )]->ptrDown  = gpu_node[cudaIndex3D( i, j-1, k, NODES_X )];
				/* up */
				if ( j <= NODES_X - 2 )
					gpu_node[cudaIndex3D( i, j, k, NODES_X )]->ptrUp    = gpu_node[cudaIndex3D( i, j+1, k, NODES_X )];
				/* back */
				if ( k >= 1 )
					gpu_node[cudaIndex3D( i, j, k, NODES_X )]->ptrBack  = gpu_node[cudaIndex3D( i, j, k-1, NODES_X )];
				/* front */
				if ( k <= NODES_X - 2 )
					gpu_node[cudaIndex3D( i, j, k, NODES_X )]->ptrFront = gpu_node[cudaIndex3D( i, j, k+1, NODES_X )];

				gpu_node[cudaIndex3D( i, j, k, NODES_X )]->nodeIX.x = i;
				gpu_node[cudaIndex3D( i, j, k, NODES_X )]->nodeIX.y = j;
				gpu_node[cudaIndex3D( i, j, k, NODES_X )]->nodeIX.z = k;
			}
		}
	}
};

void FluidSimProc::PrintMSG( void )
{
	using namespace std;

	system( "cls" );
	cout 
		<< "**************** operation to confirm *******************" << endl
		<< "mouse wheel ------------ to rotate the observation matrix" << endl
		<< "keyboard: Q ------------ to quit the program" << endl
		<< "keyboard: Esc ---------- to quit the program" << endl
		<< "keyboard: C ------------ to clear the data of stage" << endl
		<< "**************** fluid simulation info ******************" << endl
		<< "number of GPU nodes for fluid simulation: " << gpu_node.size() << endl
		<< "number of HOST nodes for fluid simulation: " << host_node.size() << endl
		<< "grid size per computation node : 64 x 64 x 64" << endl;
};

void FluidSimProc::IO_ReadBuffers( void )
{
	for ( int i = 0; i < NODES_X * NODES_X * NODES_X; i++ )
	{
		hipMemcpy( dev_density[i],    host_density[i],    m_node_size, hipMemcpyHostToDevice );
		hipMemcpy( dev_velocity_u[i], host_velocity_u[i], m_node_size, hipMemcpyHostToDevice );
		hipMemcpy( dev_velocity_v[i], host_velocity_v[i], m_node_size, hipMemcpyHostToDevice );
		hipMemcpy( dev_velocity_w[i], host_velocity_w[i], m_node_size, hipMemcpyHostToDevice );
		hipMemcpy( dev_obstacle[i],     host_obstacle[i], m_node_size, hipMemcpyHostToDevice );

		if ( helper.GetCUDALastError( "host function: hipMemcpy failed", __FILE__, __LINE__ ) )
		{
			FreeResource();
			exit( 1 );
		}
	}
};

void FluidSimProc::IO_WriteBuffers( void )
{
	for ( int i = 0; i < NODES_X * NODES_X * NODES_X; i++ )
	{
		hipMemcpy( host_density[i],    dev_density[i],    m_node_size, hipMemcpyDeviceToHost );
		hipMemcpy( host_velocity_u[i], dev_velocity_u[i], m_node_size, hipMemcpyDeviceToHost );
		hipMemcpy( host_velocity_v[i], dev_velocity_v[i], m_node_size, hipMemcpyDeviceToHost );
		hipMemcpy( host_velocity_w[i], dev_velocity_w[i], m_node_size, hipMemcpyDeviceToHost );

		if ( helper.GetCUDALastError( "host function: hipMemcpy failed", __FILE__, __LINE__ ) )
		{
			FreeResource();
			exit( 1 );
		}
	}
};

bool FluidSimProc::AllocateResource ( FLUIDSPARAM *fluid )
{
	/* choose which GPU to run on, change this on a multi-GPU system. */
	if ( hipSetDevice ( 0 ) != hipSuccess ) return false; 

	/* tempororay buffers reserved */
	if ( helper.CreateDeviceBuffers( TPBUFFER_X*sizeof(double), 1, &dev_dtpbuf ) not_eq SG_RUNTIME_OK ) return false;
	if ( helper.CreateDeviceBuffers( TPBUFFER_X*sizeof(int), 1, &dev_ntpbuf ) not_eq SG_RUNTIME_OK ) return false;
	if ( helper.CreateHostBuffers( TPBUFFER_X*sizeof(double), 1, &host_dtpbuf ) not_eq SG_RUNTIME_OK ) return false;
	if ( helper.CreateHostBuffers( TPBUFFER_X*sizeof(int), 1, &host_ntpbuf ) not_eq SG_RUNTIME_OK ) return false;

	/* vector of fluid simulation buffers */
	for ( int i = 0; i < NODES_X * NODES_X * NODES_X; i++ )
	{
		double *ptrDens, *ptrU, *ptrV, *ptrW, *ptrObs;

		if ( helper.CreateHostBuffers( m_node_size, 1, &ptrDens ) not_eq SG_RUNTIME_OK ) return false;
		if ( helper.CreateHostBuffers( m_node_size, 1, &ptrObs ) not_eq SG_RUNTIME_OK ) return false;
		if ( helper.CreateHostBuffers( m_node_size, 1, &ptrU ) not_eq SG_RUNTIME_OK ) return false;
		if ( helper.CreateHostBuffers( m_node_size, 1, &ptrV ) not_eq SG_RUNTIME_OK ) return false;
		if ( helper.CreateHostBuffers( m_node_size, 1, &ptrW ) not_eq SG_RUNTIME_OK ) return false;

		host_density.push_back( ptrDens );
		host_velocity_u.push_back( ptrU );
		host_velocity_v.push_back( ptrV );
		host_velocity_w.push_back( ptrW );
		host_obstacle.push_back( ptrObs );
	}

	/* vector of node topology */
	for ( int i = 0; i < NODES_X * NODES_X * NODES_X; i++ )
	{		
		SimNode *node  = (SimNode*)malloc(sizeof(SimNode));
		node->ptrFront = node->ptrBack = nullptr;
		node->ptrLeft  = node->ptrRight = nullptr;
		node->ptrDown  = node->ptrUp = nullptr;
		node->updated  = false;
		gpu_node.push_back( node );
	}

	/* create GPU node for fluid simulation */
	for ( int i = 0; i < NODES_X * NODES_X * NODES_X; i++ )
	{
		double *ptrDens, *ptrU, *ptrV, *ptrW, *ptrObs;

		if ( helper.CreateDeviceBuffers( m_node_size, 1, &ptrDens ) not_eq SG_RUNTIME_OK ) return false;
		if ( helper.CreateDeviceBuffers( m_node_size, 1, &ptrObs ) not_eq SG_RUNTIME_OK ) return false;
		if ( helper.CreateDeviceBuffers( m_node_size, 1, &ptrU ) not_eq SG_RUNTIME_OK ) return false;
		if ( helper.CreateDeviceBuffers( m_node_size, 1, &ptrV ) not_eq SG_RUNTIME_OK ) return false;
		if ( helper.CreateDeviceBuffers( m_node_size, 1, &ptrW ) not_eq SG_RUNTIME_OK ) return false;

		dev_density.push_back( ptrDens );
		dev_velocity_u.push_back( ptrU );
		dev_velocity_v.push_back( ptrV );
		dev_velocity_w.push_back( ptrW );

		dev_obstacle.push_back( ptrObs );
	}

	/* allocate memory on GPU devices */
	for ( int i = 0; i < dev_buffers_num; i++ )
	{
		double *ptr;
		if ( helper.CreateDeviceBuffers( m_node_size, 1, &ptr ) not_eq SG_RUNTIME_OK )
			return false;

		dev_buffers.push_back(ptr);
	}

	/* allocate visual buffers */
	if ( helper.CreateDeviceBuffers( m_volm_size, 1, &dev_visual ) not_eq SG_RUNTIME_OK ) return false;
	if ( helper.CreateHostBuffers( m_volm_size, 1, &host_visual ) not_eq SG_RUNTIME_OK )  return false;

	/* check the CUDA device if something occured */
	if ( helper.GetCUDALastError( "memory allocation failed, check the code", __FILE__, __LINE__ ) )
	{
		FreeResource();
		exit( 1 );
	}
	
	/* finally */
	return true;
}  

void FluidSimProc::FreeResource ( void )
{
	/* free node resource */
	for ( int i = 0; i < NODES_X * NODES_X * NODES_X; i++ )
	{
		/* release host resource */
		helper.FreeHostBuffers( 1, &host_density[i] );
		helper.FreeHostBuffers( 1, &host_velocity_u[i] );
		helper.FreeHostBuffers( 1, &host_velocity_v[i] );
		helper.FreeHostBuffers( 1, &host_velocity_w[i] );
		helper.FreeHostBuffers( 1, &host_obstacle[i] );

		/* release device resource */
		helper.FreeDeviceBuffers( 1, &dev_obstacle[i] );
		helper.FreeDeviceBuffers( 1, &dev_density[i] );
		helper.FreeDeviceBuffers( 1, &dev_velocity_u[i] );
		helper.FreeDeviceBuffers( 1, &dev_velocity_v[i] );
		helper.FreeDeviceBuffers( 1, &dev_velocity_w[i] );	
	}

	/* free device resource */
	for ( int i = 0; i < dev_buffers_num; i++ ) 
		helper.FreeDeviceBuffers( 1, &dev_buffers[i] );
}

void FluidSimProc::SolveNavierStokers( void )
{
	/* updating */
	for ( int i = 0; i < GNODES_X; i++ )
	{
		for ( int j = 0; j < GNODES_X; j++ )
		{
			for ( int k = 0; k < GNODES_X; k++ )
			{
				LoadNode(i,j,k) ;

				Interaction(i,j,k);

				AddSource();

				VelocitySolver();

				DensitySolver();
					
				SaveNode(i,j,k);
			}
		}
	}
};

void FluidSimProc::FluidSimSolver( FLUIDSPARAM *fluid )
{
	if ( !fluid->run ) return;

	/* read host nodes */
	IO_ReadBuffers();
	
	/* solving NS equations */
	SolveNavierStokers();

	/* waiting for all kernels end */
	if ( hipDeviceSynchronize() not_eq hipSuccess )
	{
		printf( "hipDeviceSynchronize failed\n" );
		FreeResource();
		exit( 1 );
	}

	/* finally, generate volumetric image */
	Finally( fluid );

	/* save updated nodes */
	IO_WriteBuffers();
};

void FluidSimProc::LoadNode( int i, int j, int k )
{
	cudaDeviceDim3D();
	SimNode *ptr = gpu_node[cudaIndex3D( i, j, k, NODES_X )];

	/* upload center node to GPU device */
	kernelCopyGrids __device_func__ ( dev_u, dev_velocity_u[cudaIndex3D( i, j, k, NODES_X )] );
	kernelCopyGrids __device_func__ ( dev_v, dev_velocity_v[cudaIndex3D( i, j, k, NODES_X )] );
	kernelCopyGrids __device_func__ ( dev_w, dev_velocity_w[cudaIndex3D( i, j, k, NODES_X )] );
	kernelCopyGrids __device_func__ ( dev_den,  dev_density[cudaIndex3D( i, j, k, NODES_X )] );
	kernelCopyGrids __device_func__ ( dev_obs, dev_obstacle[cudaIndex3D( i, j, k, NODES_X )] );

	if ( helper.GetCUDALastError( "device kernel: kernelCopyGrids failed", __FILE__, __LINE__ ) )
	{
		FreeResource();
		exit( 1 );
	}

	/* upload neighbouring buffers to GPU device */
	if ( ptr->ptrLeft not_eq nullptr )
	{
		kernelCopyGrids __device_func__( velu_L, dev_velocity_u[cudaIndex3D( i-1, j, k, NODES_X )] );
		kernelCopyGrids __device_func__( velv_L, dev_velocity_v[cudaIndex3D( i-1, j, k, NODES_X )] );
		kernelCopyGrids __device_func__( velw_L, dev_velocity_w[cudaIndex3D( i-1, j, k, NODES_X )] );
		kernelCopyGrids __device_func__( dens_L,    dev_density[cudaIndex3D( i-1, j, k, NODES_X )] );

		if ( helper.GetCUDALastError( "device kernel: kernelCopyGrids failed", __FILE__, __LINE__ ) )
		{
			FreeResource();
			exit( 1 );
		}
	}
	else
	{
		kernelZeroGrids __device_func__ ( velu_L );
		kernelZeroGrids __device_func__ ( velv_L );
		kernelZeroGrids __device_func__ ( velw_L );
		kernelZeroGrids __device_func__ ( dens_L );

		if ( helper.GetCUDALastError( "device kernel: kernelZeroGrids failed", __FILE__, __LINE__ ) )
		{
			FreeResource();
			exit( 1 );
		}
	}

	if ( ptr->ptrRight not_eq nullptr )
	{
		kernelCopyGrids __device_func__( velu_R, dev_velocity_u[cudaIndex3D( i+1, j, k, NODES_X )] );
		kernelCopyGrids __device_func__( velv_R, dev_velocity_v[cudaIndex3D( i+1, j, k, NODES_X )] );
		kernelCopyGrids __device_func__( velw_R, dev_velocity_w[cudaIndex3D( i+1, j, k, NODES_X )] );
		kernelCopyGrids __device_func__( dens_R,    dev_density[cudaIndex3D( i+1, j, k, NODES_X )] );

		if ( helper.GetCUDALastError( "device kernel: kernelCopyGrids failed", __FILE__, __LINE__ ) )
		{
			FreeResource();
			exit( 1 );
		}
	}
	else
	{
		kernelZeroGrids __device_func__ ( velu_R );
		kernelZeroGrids __device_func__ ( velv_R );
		kernelZeroGrids __device_func__ ( velw_R );
		kernelZeroGrids __device_func__ ( dens_R );

		if ( helper.GetCUDALastError( "device kernel: kernelZeroGrids failed", __FILE__, __LINE__ ) )
		{
			FreeResource();
			exit( 1 );
		}
	}

	if ( ptr->ptrUp not_eq nullptr )
	{
		kernelCopyGrids __device_func__( velu_U, dev_velocity_u[cudaIndex3D( i, j+1, k, NODES_X )] );
		kernelCopyGrids __device_func__( velv_U, dev_velocity_v[cudaIndex3D( i, j+1, k, NODES_X )] );
		kernelCopyGrids __device_func__( velw_U, dev_velocity_w[cudaIndex3D( i, j+1, k, NODES_X )] );
		kernelCopyGrids __device_func__( dens_U,    dev_density[cudaIndex3D( i, j+1, k, NODES_X )] );

		if ( helper.GetCUDALastError( "device kernel: kernelCopyGrids failed", __FILE__, __LINE__ ) )
		{
			FreeResource();
			exit( 1 );
		}
	}
	else
	{
		kernelZeroGrids __device_func__ ( velu_U );
		kernelZeroGrids __device_func__ ( velv_U );
		kernelZeroGrids __device_func__ ( velw_U );
		kernelZeroGrids __device_func__ ( dens_U );

		if ( helper.GetCUDALastError( "device kernel: kernelZeroGrids failed", __FILE__, __LINE__ ) )
		{
			FreeResource();
			exit( 1 );
		}
	}

	if ( ptr->ptrDown not_eq nullptr )
	{
		kernelCopyGrids __device_func__( velu_D, dev_velocity_u[cudaIndex3D( i, j-1, k, NODES_X )] );
		kernelCopyGrids __device_func__( velv_D, dev_velocity_v[cudaIndex3D( i, j-1, k, NODES_X )] );
		kernelCopyGrids __device_func__( velw_D, dev_velocity_w[cudaIndex3D( i, j-1, k, NODES_X )] );
		kernelCopyGrids __device_func__( dens_D,    dev_density[cudaIndex3D( i, j-1, k, NODES_X )] );

		if ( helper.GetCUDALastError( "device kernel: kernelCopyGrids failed", __FILE__, __LINE__ ) )
		{
			FreeResource();
			exit( 1 );
		}
	}
	else
	{
		kernelZeroGrids __device_func__ ( velu_D );
		kernelZeroGrids __device_func__ ( velv_D );
		kernelZeroGrids __device_func__ ( velw_D );
		kernelZeroGrids __device_func__ ( dens_D );

		if ( helper.GetCUDALastError( "device kernel: kernelZeroGrids failed", __FILE__, __LINE__ ) )
		{
			FreeResource();
			exit( 1 );
		}
	}

	if ( ptr->ptrFront not_eq nullptr )
	{
		kernelCopyGrids __device_func__( velu_F, dev_velocity_u[cudaIndex3D( i, j, k+1, NODES_X )] );
		kernelCopyGrids __device_func__( velv_F, dev_velocity_v[cudaIndex3D( i, j, k+1, NODES_X )] );
		kernelCopyGrids __device_func__( velw_F, dev_velocity_w[cudaIndex3D( i, j, k+1, NODES_X )] );
		kernelCopyGrids __device_func__( dens_F,    dev_density[cudaIndex3D( i, j, k+1, NODES_X )] );

		if ( helper.GetCUDALastError( "device kernel: kernelCopyGrids failed", __FILE__, __LINE__ ) )
		{
			FreeResource();
			exit( 1 );
		}
	}
	else
	{
		kernelZeroGrids __device_func__ ( velu_F );
		kernelZeroGrids __device_func__ ( velv_F );
		kernelZeroGrids __device_func__ ( velw_F );
		kernelZeroGrids __device_func__ ( dens_F );

		if ( helper.GetCUDALastError( "device kernel: kernelZeroGrids failed", __FILE__, __LINE__ ) )
		{
			FreeResource();
			exit( 1 );
		}
	}

	if ( ptr->ptrBack not_eq nullptr )
	{
		kernelCopyGrids __device_func__( velu_B, dev_velocity_u[cudaIndex3D( i, j, k-1, NODES_X )] );
		kernelCopyGrids __device_func__( velv_B, dev_velocity_v[cudaIndex3D( i, j, k-1, NODES_X )] );
		kernelCopyGrids __device_func__( velw_B, dev_velocity_w[cudaIndex3D( i, j, k-1, NODES_X )] );
		kernelCopyGrids __device_func__( dens_B,    dev_density[cudaIndex3D( i, j, k-1, NODES_X )] );

		if ( helper.GetCUDALastError( "device kernel: kernelCopyGrids failed", __FILE__, __LINE__ ) )
		{
			FreeResource();
			exit( 1 );
		}
	}
	else
	{
		kernelZeroGrids __device_func__ ( velu_B );
		kernelZeroGrids __device_func__ ( velv_B );
		kernelZeroGrids __device_func__ ( velw_B );
		kernelZeroGrids __device_func__ ( dens_B );

		if ( helper.GetCUDALastError( "device kernel: kernelZeroGrids failed", __FILE__, __LINE__ ) )
		{
			FreeResource();
			exit( 1 );
		}
	}
};

void FluidSimProc::SaveNode( int i, int j, int k )
{
	cudaDeviceDim3D();
	SimNode *ptr = gpu_node[cudaIndex3D( i, j, k, NODES_X )];

	/* draw data back */
	kernelCopyGrids __device_func__( dev_velocity_u[cudaIndex3D(i,j,k,NODES_X)], velu_C );
	kernelCopyGrids __device_func__( dev_velocity_v[cudaIndex3D(i,j,k,NODES_X)], velv_C );
	kernelCopyGrids __device_func__( dev_velocity_w[cudaIndex3D(i,j,k,NODES_X)], velw_C );
	kernelCopyGrids __device_func__(    dev_density[cudaIndex3D(i,j,k,NODES_X)], dens_C );

	if ( helper.GetCUDALastError( "device kernel: kernelCopyGrids failed", __FILE__, __LINE__ ) )
	{
		FreeResource();
		exit( 1 );
	}

	/* draw volumetric data back */	
	kernelPickData __device_func__( dev_visual, dev_den, i * GRIDS_X, j * GRIDS_X, k * GRIDS_X );

	if ( helper.GetCUDALastError( "device kernel: kernelPickData failed", __FILE__, __LINE__ ) )
	{
		FreeResource();
		exit( 1 );
	}

	/* ����ǰ�ڵ�ı������Ϊ�Ѹ��� */
	ptr->updated = true;
};

void FluidSimProc::AddSource( void )
{
	if ( decrease_times eqt 0 )
	{
		cudaDeviceDim3D();
		kernelAddSource __device_func__ ( dev_den, dev_u, dev_v, dev_w, dev_obs );

		if ( helper.GetCUDALastError( "device kernel: kernelPickData failed", __FILE__, __LINE__ ) )
		{
			FreeResource();
			exit( 1 );
		}

		increase_times++;

		if ( increase_times eqt 200 )
		{
			decrease_times = increase_times;
			increase_times = 0;
		}
	}
	else
	{
		decrease_times--;
	}
};

void FluidSimProc::InitBoundary( void )
{
	cudaDeviceDim3D();

	/* zero boundary buffers */
	kernelZeroGrids __device_func__ ( dev_obs );

	for ( int i = 0; i < host_obstacle.size(); i++ )
	{
		if ( hipMemcpy( host_obstacle[i], dev_obs, m_node_size, hipMemcpyDeviceToHost ) not_eq hipSuccess )
		{
			helper.GetCUDALastError( "hipMemcpy failed", __FILE__, __LINE__ );
			FreeResource();
			exit( 1 );
		}
	}	
	
	/* set boundary condition */
	kernelSetBoundary __device_func__( dev_obs );
	if ( hipMemcpy( host_obstacle[cudaIndex3D(0,0,0,NODES_X)], dev_obs, m_node_size, hipMemcpyDeviceToHost) not_eq hipSuccess )
	{
		helper.GetCUDALastError( "hipMemcpy failed", __FILE__, __LINE__ );
		FreeResource();
		exit( 1 );
	}
};

void FluidSimProc::VelocitySolver( void )
{
	// diffuse the velocity field (per axis):
	hostDiffusion( dev_u0, dev_u, VISOCITY, dev_obs, MACRO_VELOCITY_U );
	hostDiffusion( dev_v0, dev_v, VISOCITY, dev_obs, MACRO_VELOCITY_V );
	hostDiffusion( dev_w0, dev_w, VISOCITY, dev_obs, MACRO_VELOCITY_W );
	
	if ( helper.GetCUDALastError( "host function failed: hostDiffusion", __FILE__, __LINE__ ) )
	{
		FreeResource();
		exit( 1 );
	}

	std::swap( dev_u0, dev_u );
	std::swap( dev_v0, dev_v );
	std::swap( dev_w0, dev_w );

	// stabilize it: (vx0, vy0 are whatever, being used as temporaries to store gradient field)
	hostProject( dev_u, dev_v, dev_w, dev_div, dev_p, dev_obs );

	if ( helper.GetCUDALastError( "host function failed: hostProject", __FILE__, __LINE__ ) )
	{
		FreeResource();
		exit( 1 );
	}
	
	// advect the velocity field (per axis):
	hostAdvection( dev_u0, dev_u, dev_obs, MACRO_VELOCITY_U, dev_u, dev_v, dev_w );
	hostAdvection( dev_v0, dev_v, dev_obs, MACRO_VELOCITY_V, dev_u, dev_v, dev_w );
	hostAdvection( dev_w0, dev_w, dev_obs, MACRO_VELOCITY_W, dev_u, dev_v, dev_w );

	if ( helper.GetCUDALastError( "host function failed: hostAdvection", __FILE__, __LINE__ ) )
	{
		FreeResource();
		exit( 1 );
	}

	std::swap( dev_u0, dev_u );
	std::swap( dev_v0, dev_v );
	std::swap( dev_w0, dev_w );
	
	// stabilize it: (vx0, vy0 are whatever, being used as temporaries to store gradient field)
	hostProject( dev_u, dev_v, dev_w, dev_div, dev_p, dev_obs );
};

void FluidSimProc::DensitySolver( void )
{
	hostDiffusion( dev_den0, dev_den, DIFFUSION, dev_obs, MACRO_DENSITY );
	std::swap( dev_den0, dev_den );
	hostAdvection ( dev_den, dev_den0, dev_obs, MACRO_DENSITY, dev_u, dev_v, dev_w );

	if ( helper.GetCUDALastError( "host function failed: DensitySolver", __FILE__, __LINE__ ) )
	{
		FreeResource();
		exit( 1 );
	}
};

void FluidSimProc::ZeroBuffers( void )
{
	cudaDeviceDim3D();

	/* zero GPU buffer */
	for ( int i = 0; i < dev_buffers_num; i++ )
		kernelZeroGrids  __device_func__ ( dev_buffers[i] );

	/* zero host buffer */
	for ( int i = 0; i < NODES_X * NODES_X * NODES_X; i++ )
	{
		kernelZeroGrids __device_func__ ( dev_density[i] );
		kernelZeroGrids __device_func__ ( dev_velocity_u[i] );
		kernelZeroGrids __device_func__ ( dev_velocity_v[i] );
		kernelZeroGrids __device_func__ ( dev_velocity_w[i] );

		if ( helper.GetCUDALastError( "device failed: kernelZeroGrids", __FILE__, __LINE__ ) )
		{
			FreeResource();
			exit( 1 );
		}
	}

	/* zero visual buffer */
	kernelZeroVolumetric __device_func__ ( dev_visual );
	hipMemcpy( host_visual, dev_visual, m_volm_size, hipMemcpyDeviceToHost );

	if ( helper.GetCUDALastError( "host function failed: ZeroBuffers", __FILE__, __LINE__ ) )
	{
		FreeResource();
		exit( 1 );
	}
};

void FluidSimProc::Interaction( int i, int j, int k )
{
	SimNode *ptr = gpu_node[cudaIndex3D(i,j,k,NODES_X)];
	int left, right, up, down, front, back;

	left = right = up = down = front = back = MACRO_FALSE;
	
	if ( ptr->ptrLeft  not_eq nullptr ) left  = ( (ptr->ptrLeft->updated) ? MACRO_TRUE : MACRO_FALSE );
	if ( ptr->ptrRight not_eq nullptr )	right = ( (ptr->ptrRight->updated)? MACRO_TRUE : MACRO_FALSE );
	if ( ptr->ptrUp    not_eq nullptr ) up    = ( (ptr->ptrUp->updated)   ? MACRO_TRUE : MACRO_FALSE );
	if ( ptr->ptrDown  not_eq nullptr )	down  = ( (ptr->ptrDown->updated) ? MACRO_TRUE : MACRO_FALSE );
	if ( ptr->ptrFront not_eq nullptr )	front = ( (ptr->ptrFront->updated)? MACRO_TRUE : MACRO_FALSE );
	if ( ptr->ptrBack  not_eq nullptr ) back  = ( (ptr->ptrBack->updated) ? MACRO_TRUE : MACRO_FALSE );

	cudaDeviceDim3D();
	kernelInteractNodes __device_func__
		( dens_C, dens_L, dens_R, dens_U, dens_D, dens_F, dens_B, left, right, up, down, front, back );
	kernelInteractNodes __device_func__
		( velu_C, velu_L, velu_R, velu_U, velu_D, velu_F, velu_B, left, right, up, down, front, back );
	kernelInteractNodes __device_func__
		( velv_C, velv_L, velv_R, velv_U, velv_D, velv_F, velv_B, left, right, up, down, front, back );
	kernelInteractNodes __device_func__
		( velw_C, velw_L, velw_R, velw_U, velw_D, velw_F, velw_B, left, right, up, down, front, back );
};

void FluidSimProc::Finally( FLUIDSPARAM *fluid )
{
	/* ���½ڵ����� */
	cudaDeviceDim3D();	
	for ( int i = 0; i < NODES_X * NODES_X * NODES_X; i++ )
		gpu_node[i]->updated = false;

	/* ��ȡ���º��ͼ������ */
	hipMemcpy( host_visual, dev_visual, m_volm_size, hipMemcpyDeviceToHost );
	fluid->volume.ptrData = host_visual;

	/* counting FPS */
	fluid->fps.dwFrames ++;
	fluid->fps.dwCurrentTime = GetTickCount();
	fluid->fps.dwElapsedTime = fluid->fps.dwCurrentTime - fluid->fps.dwLastUpdateTime;

	/* 1 second */
	if ( fluid->fps.dwElapsedTime >= 1000 )
	{
		fluid->fps.uFPS     = fluid->fps.dwFrames * 1000 / fluid->fps.dwElapsedTime;
		fluid->fps.dwFrames = 0;
		fluid->fps.dwLastUpdateTime = fluid->fps.dwCurrentTime;
	}
};