/**
* <Author>        Orlando Chen
* <Email>         seagochen@gmail.com
* <First Time>    Dec 15, 2013
* <Last Time>     Mar 25, 2014
* <File Name>     FluidSimProc.cu
*/

#include <time.h>
#include <iostream>
#include <utility>
#include <hip/hip_runtime_api.h>
#include <>
#include "MacroDefinition.h"
#include "FluidSimProc.h"
#include "MacroDefinition.h"
#include "Kernels.h"

using namespace sge;
using std::cout;
using std::endl;

FluidSimProc::FluidSimProc( FLUIDSPARAM *fluid )
{
	/* choose which GPU to run on, change this on a multi-GPU system. */
	if ( hipSetDevice ( 0 ) != hipSuccess )
	{
		m_scHelper.GetCUDALastError( "cannot set device", __FILE__, __LINE__ );
		exit(1);
	}

	/* initialize FPS */
	InitParams( fluid );

	/* allocate resources */
	AllocateResource();
	
	/* clear buffer */
	ClearBuffers();

	/* create boundary condition */
	InitBoundary();

	/* finally, print message */
	printf( "fluid simulation ready...\n" );
};


void FluidSimProc::InitParams( FLUIDSPARAM *fluid )
{
	fluid->fps.dwCurrentTime = 0;
	fluid->fps.dwElapsedTime = 0;
	fluid->fps.dwFrames = 0;
	fluid->fps.dwLastUpdateTime = 0;
	fluid->fps.uFPS = 0;

	srand(time(NULL));

	m_szTitle = APP_TITLE;
};


void FluidSimProc::AllocateResource( void )
{
	if ( not m_scHelper.CreateCompNodesForDevice( &m_vectgGrids,
		gGRIDS_X * gGRIDS_Y * gGRIDS_Z * sizeof(double), 5 ) ) goto Error;

	if ( not m_scHelper.CreateCompNodesForDevice( &m_vectgBullets,
		gBULLET_X * gBULLET_Y * gBULLET_Z * sizeof(double), 11 ) ) goto Error;

	if ( not m_scHelper.CreateCompNodesForDevice( &m_vectBigBuffers, 
		BIG_X * BIG_Y * BIG_Z * sizeof(double), 5 ) ) goto Error;

	if ( not m_scHelper.CreateCompNodesForDevice( &m_vectsGrids, 
		sGRIDS_X * sGRIDS_Y * sGRIDS_Z * sizeof(double), 5 ) ) goto Error;

	if ( not m_scHelper.CreateCompNodesForDevice( &m_vectsBullets, 
		sBULLET_X * sBULLET_Y * sBULLET_Z * sizeof(double), 11 ) ) goto Error;

	m_scHelper.CreateDeviceBuffers( VOLUME_X * VOLUME_Y * VOLUME_Z * sizeof(SGUCHAR),
		1, &m_ptrDeviceVisual );
	m_scHelper.CreateHostBuffers( VOLUME_X * VOLUME_Y * VOLUME_Z * sizeof(SGUCHAR),
		1, &m_ptrHostVisual );

	goto Success;

Error:
		cout << "create computation buffers device failed" << endl;
		FreeResource();
		exit(1);

Success:
		;
};


void FluidSimProc::FreeResource( void )
{
	int i;

	for ( i = 0; i < m_vectgGrids.size(); i++ ) m_scHelper.FreeDeviceBuffers( 1, & m_vectgGrids[i] );
	for ( i = 0; i < m_vectgBullets.size(); i++ ) m_scHelper.FreeDeviceBuffers( 1, &m_vectgBullets[i] );
	for ( i = 0; i < m_vectBigBuffers.size(); i++ ) m_scHelper.FreeDeviceBuffers( 1, &m_vectBigBuffers[i] );
	for ( i = 0; i < m_vectsGrids.size(); i++ ) m_scHelper.FreeDeviceBuffers( 1, &m_vectsGrids[i] );
	for ( i = 0; i < m_vectsBullets.size(); i++ ) m_scHelper.FreeDeviceBuffers( 1, &m_vectsBullets[i] );

	m_scHelper.FreeDeviceBuffers( 1, &m_ptrDeviceVisual );
	m_scHelper.FreeHostBuffers( 1, &m_ptrHostVisual );
}


void FluidSimProc::RefreshStatus( FLUIDSPARAM *fluid )
{
	/* waiting for all kernels end */
	if ( hipDeviceSynchronize() not_eq hipSuccess )
	{
		printf( "hipDeviceSynchronize failed\n" );
		FreeResource();
		exit( 1 );
	}

	/* counting FPS */
	fluid->fps.dwFrames ++;
	fluid->fps.dwCurrentTime = GetTickCount();
	fluid->fps.dwElapsedTime = fluid->fps.dwCurrentTime - fluid->fps.dwLastUpdateTime;

	/* 1 second */
	if ( fluid->fps.dwElapsedTime >= 1000 )
	{
		fluid->fps.uFPS     = fluid->fps.dwFrames * 1000 / fluid->fps.dwElapsedTime;
		fluid->fps.dwFrames = 0;
		fluid->fps.dwLastUpdateTime = fluid->fps.dwCurrentTime;
	}

	/* updating image */
	if ( hipMemcpy( m_ptrHostVisual, m_ptrDeviceVisual, 
		VOLUME_X * VOLUME_Y * VOLUME_Z * sizeof(SGUCHAR), hipMemcpyDeviceToHost ) not_eq hipSuccess )
	{
		m_scHelper.GetCUDALastError( "host function: hipMemcpy failed", __FILE__, __LINE__ );
		FreeResource();
		exit( 1 );
	}
	fluid->volume.ptrData = m_ptrHostVisual;
};


void FluidSimProc::ClearBuffers( void )
{
	int i;

	m_scHelper.DeviceParamDim( &gridDim, &blockDim, THREADS_S,
		gGRIDS_X / 2, gGRIDS_Y / 2, gGRIDS_X, gGRIDS_Y, gGRIDS_Z );

	for ( i = 0; i < m_vectgGrids.size(); i++ )
		kernelZeroBuffers __device_func__ ( m_vectgGrids[i], gGRIDS_X, gGRIDS_Y, gGRIDS_Z );


	m_scHelper.DeviceParamDim( &gridDim, &blockDim, THREADS_S, 
		gBULLET_X / 2, gBULLET_Y / 3, gBULLET_X, gBULLET_Y, gBULLET_Z );

	for ( i = 0; i < m_vectgBullets.size(); i++ )
		kernelZeroBuffers __device_func__ ( m_vectgBullets[i], gBULLET_X, gBULLET_Y, gBULLET_Z );


	m_scHelper.DeviceParamDim( &gridDim, &blockDim, THREADS_S,
		sGRIDS_X / 4, sGRIDS_Y / 4, sGRIDS_X, sGRIDS_Y, sGRIDS_Z );

	for ( i = 0; i < m_vectsGrids.size(); i++ )
		kernelZeroBuffers __device_func__ ( m_vectsGrids[i], sGRIDS_X, sGRIDS_Y, sGRIDS_Z );


	m_scHelper.DeviceParamDim( &gridDim, &blockDim, THREADS_S,
		sBULLET_X / 5, sBULLET_Y / 5, sBULLET_X, sBULLET_Y, sBULLET_Z );

	for ( i = 0; i < m_vectsBullets.size(); i++ )
		kernelZeroBuffers __device_func__ ( m_vectsBullets[i], sBULLET_X, sBULLET_Y, sBULLET_Z );


	m_scHelper.DeviceParamDim( &gridDim, &blockDim, THREADS_S,
		BIG_X / 8, BIG_Y / 8, BIG_X, BIG_Y, BIG_Z );
	for ( i = 0; i < m_vectBigBuffers.size(); i++ )
		kernelZeroBuffers __device_func__ ( m_vectBigBuffers[i], BIG_X, BIG_Y, BIG_Z );

	kernelZeroBuffers __device_func__ ( m_ptrDeviceVisual, VOLUME_X, VOLUME_Y, VOLUME_Z );


	if ( m_scHelper.GetCUDALastError( "call member function ClearBuffers failed", __FILE__, __LINE__ ) )
	{
		FreeResource();
		exit(1);
	}
}


void FluidSimProc::InitBoundary( void )
{
	m_scHelper.DeviceParamDim( &gridDim, &blockDim, THREADS_S,
		TILE_X, TILE_Y, gGRIDS_X, gGRIDS_Y, gGRIDS_Z );

	kernelSetBound __device_func__ ( m_vectgGrids[DEV_OBSTACLE], gGRIDS_X, gGRIDS_Y, gGRIDS_Z );

	kernelLoadBullet __device_func__
		( m_vectgBullets[DEV_OBSTACLE], m_vectgGrids[DEV_OBSTACLE],
		gBULLET_X, gBULLET_Y, gBULLET_Z, gGRIDS_X, gGRIDS_Y, gGRIDS_Z );

	if ( m_scHelper.GetCUDALastError( "call member function InitBoundary failed", __FILE__, __LINE__ ) )
	{
		FreeResource();
		exit(1);
	}
};


void FluidSimProc::GenerVolumeImg( void )
{
	m_scHelper.DeviceParamDim( &gridDim, &blockDim, THREADS_S, 32, 32, 256, 256, 256 );

//	kernelPickData __device_func__ ( m_ptrDeviceVisual, m_vectBigBuffers[DEV_DENSITY],
//		VOLUME_X, VOLUME_Y, VOLUME_Z );


	m_scHelper.DeviceParamDim( &gridDim, &blockDim, THREADS_S, 32, 32, 128, 128, 128 );

	kernelExitBullet __device_func__ ( m_vectsGrids[DEV_DENSITY], m_vectsBullets[DEV_DENSITY],
		128, 128, 128, 130, 130, 130 );

	kernelPickData __device_func__ ( m_ptrDeviceVisual, m_vectsGrids[DEV_DENSITY], 
		VOLUME_X, VOLUME_Y, VOLUME_Z,
		128, 128, 128,
		1, 1, 1 );

	if ( m_scHelper.GetCUDALastError( "call member function GenerVolumeImg failed", __FILE__, __LINE__ ) )
	{
		FreeResource();
		exit(1);
	}
};

#if 0
#define dev_den m_vectgBullets[DEV_DENSITY]
#define	dev_u   m_vectgBullets[DEV_VELOCITY_U]
#define	dev_v   m_vectgBullets[DEV_VELOCITY_V]
#define	dev_w   m_vectgBullets[DEV_VELOCITY_W]
#define	dev_p   m_vectgBullets[DEV_PRESSURE]
#define	dev_div m_vectgBullets[DEV_DIVERGENCE]
#define	dev_obs m_vectgBullets[DEV_OBSTACLE]
#define	dev_den0 m_vectgBullets[DEV_DENSITY0]
#define	dev_u0   m_vectgBullets[DEV_VELOCITY_U0]
#define	dev_v0   m_vectgBullets[DEV_VELOCITY_V0]
#define	dev_w0   m_vectgBullets[DEV_VELOCITY_W0]
#endif

void FluidSimProc::SolveGlobalFlux( void )
{
#if 1
	dev_den  = &m_vectgBullets[DEV_DENSITY];
	dev_u    = &m_vectgBullets[DEV_VELOCITY_U];
	dev_v    = &m_vectgBullets[DEV_VELOCITY_V];
	dev_w    = &m_vectgBullets[DEV_VELOCITY_W];
	dev_p    = &m_vectgBullets[DEV_PRESSURE];
	dev_div  = &m_vectgBullets[DEV_DIVERGENCE];
	dev_obs  = &m_vectgBullets[DEV_OBSTACLE];
	dev_den0 = &m_vectgBullets[DEV_DENSITY0];
	dev_u0   = &m_vectgBullets[DEV_VELOCITY_U0];
	dev_v0   = &m_vectgBullets[DEV_VELOCITY_V0];
	dev_w0   = &m_vectgBullets[DEV_VELOCITY_W0];


	if ( m_scHelper.GetCUDALastError( "call member function SolveGlobalFlux failed", __FILE__, __LINE__ ) )
	{
		FreeResource();
		exit(1);
	}
#endif
};


void FluidSimProc::RefinementFlux( void )
{

	m_scHelper.DeviceParamDim( &gridDim, &blockDim, THREADS_S, 32, 32, 64, 64, 64 );

	kernelExitBullet __device_func__ ( m_vectgGrids[DEV_DENSITY], m_vectgBullets[DEV_DENSITY], 
		64, 64, 64,  66, 66, 66 );

	kernelExitBullet __device_func__ ( m_vectgGrids[DEV_VELOCITY_U], m_vectgBullets[DEV_VELOCITY_U], 
		64, 64, 64,  66, 66, 66 );
	
	kernelExitBullet __device_func__ ( m_vectgGrids[DEV_VELOCITY_V], m_vectgBullets[DEV_VELOCITY_V], 
		64, 64, 64,  66, 66, 66 );

	kernelExitBullet __device_func__ ( m_vectgGrids[DEV_VELOCITY_W], m_vectgBullets[DEV_VELOCITY_W], 
		64, 64, 64,  66, 66, 66 );


	if ( m_scHelper.GetCUDALastError( "call member function GenerVolumeImg failed", __FILE__, __LINE__ ) )
	{
		FreeResource();
		exit(1);
	}

	
	m_scHelper.DeviceParamDim( &gridDim, &blockDim, THREADS_S, 32, 32, BIG_X, BIG_Y, BIG_Z );

	kernelUpScalingInterpolation __device_func__
		( m_vectBigBuffers[DEV_DENSITY], m_vectgGrids[DEV_DENSITY],
		64, 64, 64, 256, 256, 256, 4, 4, 4 );

	kernelUpScalingInterpolation __device_func__
		( m_vectBigBuffers[DEV_VELOCITY_U], m_vectgGrids[DEV_VELOCITY_U],
		64, 64, 64, 256, 256, 256, 4, 4, 4 );

	kernelUpScalingInterpolation __device_func__
		( m_vectBigBuffers[DEV_VELOCITY_V], m_vectgGrids[DEV_VELOCITY_V],
		64, 64, 64, 256, 256, 256, 4, 4, 4 );

	kernelUpScalingInterpolation __device_func__
		( m_vectBigBuffers[DEV_VELOCITY_W], m_vectgGrids[DEV_VELOCITY_W],
		64, 64, 64, 256, 256, 256, 4, 4, 4 );

	kernelUpScalingInterpolation __device_func__
		( m_vectBigBuffers[DEV_OBSTACLE], m_vectgGrids[DEV_OBSTACLE],
		64, 64, 64, 256, 256, 256, 4, 4, 4 );

	if ( m_scHelper.GetCUDALastError( "call member function GenerVolumeImg failed", __FILE__, __LINE__ ) )
	{
		FreeResource();
		exit(1);
	}


	m_scHelper.DeviceParamDim 
		( &gridDim, &blockDim, THREADS_S, 26, 26, sBULLET_X, sBULLET_Y, sBULLET_Z );

	kernelFillBullet __device_func__ ( m_vectsBullets[DEV_DENSITY], m_vectBigBuffers[DEV_DENSITY],
		256, 256, 256, 
		130, 130, 130,
		128, 128, 128,
		1, 1, 1 );

	kernelFillBullet __device_func__ ( m_vectsBullets[DEV_VELOCITY_U], m_vectBigBuffers[DEV_VELOCITY_U],
		256, 256, 256, 
		130, 130, 130,
		128, 128, 128,
		1, 1, 1 );

	kernelFillBullet __device_func__ ( m_vectsBullets[DEV_VELOCITY_V], m_vectBigBuffers[DEV_VELOCITY_V],
		256, 256, 256, 
		130, 130, 130,
		128, 128, 128,
		1, 1, 1 );

	kernelFillBullet __device_func__ ( m_vectsBullets[DEV_VELOCITY_W], m_vectBigBuffers[DEV_VELOCITY_W],
		256, 256, 256, 
		130, 130, 130,
		128, 128, 128,
		1, 1, 1 );

	kernelFillBullet __device_func__ ( m_vectsBullets[DEV_OBSTACLE], m_vectBigBuffers[DEV_OBSTACLE],
		256, 256, 256, 
		130, 130, 130,
		128, 128, 128,
		1, 1, 1 );

	if ( m_scHelper.GetCUDALastError( "call member function GenerVolumeImg failed", __FILE__, __LINE__ ) )
	{
		FreeResource();
		exit(1);
	}

	dev_den  = &m_vectsBullets[DEV_DENSITY];
	dev_u    = &m_vectsBullets[DEV_VELOCITY_U];
	dev_v    = &m_vectsBullets[DEV_VELOCITY_V];
	dev_w    = &m_vectsBullets[DEV_VELOCITY_W];
	dev_p    = &m_vectsBullets[DEV_PRESSURE];
	dev_div  = &m_vectsBullets[DEV_DIVERGENCE];
	dev_obs  = &m_vectsBullets[DEV_OBSTACLE];
	dev_den0 = &m_vectsBullets[DEV_DENSITY0];
	dev_u0   = &m_vectsBullets[DEV_VELOCITY_U0];
	dev_v0   = &m_vectsBullets[DEV_VELOCITY_V0];
	dev_w0   = &m_vectsBullets[DEV_VELOCITY_W0];

};


void FluidSimProc::FluidSimSolver( FLUIDSPARAM *fluid )
{
	if ( not fluid->run ) return;

	SolveGlobalFlux();

	SolveNavierStokesEquation( DELTATIME, true, true, true,
		32, 32, gGRIDS_X, gGRIDS_Y, gGRIDS_Z, gBULLET_X, gBULLET_Y, gBULLET_Z );

	RefinementFlux();
	
	SolveNavierStokesEquation( DELTATIME, false, true, true,
		32, 32, sGRIDS_X, sGRIDS_Y, sGRIDS_Z, sBULLET_X, sBULLET_Y, sBULLET_Z );

	GenerVolumeImg();

	RefreshStatus( fluid );
};


void FluidSimProc::SolveNavierStokesEquation
			( cdouble dt, bool add, bool vel, bool den,
			cint tx, cint ty,
			cint gx, cint gy, cint gz,
			cint bx, cint by, cint bz )
{
	m_scHelper.DeviceParamDim( &gridDim, &blockDim, THREADS_S, tx, ty, gx, gy, gz );

#if 0
	double *ptr;
	m_scHelper.CreateHostBuffers( gBULLET_X * gBULLET_Y * gBULLET_Z * sizeof(double), 1, &ptr );

//	hipMemcpy( ptr, dev_den, gBULLET_X * gBULLET_Y * gBULLET_Z * sizeof(double), hipMemcpyDeviceToHost );
//
//	for ( int j = 0; j < gBULLET_Y; j++ )
//	{
//		for ( int k = 0; k < gBULLET_Z; k++ )
//		{
//			for ( int i = 0; i < gBULLET_X; i++ )
//			{
//				printf( "%d ", (int)ptr[ix(i,j,k,gBULLET_X,gBULLET_Y)] );
//			}
//			printf("\n");
//		}
//		printf("\n");
//	}

	cout << "----------------------------------------------------------------------" << endl;

#endif

	if (add) SourceSolver( dt, bx, by, bz );
	if (vel) VelocitySolver( dt, bx, by, bz );
	if (den) DensitySolver( dt, bx, by, bz );


#if 0

	hipMemcpy( ptr, dev_p, gBULLET_X * gBULLET_Y * gBULLET_Z * sizeof(double), hipMemcpyDeviceToHost );

	for ( int j = 0; j < gBULLET_Y; j++ )
	{
		for ( int k = 0; k < gBULLET_Z; k++ )
		{
			for ( int i = 0; i < gBULLET_X; i++ )
			{
				printf( "%f ", ptr[ix(i,j,k,gBULLET_X,gBULLET_Y)] );
			}
			printf("\n");
		}
		printf("\n");
	}

	free(ptr);
	FreeResource();
	exit(0);

#endif
};


void FluidSimProc::SourceSolver( cdouble dt, cint bx, cint by, cint bz )
{
	kernelAddSource __device_func__
		( *dev_den, *dev_v, bx, by, bz, 
			*dev_obs, dt, (double)(rand() % 300 + 1) / 100.f );
};


void FluidSimProc::VelocitySolver( cdouble dt, cint bx, cint by, cint bz )
{
	// diffuse the velocity field (per axis):
	Diffusion( *dev_u0, *dev_u, VISOCITY, bx, by, bz );
	Diffusion( *dev_v0, *dev_v, VISOCITY, bx, by, bz );
	Diffusion( *dev_w0, *dev_w, VISOCITY, bx, by, bz );
	
	if ( m_scHelper.GetCUDALastError( "host function failed: Diffusion", __FILE__, __LINE__ ) )
	{
		FreeResource();
		exit( 1 );
	}

	std::swap( *dev_u0, *dev_u );
	std::swap( *dev_v0, *dev_v );
	std::swap( *dev_w0, *dev_w );

	// stabilize it: (vx0, vy0 are whatever, being used as temporaries to store gradient field)
	Projection( *dev_u, *dev_v, *dev_w, *dev_div, *dev_p, bx, by, bz );

	if ( m_scHelper.GetCUDALastError( "host function failed: Projection", __FILE__, __LINE__ ) )
	{
		FreeResource();
		exit( 1 );
	}
	
	// advect the velocity field (per axis):
	Advection( *dev_u0, *dev_u, *dev_u, *dev_v, *dev_w, dt, bx, by, bz );
	Advection( *dev_v0, *dev_v, *dev_u, *dev_v, *dev_w, dt, bx, by, bz );
	Advection( *dev_w0, *dev_w, *dev_u, *dev_v, *dev_w, dt, bx, by, bz );

	if ( m_scHelper.GetCUDALastError( "host function failed: Advection", __FILE__, __LINE__ ) )
	{
		FreeResource();
		exit( 1 );
	}

	std::swap( *dev_u0, *dev_u );
	std::swap( *dev_v0, *dev_v );
	std::swap( *dev_w0, *dev_w );
	
	// stabilize it: (vx0, vy0 are whatever, being used as temporaries to store gradient field)
	Projection( *dev_u, *dev_v, *dev_w, *dev_div, *dev_p, bx, by, bz );
};


void FluidSimProc::DensitySolver( cdouble dt, cint bx, cint by, cint bz )
{
	Diffusion( *dev_den0, *dev_den, DIFFUSION, bx, by, bz );
	std::swap( *dev_den0, *dev_den );
	Advection( *dev_den, *dev_den0, *dev_u, *dev_v, *dev_w, dt, bx, by, bz );

	if ( m_scHelper.GetCUDALastError( "host function failed: DensitySolver", __FILE__, __LINE__ ) )
	{
		FreeResource();
		exit( 1 );
	}
};


void FluidSimProc::Jacobi
	( double *out, cdouble *in, cdouble diff, cdouble divisor, cint bx, cint by, cint bz )
{
	for ( int k=0; k<20; k++)
		kernelJacobi __device_func__ ( out, in, bx, by, bz, diff, divisor );
};

void FluidSimProc::Advection
	( double *out, cdouble *in, cdouble *u, cdouble *v, cdouble *w, cdouble dt, cint bx, cint by, cint bz )
{	
	kernelAdvection __device_func__ ( out, in, bx, by, bz, dt, u, v, w );
};

void FluidSimProc::Diffusion( double *out, cdouble *in, cdouble diff, cint bx, cint by, cint bz )
{
	double rate = diff * bx * by * bz;
	Jacobi( out, in, rate, 1+6*rate, bx, by, bz );
};

void FluidSimProc::Projection
	( double *u, double *v, double *w, double *div, double *p, cint bx, cint by, cint bz )
{
	// the velocity gradient
	kernelGradient __device_func__ ( div, p, bx, by, bz, u, v, w );

	// reuse the Gauss-Seidel relaxation solver to safely diffuse the velocity gradients from p to div
	Jacobi(p, div, 1.f, 6.f, bx, by, bz);

	// now subtract this gradient from our current velocity field
	kernelSubtract __device_func__ ( u, v, w, p, bx, by, bz );
};