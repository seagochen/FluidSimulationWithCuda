#include "hip/hip_runtime.h"
/**
* <Author>        Orlando Chen
* <Email>         seagochen@gmail.com
* <First Time>    Dec 15, 2013
* <Last Time>     Mar 25, 2014
* <File Name>     FluidSimProc.cu
*/

#include <time.h>
#include <iostream>
#include <utility>
#include <hip/hip_runtime_api.h>
#include <>
#include "MacroDefinition.h"
#include "FluidSimProc.h"
#include "MacroDefinition.h"
#include "Kernels.h"

using namespace sge;
using std::cout;
using std::endl;

inline __device__ __host__ int ix( cint x, cint tilex)
{
	if ( x >= tilex or x < 0 ) return -1;
	return x;
};

inline __device__ __host__ int ix( cint i, cint j, cint tilex, cint tiley )
{
	if ( i < 0 or j < 0 ) return -1;

	int x; if ( i >= tilex ) x = tilex - 1;
	int y; if ( j >= tiley ) y = tiley - 1;

	x = i; y = j;
	return x + y * tilex;
};

inline __host__ __device__ int ix
	( cint i, cint j, cint k, cint tilex, cint tiley, cint tilez )
{
	if ( i < 0 or i >= tilex ) return -1;
	if ( j < 0 or j >= tiley ) return -1;
	if ( k < 0 or k >= tilez ) return -1;

	return i + j * tilex + k * tilex * tiley;
};

extern // updated: 2014/3/28
__global__ void kernelSetBound( double *dst, cint tilex, cint tiley, cint tilez );

extern // updated: 2014/3/28
__global__ void kernelDeassemble( double *dst, cdouble *src, 
								 cint srcx, cint srcy, cint srcz, 
								 cint dstx, cint dsty, cint dstz, 
								 cint offi, cint offj, cint offk );

extern // updated: 2014/3/28
__global__ void kernelGlobalToBullet( double *dst, cdouble *src,
									 cint srcx, cint srcy, cint srcz, 
									 cint dstx, cint dsty, cint dstz, 
									 cint grdx, cint grdy, cint grdz,
									 cint offi, cint offj, cint offk );


FluidSimProc::FluidSimProc( FLUIDSPARAM *fluid )
{
	/* choose which GPU to run on, change this on a multi-GPU system. */
	if ( hipSetDevice ( 0 ) != hipSuccess )
	{
		m_scHelper.GetCUDALastError( "cannot set device", __FILE__, __LINE__ );
		exit(1);
	}

	/* initialize FPS */
	InitParams( fluid );

	/* allocate resources */
	AllocateResource();
	
	/* clear buffer */
	ClearBuffers();

	/* create boundary condition */
	InitBoundary();

	/* finally, print message */
	printf( "fluid simulation ready...\n" );
};

void FluidSimProc::InitParams( FLUIDSPARAM *fluid )
{
	fluid->fps.dwCurrentTime = 0;
	fluid->fps.dwElapsedTime = 0;
	fluid->fps.dwFrames = 0;
	fluid->fps.dwLastUpdateTime = 0;
	fluid->fps.uFPS = 0;

	srand(time(NULL));

	m_szTitle = APP_TITLE;
};

void FluidSimProc::AllocateResource( void )
{

	if ( not m_scHelper.CreateCompNodesForDevice( &m_vectCompBufs, 
		GRIDS_X * GRIDS_Y * GRIDS_Z * sizeof(double), COMP_BUFS ) ) goto Error;

	if ( not m_scHelper.CreateCompNodesForDevice( &m_vectBulletBufs, 
		BULLET_X * BULLET_Y * BULLET_Z * sizeof(double), BUL_BUFS ) ) goto Error;

	if ( not m_scHelper.CreateCompNodesForDevice( &m_vectBigBufs,
		VOLUME_X * VOLUME_Y * VOLUME_Z * sizeof(double), BIG_BUFS ) ) goto Error;



	if ( not m_scHelper.CreateCompNodesForDevice( &m_vectSmallDens, 
		GRIDS_X * GRIDS_Y * GRIDS_Z * sizeof(double), NODES_X * NODES_Y * NODES_Z ) ) goto Error;

	if ( not m_scHelper.CreateCompNodesForDevice( &m_vectSmallVelU, 
		GRIDS_X * GRIDS_Y * GRIDS_Z * sizeof(double), NODES_X * NODES_Y * NODES_Z ) ) goto Error;

	if ( not m_scHelper.CreateCompNodesForDevice( &m_vectSmallVelV, 
		GRIDS_X * GRIDS_Y * GRIDS_Z * sizeof(double), NODES_X * NODES_Y * NODES_Z ) ) goto Error;

	if ( not m_scHelper.CreateCompNodesForDevice( &m_vectSmallVelW, 
		GRIDS_X * GRIDS_Y * GRIDS_Z * sizeof(double), NODES_X * NODES_Y * NODES_Z ) ) goto Error;



	m_scHelper.CreateDeviceBuffers( VOLUME_X * VOLUME_Y * VOLUME_Z * sizeof(SGUCHAR),
		1, &m_ptrDeviceVisual );
	m_scHelper.CreateHostBuffers( VOLUME_X * VOLUME_Y * VOLUME_Z * sizeof(SGUCHAR),
		1, &m_ptrHostVisual );

	goto Success;

Error:
		cout << "create computation buffers device failed" << endl;
		FreeResource();
		exit(1);

Success:
		cout << "size of m_vectBulletBufs: " << m_vectBulletBufs.size() << endl
			<< "size of m_vectCompBufs: " << m_vectCompBufs.size() << endl;
		
		cout << "size of m_vectSmallDens: " << m_vectSmallDens.size() << endl
			<< "size of m_vectSmallVelU: " << m_vectSmallVelU.size() << endl
			<< "size of m_vectSmallVelV: " << m_vectSmallVelV.size() << endl
			<< "size of m_vectSmallVelW: " << m_vectSmallVelW.size() << endl;
};

void FluidSimProc::FreeResource( void )
{
	for ( int i = 0; i < m_vectBulletBufs.size(); i++ )
		m_scHelper.FreeDeviceBuffers( 1, &m_vectBulletBufs[i] );

	for ( int i = 0; i < m_vectCompBufs.size(); i++ )
		m_scHelper.FreeDeviceBuffers( 1, &m_vectCompBufs[i] );

	for ( int i = 0; i < m_vectBigBufs.size(); i++ )
		m_scHelper.FreeDeviceBuffers( 1, &m_vectBigBufs[i] );

	for ( int i = 0; i < NODES_X * NODES_Y * NODES_Z; i++ )
	{
		m_scHelper.FreeDeviceBuffers( 1, &m_vectSmallDens[i] );
		m_scHelper.FreeDeviceBuffers( 1, &m_vectSmallVelU[i] );
		m_scHelper.FreeDeviceBuffers( 1, &m_vectSmallVelV[i] );
		m_scHelper.FreeDeviceBuffers( 1, &m_vectSmallVelW[i] );
	}

	m_scHelper.FreeDeviceBuffers( 1, &m_ptrDeviceVisual );
	m_scHelper.FreeHostBuffers( 1, &m_ptrHostVisual );
};

void FluidSimProc::RefreshStatus( FLUIDSPARAM *fluid )
{
	/* waiting for all kernels end */
	if ( hipDeviceSynchronize() not_eq hipSuccess )
	{
		printf( "hipDeviceSynchronize failed\n" );
		FreeResource();
		exit( 1 );
	}

	/* counting FPS */
	fluid->fps.dwFrames ++;
	fluid->fps.dwCurrentTime = GetTickCount();
	fluid->fps.dwElapsedTime = fluid->fps.dwCurrentTime - fluid->fps.dwLastUpdateTime;

	/* 1 second */
	if ( fluid->fps.dwElapsedTime >= 1000 )
	{
		fluid->fps.uFPS     = fluid->fps.dwFrames * 1000 / fluid->fps.dwElapsedTime;
		fluid->fps.dwFrames = 0;
		fluid->fps.dwLastUpdateTime = fluid->fps.dwCurrentTime;
	}

	/* updating image */
	if ( hipMemcpy( m_ptrHostVisual, m_ptrDeviceVisual, 
		VOLUME_X * VOLUME_Y * VOLUME_Z * sizeof(SGUCHAR), hipMemcpyDeviceToHost ) not_eq hipSuccess )
	{
		m_scHelper.GetCUDALastError( "host function: hipMemcpy failed", __FILE__, __LINE__ );
		FreeResource();
		exit( 1 );
	}
	fluid->volume.ptrData = m_ptrHostVisual;
};

void FluidSimProc::ClearBuffers( void )
{
	m_scHelper.DeviceParamDim( &gridDim, &blockDim, THREADS_S, 33, 22, BULLET_X, BULLET_Y, BULLET_Z );
	for ( int i = 0; i < m_vectBulletBufs.size(); i++ )
		kernelZeroBuffers __device_func__ ( m_vectBulletBufs[i], BULLET_X, BULLET_Y, BULLET_Z );

	if ( m_scHelper.GetCUDALastError( "host function failed: ZeroBuffers", __FILE__, __LINE__ ) )
	{
		FreeResource();
		exit( 1 );
	}
};

void FluidSimProc::InitBoundary( void )
{
	m_scHelper.DeviceParamDim( &gridDim, &blockDim, THREADS_S, TILE_X, TILE_Y, GRIDS_X, GRIDS_Y, GRIDS_Z );

	kernelSetBound __device_func__ ( comp_obst, GRIDS_X, GRIDS_Y, GRIDS_Z );

	kernelLoadBullet __device_func__
		( dev_obs, comp_obst,  BULLET_X, BULLET_Y, BULLET_Z, GRIDS_X, GRIDS_Y, GRIDS_Z );

	if ( m_scHelper.GetCUDALastError( "call member function InitBound failed", __FILE__, __LINE__ ) )
	{
		FreeResource();
		exit(1);
	}
};

void FluidSimProc::FluidSimSolver( FLUIDSPARAM *fluid )
{
	if ( not fluid->run ) return;
	
	SolveGlobal( DELTATIME, true, true, true );

	SolveLocal( DELTATIME/2.f, true, true );

	GenerVolumeImg();

	RefreshStatus( fluid );
};

void FluidSimProc::GenerVolumeImg( void )
{
#if 0

	m_scHelper.DeviceParamDim( &gridDim, &blockDim, THREADS_S, TILE_X, TILE_Y, GRIDS_X, GRIDS_Y, GRIDS_Z );
	for ( int i = 0; i < NODES_X; i++ ) for ( int j = 0; j < NODES_Y; j++ ) for ( int k = 0; k < NODES_Z; k++ )
	{
		kernelPickData __device_func__ ( m_ptrDeviceVisual, m_vectSmallDens[ix(i,j,k,NODES_X,NODES_Y)], 
			VOLUME_X, VOLUME_Y, VOLUME_Z,
			GRIDS_X, GRIDS_Y, GRIDS_Z, 
			i, j, k );
	}

#else

	m_scHelper.DeviceParamDim( &gridDim, &blockDim, THREADS_S, TILE_X, TILE_Y, VOLUME_X, VOLUME_Y, VOLUME_Z );
	kernelPickData __device_func__ ( m_ptrDeviceVisual, big_den, VOLUME_X, VOLUME_Y, VOLUME_Z );
	
	for ( int i = 0; i < NODES_X; i++ ) for ( int j = 0; j < NODES_Y; j++ ) for ( int k = 0; k < NODES_Z; k++ )
	{
		if ( j > 0 )
		{
			m_scHelper.DeviceParamDim
				( &gridDim, &blockDim, THREADS_S, TILE_X, TILE_Y, GRIDS_X, GRIDS_Y, GRIDS_Z );
			kernelPickData __device_func__
				( m_ptrDeviceVisual, m_vectSmallDens[ix(i, j, k, NODES_X, NODES_Y)], 
				VOLUME_X, VOLUME_Y, VOLUME_Z,
				GRIDS_X, GRIDS_Y, GRIDS_Z, 
				i, j, k );
		}
	}

#endif
};

void FluidSimProc::SolveGlobal( cdouble dt, bool add, bool vel, bool dens )
{
	if ( add ) SourceSolverGlobal( dt );
	if ( vel ) VelocitySolverGlobal( dt );
	if ( dens ) DensitySolverGlobal( dt );

	m_scHelper.DeviceParamDim( &gridDim, &blockDim, THREADS_S, TILE_X, TILE_Y, GRIDS_X, GRIDS_X, GRIDS_X );

	kernelExitBullet __device_func__
		( comp_den, dev_den, GRIDS_X, GRIDS_Y, GRIDS_Z, BULLET_X, BULLET_Y, BULLET_Z );

	kernelExitBullet __device_func__
		( comp_u, dev_u, GRIDS_X, GRIDS_Y, GRIDS_Z, BULLET_X, BULLET_Y, BULLET_Z );

	kernelExitBullet __device_func__
		( comp_v, dev_v, GRIDS_X, GRIDS_Y, GRIDS_Z, BULLET_X, BULLET_Y, BULLET_Z );

	kernelExitBullet __device_func__
		( comp_w, dev_w, GRIDS_X, GRIDS_Y, GRIDS_Z, BULLET_X, BULLET_Y, BULLET_Z );
};

void FluidSimProc::SolveLocal( cdouble dt, bool vel, bool dens )
{
	m_scHelper.DeviceParamDim( &gridDim, &blockDim, THREADS_S, TILE_X, TILE_Y, VOLUME_X, VOLUME_Y, VOLUME_Z );

	kernelUpScalingInterpolation __device_func__ ( big_den, comp_den, 
		GRIDS_X, GRIDS_Y, GRIDS_Z, 
		VOLUME_X, VOLUME_Y, VOLUME_Z,
		2, 2, 2 );

	kernelUpScalingInterpolation __device_func__ ( big_u, comp_u, 
		GRIDS_X, GRIDS_Y, GRIDS_Z, 
		VOLUME_X, VOLUME_Y, VOLUME_Z,
		2, 2, 2 );

	kernelUpScalingInterpolation __device_func__ ( big_v, comp_v, 
		GRIDS_X, GRIDS_Y, GRIDS_Z, 
		VOLUME_X, VOLUME_Y, VOLUME_Z,
		2, 2, 2 );

	kernelUpScalingInterpolation __device_func__ ( big_w, comp_w, 
		GRIDS_X, GRIDS_Y, GRIDS_Z, 
		VOLUME_X, VOLUME_Y, VOLUME_Z,
		2, 2, 2 );

#if 0

	m_scHelper.DeviceParamDim( &gridDim, &blockDim, THREADS_S, TILE_X, TILE_Y, GRIDS_X, GRIDS_Y, GRIDS_Z );

	for ( int k = 0; k < NODES_Z; k++ ) for ( int j = 0; j < NODES_Y; j++ ) for ( int i = 0; i < NODES_X; i++ )
	{
		kernelDeassemble __device_func__ ( m_vectSmallDens[ix(i,j,k,NODES_X,NODES_Y)], big_den, 
			VOLUME_X, VOLUME_Y, VOLUME_Z,
			GRIDS_X, GRIDS_Y, GRIDS_Z,
			i, j, k );
	}

//#else

	for ( int i = 0; i < NODES_X; i++ ) for ( int j = 0; j < NODES_Y; j++ ) for ( int k = 0; k < NODES_Z; k++ )
	{
		if ( j > 0 )
		{
			m_scHelper.DeviceParamDim
				( &gridDim, &blockDim, THREADS_S, TILE_X, TILE_Y, GRIDS_X, GRIDS_Y, GRIDS_Z );
			
			kernelDeassemble __device_func__ ( m_vectSmallDens[ix(i, j, k, NODES_X, NODES_Y)], big_den, 
				VOLUME_X, VOLUME_Y, VOLUME_Z,
				GRIDS_X, GRIDS_Y, GRIDS_Z,
				i, j, k );
			
			kernelDeassemble __device_func__ ( m_vectSmallVelU[ix(i, j, k, NODES_X, NODES_Y)], big_u, 
				VOLUME_X, VOLUME_Y, VOLUME_Z,
				GRIDS_X, GRIDS_Y, GRIDS_Z,
				i, j, k );

			kernelDeassemble __device_func__ ( m_vectSmallVelV[ix(i, j, k, NODES_X, NODES_Y)], big_v, 
				VOLUME_X, VOLUME_Y, VOLUME_Z,
				GRIDS_X, GRIDS_Y, GRIDS_Z,
				i, j, k );

			kernelDeassemble __device_func__ ( m_vectSmallVelW[ix(i, j, k, NODES_X, NODES_Y)], big_w, 
				VOLUME_X, VOLUME_Y, VOLUME_Z,
				GRIDS_X, GRIDS_Y, GRIDS_Z,
				i, j, k );
		}
	}
#endif

	for ( int i = 0; i < NODES_X; i++ ) for ( int j = 0; j < NODES_Y; j++ ) for ( int k = 0; k < NODES_Z; k++ )
	{
		if ( j > 0 )
		{
			m_scHelper.DeviceParamDim( &gridDim, &blockDim, THREADS_S, 33, 22, BULLET_X, BULLET_Y, BULLET_Z );

			kernelGlobalToBullet __device_func__ ( loc_den, dev_den, 
				VOLUME_X, VOLUME_Y, VOLUME_Z,
				BULLET_X, BULLET_Y, BULLET_Z,
				GRIDS_X, GRIDS_Y, GRIDS_Z,
				i, j, k );

			kernelGlobalToBullet __device_func__ ( loc_u, dev_u, 
				VOLUME_X, VOLUME_Y, VOLUME_Z,
				BULLET_X, BULLET_Y, BULLET_Z,
				GRIDS_X, GRIDS_Y, GRIDS_Z,
				i, j, k );

			kernelGlobalToBullet __device_func__ ( loc_v, dev_v, 
				VOLUME_X, VOLUME_Y, VOLUME_Z,
				BULLET_X, BULLET_Y, BULLET_Z,
				GRIDS_X, GRIDS_Y, GRIDS_Z,
				i, j, k );

			kernelGlobalToBullet __device_func__ ( loc_w, dev_w, 
				VOLUME_X, VOLUME_Y, VOLUME_Z,
				BULLET_X, BULLET_Y, BULLET_Z,
				GRIDS_X, GRIDS_Y, GRIDS_Z,
				i, j, k );

			m_scHelper.DeviceParamDim( &gridDim, &blockDim, THREADS_S, TILE_X, TILE_Y, GRIDS_X, GRIDS_Y, GRIDS_Z );

			if ( vel ) VelocitySolverLocal( dt );
			if ( dens ) DensitySolverLocal( dt );

			kernelExitBullet __device_func__ 
				( m_vectSmallDens[ix(i, j, k, NODES_X, NODES_Y)], loc_den,
				GRIDS_X, GRIDS_Y, GRIDS_Z, 
				BULLET_X, BULLET_Y, BULLET_Z );
		}
	}
};