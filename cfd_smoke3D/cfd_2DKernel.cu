#include "hip/hip_runtime.h"
/**
*
* Copyright (C) <2013> <Orlando Chen>
* Permission is hereby granted, free of charge, to any person obtaining a copy of this software and
* associated documentation files (the "Software"), to deal in the Software without restriction, 
* including without limitation the rights to use, copy, modify, merge, publish, distribute, sublicense, 
* and/or sell copies of the Software, and to permit persons to whom the Software is furnished to do so, 
* subject to the following conditions:
*
* The above copyright notice and this permission notice shall be included in all copies or substantial
* portions of the Software.
*
* THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR IMPLIED, INCLUDING BUT 
* NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.
* IN NO EVENT SHALL THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER LIABILITY, 
* WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM, OUT OF OR IN CONNECTION WITH THE
* SOFTWARE OR THE USE OR OTHER DEALINGS IN THE SOFTWARE.
 */

/**
* <Author>      Orlando Chen
* <First>       Oct 22, 2013
* <Last>		Oct 22, 2013
* <File>        cfd_kernel.cu
*/

#ifndef __cfd_kernel_cu_
#define __cfd_kernel_cu_

#include <SGE\SGUtils.h>

using namespace sge;
using namespace sge::SGTOOL;

#include <hip/hip_runtime.h>
#include <>

#include "macro_def.h"

extern FileManager logfile;
extern char * msg;

///////////////////////////////////////////////////////////////////////////////////////////////////
///

extern hipError_t cudaStatus;

__global__ void add_source_kernel ( float *ptr_out, float *ptr_in )
{
	// Get index of GPU-thread
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	int ind = index(i, j);

	// Yield value
	ptr_out[ind] += DELTA_TIME * ptr_in[ind];
};


__global__ void set_bnd_kernel ( float *grid_out, int boundary )
{
#define is ==
	// Get index of GPU-thread
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;

	// Boundary condition
	if ( i >= 1 && i <= SimArea_X && j >= 1 && j <= SimArea_X )
	{
		// Slove line (0, y)
		grid_out[index(0, j)]  = boundary is 1 ? -grid_out[index(1, j)] : grid_out[index(1, j)];
		// Slove line (65, y)
		grid_out[index(65, j)] = boundary is 1 ? -grid_out[index(64,j)] : grid_out[index(64,j)];
		// Slove line (x, 0)
		grid_out[index(i, 0)]  = boundary is 2 ? -grid_out[index(i, 1)] : grid_out[index(i, 1)];
		// Slove line (x, 65)
		grid_out[index(i, 65)] = boundary is 2 ? -grid_out[index(i,64)] : grid_out[index(i,64)];
	}
	// Slove ghost cell (0, 0)
	grid_out[index(0, 0)] = 0.5f * ( grid_out[index(1, 0)]  + grid_out[index(0, 1)] );
	// Slove ghost cell (0, 65)
	grid_out[index(0, 65)] = 0.5f * ( grid_out[index(1, 65)] + grid_out[index(0, 64)] );
	// Slove ghost cell (65, 0)
	grid_out[index(65, 0)] = 0.5f * ( grid_out[index(64, 0)] + grid_out[index(65, 1)] );
	// Slove ghost cell (65, 65)
	grid_out[index(65, 65)] = 0.5f * ( grid_out[index(64, 65)] + grid_out[index(65, 64)]);

#undef is
}


__global__ void lin_solve_kernel ( float *grid_inout, float *grid0_in, int boundary, float a, float c )
{
	// Get index of GPU-thread
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;

	if ( i >= 1 && i <= SimArea_X && j >= 1 && j <= SimArea_X )
	{	
		grid_inout[index(i,j)] = (grid0_in[index(i,j)] + a * ( grid_inout[index(i-1,j)] + 
			grid_inout[index(i+1,j)] + grid_inout[index(i,j-1)] + grid_inout[index(i,j+1)] ) ) / c;	
	}
}


__global__ void advect_kernel(float *density_out, float *density0_in, float *u_in, float *v_in, float dt0)
{
	// Get index of GPU-thread
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;

	int i0, j0, i1, j1;
	float x, y, s0, t0, s1, t1;

	if ( i >= 1 && i <= SimArea_X && j >= 1 && j <= SimArea_X )
	{
		x = i - dt0 * u_in[index(i,j)];
		y = j - dt0 * v_in[index(i,j)];
		if (x < 0.5f) x = 0.5f;
		if (x > SimArea_X + 0.5f) x = SimArea_X+0.5f;

		i0 = (int)x; 
		i1 = i0+1;
		
		if (y < 0.5f) y=0.5f;
		if (y > SimArea_X+0.5f) y = SimArea_X+0.5f;
		
		j0 = (int)y;
		j1 = j0 + 1;
		s1 = x - i0;
		s0 = 1 - s1;
		t1 = y - j0;
		t0 = 1 - t1;

		density_out[index(i,j)] = s0 * ( t0 * density0_in[index(i0,j0)] +
			t1 * density0_in[index(i0,j1)]) + s1 * ( t0 * density0_in[index(i1,j0)] + 
			t1 * density0_in[index(i1,j1)]);
	}
};


__global__ void project_kernel_pt1(float * u, float * v, float * p, float * div)
{
	// Get index of GPU-thread
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	
	if ( i >= 1 && i <= SimArea_X && j >= 1 && j <= SimArea_X )
	{
		div[index(i,j)] = -0.5f*(u[index(i+1,j)]-u[index(i-1,j)]+v[index(i,j+1)]-v[index(i,j-1)])/SimArea_X;
		p[index(i,j)] = 0;
	}
}


__global__ void project_kernel_pt2(float * u, float * v, float * p, float * div)
{
	// Get index of GPU-thread
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	
	if ( i >= 1 && i <= SimArea_X && j >= 1 && j <= SimArea_X )
	{
			u[index(i,j)] -= 0.5f*SimArea_X*(p[index(i+1,j)]-p[index(i-1,j)]);
			v[index(i,j)] -= 0.5f*SimArea_X*(p[index(i,j+1)]-p[index(i,j-1)]);
	}
}


void cuda_add_source ( float *grid, float *grid0, dim3 *grid_size, dim3 *block_size )
{
    // Launch a kernel on the GPU with one thread for each element.
	add_source_kernel cuda_device(*grid_size, *block_size) (grid, grid0);
};


void cuda_lin_solve (float *grid, float *grid0, int boundary, float a, float c, dim3 *grid_size, dim3 *block_size)
{
    // Launch a kernel on the GPU with one thread for each element.
	for (int i=0; i<20; i++)
	{
		lin_solve_kernel cuda_device(*grid_size, *block_size) (grid, grid0, boundary, a, c);
	}
	set_bnd_kernel cuda_device(*grid_size, *block_size)  (grid, boundary);
}


void cuda_diffuse ( float *grid, float *grid0, int boundary, float diff, dim3 *grid_size, dim3 *block_size )
{
	float a=DELTA_TIME*diff*SimArea_X*SimArea_X;
	cuda_lin_solve ( grid, grid0, boundary, a, 1+4*a, grid_size, block_size );
}


void cuda_advect( float *density, float *density0, float *u, float *v,  int boundary, dim3 *grid_size, dim3 *block_size )
{
    // Launch a kernel on the GPU with one thread for each element.
	float dt0 = DELTA_TIME*SimArea_X;
	advect_kernel cuda_device(*grid_size, *block_size) (density, density0, u, v, dt0);
	set_bnd_kernel cuda_device(*grid_size, *block_size) (density, boundary);
}


void cuda_project ( float * u, float * v, float * p, float * div, dim3 *grid_size, dim3 *block_size )
{
	project_kernel_pt1  cuda_device(*grid_size, *block_size)  (u, v, p, div);
	set_bnd_kernel  cuda_device(*grid_size, *block_size)  (div, 0); 
	set_bnd_kernel  cuda_device(*grid_size, *block_size)  (p, 0);
	lin_solve_kernel  cuda_device(*grid_size, *block_size)  (p, div, 0, 1, 4);
	project_kernel_pt2  cuda_device(*grid_size, *block_size)  (u, v, p, div);
	set_bnd_kernel  cuda_device(*grid_size, *block_size)  ( u, 1 );
	set_bnd_kernel  cuda_device(*grid_size, *block_size)  ( v, 2 );
}


void dens_step ( float *grid, float *grid0, float *u, float *v )
{
	// Define the computing unit size
	dim3 block_size;
	dim3 grid_size;
	block_size.x = 16;
	block_size.y = 16;
	grid_size.x  = Grids_X / block_size.x;
	grid_size.y  = Grids_X / block_size.y;

	size_t size = Grids_X * Grids_X;

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_grid, grid, size * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
		sprintf(msg, "hipMemcpy failed! %d %s", __LINE__, __FILE__);
		logfile.SetDataToFile(msg, "logfile.txt", SGFILEOPENMODE::SG_FILE_OPEN_APPEND);
    }

	cudaStatus = hipMemcpy(dev_grid0, grid0, size * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
		sprintf(msg, "hipMemcpy failed! %d %s", __LINE__, __FILE__);
		logfile.SetDataToFile(msg, "logfile.txt", SGFILEOPENMODE::SG_FILE_OPEN_APPEND);
    }

	cudaStatus = hipMemcpy(dev_u, u, size * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
		sprintf(msg, "hipMemcpy failed! %d %s", __LINE__, __FILE__);
		logfile.SetDataToFile(msg, "logfile.txt", SGFILEOPENMODE::SG_FILE_OPEN_APPEND);
    }

	cudaStatus = hipMemcpy(dev_v, v, size * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
		sprintf(msg, "hipMemcpy failed! %d %s", __LINE__, __FILE__);
		logfile.SetDataToFile(msg, "logfile.txt", SGFILEOPENMODE::SG_FILE_OPEN_APPEND);
    }


	cuda_add_source(dev_grid, dev_grid0, &grid_size, &block_size);
	swap ( dev_grid0, dev_grid ); cuda_diffuse ( dev_grid, dev_grid0, 0, DIFFUSION, &grid_size, &block_size );
	swap ( dev_grid0, dev_grid ); cuda_advect ( dev_grid, dev_grid0, dev_u, dev_v, 0, &grid_size, &block_size );
	
	
	// Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "add_source_kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		sprintf(msg, "add_source_kernel launch failed: %s %d %s", hipGetErrorString(cudaStatus), __LINE__, __FILE__);
		logfile.SetDataToFile(msg, "logfile.txt", SGFILEOPENMODE::SG_FILE_OPEN_APPEND);
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		sprintf(msg, "hipDeviceSynchronize returned error code %d after launching addKernel! %s %d %s", cudaStatus, hipGetErrorString(cudaStatus), __LINE__, __FILE__);
		logfile.SetDataToFile(msg, "logfile.txt", SGFILEOPENMODE::SG_FILE_OPEN_APPEND);
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(grid, dev_grid, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
		sprintf(msg, "hipMemcpy failed! %d %s", __LINE__, __FILE__);
		logfile.SetDataToFile(msg, "logfile.txt", SGFILEOPENMODE::SG_FILE_OPEN_APPEND);
    }

	cudaStatus = hipMemcpy(grid0, dev_grid0, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
		sprintf(msg, "hipMemcpy failed!  %d %s", __LINE__, __FILE__);
		logfile.SetDataToFile(msg, "logfile.txt", SGFILEOPENMODE::SG_FILE_OPEN_APPEND);
    }
	
	cudaStatus = hipMemcpy(u, dev_u, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
		sprintf(msg, "hipMemcpy failed!  %d %s", __LINE__, __FILE__);
		logfile.SetDataToFile(msg, "logfile.txt", SGFILEOPENMODE::SG_FILE_OPEN_APPEND);
    }

	cudaStatus = hipMemcpy(v, dev_v, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
		sprintf(msg, "hipMemcpy failed!  %d %s", __LINE__, __FILE__);
		logfile.SetDataToFile(msg, "logfile.txt", SGFILEOPENMODE::SG_FILE_OPEN_APPEND);
    }
}


void vel_step ( float * u, float * v, float * u0, float * v0 )
{
	// Define the computing unit size
	dim3 block_size;
	dim3 grid_size;
	block_size.x = 16;
	block_size.y = 16;
	grid_size.x  = Grids_X / block_size.x;
	grid_size.y  = Grids_X / block_size.y;

	size_t size = Grids_X * Grids_X;

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_u0, u0, size * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
		sprintf(msg, "hipMemcpy failed!  %d %s", __LINE__, __FILE__);
		logfile.SetDataToFile(msg, "logfile.txt", SGFILEOPENMODE::SG_FILE_OPEN_APPEND);
    }

	cudaStatus = hipMemcpy(dev_v0, v0, size * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
		sprintf(msg, "hipMemcpy failed!  %d %s", __LINE__, __FILE__);
		logfile.SetDataToFile(msg, "logfile.txt", SGFILEOPENMODE::SG_FILE_OPEN_APPEND);
    }

	cudaStatus = hipMemcpy(dev_u, u, size * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
		sprintf(msg, "hipMemcpy failed!  %d %s", __LINE__, __FILE__);
		logfile.SetDataToFile(msg, "logfile.txt", SGFILEOPENMODE::SG_FILE_OPEN_APPEND);
    }

	cudaStatus = hipMemcpy(dev_v, v, size * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
		sprintf(msg, "hipMemcpy failed! %d %s", __LINE__, __FILE__);
		logfile.SetDataToFile(msg, "logfile.txt", SGFILEOPENMODE::SG_FILE_OPEN_APPEND);
    }


	cuda_add_source ( dev_u, dev_u0, &grid_size, &block_size ); cuda_add_source ( dev_v, dev_v0, &grid_size, &block_size );
	swap ( dev_u0, dev_u ); cuda_diffuse ( dev_u, dev_u0, 1, VISCOSITY, &grid_size, &block_size );
	swap ( dev_v0, dev_v ); cuda_diffuse ( dev_v, dev_v0, 2, VISCOSITY, &grid_size, &block_size );
	cuda_project ( dev_u, dev_v, dev_u0, dev_v0, &grid_size, &block_size );
	swap ( dev_u0, dev_u ); swap ( dev_v0, dev_v );
	cuda_advect ( dev_u, dev_u0, dev_u0, dev_v0, 1, &grid_size, &block_size );
	cuda_advect ( dev_v, dev_v0, dev_u0, dev_v0, 2, &grid_size, &block_size );
	cuda_project ( dev_u, dev_v, dev_u0, dev_v0, &grid_size, &block_size );


	// Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "add_source_kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		sprintf(msg, "add_source_kernel launch failed: %s %d %s", hipGetErrorString(cudaStatus), __LINE__, __FILE__);
		logfile.SetDataToFile(msg, "logfile.txt", SGFILEOPENMODE::SG_FILE_OPEN_APPEND);
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		sprintf(msg, "hipDeviceSynchronize returned error code %d after launching addKernel! %d %s", cudaStatus, __LINE__, __FILE__);
		logfile.SetDataToFile(msg, "logfile.txt", SGFILEOPENMODE::SG_FILE_OPEN_APPEND);
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(u0, dev_u0, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
		sprintf(msg, "hipMemcpy failed!  %d %s", __LINE__, __FILE__);
		logfile.SetDataToFile(msg, "logfile.txt", SGFILEOPENMODE::SG_FILE_OPEN_APPEND);
    }

	cudaStatus = hipMemcpy(v0, dev_v0, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
		sprintf(msg, "hipMemcpy failed!  %d %s", __LINE__, __FILE__);
		logfile.SetDataToFile(msg, "logfile.txt", SGFILEOPENMODE::SG_FILE_OPEN_APPEND);
    }
	
	cudaStatus = hipMemcpy(u, dev_u, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
		sprintf(msg, "hipMemcpy failed!  %d %s", __LINE__, __FILE__);
		logfile.SetDataToFile(msg, "logfile.txt", SGFILEOPENMODE::SG_FILE_OPEN_APPEND);
    }

	cudaStatus = hipMemcpy(v, dev_v, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
		sprintf(msg, "hipMemcpy failed!  %d %s", __LINE__, __FILE__);
		logfile.SetDataToFile(msg, "logfile.txt", SGFILEOPENMODE::SG_FILE_OPEN_APPEND);
    }
}

///
///////////////////////////////////////////////////////////////////////////////////////////////////

#endif