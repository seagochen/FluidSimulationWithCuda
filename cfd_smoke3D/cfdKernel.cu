#include "hip/hip_runtime.h"
/**
*
* Copyright (C) <2013> <Orlando Chen>
* Permission is hereby granted, free of charge, to any person obtaining a copy of this software and
* associated documentation files (the "Software"), to deal in the Software without restriction, 
* including without limitation the rights to use, copy, modify, merge, publish, distribute, sublicense, 
* and/or sell copies of the Software, and to permit persons to whom the Software is furnished to do so, 
* subject to the following conditions:
*
* The above copyright notice and this permission notice shall be included in all copies or substantial
* portions of the Software.
*
* THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR IMPLIED, INCLUDING BUT 
* NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.
* IN NO EVENT SHALL THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER LIABILITY, 
* WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM, OUT OF OR IN CONNECTION WITH THE
* SOFTWARE OR THE USE OR OTHER DEALINGS IN THE SOFTWARE.
 */

/**
* <Author>      Orlando Chen
* <First>       Oct 12, 2013
* <Last>		Nov 5, 2013
* <File>        cfdKernel.cu
*/

#ifndef __cfd_2DKernel_cu_
#define __cfd_2DKernel_cu_

#include "macroDef.h"
#include "cudaHelper.h"

using namespace std;

#define cudaKernel3DIndex(i, j, k) \
	i = blockIdx.x * blockDim.x + threadIdx.x; \
	j = blockIdx.y * blockDim.y + threadIdx.y; \
	k = 0; \
	cudaTrans2DTo3D(i, j, k, Grids_X); \

#define cudaKernel2DIndex(i, j) \
	i = blockIdx.x * blockDim.x + threadIdx.x; \
	j = blockIdx.y * blockDim.y + threadIdx.y; \


__global__ void add_source_kernel ( float *ptr_out, float *ptr_in )
{
	// Get index of GPU-thread
	int i, j, k;
	cudaKernel3DIndex(i, j, k);

	if (i == Grids_X / 2 && j ==  Grids_X / 2)
	{
		int ind = cudaIndex3D(i, j, k, Grids_X);
		ptr_out[ind] += DELTA_TIME * 1.f;
	}
};


__global__ void set_bnd_kernel ( float *grid_out, int boundary )
{
#define is ==
	// Get index of GPU-thread
	int i, j, k;
	cudaKernel3DIndex(i, j, k);

	// Boundary condition
	if ( i >= 1 && i <= SimArea_X && j >= 1 && j <= SimArea_X )
	{
		// Slove line (0, y)
		grid_out[cudaIndex2D(0, j, Grids_X)]  = boundary is 1 ? -grid_out[cudaIndex2D(1, j, Grids_X)] : grid_out[cudaIndex2D(1, j, Grids_X)];
		// Slove line (65, y)
		grid_out[cudaIndex2D(65, j, Grids_X)] = boundary is 1 ? -grid_out[cudaIndex2D(64,j, Grids_X)] : grid_out[cudaIndex2D(64,j, Grids_X)];
		// Slove line (x, 0)
		grid_out[cudaIndex2D(i, 0, Grids_X)]  = boundary is 2 ? -grid_out[cudaIndex2D(i, 1, Grids_X)] : grid_out[cudaIndex2D(i, 1, Grids_X)];
		// Slove line (x, 65)
		grid_out[cudaIndex2D(i, 65, Grids_X)] = boundary is 2 ? -grid_out[cudaIndex2D(i,64, Grids_X)] : grid_out[cudaIndex2D(i,64, Grids_X)];
	}
	// Slove ghost cell (0, 0)
	grid_out[cudaIndex2D(0, 0, Grids_X)] = 0.5f * ( grid_out[cudaIndex2D(1, 0, Grids_X)]  + grid_out[cudaIndex2D(0, 1, Grids_X)] );
	// Slove ghost cell (0, 65)
	grid_out[cudaIndex2D(0, 65, Grids_X)] = 0.5f * ( grid_out[cudaIndex2D(1, 65, Grids_X)] + grid_out[cudaIndex2D(0, 64, Grids_X)] );
	// Slove ghost cell (65, 0)
	grid_out[cudaIndex2D(65, 0, Grids_X)] = 0.5f * ( grid_out[cudaIndex2D(64, 0, Grids_X)] + grid_out[cudaIndex2D(65, 1, Grids_X)] );
	// Slove ghost cell (65, 65)
	grid_out[cudaIndex2D(65, 65, Grids_X)] = 0.5f * ( grid_out[cudaIndex2D(64, 65, Grids_X)] + grid_out[cudaIndex2D(65, 64, Grids_X)]);

#undef is
}


__global__ void lin_solve_kernel ( float *grid_inout, float *grid0_in, int boundary, float a, float c )
{
	// Get index of GPU-thread
	int i, j, k;
	cudaKernel2DIndex(i, j);

	if ( i >= 1 && i <= SimArea_X && j >= 1 && j <= SimArea_X )
	{	
		grid_inout[cudaIndex2D(i,j,Grids_X)] = (grid0_in[cudaIndex2D(i,j,Grids_X)] + a * ( grid_inout[cudaIndex2D(i-1,j,Grids_X)] + 
			grid_inout[cudaIndex2D(i+1,j,Grids_X)] + grid_inout[cudaIndex2D(i,j-1,Grids_X)] + grid_inout[cudaIndex2D(i,j+1,Grids_X)] ) ) / c;	
	}
}


__global__ void advect_kernel(float *density_out, float *density0_in, float *u_in, float *v_in, float dt0)
{
	// Get index of GPU-thread
	int i, j, k;
	cudaKernel2DIndex(i, j);

	int i0, j0, i1, j1;
	float x, y, s0, t0, s1, t1;

	if ( i >= 1 && i <= SimArea_X && j >= 1 && j <= SimArea_X )
	{
		x = i - dt0 * u_in[cudaIndex2D(i,j,Grids_X)];
		y = j - dt0 * v_in[cudaIndex2D(i,j,Grids_X)];
		if (x < 0.5f) x = 0.5f;
		if (x > SimArea_X + 0.5f) x = SimArea_X+0.5f;

		i0 = (int)x; 
		i1 = i0+1;
		
		if (y < 0.5f) y=0.5f;
		if (y > SimArea_X+0.5f) y = SimArea_X+0.5f;
		
		j0 = (int)y;
		j1 = j0 + 1;
		s1 = x - i0;
		s0 = 1 - s1;
		t1 = y - j0;
		t0 = 1 - t1;

		density_out[cudaIndex2D(i,j,Grids_X)] = s0 * ( t0 * density0_in[cudaIndex2D(i0,j0,Grids_X)] +
			t1 * density0_in[cudaIndex2D(i0,j1,Grids_X)]) + s1 * ( t0 * density0_in[cudaIndex2D(i1,j0,Grids_X)] + 
			t1 * density0_in[cudaIndex2D(i1,j1,Grids_X)]);
	}
};


__global__ void project_kernel_pt1(float * u, float * v, float * p, float * div)
{
	// Get index of GPU-thread
	int i, j, k;
	cudaKernel2DIndex(i, j);
	
	if ( i >= 1 && i <= SimArea_X && j >= 1 && j <= SimArea_X )
	{
		div[cudaIndex2D(i,j,Grids_X)] = -0.5f*(u[cudaIndex2D(i+1,j,Grids_X)]-u[cudaIndex2D(i-1,j,Grids_X)]+
			v[cudaIndex2D(i,j+1,Grids_X)]-v[cudaIndex2D(i,j-1,Grids_X)])/SimArea_X;
		p[cudaIndex2D(i,j,Grids_X)] = 0;
	}
}


__global__ void project_kernel_pt2(float * u, float * v, float * p, float * div)
{
	// Get index of GPU-thread
	int i, j, k;
	cudaKernel2DIndex(i, j);
	
	if ( i >= 1 && i <= SimArea_X && j >= 1 && j <= SimArea_X )
	{
			u[cudaIndex2D(i,j,Grids_X)] -= 0.5f*SimArea_X*(p[cudaIndex2D(i+1,j,Grids_X)]-p[cudaIndex2D(i-1,j,Grids_X)]);
			v[cudaIndex2D(i,j,Grids_X)] -= 0.5f*SimArea_X*(p[cudaIndex2D(i,j+1,Grids_X)]-p[cudaIndex2D(i,j-1,Grids_X)]);
	}
}


void cuda_add_source ( float *grid, float *grid0, dim3 *gridDim, dim3 *blockDim )
{
    // Launch a kernel on the GPU with one thread for each element.
	add_source_kernel cudaDevice(*gridDim,  *blockDim) (grid, grid0);
};


void cuda_lin_solve (float *grid, float *grid0, int boundary, float a, float c, dim3 *gridDim, dim3 *blockDim)
{
    // Launch a kernel on the GPU with one thread for each element.
	for (int i=0; i<20; i++)
	{
		lin_solve_kernel cudaDevice(*gridDim,  *blockDim) (grid, grid0, boundary, a, c);
	}
	set_bnd_kernel cudaDevice(*gridDim,  *blockDim)  (grid, boundary);
}


void cuda_diffuse ( float *grid, float *grid0, int boundary, float diff, dim3 *gridDim, dim3 *blockDim )
{
	float a=DELTA_TIME*diff*SimArea_X*SimArea_X;
	cuda_lin_solve ( grid, grid0, boundary, a, 1+4*a, gridDim, blockDim );
}


void cuda_advect( float *density, float *density0, float *u, float *v,  int boundary, dim3 *gridDim, dim3 *blockDim )
{
    // Launch a kernel on the GPU with one thread for each element.
	float dt0 = DELTA_TIME*SimArea_X;
	advect_kernel cudaDevice(*gridDim,  *blockDim) (density, density0, u, v, dt0);
	set_bnd_kernel cudaDevice(*gridDim,  *blockDim) (density, boundary);
}


void cuda_project ( float * u, float * v, float * p, float * div, dim3 *gridDim, dim3 *blockDim )
{
	project_kernel_pt1  cudaDevice(*gridDim,  *blockDim)  (u, v, p, div);
	set_bnd_kernel  cudaDevice(*gridDim,  *blockDim)  (div, 0); 
	set_bnd_kernel  cudaDevice(*gridDim,  *blockDim)  (p, 0);
	lin_solve_kernel  cudaDevice(*gridDim,  *blockDim)  (p, div, 0, 1, 4);
	project_kernel_pt2  cudaDevice(*gridDim,  *blockDim)  (u, v, p, div);
	set_bnd_kernel  cudaDevice(*gridDim,  *blockDim)  ( u, 1 );
	set_bnd_kernel  cudaDevice(*gridDim,  *blockDim)  ( v, 2 );
}


void dens_step ( float *grid, float *grid0, float *u, float *v )
{
	// Define the computing unit size
	cudaDeviceDim2D();

	size_t size = Grids_X * Grids_X;

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_grid, grid, size * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
		Logfile.SaveStringToFile("errormsg.log", sge::SG_FILE_OPEN_APPEND, "hipMemcpy was failed, at line: %d of file %s", __LINE__, __FILE__);
		Logfile.SaveStringToFile("errormsg.log", sge::SG_FILE_OPEN_APPEND, ">>>> Error Message: %s", hipGetErrorString(cudaStatus));
		exit(0);
    }

	cudaStatus = hipMemcpy(dev_grid0, grid0, size * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
		Logfile.SaveStringToFile("errormsg.log", sge::SG_FILE_OPEN_APPEND, "hipMemcpy was failed, at line: %d of file %s", __LINE__, __FILE__);
		Logfile.SaveStringToFile("errormsg.log", sge::SG_FILE_OPEN_APPEND, ">>>> Error Message: %s", hipGetErrorString(cudaStatus));
		exit(0);
    }

	cudaStatus = hipMemcpy(dev_u, u, size * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
		Logfile.SaveStringToFile("errormsg.log", sge::SG_FILE_OPEN_APPEND, "hipMemcpy was failed, at line: %d of file %s", __LINE__, __FILE__);
		Logfile.SaveStringToFile("errormsg.log", sge::SG_FILE_OPEN_APPEND, ">>>> Error Message: %s", hipGetErrorString(cudaStatus));
		exit(0);    
	}

	cudaStatus = hipMemcpy(dev_v, v, size * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
		Logfile.SaveStringToFile("errormsg.log", sge::SG_FILE_OPEN_APPEND, "hipMemcpy was failed, at line: %d of file %s", __LINE__, __FILE__);
		Logfile.SaveStringToFile("errormsg.log", sge::SG_FILE_OPEN_APPEND, ">>>> Error Message: %s", hipGetErrorString(cudaStatus));
		exit(0);
    }


	cuda_add_source(dev_grid, dev_grid0, &gridDim, &blockDim);
	swap ( dev_grid0, dev_grid ); cuda_diffuse ( dev_grid, dev_grid0, 0, DIFFUSION, &gridDim, &blockDim );
	swap ( dev_grid0, dev_grid ); cuda_advect ( dev_grid, dev_grid0, dev_u, dev_v, 0, &gridDim, &blockDim );
	
	
	// Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
		Logfile.SaveStringToFile("errormsg.log", sge::SG_FILE_OPEN_APPEND, "CUDA encountered an error, at line: %d of file %s", __LINE__, __FILE__);
		Logfile.SaveStringToFile("errormsg.log", sge::SG_FILE_OPEN_APPEND, ">>>> Error Message: %s", hipGetErrorString(cudaStatus));
		exit(0);
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
		Logfile.SaveStringToFile("errormsg.log", sge::SG_FILE_OPEN_APPEND, "hipDeviceSynchronize was failed, at line: %d of file %s", __LINE__, __FILE__);
		Logfile.SaveStringToFile("errormsg.log", sge::SG_FILE_OPEN_APPEND, ">>>> Error Message: %s", hipGetErrorString(cudaStatus));
		exit(0);
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(grid, dev_grid, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
		Logfile.SaveStringToFile("errormsg.log", sge::SG_FILE_OPEN_APPEND, "hipMemcpy was failed, at line: %d of file %s", __LINE__, __FILE__);
		Logfile.SaveStringToFile("errormsg.log", sge::SG_FILE_OPEN_APPEND, ">>>> Error Message: %s", hipGetErrorString(cudaStatus));
		exit(0);
    }

	cudaStatus = hipMemcpy(grid0, dev_grid0, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
		Logfile.SaveStringToFile("errormsg.log", sge::SG_FILE_OPEN_APPEND, "hipMemcpy was failed, at line: %d of file %s", __LINE__, __FILE__);
		Logfile.SaveStringToFile("errormsg.log", sge::SG_FILE_OPEN_APPEND, ">>>> Error Message: %s", hipGetErrorString(cudaStatus));
		exit(0);
	}
	
	cudaStatus = hipMemcpy(u, dev_u, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
		Logfile.SaveStringToFile("errormsg.log", sge::SG_FILE_OPEN_APPEND, "hipMemcpy was failed, at line: %d of file %s", __LINE__, __FILE__);
		Logfile.SaveStringToFile("errormsg.log", sge::SG_FILE_OPEN_APPEND, ">>>> Error Message: %s", hipGetErrorString(cudaStatus));
		exit(0);
	}

	cudaStatus = hipMemcpy(v, dev_v, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
		Logfile.SaveStringToFile("errormsg.log", sge::SG_FILE_OPEN_APPEND, "hipMemcpy was failed, at line: %d of file %s", __LINE__, __FILE__);
		Logfile.SaveStringToFile("errormsg.log", sge::SG_FILE_OPEN_APPEND, ">>>> Error Message: %s", hipGetErrorString(cudaStatus));
		exit(0);
	}
}


void vel_step ( float * u, float * v, float * u0, float * v0 )
{
	// Define the computing unit size
	cudaDeviceDim2D();

	size_t size = Grids_X * Grids_X;

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_u0, u0, size * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
		Logfile.SaveStringToFile("errormsg.log", sge::SG_FILE_OPEN_APPEND, "hipMemcpy was failed, at line: %d of file %s", __LINE__, __FILE__);
		Logfile.SaveStringToFile("errormsg.log", sge::SG_FILE_OPEN_APPEND, ">>>> Error Message: %s", hipGetErrorString(cudaStatus));
		exit(0);
    }

	cudaStatus = hipMemcpy(dev_v0, v0, size * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
		Logfile.SaveStringToFile("errormsg.log", sge::SG_FILE_OPEN_APPEND, "hipMemcpy was failed, at line: %d of file %s", __LINE__, __FILE__);
		Logfile.SaveStringToFile("errormsg.log", sge::SG_FILE_OPEN_APPEND, ">>>> Error Message: %s", hipGetErrorString(cudaStatus));
		exit(0);
    }

	cudaStatus = hipMemcpy(dev_u, u, size * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
		Logfile.SaveStringToFile("errormsg.log", sge::SG_FILE_OPEN_APPEND, "hipMemcpy was failed, at line: %d of file %s", __LINE__, __FILE__);
		Logfile.SaveStringToFile("errormsg.log", sge::SG_FILE_OPEN_APPEND, ">>>> Error Message: %s", hipGetErrorString(cudaStatus));
		exit(0);
    }

	cudaStatus = hipMemcpy(dev_v, v, size * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
		Logfile.SaveStringToFile("errormsg.log", sge::SG_FILE_OPEN_APPEND, "hipMemcpy was failed, at line: %d of file %s", __LINE__, __FILE__);
		Logfile.SaveStringToFile("errormsg.log", sge::SG_FILE_OPEN_APPEND, ">>>> Error Message: %s", hipGetErrorString(cudaStatus));
		exit(0);
    }


	cuda_add_source ( dev_u, dev_u0, &gridDim, &blockDim ); cuda_add_source ( dev_v, dev_v0, &gridDim, &blockDim );
	swap ( dev_u0, dev_u ); cuda_diffuse ( dev_u, dev_u0, 1, VISCOSITY, &gridDim, &blockDim );
	swap ( dev_v0, dev_v ); cuda_diffuse ( dev_v, dev_v0, 2, VISCOSITY, &gridDim, &blockDim );
	cuda_project ( dev_u, dev_v, dev_u0, dev_v0, &gridDim, &blockDim );
	swap ( dev_u0, dev_u ); swap ( dev_v0, dev_v );
	cuda_advect ( dev_u, dev_u0, dev_u0, dev_v0, 1, &gridDim, &blockDim );
	cuda_advect ( dev_v, dev_v0, dev_u0, dev_v0, 2, &gridDim, &blockDim );
	cuda_project ( dev_u, dev_v, dev_u0, dev_v0, &gridDim, &blockDim );


	// Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
		Logfile.SaveStringToFile("errormsg.log", sge::SG_FILE_OPEN_APPEND, "CUDA encountered an error, at line: %d of file %s", __LINE__, __FILE__);
		Logfile.SaveStringToFile("errormsg.log", sge::SG_FILE_OPEN_APPEND, ">>>> Error Message: %s", hipGetErrorString(cudaStatus));
		exit(0);
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
		Logfile.SaveStringToFile("errormsg.log", sge::SG_FILE_OPEN_APPEND, "hipDeviceSynchronize was failed, at line: %d of file %s", __LINE__, __FILE__);
		Logfile.SaveStringToFile("errormsg.log", sge::SG_FILE_OPEN_APPEND, ">>>> Error Message: %s", hipGetErrorString(cudaStatus));
		exit(0);
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(u0, dev_u0, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
		Logfile.SaveStringToFile("errormsg.log", sge::SG_FILE_OPEN_APPEND, "hipMemcpy was failed, at line: %d of file %s", __LINE__, __FILE__);
		Logfile.SaveStringToFile("errormsg.log", sge::SG_FILE_OPEN_APPEND, ">>>> Error Message: %s", hipGetErrorString(cudaStatus));
		exit(0);
    }

	cudaStatus = hipMemcpy(v0, dev_v0, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
		Logfile.SaveStringToFile("errormsg.log", sge::SG_FILE_OPEN_APPEND, "hipMemcpy was failed, at line: %d of file %s", __LINE__, __FILE__);
		Logfile.SaveStringToFile("errormsg.log", sge::SG_FILE_OPEN_APPEND, ">>>> Error Message: %s", hipGetErrorString(cudaStatus));
		exit(0);
    }
	
	cudaStatus = hipMemcpy(u, dev_u, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
		Logfile.SaveStringToFile("errormsg.log", sge::SG_FILE_OPEN_APPEND, "hipMemcpy was failed, at line: %d of file %s", __LINE__, __FILE__);
		Logfile.SaveStringToFile("errormsg.log", sge::SG_FILE_OPEN_APPEND, ">>>> Error Message: %s", hipGetErrorString(cudaStatus));
		exit(0);
    }

	cudaStatus = hipMemcpy(v, dev_v, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
		Logfile.SaveStringToFile("errormsg.log", sge::SG_FILE_OPEN_APPEND, "hipMemcpy was failed, at line: %d of file %s", __LINE__, __FILE__);
		Logfile.SaveStringToFile("errormsg.log", sge::SG_FILE_OPEN_APPEND, ">>>> Error Message: %s", hipGetErrorString(cudaStatus));
		exit(0);
    }
}

#endif