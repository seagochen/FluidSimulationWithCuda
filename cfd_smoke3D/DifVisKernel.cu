#include "hip/hip_runtime.h"
/**
*
* Copyright (C) <2013> <Orlando Chen>
* Permission is hereby granted, free of charge, to any person obtaining a copy of this software and
* associated documentation files (the "Software"), to deal in the Software without restriction, 
* including without limitation the rights to use, copy, modify, merge, publish, distribute, sublicense, 
* and/or sell copies of the Software, and to permit persons to whom the Software is furnished to do so, 
* subject to the following conditions:
*
* The above copyright notice and this permission notice shall be included in all copies or substantial
* portions of the Software.
*
* THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR IMPLIED, INCLUDING BUT 
* NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.
* IN NO EVENT SHALL THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER LIABILITY, 
* WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM, OUT OF OR IN CONNECTION WITH THE
* SOFTWARE OR THE USE OR OTHER DEALINGS IN THE SOFTWARE.
 */

/**
* <Author>      Orlando Chen
* <First>       Nov 25, 2013
* <Last>		Nov 25, 2013
* <File>        DifVisKernel.cu
*/

#ifndef __diffuse_viscosity_kernel_cu_
#define __diffuse_viscosity_kernel_cu_

#include "cfdHeader.h"

extern void cudaSetBoundary ( float *grid_out, int boundary, dim3 *gridDim, dim3 *blockDim );

/*
-----------------------------------------------------------------------------------------------------------
* @function kernelDiffuse
* @author   Orlando Chen
* @date     Nov 25, 2013
* @input    float *grid_out, float const *grid_in
* @return   NULL
* @bref     To diffuse (smooth) the simulation result
-----------------------------------------------------------------------------------------------------------
*/
__global__ void kernelLineSolver ( float *grid_out, float const *grid_in, float const ratio, float const div )
{
	// Get index of GPU-thread
	GetIndex ( );

	BeginSimArea ( );
	{
		grid_out [ Index(i, j, k) ] = (grid_in [ Index(i, j, k) ] + ratio * ( grid_out [ Index(i-1, j, k) ] +  grid_out [ Index( i+1, j, k) ] +
			grid_out [ Index(i, j-1, k) ] + grid_out [ Index(i, j+1, k) ] )) / div;
	}
	EndSimArea ( );
};


/*
-----------------------------------------------------------------------------------------------------------
* @function cudaViscosity
* @author   Orlando Chen
* @date     Nov 25, 2013
* @input    float *grid_out, float const *grid_in, int boundary, dim3 *gridDim, dim3 *blockDim
* @return   NULL
* @bref     Encapsulation the CUDA routine (diffuse)
-----------------------------------------------------------------------------------------------------------
*/
__host__ void cudaViscosity ( float *grid_out, float const *grid_in, int boundary, dim3 *gridDim, dim3 *blockDim )
{
	float ratio = DELTA_TIME * VISCOSITY * SimArea_X * SimArea_X;
	float div   = 1.f + 4.f * ratio;
	for ( int i = 0; i < 20; i++ )
	{
		kernelLineSolver cudaDevice(*gridDim, *blockDim) ( grid_out, grid_in, ratio, div );
		cudaSetBoundary  ( grid_out, boundary, gridDim, blockDim );
	}
};


/*
-----------------------------------------------------------------------------------------------------------
* @function cudaDiffuse
* @author   Orlando Chen
* @date     Nov 25, 2013
* @input    float *grid_out, float const *grid_in, int boundary, dim3 *gridDim, dim3 *blockDim
* @return   NULL
* @bref     Encapsulation the CUDA routine (diffuse)
-----------------------------------------------------------------------------------------------------------
*/
__host__ void cudaDiffuse ( float *grid_out, float const *grid_in, int boundary, dim3 *gridDim, dim3 *blockDim )
{
	float ratio = DELTA_TIME * DIFFUSION * SimArea_X * SimArea_X;
	float div   = 1.f + 4.f * ratio;

	for ( int i = 0; i < 20; i++ )
	{
		kernelLineSolver cudaDevice(*gridDim, *blockDim) ( grid_out, grid_in, ratio, div );
		cudaSetBoundary  ( grid_out, boundary, gridDim, blockDim );
	}
};

#endif