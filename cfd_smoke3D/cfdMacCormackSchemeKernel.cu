#include "hip/hip_runtime.h"
/**
*
* Copyright (C) <2013> <Orlando Chen>
* Permission is hereby granted, free of charge, to any person obtaining a copy of this software and
* associated documentation files (the "Software"), to deal in the Software without restriction, 
* including without limitation the rights to use, copy, modify, merge, publish, distribute, sublicense, 
* and/or sell copies of the Software, and to permit persons to whom the Software is furnished to do so, 
* subject to the following conditions:
*
* The above copyright notice and this permission notice shall be included in all copies or substantial
* portions of the Software.
*
* THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR IMPLIED, INCLUDING BUT 
* NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.
* IN NO EVENT SHALL THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER LIABILITY, 
* WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM, OUT OF OR IN CONNECTION WITH THE
* SOFTWARE OR THE USE OR OTHER DEALINGS IN THE SOFTWARE.
 */

/**
* <Author>      Orlando Chen
* <First>       Nov 19, 2013
* <Last>		Nov 20, 2013
* <File>        cfdMacCormackSchemeKernel.cu
*/

#ifndef __cfd_MacCormack_Scheme_Kernel_cu_
#define __cfd_MacCormack_Scheme_Kernel_cu_

#include "cfdHeader.h"

/*
  -----------------------------------------------------------------------------------------------------------
   Define something
  -----------------------------------------------------------------------------------------------------------
*/

#define eqt              ==            /* equal to */
#define and              &&            /* logical and */
#define or               ||            /* logical or */

#define gst_header       0             /* (ghost, halo) the header cell of grid */
#define sim_header       1             /* (actually) the second cell of grid */
#define gst_tailer       Grids_X - 1   /* (ghost, halo) the last cell of grid */
#define sim_tailer       Grids_X - 2   /* (actually) the second last cell of grid */

#define BeginSimArea() \
	if ( i >= sim_header and i <= sim_tailer ) \
	if ( j >= sim_header and j <= sim_tailer ) \
	if ( k >= sim_header and k <= sim_tailer ) {

#define EndSimArea() }


/*
-----------------------------------------------------------------------------------------------------------
* @function kernelAddSourceMacCormack
* @author   Orlando Chen
* @date     Nov 19, 2013
* @input    float *density_inout, float *velU_inout, float *velV_inout, float *velW_inout
* @return   NULL
* @bref     Add source to simulation grid      
-----------------------------------------------------------------------------------------------------------
*/
__global__ void kernelAddSourceMacCormack ( float *density_inout, float *velU_inout, float *velV_inout, float *velW_inout )
{
	// Get index of GPU-thread
	GetIndex ( );

	// Coordinates arround the (64, 64, 64), r is 5
	if ( i > 54 and i < 74 ) if ( k > 54 and k < 74 )
	{
		int x = i - 64;
		int y = k - 64;
		float r = sqrtf ( x * x + y * y );
		
		// Within the correct distance
		if ( r >= 0 && r <= 5  )
		{
			// Add source from layer 0 - 4
			if ( j < 5 )
			{
				density_inout [ Index (i, j, k) ] += SOURCE * DELTA_TIME;
				velV_inout [ Index (i, j, k) ] = SOURCE * DELTA_TIME;
			}
		}
	}
};


/*
-----------------------------------------------------------------------------------------------------------
* @function cudaAddSourceMacCormack
* @author   Orlando Chen
* @date     Nov 22, 2013
* @input    float *density_inout, float *velU_inout, float *velV_inout, float *velW_inout
* @return   NULL
* @bref     Encapsulation of kernelAddSourceMacCormack    
-----------------------------------------------------------------------------------------------------------
*/
__host__ void cudaAddSourceMacCormack ( float *density_inout, float *velU_inout, float *velV_inout, float *velW_inout )
{
	// Define the computing unit size
	cudaDeviceDim3D ( );

	kernelAddSourceMacCormack <<< gridDim, blockDim >>> 
		( density_inout, velU_inout, velV_inout, velW_inout );
};


/*
-----------------------------------------------------------------------------------------------------------
* @function subkernelBoundaryMacCormack
* @author   Orlando Chen
* @date     Nov 22, 2013
* @input    float *density_inout, float *velU_inout, float *velV_inout, float *velW_inout
* @return   NULL
* @bref     Check and set boundary condition      
-----------------------------------------------------------------------------------------------------------
*/
__device__ void subkernelBoundaryMacCormack ( float *density_inout, float *velU_inout, float *velV_inout, float *velW_inout )
{
	// Get index of GPU-thread
	GetIndex ( );

	// Boundary condition
	BeginSimArea();
	{
		// Solve the condition of density
		density_inout [ Index (gst_header, j, k) ] = density_inout [ Index (sim_header, j, k) ]; // bottom side of simulation grid
		density_inout [ Index (gst_tailer, j, k) ] = density_inout [ Index (sim_tailer, j, k) ]; // top side of simulation grid
		density_inout [ Index (i, gst_header, k) ] = density_inout [ Index (i, sim_header, k) ]; // left side of simulation grid
		density_inout [ Index (i, gst_tailer, k) ] = density_inout [ Index (i, sim_tailer, k) ]; // right side of simulation grid
		density_inout [ Index (i, j, gst_header) ] = density_inout [ Index (i, j, sim_header) ]; // front side of simulation grid
		density_inout [ Index (i, j, gst_tailer) ] = density_inout [ Index (i, j, sim_tailer) ]; // back side of simulation grid

		// Solve the condition of sub-flow U
		velU_inout [ Index (gst_header, j, k) ] = -velU_inout [ Index (sim_header, j, k) ]; // bottom side of simulation grid
		velU_inout [ Index (gst_tailer, j, k) ] = -velU_inout [ Index (sim_tailer, j, k) ]; // top side of simulation grid
		velU_inout [ Index (i, gst_header, k) ] =  velU_inout [ Index (i, sim_header, k) ]; // left side of simulation grid
		velU_inout [ Index (i, gst_tailer, k) ] =  velU_inout [ Index (i, sim_tailer, k) ]; // right side of simulation grid
		velU_inout [ Index (i, j, gst_header) ] =  velU_inout [ Index (i, j, sim_header) ]; // front side of simulation grid
		velU_inout [ Index (i, j, gst_tailer) ] =  velU_inout [ Index (i, j, sim_tailer) ]; // back side of simulation grid
		
		// Solve the condition of sub-flow v
		velV_inout [ Index (gst_header, j, k) ] =  velV_inout [ Index (sim_header, j, k) ]; // bottom side of simulation grid
		velV_inout [ Index (gst_tailer, j, k) ] =  velV_inout [ Index (sim_tailer, j, k) ]; // top side of simulation grid
		velV_inout [ Index (i, gst_header, k) ] = -velV_inout [ Index (i, sim_header, k) ]; // left side of simulation grid
		velV_inout [ Index (i, gst_tailer, k) ] =  velV_inout [ Index (i, sim_tailer, k) ]; // right side of simulation grid
		velV_inout [ Index (i, j, gst_header) ] =  velV_inout [ Index (i, j, sim_header) ]; // front side of simulation grid
		velV_inout [ Index (i, j, gst_tailer) ] =  velV_inout [ Index (i, j, sim_tailer) ]; // back side of simulation grid

		// Solve the condition of sub-flow w
		velW_inout [ Index (gst_header, j, k) ] =  velW_inout [ Index (sim_header, j, k) ]; // bottom side of simulation grid
		velW_inout [ Index (gst_tailer, j, k) ] =  velW_inout [ Index (sim_tailer, j, k) ]; // top side of simulation grid
		velW_inout [ Index (i, gst_header, k) ] =  velW_inout [ Index (i, sim_header, k) ]; // left side of simulation grid
		velW_inout [ Index (i, gst_tailer, k) ] =  velW_inout [ Index (i, sim_tailer, k) ]; // right side of simulation grid
		velW_inout [ Index (i, j, gst_header) ] = -velW_inout [ Index (i, j, sim_header) ]; // front side of simulation grid
		velW_inout [ Index (i, j, gst_tailer) ] = -velW_inout [ Index (i, j, sim_tailer) ]; // back side of simulation grid
	}
	EndSimArea();

	// Condition of density
	density_inout [ Index (i, gst_tailer, k) ] *= ANNIHILATION;
};


/*
-----------------------------------------------------------------------------------------------------------
* @function subkernelAdvectDensity
* @author   Orlando Chen
* @date     Nov 22, 2013
* @input    float *den_out, float *den_in, float *velU_in, float *velV_in, float *velW_in, int opt
* @return   NULL
* @bref     Update density state.
* --------- When opt is 0, the prediction is calculated, opt is 1, the checksum value is calculated
-----------------------------------------------------------------------------------------------------------
*/
__device__ void subkernelAdvectDensity ( float *den_out, float *den_in, float *velU_in, float *velV_in, float *velW_in, int opt )
{
	GetIndex();

	BeginSimArea();
	{

	}
	EndSimArea();
};


/*
-----------------------------------------------------------------------------------------------------------
* @function subkernelAdvectFlow
* @author   Orlando Chen
* @date     Nov 22, 2013
* @input    float *vel_out, float *den_in, float *velU_in, float *velV_in, float *velW_in, int opt
* @return   NULL
* @bref     Update a component of flow state
* --------- When opt is 0, the prediction is calculated, opt is 1, the checksum value is calculated
-----------------------------------------------------------------------------------------------------------
*/
__device__ void subkernelAdvectFlow ( float *vel_out, float *den_in, float *velU_in, float *velV_in, float *velW_in, int opt )
{
	GetIndex();

	BeginSimArea();
	{

	}
	EndSimArea();
};


/*
-----------------------------------------------------------------------------------------------------------
* @function kernelPredicateAdvectionMacCormack
* @author   Orlando Chen
* @date     Nov 22, 2013
* @input    float *den_out, float *velU_out, float *velV_out, float *velW_out, 
* --------- float *den_in, float *velU_in, float *velV_in,float *velW_in
* @return   NULL
* @bref     Update flow status      
-----------------------------------------------------------------------------------------------------------
*/
__global__ void kernelPredicateAdvectionMacCormack ( float *den_out, float *velU_out, float *velV_out, float *velW_out, 
										float *den_in, float *velU_in, float *velV_in,float *velW_in )
{
	subkernelAdvectFlow ( velU_out, den_in, velU_in, velV_in, velW_in, 0 );
	subkernelAdvectFlow ( velV_out, den_in, velU_in, velV_in, velW_in, 0 );
	subkernelAdvectFlow ( velW_out, den_in, velU_in, velV_in, velW_in, 0 );
	subkernelAdvectDensity ( den_out, den_in, velU_in, velV_in, velW_in, 0 );
};


/*
-----------------------------------------------------------------------------------------------------------
* @function kernelChecksumAdvectionMacCormack
* @author   Orlando Chen
* @date     Nov 22, 2013
* @input    float *den_out, float *velU_out, float *velV_out, float *velW_out, 
* --------- float *den_in, float *velU_in, float *velV_in,float *velW_in
* @return   NULL
* @bref     Update flow status      
-----------------------------------------------------------------------------------------------------------
*/
__global__ void kernelChecksumAdvectionMacCormack ( float *den_out, float *velU_out, float *velV_out, float *velW_out, 
										float *den_in, float *velU_in, float *velV_in,float *velW_in )
{
	subkernelAdvectFlow ( velU_in, den_out, velU_out, velV_out, velW_out, 1 );
	subkernelAdvectFlow ( velV_in, den_out, velU_out, velV_out, velW_out, 1 );
	subkernelAdvectFlow ( velW_in, den_out, velU_out, velV_out, velW_out, 1 );
	subkernelAdvectDensity ( den_in, den_out, velU_out, velV_out, velW_out, 1 );
};


__global__ void kernelFinalAdvectionMacCormack ( float *den_out, float *velU_out, float *velV_out, float *velW_out, 
										float *den_in, float *velU_in, float *velV_in,float *velW_in )
{
	GetIndex ( );

	den_out [ Index(i,j,k) ] = 0.5f * ( den_out [ Index(i,j,k) ] + den_in [ Index(i,j,k) ] );
	velU_out [ Index(i,j,k) ] = 0.5f * ( velU_out [ Index(i,j,k) ] + velU_in [ Index(i,j,k) ] );
	velV_out [ Index(i,j,k) ] = 0.5f * ( velV_out [ Index(i,j,k) ] + velV_in [ Index(i,j,k) ] );
	velW_out [ Index(i,j,k) ] = 0.5f * ( velW_out [ Index(i,j,k) ] + velW_in [ Index(i,j,k) ] );

	// Check boundary condition
	subkernelBoundaryMacCormack ( den_out, velU_out, velV_out, velW_out );
};

/*
-----------------------------------------------------------------------------------------------------------
* @function cudaAdvectMacCormack
* @author   Orlando Chen
* @date     Nov 22, 2013
* @input    float *den_out, float *velU_out, float *velV_out, float *velW_out, 
* --------- float *den_in, float *velU_in, float *velV_in,float *velW_in
* @return   NULL
* @bref     Update flow status      
-----------------------------------------------------------------------------------------------------------
*/
__host__ void cudaAdvectMacCormack ( float *den_out, float *velU_out, float *velV_out, float *velW_out, 
										float *den_in, float *velU_in, float *velV_in,float *velW_in )
{
	// Define the computing unit size
	cudaDeviceDim3D ( );

	// Calculate the advection
	kernelPredicateAdvectionMacCormack <<<gridDim, blockDim>>> 
		( den_out, velU_out, velV_out, velW_out, den_in, velU_in, velV_in, velW_in );
	kernelChecksumAdvectionMacCormack  <<<gridDim, blockDim>>>
		( den_out, velU_out, velV_out, velW_out, den_in, velU_in, velV_in, velW_in );
};

/*
-----------------------------------------------------------------------------------------------------------
* @function MacCormackSchemeSolver
* @author   Orlando Chen
* @date     Nov 19, 2013
* @input    u, v, w, u0, v0, w0, grid, grid0
* @return   NULL
* @bref     Calculate the advection of flow by MackCormack Scheme
-----------------------------------------------------------------------------------------------------------
*/
__host__ void MacCormackSchemeSolver ( float *u, float *v, float *w, float *u0, float *v0, float *w0, float *grid, float *grid0 )
{
	// Define the computing unit size
	cudaDeviceDim3D ( );
	
    // Copy input vectors from host memory to GPU buffers.
	if ( hipMemcpy ( dev_grid, grid, SIM_SIZE * sizeof(float), hipMemcpyHostToDevice ) != hipSuccess )
		cudaCheckRuntimeErrors ( "hipMemcpy was failed" );

	if ( hipMemcpy ( dev_grid0, grid0, SIM_SIZE * sizeof(float), hipMemcpyHostToDevice ) != hipSuccess )
		cudaCheckRuntimeErrors ( "hipMemcpy was failed" );

	if ( hipMemcpy ( dev_u0, u0, SIM_SIZE * sizeof(float), hipMemcpyHostToDevice ) != hipSuccess )
		cudaCheckRuntimeErrors ( "hipMemcpy was failed" );

	if ( hipMemcpy ( dev_v0, v0, SIM_SIZE * sizeof(float), hipMemcpyHostToDevice ) != hipSuccess )
		cudaCheckRuntimeErrors ( "hipMemcpy was failed" );

	if ( hipMemcpy ( dev_w0, w0, SIM_SIZE * sizeof(float), hipMemcpyHostToDevice ) != hipSuccess )
		cudaCheckRuntimeErrors ( "hipMemcpy was failed" );

	if ( hipMemcpy ( dev_u, u, SIM_SIZE * sizeof(float), hipMemcpyHostToDevice ) != hipSuccess )
		cudaCheckRuntimeErrors ( "hipMemcpy was failed" );

	if ( hipMemcpy ( dev_v, v, SIM_SIZE * sizeof(float), hipMemcpyHostToDevice ) != hipSuccess )
		cudaCheckRuntimeErrors ( "hipMemcpy was failed" );

	if ( hipMemcpy ( dev_w, w, SIM_SIZE * sizeof(float), hipMemcpyHostToDevice ) != hipSuccess )
		cudaCheckRuntimeErrors ( "hipMemcpy was failed" );

	// Launch kernels
	// Add source to background for further simulation
	cudaAddSourceMacCormack (dev_den0, dev_u0, dev_v0, dev_w0);
	cudaAdvectMacCormack (dev_den, dev_u, dev_v, dev_w, dev_den0, dev_u0, dev_v0, dev_w0);
	// ...
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
	if ( hipDeviceSynchronize ( ) != hipSuccess )
		cudaCheckRuntimeErrors ( "hipDeviceSynchronize was failed" );

    // Copy output vector from GPU buffer to host memory.
	if ( hipMemcpy ( grid, dev_grid, SIM_SIZE * sizeof(float), hipMemcpyDeviceToHost ) != hipSuccess )
		cudaCheckRuntimeErrors ( "hipMemcpy was failed" );

	if ( hipMemcpy ( grid0, dev_grid0, SIM_SIZE * sizeof(float), hipMemcpyDeviceToHost ) != hipSuccess )
		cudaCheckRuntimeErrors ( "hipMemcpy was failed" );

	if ( hipMemcpy ( u0, dev_u0, SIM_SIZE * sizeof(float), hipMemcpyDeviceToHost ) != hipSuccess )
		cudaCheckRuntimeErrors ( "hipMemcpy was failed" );

	if ( hipMemcpy ( v0, dev_v0, SIM_SIZE * sizeof(float), hipMemcpyDeviceToHost ) != hipSuccess )
		cudaCheckRuntimeErrors ( "hipMemcpy was failed" );

	if ( hipMemcpy ( w0, dev_w0, SIM_SIZE * sizeof(float), hipMemcpyDeviceToHost ) != hipSuccess )
		cudaCheckRuntimeErrors ( "hipMemcpy was failed" );
	
	if ( hipMemcpy ( u, dev_u, SIM_SIZE * sizeof(float), hipMemcpyDeviceToHost ) != hipSuccess )
		cudaCheckRuntimeErrors ( "hipMemcpy was failed" );

	if ( hipMemcpy ( v, dev_v, SIM_SIZE * sizeof(float), hipMemcpyDeviceToHost ) != hipSuccess )
		cudaCheckRuntimeErrors ( "hipMemcpy was failed" );

	if ( hipMemcpy ( w, dev_w, SIM_SIZE * sizeof(float), hipMemcpyDeviceToHost ) != hipSuccess )
		cudaCheckRuntimeErrors ( "hipMemcpy was failed" );
};

/*
  -----------------------------------------------------------------------------------------------------------
   Undefine something
  -----------------------------------------------------------------------------------------------------------
*/

#undef eqt   /* equal to */
#undef and   /* logical and */
#undef or    /* logical or */

#undef gst_header  /* (ghost, halo) the header cell of grid */
#undef sim_header  /* (actually) the second cell of grid */
#undef gst_tailer  /* (ghost, halo) the last cell of grid */
#undef sim_tailer  /* (actually) the second last cell of grid */

#endif