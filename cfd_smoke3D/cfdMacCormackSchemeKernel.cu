#include "hip/hip_runtime.h"
/**
*
* Copyright (C) <2013> <Orlando Chen>
* Permission is hereby granted, free of charge, to any person obtaining a copy of this software and
* associated documentation files (the "Software"), to deal in the Software without restriction, 
* including without limitation the rights to use, copy, modify, merge, publish, distribute, sublicense, 
* and/or sell copies of the Software, and to permit persons to whom the Software is furnished to do so, 
* subject to the following conditions:
*
* The above copyright notice and this permission notice shall be included in all copies or substantial
* portions of the Software.
*
* THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR IMPLIED, INCLUDING BUT 
* NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.
* IN NO EVENT SHALL THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER LIABILITY, 
* WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM, OUT OF OR IN CONNECTION WITH THE
* SOFTWARE OR THE USE OR OTHER DEALINGS IN THE SOFTWARE.
 */

/**
* <Author>      Orlando Chen
* <First>       Nov 19, 2013
* <Last>		Nov 20, 2013
* <File>        cfdMacCormackSchemeKernel.cu
*/

#ifndef __cfd_MacCormack_Scheme_Kernel_cu_
#define __cfd_MacCormack_Scheme_Kernel_cu_

#include "cfdHeader.h"

/*
  -----------------------------------------------------------------------------------------------------------
   Define something
  -----------------------------------------------------------------------------------------------------------
*/

#define eqt              ==            /* equal to */
#define and              &&            /* logical and */
#define or               ||            /* logical or */

#define gst_header       0             /* (ghost, halo) the header cell of grid */
#define sim_header       1             /* (actually) the second cell of grid */
#define gst_tailer       Grids_X - 1   /* (ghost, halo) the last cell of grid */
#define sim_tailer       Grids_X - 2   /* (actually) the second last cell of grid */

#define BeginSimArea() \
	if ( i >= sim_header and i <= sim_tailer ) \
	if ( j >= sim_header and j <= sim_tailer ) \
	if ( k >= sim_header and k <= sim_tailer ) {

#define EndSimArea() }


/*
-----------------------------------------------------------------------------------------------------------
* @function kernelAddSourceMacCormack
* @author   Orlando Chen
* @date     Nov 19, 2013
* @input    float *density_inout, float *velU_inout, float *velV_inout, float *velW_inout
* @return   NULL
* @bref     Add source to simulation grid      
-----------------------------------------------------------------------------------------------------------
*/
__global__ void kernelAddSourceMacCormack ( float *density_inout, float *velU_inout, float *velV_inout, float *velW_inout )
{
	// Get index of GPU-thread
	GetIndex ( );

	// Coordinates arround the (64, 64, 64), r is 5
	if ( i > 54 and i < 74 ) if ( k > 54 and k < 74 )
	{
		int x = i - 64;
		int y = k - 64;
		float r = sqrtf ( x * x + y * y );
		
		// Within the correct distance
		if ( r >= 0 && r <= 5  )
		{
			// Add source from layer 0 - 4
			if ( j < 5 )
			{
				density_inout [ Index (i, j, k) ] += SOURCE * DELTA_TIME;
				velV_inout [ Index (i, j, k) ] = SOURCE * DELTA_TIME;
			}
		}
	}
};


/*
-----------------------------------------------------------------------------------------------------------
* @function subkernelBoundaryMacCormack
* @author   Orlando Chen
* @date     Nov 22, 2013
* @input    float *density_inout, float *velU_inout, float *velV_inout, float *velW_inout
* @return   NULL
* @bref     Check and set boundary condition      
-----------------------------------------------------------------------------------------------------------
*/
__device__ void subkernelBoundaryMacCormack ( float *density_inout, float *velU_inout, float *velV_inout, float *velW_inout )
{
	// Get index of GPU-thread
	GetIndex ( );

	// Boundary condition
	BeginSimArea();
	{
		// Solve the condition of density
		density_inout [ Index (gst_header, j, k) ] = density_inout [ Index (sim_header, j, k) ]; // bottom side of simulation grid
		density_inout [ Index (gst_tailer, j, k) ] = density_inout [ Index (sim_tailer, j, k) ]; // top side of simulation grid
		density_inout [ Index (i, gst_header, k) ] = density_inout [ Index (i, sim_header, k) ]; // left side of simulation grid
		density_inout [ Index (i, gst_tailer, k) ] = density_inout [ Index (i, sim_tailer, k) ]; // right side of simulation grid
		density_inout [ Index (i, j, gst_header) ] = density_inout [ Index (i, j, sim_header) ]; // front side of simulation grid
		density_inout [ Index (i, j, gst_tailer) ] = density_inout [ Index (i, j, sim_tailer) ]; // back side of simulation grid

		// Solve the condition of sub-flow U
		velU_inout [ Index (gst_header, j, k) ] = -velU_inout [ Index (sim_header, j, k) ]; // bottom side of simulation grid
		velU_inout [ Index (gst_tailer, j, k) ] = -velU_inout [ Index (sim_tailer, j, k) ]; // top side of simulation grid
		velU_inout [ Index (i, gst_header, k) ] =  velU_inout [ Index (i, sim_header, k) ]; // left side of simulation grid
		velU_inout [ Index (i, gst_tailer, k) ] =  velU_inout [ Index (i, sim_tailer, k) ]; // right side of simulation grid
		velU_inout [ Index (i, j, gst_header) ] =  velU_inout [ Index (i, j, sim_header) ]; // front side of simulation grid
		velU_inout [ Index (i, j, gst_tailer) ] =  velU_inout [ Index (i, j, sim_tailer) ]; // back side of simulation grid
		
		// Solve the condition of sub-flow v
		velV_inout [ Index (gst_header, j, k) ] =  velV_inout [ Index (sim_header, j, k) ]; // bottom side of simulation grid
		velV_inout [ Index (gst_tailer, j, k) ] =  velV_inout [ Index (sim_tailer, j, k) ]; // top side of simulation grid
		velV_inout [ Index (i, gst_header, k) ] = -velV_inout [ Index (i, sim_header, k) ]; // left side of simulation grid
		velV_inout [ Index (i, gst_tailer, k) ] =  velV_inout [ Index (i, sim_tailer, k) ]; // right side of simulation grid
		velV_inout [ Index (i, j, gst_header) ] =  velV_inout [ Index (i, j, sim_header) ]; // front side of simulation grid
		velV_inout [ Index (i, j, gst_tailer) ] =  velV_inout [ Index (i, j, sim_tailer) ]; // back side of simulation grid

		// Solve the condition of sub-flow w
		velW_inout [ Index (gst_header, j, k) ] =  velW_inout [ Index (sim_header, j, k) ]; // bottom side of simulation grid
		velW_inout [ Index (gst_tailer, j, k) ] =  velW_inout [ Index (sim_tailer, j, k) ]; // top side of simulation grid
		velW_inout [ Index (i, gst_header, k) ] =  velW_inout [ Index (i, sim_header, k) ]; // left side of simulation grid
		velW_inout [ Index (i, gst_tailer, k) ] =  velW_inout [ Index (i, sim_tailer, k) ]; // right side of simulation grid
		velW_inout [ Index (i, j, gst_header) ] = -velW_inout [ Index (i, j, sim_header) ]; // front side of simulation grid
		velW_inout [ Index (i, j, gst_tailer) ] = -velW_inout [ Index (i, j, sim_tailer) ]; // back side of simulation grid
	}
	EndSimArea();

	// Condition of density
	density_inout [ Index (i, gst_tailer, k) ] *= ANNIHILATION;
};


/*
-----------------------------------------------------------------------------------------------------------
* @function subkernelAdvectDensity
* @author   Orlando Chen
* @date     Nov 22, 2013
* @input    float *den_out, float *den_in, float *velU_in, float *velV_in, float *velW_in
* @return   NULL
* @bref     Update density state      
-----------------------------------------------------------------------------------------------------------
*/
__device__ void subkernelAdvectDensity ( float *den_out, float *den_in, float *velU_in, float *velV_in, float *velW_in )
{
	GetIndex();

	BeginSimArea();
	{

	}
	EndSimArea();
};


/*
-----------------------------------------------------------------------------------------------------------
* @function subkernelAdvectFlowU
* @author   Orlando Chen
* @date     Nov 22, 2013
* @input    float *velU_out, float *den_in, float *velU_in, float *velV_in, float *velW_in
* @return   NULL
* @bref     Update component u of flow state      
-----------------------------------------------------------------------------------------------------------
*/
__device__ void subkernelAdvectFlowU ( float *velU_out, float *den_in, float *velU_in, float *velV_in, float *velW_in )
{
	GetIndex();

	BeginSimArea();
	{

	}
	EndSimArea();
};


/*
-----------------------------------------------------------------------------------------------------------
* @function subkernelAdvectFlowV
* @author   Orlando Chen
* @date     Nov 22, 2013
* @input    float *velU_out, float *den_in, float *velU_in, float *velV_in, float *velW_in
* @return   NULL
* @bref     Update component v of flow state      
-----------------------------------------------------------------------------------------------------------
*/
__device__ void subkernelAdvectFlowV ( float *velV_out, float *den_in, float *velU_in, float *velV_in, float *velW_in )
{
	GetIndex();

	BeginSimArea();
	{

	}
	EndSimArea();
};


/*
-----------------------------------------------------------------------------------------------------------
* @function subkernelAdvectFlowW
* @author   Orlando Chen
* @date     Nov 22, 2013
* @input    float *velW_out, float *den_in, float *velU_in, float *velV_in, float *velW_in
* @return   NULL
* @bref     Update component w of flow state      
-----------------------------------------------------------------------------------------------------------
*/
__device__ void subkernelAdvectFlowW ( float *velW_out, float *den_in, float *velU_in, float *velV_in, float *velW_in )
{
	GetIndex();

	BeginSimArea();
	{

	}
	EndSimArea();
};


/*
-----------------------------------------------------------------------------------------------------------
* @function kernelAdvectMacCormack
* @author   Orlando Chen
* @date     Nov 22, 2013
* @input    float *den_out, float *velU_out, float *velV_out, float *velW_out, 
* --------- float *den_in, float *velU_in, float *velV_in,float *velW_in
* @return   NULL
* @bref     Update flow status      
-----------------------------------------------------------------------------------------------------------
*/
__global__ void kernelAdvectMacCormack ( float *den_out, float *velU_out, float *velV_out, float *velW_out, 
										float *den_in, float *velU_in, float *velV_in,float *velW_in )
{
	// Update state of flow
	subkernelAdvectFlowU ( velU_out, den_in, velU_in, velV_in, velW_in );
	subkernelAdvectFlowV ( velV_out, den_in, velU_in, velV_in, velW_in );
	subkernelAdvectFlowW ( velW_out, den_in, velU_in, velV_in, velW_in );
	subkernelAdvectDensity ( den_out, den_in, velU_in, velV_in, velW_in );

	// Check boundary condition
	subkernelBoundaryMacCormack ( den_out, velU_out, velV_out, velW_out);
};

/*
-----------------------------------------------------------------------------------------------------------
* @function MacCormackSchemeSolver
* @author   Orlando Chen
* @date     Nov 19, 2013
* @input    u, v, w, u0, v0, w0, grid, grid0
* @return   NULL
* @bref     Calculate the advection of flow by MackCormack Scheme
-----------------------------------------------------------------------------------------------------------
*/
void MacCormackSchemeSolver ( float *u, float *v, float *w, float *u0, float *v0, float *w0, float *grid, float *grid0 )
{

	// Define the computing unit size
	cudaDeviceDim3D ( );
	
    // Copy input vectors from host memory to GPU buffers.
	if ( hipMemcpy ( dev_grid, grid, SIM_SIZE * sizeof(float), hipMemcpyHostToDevice ) != hipSuccess )
		cudaCheckRuntimeErrors ( "hipMemcpy was failed" );

	if ( hipMemcpy ( dev_grid0, grid0, SIM_SIZE * sizeof(float), hipMemcpyHostToDevice ) != hipSuccess )
		cudaCheckRuntimeErrors ( "hipMemcpy was failed" );

	if ( hipMemcpy ( dev_u0, u0, SIM_SIZE * sizeof(float), hipMemcpyHostToDevice ) != hipSuccess )
		cudaCheckRuntimeErrors ( "hipMemcpy was failed" );

	if ( hipMemcpy ( dev_v0, v0, SIM_SIZE * sizeof(float), hipMemcpyHostToDevice ) != hipSuccess )
		cudaCheckRuntimeErrors ( "hipMemcpy was failed" );

	if ( hipMemcpy ( dev_w0, w0, SIM_SIZE * sizeof(float), hipMemcpyHostToDevice ) != hipSuccess )
		cudaCheckRuntimeErrors ( "hipMemcpy was failed" );

	if ( hipMemcpy ( dev_u, u, SIM_SIZE * sizeof(float), hipMemcpyHostToDevice ) != hipSuccess )
		cudaCheckRuntimeErrors ( "hipMemcpy was failed" );

	if ( hipMemcpy ( dev_v, v, SIM_SIZE * sizeof(float), hipMemcpyHostToDevice ) != hipSuccess )
		cudaCheckRuntimeErrors ( "hipMemcpy was failed" );

	if ( hipMemcpy ( dev_w, w, SIM_SIZE * sizeof(float), hipMemcpyHostToDevice ) != hipSuccess )
		cudaCheckRuntimeErrors ( "hipMemcpy was failed" );

	// Launch kernels
	// Add source to background for further simulation
	kernelAddSourceMacCormack <<< gridDim, blockDim >>> (dev_den0, dev_u0, dev_v0, dev_w0);
	// ...
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
	if ( hipDeviceSynchronize ( ) != hipSuccess )
		cudaCheckRuntimeErrors ( "hipDeviceSynchronize was failed" );

    // Copy output vector from GPU buffer to host memory.
	if ( hipMemcpy ( grid, dev_grid, SIM_SIZE * sizeof(float), hipMemcpyDeviceToHost ) != hipSuccess )
		cudaCheckRuntimeErrors ( "hipMemcpy was failed" );

	if ( hipMemcpy ( grid0, dev_grid0, SIM_SIZE * sizeof(float), hipMemcpyDeviceToHost ) != hipSuccess )
		cudaCheckRuntimeErrors ( "hipMemcpy was failed" );

	if ( hipMemcpy ( u0, dev_u0, SIM_SIZE * sizeof(float), hipMemcpyDeviceToHost ) != hipSuccess )
		cudaCheckRuntimeErrors ( "hipMemcpy was failed" );

	if ( hipMemcpy ( v0, dev_v0, SIM_SIZE * sizeof(float), hipMemcpyDeviceToHost ) != hipSuccess )
		cudaCheckRuntimeErrors ( "hipMemcpy was failed" );

	if ( hipMemcpy ( w0, dev_w0, SIM_SIZE * sizeof(float), hipMemcpyDeviceToHost ) != hipSuccess )
		cudaCheckRuntimeErrors ( "hipMemcpy was failed" );
	
	if ( hipMemcpy ( u, dev_u, SIM_SIZE * sizeof(float), hipMemcpyDeviceToHost ) != hipSuccess )
		cudaCheckRuntimeErrors ( "hipMemcpy was failed" );

	if ( hipMemcpy ( v, dev_v, SIM_SIZE * sizeof(float), hipMemcpyDeviceToHost ) != hipSuccess )
		cudaCheckRuntimeErrors ( "hipMemcpy was failed" );

	if ( hipMemcpy ( w, dev_w, SIM_SIZE * sizeof(float), hipMemcpyDeviceToHost ) != hipSuccess )
		cudaCheckRuntimeErrors ( "hipMemcpy was failed" );
};

/*
  -----------------------------------------------------------------------------------------------------------
   Undefine something
  -----------------------------------------------------------------------------------------------------------
*/

#undef eqt   /* equal to */
#undef and   /* logical and */
#undef or    /* logical or */

#undef gst_header  /* (ghost, halo) the header cell of grid */
#undef sim_header  /* (actually) the second cell of grid */
#undef gst_tailer  /* (ghost, halo) the last cell of grid */
#undef sim_tailer  /* (actually) the second last cell of grid */

#endif