#include "hip/hip_runtime.h"
/**
*
* Copyright (C) <2013> <Orlando Chen>
* Permission is hereby granted, free of charge, to any person obtaining a copy of this software and
* associated documentation files (the "Software"), to deal in the Software without restriction, 
* including without limitation the rights to use, copy, modify, merge, publish, distribute, sublicense, 
* and/or sell copies of the Software, and to permit persons to whom the Software is furnished to do so, 
* subject to the following conditions:
*
* The above copyright notice and this permission notice shall be included in all copies or substantial
* portions of the Software.
*
* THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR IMPLIED, INCLUDING BUT 
* NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.
* IN NO EVENT SHALL THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER LIABILITY, 
* WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM, OUT OF OR IN CONNECTION WITH THE
* SOFTWARE OR THE USE OR OTHER DEALINGS IN THE SOFTWARE.
 */

/**
* <Author>      Orlando Chen
* <First>       Nov 15, 2013
* <Last>		Nov 15, 2013
* <File>        renderingKernel.cu
*/

#ifndef __rendering_Kernel_cu_
#define __rendering_Kernel_cu_

#include "cfdHeaders.h"
#include "cudaHelper.h"
#include "macroDef.h"

#define is       ==            /* equal to */
#define like     ==            /* equal to */
#define gte      >=            /* greater than or equal to  */
#define gt       >             /* greater than */
#define lse      <=            /* less than or equal to */
#define ls       <             /* less than */
#define and      &&            /* logical and */
#define or       ||            /* logical or */


#define GetIndex()  \
	int i = blockIdx.x * blockDim.x + threadIdx.x; \
	int j = blockIdx.y * blockDim.y + threadIdx.y; \
	int k = 0; \
	cudaTrans2DTo3D ( i, j, k, Grids_X );


__global__ void kernelZeroBuffer ( float *buff_inout )
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;

	buff_inout [ cudaIndex2D (i, j, Grids_X) ] = 0.f;
};


__global__ void kernelDensityInterpolate ( float *den3D_in, float *den2D_out )
{
	GetIndex ( );
	
	den2D_out [ cudaIndex2D (i, j, Grids_X) ] += den3D_in [ cudaIndex3D (i, j, k, Grids_X) ];
};


__global__ void kernelVelocityInterpolate ( float *u3D_in, float *v3D_in, float *u2D_out, float *v2D_out )
{
	GetIndex ( );

	u2D_out [ cudaIndex2D (i, j, Grids_X) ] += u3D_in [ cudaIndex3D (i, j, k, Grids_X) ];
	v2D_out [ cudaIndex2D (i, j, Grids_X) ] += v3D_in [ cudaIndex3D (i, j, k, Grids_X) ];
};


void cudaCheckRuntimeErrors ( char *msg )
{
	extern void FreeResources (void);

	Logfile.SaveStringToFile ( "errormsg.log", sge::SG_FILE_OPEN_APPEND, 
		"%s, at line: %d of file %s", msg, __LINE__, __FILE__ ); 
	Logfile.SaveStringToFile ( "errormsg.log", sge::SG_FILE_OPEN_APPEND, 
		">>>> Error Message: %s", hipGetErrorString ( cudaStatus ) );
	FreeResources ( ); exit ( 0 );
};


void DensityInterpolate ( void )
{
	extern void FreeResources  ( void );

	// Define the computing unit size
	cudaDeviceDim3D ( );
	
    // Copy input vectors from host memory to GPU buffers.
	if ( hipMemcpy ( dev_grid, host_den, SIM_SIZE * sizeof(float), hipMemcpyHostToDevice ) != hipSuccess )
		cudaCheckRuntimeErrors ( "hipMemcpy was failed" );

	if ( hipMemcpy ( dev_2DRender, host_disD, DIS_SIZE * sizeof(float), hipMemcpyHostToDevice ) != hipSuccess )
		cudaCheckRuntimeErrors ( "hipMemcpy was failed" );


	kernelZeroBuffer cudaDevice(gridDim, blockDim) ( dev_2DRender );
	kernelDensityInterpolate cudaDevice(gridDim, blockDim) ( dev_grid, dev_2DRender );
    

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
	if ( hipDeviceSynchronize ( ) != hipSuccess )
		cudaCheckRuntimeErrors ( "hipDeviceSynchronize was failed" );

    // Copy output vector from GPU buffer to host memory.
	if ( hipMemcpy ( host_disD, dev_2DRender, DIS_SIZE * sizeof(float), hipMemcpyDeviceToHost ) != hipSuccess )
		cudaCheckRuntimeErrors ( "hipMemcpy was failed" );
};


void VelocityInterpolate ( void )
{
	for ( int i = 0; i < Grids_X; i++ )
	{
		for ( int j = 0; j < Grids_X; j++ )
		{
			float var0 = 0.f, var1 = 0.f;
			
			for ( int k = 0; k < Grids_X; k++ )
			{
				var0 = host_u [ cudaIndex3D (i, j, k, Grids_X) ];
				var1 = host_v [ cudaIndex3D (i, j, k, Grids_X) ];
			}

			host_disu [ cudaIndex2D (i, j, Grids_X)] = var0;
			host_disv [ cudaIndex2D (i, j, Grids_X)] = var1;
		}
	}
};


#undef is     /* equal to */
#undef like   /* equal to */
#undef gte    /* greater than or equal to  */
#undef gt     /* greater than */
#undef lse    /* less than or equal to */
#undef ls     /* less than */
#undef and    /* logical and */
#undef or     /* logical or */

#undef GetIndex()

#endif