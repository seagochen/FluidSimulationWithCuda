#include "hip/hip_runtime.h"
/**
*
* Copyright (C) <2013> <Orlando Chen>
* Permission is hereby granted, free of charge, to any person obtaining a copy of this software and
* associated documentation files (the "Software"), to deal in the Software without restriction, 
* including without limitation the rights to use, copy, modify, merge, publish, distribute, sublicense, 
* and/or sell copies of the Software, and to permit persons to whom the Software is furnished to do so, 
* subject to the following conditions:
*
* The above copyright notice and this permission notice shall be included in all copies or substantial
* portions of the Software.
*
* THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR IMPLIED, INCLUDING BUT 
* NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.
* IN NO EVENT SHALL THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER LIABILITY, 
* WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM, OUT OF OR IN CONNECTION WITH THE
* SOFTWARE OR THE USE OR OTHER DEALINGS IN THE SOFTWARE.
 */

/**
* <Author>      Orlando Chen
* <First>       Oct 12, 2013
* <Last>		Nov 19, 2013
* <File>        cfdMainKernel.cu
*/

#ifndef __cfd_Main_Kernel_cu_
#define __cfd_Main_Kernel_cu_

#include "cfdHeader.h"

using namespace std;

/*
  -----------------------------------------------------------------------------------------------------------
   Define something
  -----------------------------------------------------------------------------------------------------------
*/

#define is       ==            /* equal to */
#define like     ==            /* equal to */
#define gte      >=            /* greater than or equal to  */
#define gt       >             /* greater than */
#define lse      <=            /* less than or equal to */
#define ls       <             /* less than */
#define and      &&            /* logical and */
#define or       ||            /* logical or */

#define gst0   0               /* ghost cell, No. #0 */
#define gstl   Grids_X - 1     /* ghost cell, No. #last */
#define rsc0   1               /* simulation cell, No. #0 */
#define rscl   SimArea_X       /* simulation cell, No. #last */


/*
-----------------------------------------------------------------------------------------------------------
* @function kernelAddSource
* @author   Orlando Chen
* @date     Nov 19, 2013
* @input    ptr_inout
* @return   NULL
* @bref     Add source to simulation grid      
-----------------------------------------------------------------------------------------------------------
*/
__global__ void kernelAddSource ( float *ptr_inout )
{
	// Get index of GPU-thread
	GetIndex ( );

	// Coordinates arround the (64, 64, 64), r is 5
	if ( i > 54 and i < 74 ) if ( k > 54 and k < 74 )
	{
		int x = i - 64;
		int y = k - 64;
		float r = sqrtf ( x * x + y * y );
		
		// Within the correct distance
		if ( r >= 0 && r <= 5  )
		{
			// Add source from layer 0 - 4
			if ( j < 5 )
				ptr_inout [ Index (i, j, k) ] = SOURCE * DELTA_TIME * 0.1f;
		}
	}
};


/*
-----------------------------------------------------------------------------------------------------------
* @function kernelSetBoundary
* @author   Orlando Chen
* @date     Nov 15, 2013
* @input    grid_out, boundary
* @return   NULL
* @bref     Check and set boundary condition      
-----------------------------------------------------------------------------------------------------------
*/
__global__ void kernelSetBoundary ( float *grid_out, int boundary )
{
	// Get index of GPU-thread
	GetIndex ( );

	// Boundary condition
	if ( i gte 1 && i lse SimArea_X && j gte 1 && j lse SimArea_X )
	{
		// Slove line (gst0, y)
		grid_out [ Index ( gst0, j, k ) ] = boundary is 1 ? -grid_out [ Index ( rsc0, j, k ) ] : grid_out [ Index ( rsc0, j, k ) ];
		// Slove line (gstl, y)
		grid_out [ Index ( gstl, j, k ) ] = boundary is 1 ? -grid_out [ Index ( rscl, j, k ) ] : grid_out [ Index ( rscl, j, k ) ];
		// Slove line (x, gst0)
		grid_out [ Index ( i, gst0, k ) ] = boundary is 2 ? -grid_out [ Index ( i, rsc0, k ) ] : grid_out [ Index ( i, rsc0, k ) ];
		// Slove line (x, gstl)
		grid_out [ Index ( i, gstl, k ) ] = boundary is 2 ? -grid_out [ Index ( i, rscl, k ) ] : grid_out [ Index ( i, rscl, k ) ];
	}
	// Slove ghost cell (gst0, gst0)
	grid_out [ Index ( gst0, gst0, k ) ] = 0.5f * ( grid_out [ Index ( rsc0, gst0, k ) ] + grid_out [ Index ( gst0, rsc0, k ) ] );
	// Slove ghost cell (gst0, gstl)
	grid_out [ Index ( gst0, gstl, k ) ] = 0.5f * ( grid_out [ Index ( rsc0, gstl, k ) ] + grid_out [ Index ( gst0, rscl, k ) ] );
	// Slove ghost cell (gstl, gst0)
	grid_out [ Index ( gstl, gst0, k ) ] = 0.5f * ( grid_out [ Index ( rscl, gst0, k ) ] + grid_out [ Index ( gstl, rsc0, k ) ] );
	// Slove ghost cell (gstl, gstl)
	grid_out [ Index ( gstl, gstl, k ) ] = 0.5f * ( grid_out [ Index ( rscl, gstl, k ) ] + grid_out [ Index ( gstl, rscl, k ) ] );
}


/*
-----------------------------------------------------------------------------------------------------------
* @function kernelLineSolver
* @author   Orlando Chen
* @date     Nov 15, 2013
* @input    grid_inout, grid0_in, boudnary, a, c
* @return   NULL
* @bref     Line solver
-----------------------------------------------------------------------------------------------------------
*/
__global__ void kernelLineSolver ( float *grid_inout, float *grid0_in, int boundary, float a, float c )
{
	// Get index of GPU-thread
	GetIndex ( );

	if ( i gte 1 && i lse SimArea_X && j gte 1 && j lse SimArea_X )
	{
		grid_inout [ Index ( i, j, k ) ] = ( grid0_in [ Index ( i, j, k ) ] + a * ( grid_inout [ Index ( i-1, j, k ) ] + 
			grid_inout [ Index ( i+1, j, k ) ] + grid_inout [ Index ( i, j-1, k ) ] + grid_inout [ Index ( i, j+1, k ) ] ) ) / c;	
	}
}


/*
-----------------------------------------------------------------------------------------------------------
* @function kernelAdvect
* @author   Orlando Chen
* @date     Nov 18, 2013
* @input    density_out, density0_in, u_in, v_in, w_in, dt0
* @return   NULL
* @bref     Advection method      
-----------------------------------------------------------------------------------------------------------
*/
__global__ void kernelAdvect ( float *density_out, float *density0_in, float *u_in, float *v_in, float *w_in, float dt0 )
{
	// Get index of GPU-thread
	GetIndex ( );

	int i0, j0, i1, j1;
	float x, y, s0, t0, s1, t1;

	if ( i gte 1 && i lse SimArea_X && j gte 1 && j lse SimArea_X )
	{
		x = i - dt0 * u_in[Index(i, j, k)];
		y = j - dt0 * v_in[Index(i, j, k)];
		if (x < 0.5f) x = 0.5f;
		if (x > SimArea_X + 0.5f) x = SimArea_X+0.5f;

		i0 = (int)x; 
		i1 = i0+1;
		
		if (y < 0.5f) y=0.5f;
		if (y > SimArea_X+0.5f) y = SimArea_X+0.5f;
		
		j0 = (int)y;
		j1 = j0 + 1;
		s1 = x - i0;
		s0 = 1 - s1;
		t1 = y - j0;
		t0 = 1 - t1;

		density_out[Index(i, j, k)] = s0 * ( t0 * density0_in[Index(i0, j0, k)] +
			t1 * density0_in[Index(i0, j1, k)]) + s1 * ( t0 * density0_in[Index(i1, j0, k)] + 
			t1 * density0_in[Index(i1, j1, k)]);
	}
};


/*
-----------------------------------------------------------------------------------------------------------
* @function kernelProjectPt1
* @author   Orlando Chen
* @date     Nov 18, 2013
* @input    u, v, w, u0, v0, w0
* @return   NULL
* @bref     CFD projection part I      
-----------------------------------------------------------------------------------------------------------
*/
__global__ void kernelProjectPt1 ( float *u, float *v, float *w, float *u0, float *v0, float *w0 )
{
	// Get index of GPU-thread
	GetIndex ( );
	
	if ( i gte 1 && i lse SimArea_X && j gte 1 && j lse SimArea_X )
	{
		v0 [ Index (i, j, k) ] = -0.5f * ( u [ Index ( i+1, j, k ) ] - u [ Index ( i-1, j, k ) ] + v [ Index ( i, j+1, k ) ] 
		- v [ Index ( i, j-1, k ) ] ) / SimArea_X;
		u0 [ Index (i, j, k) ] = 0;
	}
}

/*
-----------------------------------------------------------------------------------------------------------
* @function kernelProjectPt2
* @author   Orlando Chen
* @date     Nov 18, 2013
* @input    u, v, w, u0, v0, w0
* @return   NULL
* @bref     CFD projection part II      
-----------------------------------------------------------------------------------------------------------
*/
__global__ void kernelProjectPt2( float *u, float *v, float *w, float *u0, float *v0, float *w0 )
{
	// Get index of GPU-thread
	GetIndex ( );
	
	if ( i gte 1 && i lse SimArea_X && j gte 1 && j lse SimArea_X )
	{
			u [ Index ( i, j, k ) ] -= 0.5f * SimArea_X * ( u0 [ Index ( i+1, j, k ) ] - u0 [ Index ( i-1, j, k ) ] );
			v [ Index ( i, j, k ) ] -= 0.5f * SimArea_X * ( u0 [ Index ( i, j+1, k ) ] - u0 [ Index ( i, j-1, k ) ] );
	}
}


/*
-----------------------------------------------------------------------------------------------------------
* @function cudaAddSource
* @author   Orlando Chen
* @date     Nov 18, 2013
* @input    grid, gridDim, blockDim
* @return   NULL
* @bref     C++ encapsulation of kernelAddSource      
-----------------------------------------------------------------------------------------------------------
*/
void cudaAddSource ( float *grid, dim3 *gridDim, dim3 *blockDim )
{
    // Launch a kernel on the GPU with one thread for each element.
	kernelAddSource cudaDevice(*gridDim,  *blockDim) (grid);
};

/*
-----------------------------------------------------------------------------------------------------------
* @function cudaLineSolver
* @author   Orlando Chen
* @date     Nov 18, 2013
* @input    grid, grid0, boundary, a, c, gridDim, blockDim
* @return   NULL
* @bref     C++ encapsulation of kernelLineSolver      
-----------------------------------------------------------------------------------------------------------
*/
void cudaLineSolver (float *grid, float *grid0, int boundary, float a, float c, dim3 *gridDim, dim3 *blockDim)
{
    // Launch a kernel on the GPU with one thread for each element.
	for (int i=0; i<20; i++)
	{
		kernelLineSolver cudaDevice(*gridDim,  *blockDim) (grid, grid0, boundary, a, c);
	}
	kernelSetBoundary cudaDevice(*gridDim,  *blockDim)  (grid, boundary);
}


/*
-----------------------------------------------------------------------------------------------------------
* @function cudaDiffuse
* @author   Orlando Chen
* @date     Nov 18, 2013
* @input    grid, grid0, boundary, diff, gridDim, blockDim
* @return   NULL
* @bref     C++ encapsulation of diffuse method      
-----------------------------------------------------------------------------------------------------------
*/
void cudaDiffuse ( float *grid, float *grid0, int boundary, float diff, dim3 *gridDim, dim3 *blockDim )
{
	float a = DELTA_TIME * diff * SimArea_X * SimArea_X;
	cudaLineSolver ( grid, grid0, boundary, a, 1+4*a, gridDim, blockDim );
}

/*
-----------------------------------------------------------------------------------------------------------
* @function cudaAdvect
* @author   Orlando Chen
* @date     Nov 18, 2013
* @input    density, density0, u, v, w, boundary, gridDim, blockDim
* @return   NULL
* @bref     C++ encapsulation of advection method      
-----------------------------------------------------------------------------------------------------------
*/
void cudaAdvect ( float *density, float *density0, float *u, float *v, float *w, int boundary, dim3 *gridDim, dim3 *blockDim )
{
    // Launch a kernel on the GPU with one thread for each element.
	float dt0 = DELTA_TIME*SimArea_X;
	kernelAdvect      cudaDevice(*gridDim,  *blockDim) ( density, density0, u, v, w, dt0 );
	kernelSetBoundary cudaDevice(*gridDim,  *blockDim) ( density, boundary );
}


/*
-----------------------------------------------------------------------------------------------------------
* @function cudaProject
* @author   Orlando Chen
* @date     Nov 18, 2013
* @input    u, v, w, u0, v0, w0, gridDim, blockDim
* @return   NULL
* @bref     C++ encapsulation of projection method
*      
-----------------------------------------------------------------------------------------------------------
*/
void cudaProject ( float *u, float *v, float *w, float *u0, float *v0, float *w0, dim3 *gridDim, dim3 *blockDim )
{
	kernelProjectPt1   cudaDevice(*gridDim,  *blockDim) ( u, v, w, u0, v0, w0 );
	kernelSetBoundary  cudaDevice(*gridDim,  *blockDim) ( v0, 0 );
	kernelSetBoundary  cudaDevice(*gridDim,  *blockDim) ( u0, 0 );
	kernelLineSolver   cudaDevice(*gridDim,  *blockDim) ( u0, v0, 0, 1, 4 );
	kernelProjectPt2   cudaDevice(*gridDim,  *blockDim) ( u, v, w, u0, v0, w0 );
	kernelSetBoundary  cudaDevice(*gridDim,  *blockDim) ( u, 1 );
	kernelSetBoundary  cudaDevice(*gridDim,  *blockDim) ( v, 2 );
}


/*
-----------------------------------------------------------------------------------------------------------
* @function DensitySolver
* @author   Orlando Chen
* @date     Nov 18, 2013
* @input    grid, grid0, u, v, w
* @return   NULL
* @bref     Calculate the advection of flow, and update the density on each cell     
-----------------------------------------------------------------------------------------------------------
*/
void DensitySolver ( float *grid, float *grid0, float *u, float *v, float *w )
{
	// Define the computing unit size
	cudaDeviceDim3D ( );
	
    // Copy input vectors from host memory to GPU buffers.
	if ( hipMemcpy ( dev_grid, grid, SIM_SIZE * sizeof(float), hipMemcpyHostToDevice ) != hipSuccess )
		cudaCheckRuntimeErrors ( "hipMemcpy was failed" );

	if ( hipMemcpy ( dev_grid0, grid0, SIM_SIZE * sizeof(float), hipMemcpyHostToDevice ) != hipSuccess )
		cudaCheckRuntimeErrors ( "hipMemcpy was failed" );

	if ( hipMemcpy ( dev_u, u, SIM_SIZE * sizeof(float), hipMemcpyHostToDevice ) != hipSuccess )
		cudaCheckRuntimeErrors ( "hipMemcpy was failed" );

	if ( hipMemcpy ( dev_v, v, SIM_SIZE * sizeof(float), hipMemcpyHostToDevice ) != hipSuccess )
		cudaCheckRuntimeErrors ( "hipMemcpy was failed" );

	if ( hipMemcpy ( dev_w, w, SIM_SIZE * sizeof(float), hipMemcpyHostToDevice ) != hipSuccess )
		cudaCheckRuntimeErrors ( "hipMemcpy was failed" );


	cudaAddSource ( dev_grid, &gridDim, &blockDim );
	swap ( dev_grid0, dev_grid ); cudaDiffuse ( dev_grid, dev_grid0, 0, DIFFUSION, &gridDim, &blockDim );
	swap ( dev_grid0, dev_grid ); cudaAdvect  ( dev_grid, dev_grid0, dev_u, dev_v, dev_w, 0, &gridDim, &blockDim );
    

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
	if ( hipDeviceSynchronize ( ) != hipSuccess )
		cudaCheckRuntimeErrors ( "hipDeviceSynchronize was failed" );

    // Copy output vector from GPU buffer to host memory.
	if ( hipMemcpy ( grid, dev_grid, SIM_SIZE * sizeof(float), hipMemcpyDeviceToHost ) != hipSuccess )
		cudaCheckRuntimeErrors ( "hipMemcpy was failed" );

	if ( hipMemcpy ( grid0, dev_grid0, SIM_SIZE * sizeof(float), hipMemcpyDeviceToHost ) != hipSuccess )
		cudaCheckRuntimeErrors ( "hipMemcpy was failed" );
	
	if ( hipMemcpy ( u, dev_u, SIM_SIZE * sizeof(float), hipMemcpyDeviceToHost ) != hipSuccess )
		cudaCheckRuntimeErrors ( "hipMemcpy was failed" );

	if ( hipMemcpy ( v, dev_v, SIM_SIZE * sizeof(float), hipMemcpyDeviceToHost ) != hipSuccess )
		cudaCheckRuntimeErrors ( "hipMemcpy was failed" );

	if ( hipMemcpy ( w, dev_w, SIM_SIZE * sizeof(float), hipMemcpyDeviceToHost ) != hipSuccess )
		cudaCheckRuntimeErrors ( "hipMemcpy was failed" );
}


/*
-----------------------------------------------------------------------------------------------------------
* @function VelocitySolver
* @author   Orlando Chen
* @date     Nov 18, 2013
* @input    u, v, w, u0, v0, w0
* @return   NULL
* @bref     Calculate the advection of flow, and update the velocity on each cell
*      
-----------------------------------------------------------------------------------------------------------
*/
void VelocitySolver ( float *u, float *v, float *w, float *u0, float *v0, float *w0 )
{
	// Define the computing unit size
	cudaDeviceDim3D ( );
	
    // Copy input vectors from host memory to GPU buffers.
	if ( hipMemcpy ( dev_u0, u0, SIM_SIZE * sizeof(float), hipMemcpyHostToDevice ) != hipSuccess )
		cudaCheckRuntimeErrors ( "hipMemcpy was failed" );

	if ( hipMemcpy ( dev_v0, v0, SIM_SIZE * sizeof(float), hipMemcpyHostToDevice ) != hipSuccess )
		cudaCheckRuntimeErrors ( "hipMemcpy was failed" );

	if ( hipMemcpy ( dev_w0, w0, SIM_SIZE * sizeof(float), hipMemcpyHostToDevice ) != hipSuccess )
		cudaCheckRuntimeErrors ( "hipMemcpy was failed" );

	if ( hipMemcpy ( dev_u, u, SIM_SIZE * sizeof(float), hipMemcpyHostToDevice ) != hipSuccess )
		cudaCheckRuntimeErrors ( "hipMemcpy was failed" );

	if ( hipMemcpy ( dev_v, v, SIM_SIZE * sizeof(float), hipMemcpyHostToDevice ) != hipSuccess )
		cudaCheckRuntimeErrors ( "hipMemcpy was failed" );

	if ( hipMemcpy ( dev_w, w, SIM_SIZE * sizeof(float), hipMemcpyHostToDevice ) != hipSuccess )
		cudaCheckRuntimeErrors ( "hipMemcpy was failed" );


	cudaAddSource ( dev_u, &gridDim, &blockDim ); cudaAddSource ( dev_v, &gridDim, &blockDim );
	swap ( dev_u0, dev_u ); cudaDiffuse ( dev_u, dev_u0, 1, VISCOSITY, &gridDim, &blockDim );
	swap ( dev_v0, dev_v ); cudaDiffuse ( dev_v, dev_v0, 2, VISCOSITY, &gridDim, &blockDim );
	cudaProject ( dev_u, dev_v, dev_w, dev_u0, dev_v0, dev_w0, &gridDim, &blockDim );
	swap ( dev_u0, dev_u ); swap ( dev_v0, dev_v );
	cudaAdvect ( dev_u, dev_u0, dev_u0, dev_v0, dev_w0, 1, &gridDim, &blockDim );
	cudaAdvect ( dev_v, dev_v0, dev_u0, dev_v0, dev_w0, 2, &gridDim, &blockDim );
	cudaProject ( dev_u, dev_v, dev_w, dev_u0, dev_v0, dev_w0, &gridDim, &blockDim );


    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
	if ( hipDeviceSynchronize ( ) != hipSuccess )
		cudaCheckRuntimeErrors ( "hipDeviceSynchronize was failed" );

    // Copy output vector from GPU buffer to host memory.
	if ( hipMemcpy ( u0, dev_u0, SIM_SIZE * sizeof(float), hipMemcpyDeviceToHost ) != hipSuccess )
		cudaCheckRuntimeErrors ( "hipMemcpy was failed" );

	if ( hipMemcpy ( v0, dev_v0, SIM_SIZE * sizeof(float), hipMemcpyDeviceToHost ) != hipSuccess )
		cudaCheckRuntimeErrors ( "hipMemcpy was failed" );

	if ( hipMemcpy ( w0, dev_w0, SIM_SIZE * sizeof(float), hipMemcpyDeviceToHost ) != hipSuccess )
		cudaCheckRuntimeErrors ( "hipMemcpy was failed" );
	
	if ( hipMemcpy ( u, dev_u, SIM_SIZE * sizeof(float), hipMemcpyDeviceToHost ) != hipSuccess )
		cudaCheckRuntimeErrors ( "hipMemcpy was failed" );

	if ( hipMemcpy ( v, dev_v, SIM_SIZE * sizeof(float), hipMemcpyDeviceToHost ) != hipSuccess )
		cudaCheckRuntimeErrors ( "hipMemcpy was failed" );

	if ( hipMemcpy ( w, dev_w, SIM_SIZE * sizeof(float), hipMemcpyDeviceToHost ) != hipSuccess )
		cudaCheckRuntimeErrors ( "hipMemcpy was failed" );
}


/*
  -----------------------------------------------------------------------------------------------------------
   Undef the definitions
  -----------------------------------------------------------------------------------------------------------
*/

#undef gst0   /* ghost cell, No. #0 */
#undef gstl   /* ghost cell, No. #last */
#undef rsc0   /* simulation cell, No. #0 */
#undef rscl   /* simulation cell, No. #last */

#undef is     /* equal to */
#undef like   /* equal to */
#undef gte    /* greater than or equal to  */
#undef gt     /* greater than */
#undef lse    /* less than or equal to */
#undef ls     /* less than */
#undef and    /* logical and */
#undef or     /* logical or */

#endif