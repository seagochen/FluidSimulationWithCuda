#include "hip/hip_runtime.h"
/**
*
* Copyright (C) <2013> <Orlando Chen>
* Permission is hereby granted, free of charge, to any person obtaining a copy of this software and
* associated documentation files (the "Software"), to deal in the Software without restriction, 
* including without limitation the rights to use, copy, modify, merge, publish, distribute, sublicense, 
* and/or sell copies of the Software, and to permit persons to whom the Software is furnished to do so, 
* subject to the following conditions:
*
* The above copyright notice and this permission notice shall be included in all copies or substantial
* portions of the Software.
*
* THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR IMPLIED, INCLUDING BUT 
* NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.
* IN NO EVENT SHALL THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER LIABILITY, 
* WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM, OUT OF OR IN CONNECTION WITH THE
* SOFTWARE OR THE USE OR OTHER DEALINGS IN THE SOFTWARE.
 */

/**
* <Author>      Orlando Chen
* <First>       Oct 12, 2013
* <Last>		Nov 6, 2013
* <File>        cfdMainKernel.cu
*/

#ifndef __cfd_Main_Kernel_cu_
#define __cfd_Main_Kernel_cu_

#include "macroDef.h"
#include "cudaHelper.h"

using namespace std;

extern void FreeResources(void);

static size_t size  = SIM_SIZE;

#define GetIndex()  \
	int i = blockIdx.x * blockDim.x + threadIdx.x; \
	int j = blockIdx.y * blockDim.y + threadIdx.y; \
	int k = 0; \
	cudaTrans2DTo3D(i, j, k, Grids_X);


__global__ void kernelAddSource ( float *ptr_out )
{
	// Get index of GPU-thread
	GetIndex ( );

	if (i == Grids_X / 2 && j == Grids_X / 2)
	{
		int ind = Index(i, j, k);
		ptr_out[ind] += DELTA_TIME * 1.f;
	}
};

#define is ==
#define gst0   0
#define gstl   Grids_X - 1
#define rsc0   1
#define rscl   SimArea_X

__global__ void kernelSetBoundary ( float *grid_out, int boundary )
{
	// Get index of GPU-thread
	GetIndex ( );

	// Boundary condition
	if ( i >= 1 && i <= SimArea_X && j >= 1 && j <= SimArea_X )
	{
		// Slove line (gst0, y)
		grid_out[Index(gst0, j, k)] = boundary is 1 ? -grid_out[Index(rsc0, j, k)] : grid_out[Index(rsc0, j, k)];
		// Slove line (gstl, y)
		grid_out[Index(gstl, j, k)] = boundary is 1 ? -grid_out[Index(rscl, j, k)] : grid_out[Index(rscl, j, k)];
		// Slove line (x, gst0)
		grid_out[Index(i, gst0, k)] = boundary is 2 ? -grid_out[Index(i, rsc0, k)] : grid_out[Index(i, rsc0, k)];
		// Slove line (x, gstl)
		grid_out[Index(i, gstl, k)] = boundary is 2 ? -grid_out[Index(i, rscl, k)] : grid_out[Index(i, rscl, k)];
	}
	// Slove ghost cell (gst0, gst0)
	grid_out[Index(gst0, gst0, k)] = 0.5f * ( grid_out[Index(rsc0, gst0, k)] + grid_out[Index(gst0, rsc0, k)] );
	// Slove ghost cell (gst0, gstl)
	grid_out[Index(gst0, gstl, k)] = 0.5f * ( grid_out[Index(rsc0, gstl, k)] + grid_out[Index(gst0, rscl, k)] );
	// Slove ghost cell (gstl, gst0)
	grid_out[Index(gstl, gst0, k)] = 0.5f * ( grid_out[Index(rscl, gst0, k)] + grid_out[Index(gstl, rsc0, k)] );
	// Slove ghost cell (gstl, gstl)
	grid_out[Index(gstl, gstl, k)] = 0.5f * ( grid_out[Index(rscl, gstl, k)] + grid_out[Index(gstl, rscl, k)]);

}

#undef gst0
#undef gstl
#undef rsc0
#undef rscl
#undef is


__global__ void kernelLineSolver ( float *grid_inout, float *grid0_in, int boundary, float a, float c )
{
	// Get index of GPU-thread
	GetIndex ( );

	if ( i >= 1 && i <= SimArea_X && j >= 1 && j <= SimArea_X )
	{
		grid_inout[Index(i, j, k)] = (grid0_in[Index(i, j, k)] + a * ( grid_inout[Index(i-1, j, k)] + 
			grid_inout[Index(i+1, j, k)] + grid_inout[Index(i, j-1, k)] + grid_inout[Index(i, j+1,k)] ) ) / c;	
	}
}


__global__ void kernelAdvect ( float *density_out, float *density0_in, float *u_in, float *v_in, float *w_in, float dt0 )
{
	// Get index of GPU-thread
	GetIndex ( );

	int i0, j0, i1, j1;
	float x, y, s0, t0, s1, t1;

	if ( i >= 1 && i <= SimArea_X && j >= 1 && j <= SimArea_X )
	{
		x = i - dt0 * u_in[Index(i, j, k)];
		y = j - dt0 * v_in[Index(i, j, k)];
		if (x < 0.5f) x = 0.5f;
		if (x > SimArea_X + 0.5f) x = SimArea_X+0.5f;

		i0 = (int)x; 
		i1 = i0+1;
		
		if (y < 0.5f) y=0.5f;
		if (y > SimArea_X+0.5f) y = SimArea_X+0.5f;
		
		j0 = (int)y;
		j1 = j0 + 1;
		s1 = x - i0;
		s0 = 1 - s1;
		t1 = y - j0;
		t0 = 1 - t1;

		density_out[Index(i, j, k)] = s0 * ( t0 * density0_in[Index(i0, j0, k)] +
			t1 * density0_in[Index(i0, j1, k)]) + s1 * ( t0 * density0_in[Index(i1, j0, k)] + 
			t1 * density0_in[Index(i1, j1, k)]);
	}
};



__global__ void kernelProjectPt1 ( float *u, float *v, float *w, float *u0, float *v0, float *w0 )
{
	// Get index of GPU-thread
	GetIndex ( );
	
	if ( i >= 1 && i <= SimArea_X && j >= 1 && j <= SimArea_X )
	{
		v0 [ Index (i, j, k) ] = -0.5f * ( u [ Index ( i+1, j, k ) ] - u [ Index ( i-1, j, k ) ] + v [ Index ( i, j+1, k ) ] 
		- v [ Index ( i, j-1, k ) ] ) / SimArea_X;
		u0 [ Index (i, j, k) ] = 0;
	}
}


__global__ void kernelProjectPt2( float *u, float *v, float *w, float *u0, float *v0, float *w0 )
{
	// Get index of GPU-thread
	GetIndex ( );
	
	if ( i >= 1 && i <= SimArea_X && j >= 1 && j <= SimArea_X )
	{
			u [ Index ( i, j, k ) ] -= 0.5f * SimArea_X * ( u0 [ Index ( i+1, j, k ) ] - u0 [ Index ( i-1, j, k ) ] );
			v [ Index ( i, j, k ) ] -= 0.5f * SimArea_X * ( u0 [ Index ( i, j+1, k ) ] - u0 [ Index ( i, j-1, k ) ] );
	}
}


void cudaAddSource ( float *grid, dim3 *gridDim, dim3 *blockDim )
{
    // Launch a kernel on the GPU with one thread for each element.
	kernelAddSource cudaDevice(*gridDim,  *blockDim) (grid);
};


void cudaLineSolver (float *grid, float *grid0, int boundary, float a, float c, dim3 *gridDim, dim3 *blockDim)
{
    // Launch a kernel on the GPU with one thread for each element.
	for (int i=0; i<20; i++)
	{
		kernelLineSolver cudaDevice(*gridDim,  *blockDim) (grid, grid0, boundary, a, c);
	}
	kernelSetBoundary cudaDevice(*gridDim,  *blockDim)  (grid, boundary);
}


void cudaDiffuse ( float *grid, float *grid0, int boundary, float diff, dim3 *gridDim, dim3 *blockDim )
{
	float a=DELTA_TIME*diff*SimArea_X*SimArea_X;
	cudaLineSolver ( grid, grid0, boundary, a, 1+4*a, gridDim, blockDim );
}


void cudaAdvect( float *density, float *density0, float *u, float *v, float *w, int boundary, dim3 *gridDim, dim3 *blockDim )
{
    // Launch a kernel on the GPU with one thread for each element.
	float dt0 = DELTA_TIME*SimArea_X;
	kernelAdvect      cudaDevice(*gridDim,  *blockDim) ( density, density0, u, v, w, dt0 );
	kernelSetBoundary cudaDevice(*gridDim,  *blockDim) ( density, boundary );
}


void cudaProject ( float *u, float *v, float *w, float *u0, float *v0, float *w0, dim3 *gridDim, dim3 *blockDim )
{
	kernelProjectPt1   cudaDevice(*gridDim,  *blockDim) ( u, v, w, u0, v0, w0 );
	kernelSetBoundary  cudaDevice(*gridDim,  *blockDim) ( v0, 0 );
	kernelSetBoundary  cudaDevice(*gridDim,  *blockDim) ( u0, 0 );
	kernelLineSolver   cudaDevice(*gridDim,  *blockDim) ( u0, v0, 0, 1, 4 );
	kernelProjectPt2   cudaDevice(*gridDim,  *blockDim) ( u, v, w, u0, v0, w0 );
	kernelSetBoundary  cudaDevice(*gridDim,  *blockDim) ( u, 1 );
	kernelSetBoundary  cudaDevice(*gridDim,  *blockDim) ( v, 2 );
}


void DensitySolver ( float *grid, float *grid0, float *u, float *v, float *w )
{
	// Define the computing unit size
	cudaDeviceDim3D ( );
	
    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy ( dev_grid, grid, size * sizeof(float), hipMemcpyHostToDevice );
    if (cudaStatus != hipSuccess) {
		Logfile.SaveStringToFile ( "errormsg.log", sge::SG_FILE_OPEN_APPEND,
			"hipMemcpy was failed, at line: %d of file %s", __LINE__, __FILE__ );
		Logfile.SaveStringToFile ( "errormsg.log", sge::SG_FILE_OPEN_APPEND, 
			">>>> Error Message: %s", hipGetErrorString ( cudaStatus ) );
		FreeResources ( ); exit ( 0 );
    }

	cudaStatus = hipMemcpy ( dev_grid0, grid0, size * sizeof(float), hipMemcpyHostToDevice );
    if (cudaStatus != hipSuccess) {
		Logfile.SaveStringToFile ( "errormsg.log", sge::SG_FILE_OPEN_APPEND, 
			"hipMemcpy was failed, at line: %d of file %s", __LINE__, __FILE__ );
		Logfile.SaveStringToFile ( "errormsg.log", sge::SG_FILE_OPEN_APPEND, 
			">>>> Error Message: %s", hipGetErrorString ( cudaStatus ) );
		FreeResources ( ); exit ( 0 );
    }

	cudaStatus = hipMemcpy ( dev_u, u, size * sizeof(float), hipMemcpyHostToDevice );
    if (cudaStatus != hipSuccess) {
		Logfile.SaveStringToFile ( "errormsg.log", sge::SG_FILE_OPEN_APPEND, 
			"hipMemcpy was failed, at line: %d of file %s", __LINE__, __FILE__ );
		Logfile.SaveStringToFile ( "errormsg.log", sge::SG_FILE_OPEN_APPEND,
			">>>> Error Message: %s", hipGetErrorString ( cudaStatus ) );
		FreeResources ( ); exit ( 0 );    
	}

	cudaStatus = hipMemcpy ( dev_v, v, size * sizeof(float), hipMemcpyHostToDevice );
    if (cudaStatus != hipSuccess) {
		Logfile.SaveStringToFile ( "errormsg.log", sge::SG_FILE_OPEN_APPEND, 
			"hipMemcpy was failed, at line: %d of file %s", __LINE__, __FILE__ );
		Logfile.SaveStringToFile ( "errormsg.log", sge::SG_FILE_OPEN_APPEND, 
			">>>> Error Message: %s", hipGetErrorString ( cudaStatus ) );
		FreeResources ( ); exit ( 0 );
    }

	cudaStatus = hipMemcpy ( dev_w, w, size * sizeof(float), hipMemcpyHostToDevice );
    if (cudaStatus != hipSuccess) {
		Logfile.SaveStringToFile ( "errormsg.log", sge::SG_FILE_OPEN_APPEND, 
			"hipMemcpy was failed, at line: %d of file %s", __LINE__, __FILE__ );
		Logfile.SaveStringToFile ( "errormsg.log", sge::SG_FILE_OPEN_APPEND, 
			">>>> Error Message: %s", hipGetErrorString ( cudaStatus ) );
		FreeResources ( ); exit ( 0 );
    }


	cudaAddSource ( dev_grid, &gridDim, &blockDim );
	swap ( dev_grid0, dev_grid ); cudaDiffuse ( dev_grid, dev_grid0, 0, DIFFUSION, &gridDim, &blockDim );
	swap ( dev_grid0, dev_grid ); cudaAdvect  ( dev_grid, dev_grid0, dev_u, dev_v, dev_w, 0, &gridDim, &blockDim );
	
	
	// Check for any errors launching the kernel
    cudaStatus = hipGetLastError ( );
    if (cudaStatus != hipSuccess) {
		Logfile.SaveStringToFile ( "errormsg.log", sge::SG_FILE_OPEN_APPEND,
			"CUDA encountered an error, at line: %d of file %s", __LINE__, __FILE__ );
		Logfile.SaveStringToFile ( "errormsg.log", sge::SG_FILE_OPEN_APPEND, 
			">>>> Error Message: %s", hipGetErrorString ( cudaStatus ) );
		FreeResources ( ); exit ( 0 );
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize ( );
    if (cudaStatus != hipSuccess) {
		Logfile.SaveStringToFile ( "errormsg.log", sge::SG_FILE_OPEN_APPEND,
			"hipDeviceSynchronize was failed, at line: %d of file %s", __LINE__, __FILE__ );
		Logfile.SaveStringToFile ( "errormsg.log", sge::SG_FILE_OPEN_APPEND,
			">>>> Error Message: %s", hipGetErrorString ( cudaStatus ) );
		FreeResources ( ); exit ( 0 );
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy ( grid, dev_grid, size * sizeof(int), hipMemcpyDeviceToHost );
    if (cudaStatus != hipSuccess) {
		Logfile.SaveStringToFile ( "errormsg.log", sge::SG_FILE_OPEN_APPEND, 
			"hipMemcpy was failed, at line: %d of file %s", __LINE__, __FILE__ );
		Logfile.SaveStringToFile ( "errormsg.log", sge::SG_FILE_OPEN_APPEND, 
			">>>> Error Message: %s", hipGetErrorString ( cudaStatus ) );
		FreeResources ( ); exit ( 0 );
    }

	cudaStatus = hipMemcpy ( grid0, dev_grid0, size * sizeof(int), hipMemcpyDeviceToHost );
    if (cudaStatus != hipSuccess) {
		Logfile.SaveStringToFile ( "errormsg.log", sge::SG_FILE_OPEN_APPEND, 
			"hipMemcpy was failed, at line: %d of file %s", __LINE__, __FILE__ );
		Logfile.SaveStringToFile ( "errormsg.log", sge::SG_FILE_OPEN_APPEND,
			">>>> Error Message: %s", hipGetErrorString ( cudaStatus ) );
		FreeResources ( ); exit ( 0 );
	}
	
	cudaStatus = hipMemcpy ( u, dev_u, size * sizeof(int), hipMemcpyDeviceToHost );
    if (cudaStatus != hipSuccess) {
		Logfile.SaveStringToFile ( "errormsg.log", sge::SG_FILE_OPEN_APPEND, 
			"hipMemcpy was failed, at line: %d of file %s", __LINE__, __FILE__ );
		Logfile.SaveStringToFile ( "errormsg.log", sge::SG_FILE_OPEN_APPEND, 
			">>>> Error Message: %s", hipGetErrorString ( cudaStatus ) );
		FreeResources ( ); exit ( 0 );
	}

	cudaStatus = hipMemcpy ( v, dev_v, size * sizeof(int), hipMemcpyDeviceToHost );
    if (cudaStatus != hipSuccess) {
		Logfile.SaveStringToFile ( "errormsg.log", sge::SG_FILE_OPEN_APPEND,
			"hipMemcpy was failed, at line: %d of file %s", __LINE__, __FILE__ );
		Logfile.SaveStringToFile ( "errormsg.log", sge::SG_FILE_OPEN_APPEND, 
			">>>> Error Message: %s", hipGetErrorString ( cudaStatus ) );
		FreeResources ( ); exit ( 0 );
	}

	cudaStatus = hipMemcpy ( w, dev_w, size * sizeof(int), hipMemcpyDeviceToHost );
    if (cudaStatus != hipSuccess) {
		Logfile.SaveStringToFile ( "errormsg.log", sge::SG_FILE_OPEN_APPEND,
			"hipMemcpy was failed, at line: %d of file %s", __LINE__, __FILE__ );
		Logfile.SaveStringToFile ( "errormsg.log", sge::SG_FILE_OPEN_APPEND, 
			">>>> Error Message: %s", hipGetErrorString ( cudaStatus ) );
		FreeResources ( ); exit ( 0 );
	}
}


void VelocitySolver ( float *u, float *v, float *w, float *u0, float *v0, float *w0 )
{
	// Define the computing unit size
	cudaDeviceDim3D ( );
	
    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy ( dev_u0, u0, size * sizeof(float), hipMemcpyHostToDevice );
    if (cudaStatus != hipSuccess) {
		Logfile.SaveStringToFile ( "errormsg.log", sge::SG_FILE_OPEN_APPEND, 
			"hipMemcpy was failed, at line: %d of file %s", __LINE__, __FILE__ );
		Logfile.SaveStringToFile ( "errormsg.log", sge::SG_FILE_OPEN_APPEND,
			">>>> Error Message: %s", hipGetErrorString ( cudaStatus ) );
		FreeResources ( ); exit ( 0 );
    }

	cudaStatus = hipMemcpy ( dev_v0, v0, size * sizeof(float), hipMemcpyHostToDevice );
    if (cudaStatus != hipSuccess) {
		Logfile.SaveStringToFile ( "errormsg.log", sge::SG_FILE_OPEN_APPEND,
			"hipMemcpy was failed, at line: %d of file %s", __LINE__, __FILE__);
		Logfile.SaveStringToFile ( "errormsg.log", sge::SG_FILE_OPEN_APPEND,
			">>>> Error Message: %s", hipGetErrorString ( cudaStatus ) );
		FreeResources ( ); exit ( 0 );
    }

	cudaStatus = hipMemcpy ( dev_w0, w0, size * sizeof(float), hipMemcpyHostToDevice );
    if (cudaStatus != hipSuccess) {
		Logfile.SaveStringToFile ( "errormsg.log", sge::SG_FILE_OPEN_APPEND,
			"hipMemcpy was failed, at line: %d of file %s", __LINE__, __FILE__);
		Logfile.SaveStringToFile ( "errormsg.log", sge::SG_FILE_OPEN_APPEND,
			">>>> Error Message: %s", hipGetErrorString ( cudaStatus ) );
		FreeResources ( ); exit ( 0 );
    }

	cudaStatus = hipMemcpy ( dev_u, u, size * sizeof(float), hipMemcpyHostToDevice );
    if (cudaStatus != hipSuccess) {
		Logfile.SaveStringToFile ( "errormsg.log", sge::SG_FILE_OPEN_APPEND,
			"hipMemcpy was failed, at line: %d of file %s", __LINE__, __FILE__ );
		Logfile.SaveStringToFile ( "errormsg.log", sge::SG_FILE_OPEN_APPEND, 
			">>>> Error Message: %s", hipGetErrorString ( cudaStatus ) );
		FreeResources ( ); exit ( 0 );
    }

	cudaStatus = hipMemcpy ( dev_v, v, size * sizeof(float), hipMemcpyHostToDevice );
    if (cudaStatus != hipSuccess) {
		Logfile.SaveStringToFile ( "errormsg.log", sge::SG_FILE_OPEN_APPEND,
			"hipMemcpy was failed, at line: %d of file %s", __LINE__, __FILE__ );
		Logfile.SaveStringToFile ( "errormsg.log", sge::SG_FILE_OPEN_APPEND,
			">>>> Error Message: %s", hipGetErrorString ( cudaStatus ) );
		FreeResources ( ); exit ( 0 );
    }

	cudaStatus = hipMemcpy ( dev_w, w, size * sizeof(float), hipMemcpyHostToDevice );
    if (cudaStatus != hipSuccess) {
		Logfile.SaveStringToFile("errormsg.log", sge::SG_FILE_OPEN_APPEND,
			"hipMemcpy was failed, at line: %d of file %s", __LINE__, __FILE__);
		Logfile.SaveStringToFile("errormsg.log", sge::SG_FILE_OPEN_APPEND, 
			">>>> Error Message: %s", hipGetErrorString(cudaStatus));
		FreeResources ( ); exit ( 0 );
    }


	cudaAddSource ( dev_u, &gridDim, &blockDim ); cudaAddSource ( dev_v, &gridDim, &blockDim );
	swap ( dev_u0, dev_u ); cudaDiffuse ( dev_u, dev_u0, 1, VISCOSITY, &gridDim, &blockDim );
	swap ( dev_v0, dev_v ); cudaDiffuse ( dev_v, dev_v0, 2, VISCOSITY, &gridDim, &blockDim );
	cudaProject ( dev_u, dev_v, dev_w, dev_u0, dev_v0, dev_w0, &gridDim, &blockDim );
	swap ( dev_u0, dev_u ); swap ( dev_v0, dev_v );
	cudaAdvect ( dev_u, dev_u0, dev_u0, dev_v0, dev_w0, 1, &gridDim, &blockDim );
	cudaAdvect ( dev_v, dev_v0, dev_u0, dev_v0, dev_w0, 2, &gridDim, &blockDim );
	cudaProject ( dev_u, dev_v, dev_w, dev_u0, dev_v0, dev_w0, &gridDim, &blockDim );


	// Check for any errors launching the kernel
    cudaStatus = hipGetLastError ( );
    if (cudaStatus != hipSuccess) {
		Logfile.SaveStringToFile ( "errormsg.log", sge::SG_FILE_OPEN_APPEND, 
			"CUDA encountered an error, at line: %d of file %s", __LINE__, __FILE__ );
		Logfile.SaveStringToFile ( "errormsg.log", sge::SG_FILE_OPEN_APPEND, 
			">>>> Error Message: %s", hipGetErrorString ( cudaStatus ) );
		FreeResources ( ); exit ( 0 );
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize ( );
    if (cudaStatus != hipSuccess) {
		Logfile.SaveStringToFile ( "errormsg.log", sge::SG_FILE_OPEN_APPEND,
			"hipDeviceSynchronize was failed, at line: %d of file %s", __LINE__, __FILE__ );
		Logfile.SaveStringToFile ( "errormsg.log", sge::SG_FILE_OPEN_APPEND,
			">>>> Error Message: %s", hipGetErrorString ( cudaStatus ) );
		FreeResources ( ); exit ( 0 );
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy ( u0, dev_u0, size * sizeof(int), hipMemcpyDeviceToHost );
    if (cudaStatus != hipSuccess) {
		Logfile.SaveStringToFile ( "errormsg.log", sge::SG_FILE_OPEN_APPEND,
			"hipMemcpy was failed, at line: %d of file %s", __LINE__, __FILE__);
		Logfile.SaveStringToFile ( "errormsg.log", sge::SG_FILE_OPEN_APPEND,
			">>>> Error Message: %s", hipGetErrorString ( cudaStatus ) );
		FreeResources ( ); exit ( 0 );
    }

	cudaStatus = hipMemcpy ( v0, dev_v0, size * sizeof(int), hipMemcpyDeviceToHost );
    if (cudaStatus != hipSuccess) {
		Logfile.SaveStringToFile ( "errormsg.log", sge::SG_FILE_OPEN_APPEND, 
			"hipMemcpy was failed, at line: %d of file %s", __LINE__, __FILE__);
		Logfile.SaveStringToFile ( "errormsg.log", sge::SG_FILE_OPEN_APPEND, 
			">>>> Error Message: %s", hipGetErrorString ( cudaStatus ) );
		FreeResources ( ); exit ( 0 );
    }

	cudaStatus = hipMemcpy ( w0, dev_w0, size * sizeof(int), hipMemcpyDeviceToHost );
    if (cudaStatus != hipSuccess) {
		Logfile.SaveStringToFile("errormsg.log", sge::SG_FILE_OPEN_APPEND, 
			"hipMemcpy was failed, at line: %d of file %s", __LINE__, __FILE__ );
		Logfile.SaveStringToFile("errormsg.log", sge::SG_FILE_OPEN_APPEND, 
			">>>> Error Message: %s", hipGetErrorString ( cudaStatus ) );
		FreeResources ( ); exit ( 0 );
    }
	
	cudaStatus = hipMemcpy ( u, dev_u, size * sizeof(int), hipMemcpyDeviceToHost );
    if (cudaStatus != hipSuccess) {
		Logfile.SaveStringToFile( "errormsg.log", sge::SG_FILE_OPEN_APPEND,
			"hipMemcpy was failed, at line: %d of file %s", __LINE__, __FILE__ );
		Logfile.SaveStringToFile( "errormsg.log", sge::SG_FILE_OPEN_APPEND, 
			">>>> Error Message: %s", hipGetErrorString ( cudaStatus ) );
		FreeResources ( ); exit ( 0 );
    }

	cudaStatus = hipMemcpy ( v, dev_v, size * sizeof(int), hipMemcpyDeviceToHost );
    if (cudaStatus != hipSuccess) {
		Logfile.SaveStringToFile ( "errormsg.log", sge::SG_FILE_OPEN_APPEND, 
			"hipMemcpy was failed, at line: %d of file %s", __LINE__, __FILE__ );
		Logfile.SaveStringToFile ( "errormsg.log", sge::SG_FILE_OPEN_APPEND, 
			">>>> Error Message: %s", hipGetErrorString ( cudaStatus ) );
		FreeResources ( ); exit ( 0 );
    }

	cudaStatus = hipMemcpy ( w, dev_w, size * sizeof(int), hipMemcpyDeviceToHost );
    if (cudaStatus != hipSuccess) {
		Logfile.SaveStringToFile ( "errormsg.log", sge::SG_FILE_OPEN_APPEND, 
			"hipMemcpy was failed, at line: %d of file %s", __LINE__, __FILE__ );
		Logfile.SaveStringToFile ( "errormsg.log", sge::SG_FILE_OPEN_APPEND, 
			">>>> Error Message: %s", hipGetErrorString ( cudaStatus ) );
		FreeResources ( ); exit ( 0 );
    }
}

#endif