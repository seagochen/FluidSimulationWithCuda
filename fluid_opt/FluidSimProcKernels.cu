#include "hip/hip_runtime.h"
/**
* <Author>        Orlando Chen
* <Email>         seagochen@gmail.com
* <First Time>    Dec 15, 2013
* <Last Time>     Feb 23, 2014
* <File Name>     FluidSimProcKernels.cu
*/

#include <iostream>
#include <utility>
#include "FluidSimulationKernels.h"
#include "CUDATracingKernels.h"

using namespace sge;

__global__ void kernelCopyGrids( double *dst, double const *src )
{
	GetIndex();

	dst[Index(i,j,k)] = src[Index(i,j,k)];
};

__global__ void kernelZeroGrids( double *grid )
{
	GetIndex ();
	grid [ Index(i,j,k) ] = 0.f;
};

__global__ void kernelZeroVolumetric( SGUCHAR *visual )
{
	GetIndex();

	for ( int ii = 0; ii < NODES_X; ii++ )
	{
		for ( int jj = 0; jj < NODES_X; jj++ )
		{
			for ( int kk = 0; kk < NODES_X; kk++ )
			{
				int di = ii * GRIDS_X + i;
				int dj = jj * GRIDS_X + j;
				int dk = kk * GRIDS_X + k;
				
				/* zero data */
				visual[ cudaIndex3D(di, dj, dk, VOLUME_X) ] = 0;
			}
		}
	}
};

__global__ void kernelZeroTemporaryBuffers( double *bufs )
{
	GetIndex();

	bufs[threadIdx.x] = 0.f;
};

__global__ void kernelZeroTemporaryBuffers( int *bufs )
{
	bufs[threadIdx.x] = 0;
};

__global__ void kernelPickData
( unsigned char *data, const double *bufs, int const offseti, int const offsetj, int const offsetk )
{
	GetIndex();

	int di = offseti + i;
	int dj = offsetj + j;
	int dk = offsetk + k;

	/* zero data first */
	data[ cudaIndex3D(di, dj, dk, VOLUME_X) ] = 0;

	/* retrieve data from grid */
	double value = bufs[ Index(i, j, k) ];

	/* append data to volume data */
	int temp = atomicRound( value );
	if ( temp > 0 and temp < 250 )
		data [ cudaIndex3D(di, dj, dk, VOLUME_X) ] = (unsigned char) temp;
};


FluidSimProc::FluidSimProc ( FLUIDSPARAM *fluid )
{
	/* initialize FPS */
	InitParams( fluid );

	/* allocate resources */
	if ( !AllocateResource ( fluid ) ) { FreeResource (); exit (1); }

	/* build order */
	BuildOrder();

#if !TESTING_MODE_SWITCH
	/* select node */
	ActiveTheNode( 1, 0, 1 );

	/* clear buffer */
	ZeroBuffers();

	/* set boundary */
	InitBoundary( 1, 0, 1 );
#else
	/* select node */
	ActiveTheNode( 1, 1, 1 );

	/* clear buffer */
	ZeroBuffers();

	/* set boundary */
	InitBoundary( 1, 1, 1 );
#endif

	/* upload buffers */
	UploadBuffers();

	/* finally, print message */
	printf( "fluid simulation ready...\n" );
};

void FluidSimProc::UploadBuffers()
{
	for ( int i = 0; i < NODES_X * NODES_X * NODES_X; i++ )
	{
		hipMemcpy( dev_density[i], host_density[i], m_node_size, hipMemcpyHostToDevice );
		hipMemcpy( dev_velocity_u[i], host_velocity_u[i], m_node_size, hipMemcpyHostToDevice );
		hipMemcpy( dev_velocity_v[i], host_velocity_v[i], m_node_size, hipMemcpyHostToDevice );
		hipMemcpy( dev_velocity_w[i], host_velocity_w[i], m_node_size, hipMemcpyHostToDevice );
		hipMemcpy( dev_obstacle[i], host_obstacle[i], m_node_size, hipMemcpyHostToDevice );

		if ( helper.GetCUDALastError( "hipMemcpy failed", __FILE__, __LINE__ ) )
		{
			FreeResource();
			exit(1);
		}
	}
};

void FluidSimProc::InitParams( FLUIDSPARAM *fluid )
{
	fluid->fps.dwCurrentTime = 0;
	fluid->fps.dwElapsedTime = 0;
	fluid->fps.dwFrames = 0;
	fluid->fps.dwLastUpdateTime = 0;
	fluid->fps.uFPS = 0;

	m_node_size = GRIDS_X * GRIDS_X * GRIDS_X * sizeof(double);
	m_volm_size = VOLUME_X * VOLUME_X * VOLUME_X * sizeof(SGUCHAR);

	increase_times = decrease_times = 0;
};

void FluidSimProc::BuildOrder( void )
{
	for ( int k = 0; k < NODES_X; k++ )
	{
		for ( int j = 0; j < NODES_X; j++ )
		{
			for ( int i = 0; i < NODES_X; i++ )
			{
				/* left */
				if ( i >= 1 )
					host_node[cudaIndex3D( i, j, k, NODES_X )]->ptrLeft  = host_node[cudaIndex3D( i-1, j, k, NODES_X )];
				/* right */
				if ( i <= NODES_X - 2 )
					host_node[cudaIndex3D( i, j, k, NODES_X )]->ptrRight = host_node[cudaIndex3D( i+1, j, k, NODES_X )];
				/* down */
				if ( j >= 1 )
					host_node[cudaIndex3D( i, j, k, NODES_X )]->ptrDown  = host_node[cudaIndex3D( i, j-1, k, NODES_X )];
				/* up */
				if ( j <= NODES_X - 2 )
					host_node[cudaIndex3D( i, j, k, NODES_X )]->ptrUp    = host_node[cudaIndex3D( i, j+1, k, NODES_X )];
				/* back */
				if ( k >= 1 )
					host_node[cudaIndex3D( i, j, k, NODES_X )]->ptrBack  = host_node[cudaIndex3D( i, j, k-1, NODES_X )];
				/* front */
				if ( k <= NODES_X - 2 )
					host_node[cudaIndex3D( i, j, k, NODES_X )]->ptrFront = host_node[cudaIndex3D( i, j, k+1, NODES_X )];

				host_node[cudaIndex3D( i, j, k, NODES_X )]->nodeIX.x = i;
				host_node[cudaIndex3D( i, j, k, NODES_X )]->nodeIX.y = j;
				host_node[cudaIndex3D( i, j, k, NODES_X )]->nodeIX.z = k;
			}
		}
	}

	for ( int i = 0; i < NODES_X; i++ )
	{
		for ( int j = 0; j < NODES_X; j++ )
		{
			for ( int k = 0; k < NODES_X; k++ )
			{
				printf ( "offset: %d %d %d | L: %d | R: %d | U: %d | D: %d | F: %d | B: %d \n",
					host_node[cudaIndex3D( i, j, k, NODES_X )]->nodeIX.x, 
					host_node[cudaIndex3D( i, j, k, NODES_X )]->nodeIX.y, 
					host_node[cudaIndex3D( i, j, k, NODES_X )]->nodeIX.z,
					host_node[cudaIndex3D( i, j, k, NODES_X )]->ptrLeft not_eq nullptr,
					host_node[cudaIndex3D( i, j, k, NODES_X )]->ptrRight not_eq nullptr,
					host_node[cudaIndex3D( i, j, k, NODES_X )]->ptrUp not_eq nullptr,
					host_node[cudaIndex3D( i, j, k, NODES_X )]->ptrDown not_eq nullptr,
					host_node[cudaIndex3D( i, j, k, NODES_X )]->ptrFront not_eq nullptr,
					host_node[cudaIndex3D( i, j, k, NODES_X )]->ptrBack not_eq nullptr );
			}
		}
	}
};

bool FluidSimProc::AllocateResource ( FLUIDSPARAM *fluid )
{
	/* choose which GPU to run on, change this on a multi-GPU system. */
	if ( hipSetDevice ( 0 ) != hipSuccess )
	{
		helper.GetCUDALastError ( "cudaSetDevices", __FILE__, __LINE__ );
		return false;
	}

	/* allocate memory on host */
	for ( int i = 0; i < NODES_X * NODES_X * NODES_X; i++ )
	{
		double *ptrDens, *ptrU, *ptrV, *ptrW, *ptrObs;

		if ( helper.CreateHostBuffers( m_node_size, 1, &ptrDens ) not_eq SG_RUNTIME_OK )
			return false;
		if ( helper.CreateHostBuffers( m_node_size, 1, &ptrU ) not_eq SG_RUNTIME_OK )
			return false;
		if ( helper.CreateHostBuffers( m_node_size, 1, &ptrV ) not_eq SG_RUNTIME_OK )
			return false;
		if ( helper.CreateHostBuffers( m_node_size, 1, &ptrW ) not_eq SG_RUNTIME_OK )
			return false;
		if ( helper.CreateHostBuffers( m_node_size, 1, &ptrObs ) not_eq SG_RUNTIME_OK )
			return false;

		/* simulation nodes */
		SimNode *node = (SimNode*)malloc(sizeof(SimNode));
		node->ptrFront = node->ptrBack = nullptr;
		node->ptrLeft = node->ptrRight = nullptr;
		node->ptrDown = node->ptrUp = nullptr;
		host_node.push_back( node );

		node->active = false;

		host_density.push_back( ptrDens );
		host_velocity_u.push_back( ptrU );
		host_velocity_v.push_back( ptrV );
		host_velocity_w.push_back( ptrW );
		host_obstacle.push_back( ptrObs );
	}

	/* allocate memory on GPU */
	for ( int i = 0; i < NODES_X * NODES_X * NODES_X; i++ )
	{
		double *ptrDens, *ptrU, *ptrV, *ptrW, *ptrObs;

		if ( helper.CreateDeviceBuffers( m_node_size, 1, &ptrDens ) not_eq SG_RUNTIME_OK )
			return false;
		if ( helper.CreateDeviceBuffers( m_node_size, 1, &ptrU ) not_eq SG_RUNTIME_OK )
			return false;
		if ( helper.CreateDeviceBuffers( m_node_size, 1, &ptrV ) not_eq SG_RUNTIME_OK )
			return false;
		if ( helper.CreateDeviceBuffers( m_node_size, 1, &ptrW ) not_eq SG_RUNTIME_OK )
			return false;
		if ( helper.CreateDeviceBuffers( m_node_size, 1, &ptrObs ) not_eq SG_RUNTIME_OK )
			return false;

		dev_density.push_back( ptrDens );
		dev_velocity_u.push_back( ptrU );
		dev_velocity_v.push_back( ptrV );
		dev_velocity_w.push_back( ptrW );
		dev_obstacle.push_back( ptrObs );
	}

	/* allocate memory on GPU devices */
	for ( int i = 0; i < dev_buffers_num; i++ )
	{
		double *ptr;
		if ( helper.CreateDeviceBuffers( m_node_size, 1, &ptr ) not_eq SG_RUNTIME_OK )
			return false;

		dev_buffers.push_back(ptr);
	}

	/* allocate visual buffers */
	if ( helper.CreateDeviceBuffers( m_volm_size, 1, &dev_visual ) not_eq SG_RUNTIME_OK )
		return false;
	if ( helper.CreateHostBuffers( m_volm_size, 1, &host_visual ) not_eq SG_RUNTIME_OK )
		return false;

	/* allocate temporary buffers */
	if ( helper.CreateDeviceBuffers( sizeof(double)*TPBUFFER_X, 1, &dev_tpbufs ) not_eq SG_RUNTIME_OK )
		return false;
	if ( helper.CreateHostBuffers(sizeof(double)*TPBUFFER_X, 1, &host_tpbufs ) not_eq SG_RUNTIME_OK )
		return false;

	/* finally */
	return true;
}  

void FluidSimProc::FreeResource ( void )
{
	/* free host resource */
	for ( int i = 0; i < NODES_X * NODES_X * NODES_X; i++ )
	{
		helper.FreeHostBuffers( 5, &host_density[i], &host_velocity_u[i], &host_velocity_v[i], &host_velocity_w[i], &host_obstacle[i] );
		helper.FreeDeviceBuffers( 5, &dev_density[i], &dev_velocity_u[i], &dev_velocity_v[i], &dev_velocity_w[i], &dev_obstacle[i] );
	}

	/* free device resource */
	for ( int i = 0; i < dev_buffers_num; i++ )
	{
		helper.FreeDeviceBuffers( 1, &dev_buffers[i] );
	}

	/* free... */
	helper.FreeHostBuffers( 2, &host_visual, &host_tpbufs );
	helper.FreeDeviceBuffers( 2, &dev_visual, &dev_tpbufs );
}

bool FluidSimProc::SelectTheNode( int i, int j, int k )
{
	if ( i >= 0 and i < NODES_X and j >= 0 and j < NODES_X and k >= 0 and k < NODES_X )
	{
		nPos.x = i;
		nPos.y = j;
		nPos.z = k;
		
#if !TESTING_MODE_SWITCH
		int ix = cudaIndex3D( i, j, k, NODES_X );
		return host_node[ix]->active;
#else
		int ix    = cudaIndex3D( i, j, k, NODES_X );
		int left  = cudaIndex3D( 0, 1, 1, NODES_X );
		int right = cudaIndex3D( 2, 1, 1, NODES_X );
		int up    = cudaIndex3D( 1, 2, 1, NODES_X );
		int down  = cudaIndex3D( 1, 0, 1, NODES_X );
		int front = cudaIndex3D( 1, 1, 2, NODES_X );
		int back  = cudaIndex3D( 1, 1, 0, NODES_X );
		int center= cudaIndex3D( 1, 1, 1, NODES_X );

		return 
			ix eqt center or
			ix eqt left  or ix eqt right or
			ix eqt up 	 or ix eqt down  or
			ix eqt front or ix eqt back;
#endif
	}

	return false;
};

bool FluidSimProc::ActiveTheNode( int i, int j, int k )
{
	int ix;
	if ( i >= 0 and i < NODES_X and j >= 0 and j < NODES_X and k >= 0 and k < NODES_X )
	{
		ix = cudaIndex3D( i, j, k, NODES_X );
		host_node[ix]->active = true;
	}

	return host_node[ix]->active == true;
};

bool FluidSimProc::DeactiveTheNode( int i, int j, int k )
{
	int ix;
	if ( i >= 0 and i < NODES_X and j >= 0 and j < NODES_X and k >= 0 and k < NODES_X )
	{
		ix = cudaIndex3D( i, j, k, NODES_X );
		host_node[ix]->active = false;
	}

	return host_node[ix]->active == false;
};

void FluidSimProc::FluidSimSolver( FLUIDSPARAM *fluid )
{
	if ( !fluid->run ) return;

	for ( int i = 0; i < NODES_X; i++ )
	{
		for ( int j = 0; j < NODES_X; j++ )
		{
			for ( int k = 0; k < NODES_X; k++ )
			{
				//if ( SelectTheNode( i, j, k ) )
				{
					/* select node */
					SelectTheNode( i, j, k );

					/* for fluid simulation, copy the data to device */
					NodeToDevice();
					
					/* Fluid process */
					AddSource();
					VelocitySolver();
					DensitySolver();
					
					/* tracing */
					TracingTheFlow();

					/* retrieve data back to host */
					DeviceToNode();

					if ( hipDeviceSynchronize() not_eq hipSuccess )
					{
						printf( "hipDeviceSynchronize failed\n" );
						FreeResource();
						exit( 1 );
					}
				}
			}
		}
	}
	/* finally, generate volumetric image */
	GetVolumetric( fluid );
};

void FluidSimProc::GetVolumetric( FLUIDSPARAM *fluid )
{
	hipMemcpy( host_visual, dev_visual, m_volm_size, hipMemcpyDeviceToHost );
	fluid->volume.ptrData = host_visual;
};

void FluidSimProc::NodeToDevice ( void )
{
	/* navigate the node's position */
	int i = nPos.x;
	int j = nPos.y;
	int k = nPos.z;

	SimNode *ptr = host_node[cudaIndex3D( i, j, k, NODES_X )];
	int ix = cudaIndex3D( i, j, k, NODES_X );

	/* upload center node to GPU device */
	cudaDeviceDim3D();
	kernelCopyGrids <<<gridDim, blockDim>>> ( dev_u, dev_velocity_u[ix] );
	kernelCopyGrids <<<gridDim, blockDim>>> ( dev_v, dev_velocity_v[ix] );
	kernelCopyGrids <<<gridDim, blockDim>>> ( dev_w, dev_velocity_w[ix] );
	kernelCopyGrids <<<gridDim, blockDim>>> ( dev_den, dev_density[ix] );
	kernelCopyGrids <<<gridDim, blockDim>>> ( dev_obs, dev_obstacle[ix] );

	if ( helper.GetCUDALastError( "hipMemcpy failed", __FILE__, __LINE__ ) )
	{
		FreeResource();
		exit( 1 );
	}

	/* upload neighbouring buffers to GPU device */
	if ( ptr->ptrLeft not_eq nullptr )
	{
		ix = cudaIndex3D( i-1, j, k, NODES_X );
		kernelCopyGrids <<<gridDim, blockDim>>> ( velu_L, dev_velocity_u[ix] );
		kernelCopyGrids <<<gridDim, blockDim>>> ( velv_L, dev_velocity_v[ix] );
		kernelCopyGrids <<<gridDim, blockDim>>> ( velw_L, dev_velocity_w[ix] );
		kernelCopyGrids <<<gridDim, blockDim>>> ( dens_L, dev_density[ix] );

		if ( helper.GetCUDALastError( "hipMemcpy failed", __FILE__, __LINE__ ) )
		{
			FreeResource();
			exit( 1 );
		}
	}

	if ( ptr->ptrRight not_eq nullptr )
	{
		ix = cudaIndex3D( i+1, j, k, NODES_X );
		kernelCopyGrids <<<gridDim, blockDim>>> ( velu_R, dev_velocity_u[ix] );
		kernelCopyGrids <<<gridDim, blockDim>>> ( velv_R, dev_velocity_v[ix] );
		kernelCopyGrids <<<gridDim, blockDim>>> ( velw_R, dev_velocity_w[ix] );
		kernelCopyGrids <<<gridDim, blockDim>>> ( dens_R, dev_density[ix] );

		if ( helper.GetCUDALastError( "hipMemcpy failed", __FILE__, __LINE__ ) )
		{
			FreeResource();
			exit( 1 );
		}
	}

	if ( ptr->ptrUp not_eq nullptr )
	{
		ix = cudaIndex3D( i, j+1, k, NODES_X );
		kernelCopyGrids <<<gridDim, blockDim>>> ( velu_U, dev_velocity_u[ix] );
		kernelCopyGrids <<<gridDim, blockDim>>> ( velv_U, dev_velocity_v[ix] );
		kernelCopyGrids <<<gridDim, blockDim>>> ( velw_U, dev_velocity_w[ix] );
		kernelCopyGrids <<<gridDim, blockDim>>> ( dens_U, dev_density[ix] );

		if ( helper.GetCUDALastError( "hipMemcpy failed", __FILE__, __LINE__ ) )
		{
			FreeResource();
			exit( 1 );
		}
	}

	if ( ptr->ptrDown not_eq nullptr )
	{
		ix = cudaIndex3D( i, j-1, k, NODES_X );
		kernelCopyGrids <<<gridDim, blockDim>>> ( velu_D, dev_velocity_u[ix] );
		kernelCopyGrids <<<gridDim, blockDim>>> ( velv_D, dev_velocity_v[ix] );
		kernelCopyGrids <<<gridDim, blockDim>>> ( velw_D, dev_velocity_w[ix] );
		kernelCopyGrids <<<gridDim, blockDim>>> ( dens_D, dev_density[ix] );

		if ( helper.GetCUDALastError( "hipMemcpy failed", __FILE__, __LINE__ ) )
		{
			FreeResource();
			exit( 1 );
		}
	}

	if ( ptr->ptrFront not_eq nullptr )
	{
		ix = cudaIndex3D( i, j, k+1, NODES_X );
		kernelCopyGrids <<<gridDim, blockDim>>> ( velu_F, dev_velocity_u[ix] );
		kernelCopyGrids <<<gridDim, blockDim>>> ( velv_F, dev_velocity_v[ix] );
		kernelCopyGrids <<<gridDim, blockDim>>> ( velw_F, dev_velocity_w[ix] );
		kernelCopyGrids <<<gridDim, blockDim>>> ( dens_F, dev_density[ix] );

		if ( helper.GetCUDALastError( "hipMemcpy failed", __FILE__, __LINE__ ) )
		{
			FreeResource();
			exit( 1 );
		}
	}

	if ( ptr->ptrBack not_eq nullptr )
	{
		ix = cudaIndex3D( i, j, k-1, NODES_X );
		kernelCopyGrids <<<gridDim, blockDim>>> ( velu_B, dev_velocity_u[ix] );
		kernelCopyGrids <<<gridDim, blockDim>>> ( velv_B, dev_velocity_v[ix] );
		kernelCopyGrids <<<gridDim, blockDim>>> ( velw_B, dev_velocity_w[ix] );
		kernelCopyGrids <<<gridDim, blockDim>>> ( dens_B, dev_density[ix] );

		if ( helper.GetCUDALastError( "hipMemcpy failed", __FILE__, __LINE__ ) )
		{
			FreeResource();
			exit( 1 );
		}
	}
};

void FluidSimProc::DeviceToNode ( void )
{
	/* navigate the node's position */
	int i = nPos.x;
	int j = nPos.y;
	int k = nPos.z;
	SimNode *ptr = host_node[cudaIndex3D( i, j, k, NODES_X )];
	int ix = cudaIndex3D( i, j, k, NODES_X );

	/* draw data back */
	cudaDeviceDim3D();

	kernelCopyGrids <<<gridDim, blockDim>>> ( dev_density[ix], dev_den );
	kernelCopyGrids <<<gridDim, blockDim>>> ( dev_velocity_u[ix], dev_u );
	kernelCopyGrids <<<gridDim, blockDim>>> ( dev_velocity_v[ix], dev_v );
	kernelCopyGrids <<<gridDim, blockDim>>> ( dev_velocity_w[ix], dev_w );

	if ( helper.GetCUDALastError( "hipMemcpy failed", __FILE__, __LINE__ ) )
	{
		FreeResource();
		exit( 1 );
	}

	/* draw neighbouring buffers back */
	if ( ptr->ptrLeft not_eq nullptr )
	{
		ix = cudaIndex3D( i-1, j, k, NODES_X );
		kernelCopyGrids <<<gridDim, blockDim>>> ( dev_density[ix], dens_L );
		kernelCopyGrids <<<gridDim, blockDim>>> ( dev_velocity_u[ix], velu_L );
		kernelCopyGrids <<<gridDim, blockDim>>> ( dev_velocity_v[ix], velv_L );
		kernelCopyGrids <<<gridDim, blockDim>>> ( dev_velocity_w[ix], velw_L );

		if ( helper.GetCUDALastError( "hipMemcpy failed", __FILE__, __LINE__ ) )
		{
			FreeResource();
			exit( 1 );
		}
	}

	if ( ptr->ptrRight not_eq nullptr )
	{
		ix = cudaIndex3D( i+1, j, k, NODES_X );
		kernelCopyGrids <<<gridDim, blockDim>>> ( dev_density[ix], dens_R );
		kernelCopyGrids <<<gridDim, blockDim>>> ( dev_velocity_u[ix], velu_R );
		kernelCopyGrids <<<gridDim, blockDim>>> ( dev_velocity_v[ix], velv_R );
		kernelCopyGrids <<<gridDim, blockDim>>> ( dev_velocity_w[ix], velw_R );

		if ( helper.GetCUDALastError( "hipMemcpy failed", __FILE__, __LINE__ ) )
		{
			FreeResource();
			exit( 1 );
		}
	}

	if ( ptr->ptrUp not_eq nullptr )
	{
		ix = cudaIndex3D( i, j+1, k, NODES_X );
		kernelCopyGrids <<<gridDim, blockDim>>> ( dev_density[ix], dens_U );
		kernelCopyGrids <<<gridDim, blockDim>>> ( dev_velocity_u[ix], velu_U );
		kernelCopyGrids <<<gridDim, blockDim>>> ( dev_velocity_v[ix], velv_U );
		kernelCopyGrids <<<gridDim, blockDim>>> ( dev_velocity_w[ix], velw_U );

		if ( helper.GetCUDALastError( "hipMemcpy failed", __FILE__, __LINE__ ) )
		{
			FreeResource();
			exit( 1 );
		}
	}

	if ( ptr->ptrDown not_eq nullptr )
	{
		ix = cudaIndex3D( i, j-1, k, NODES_X );
		kernelCopyGrids <<<gridDim, blockDim>>> ( dev_density[ix], dens_D );
		kernelCopyGrids <<<gridDim, blockDim>>> ( dev_velocity_u[ix], velu_D );
		kernelCopyGrids <<<gridDim, blockDim>>> ( dev_velocity_v[ix], velv_D );
		kernelCopyGrids <<<gridDim, blockDim>>> ( dev_velocity_w[ix], velw_D );

		if ( helper.GetCUDALastError( "hipMemcpy failed", __FILE__, __LINE__ ) )
		{
			FreeResource();
			exit( 1 );
		}
	}

	if ( ptr->ptrFront not_eq nullptr )
	{
		ix = cudaIndex3D( i, j, k+1, NODES_X );
		kernelCopyGrids <<<gridDim, blockDim>>> ( dev_density[ix], dens_F );
		kernelCopyGrids <<<gridDim, blockDim>>> ( dev_velocity_u[ix], velu_F );
		kernelCopyGrids <<<gridDim, blockDim>>> ( dev_velocity_v[ix], velv_F );
		kernelCopyGrids <<<gridDim, blockDim>>> ( dev_velocity_w[ix], velw_F );

		if ( helper.GetCUDALastError( "hipMemcpy failed", __FILE__, __LINE__ ) )
		{
			FreeResource();
			exit( 1 );
		}
	}

	if ( ptr->ptrBack not_eq nullptr )
	{
		ix = cudaIndex3D( i, j, k-1, NODES_X );
		kernelCopyGrids <<<gridDim, blockDim>>> ( dev_density[ix], dens_B );
		kernelCopyGrids <<<gridDim, blockDim>>> ( dev_velocity_u[ix], velu_B );
		kernelCopyGrids <<<gridDim, blockDim>>> ( dev_velocity_v[ix], velv_B );
		kernelCopyGrids <<<gridDim, blockDim>>> ( dev_velocity_w[ix], velw_B );

		if ( helper.GetCUDALastError( "hipMemcpy failed", __FILE__, __LINE__ ) )
		{
			FreeResource();
			exit( 1 );
		}
	}

	/* draw volumetric data back */
	kernelPickData <<<gridDim, blockDim>>>
		( dev_visual, dev_den, nPos.x * GRIDS_X, nPos.y * GRIDS_X, nPos.z * GRIDS_X );

};

void FluidSimProc::AddSource( void )
{
#if TESTING_MODE_SWITCH
	if ( decrease_times eqt 0 )
	{
		cudaDeviceDim3D();
		kernelAddSource<<<gridDim, blockDim>>> ( dev_den, dev_u, dev_v, dev_w, dev_obs );
		increase_times++;

		if ( increase_times eqt 200 )
		{
			decrease_times = increase_times;
			increase_times = 0;
		}
	}
	else
	{
		decrease_times--;
	}
#else
	cudaDeviceDim3D();
	kernelAddSource<<<gridDim, blockDim>>> ( dev_den, dev_u, dev_v, dev_w, dev_obs );
#endif
};

void FluidSimProc::InitBoundary( int i, int j, int k )
{
	cudaDeviceDim3D();

	/* zero boundary buffers */
	kernelZeroGrids<<<gridDim, blockDim>>>( dev_obs );

	for ( int i = 0; i < host_obstacle.size(); i++ )
	{
		if ( hipMemcpy( host_obstacle[i], dev_obs,
			m_node_size, hipMemcpyDeviceToHost ) not_eq hipSuccess )
		{
			helper.GetCUDALastError( "hipMemcpy failed", __FILE__, __LINE__ );
			FreeResource();
			exit( 1 );
		}
	}

	/* select middle node */
	SelectTheNode( i, j, k );

	const int ix = cudaIndex3D( nPos.x, nPos.y, nPos.z, NODES_X );

	/* set boundary */
	kernelSetBoundary<<<gridDim, blockDim>>>( dev_obs );
	
	if ( hipMemcpy( host_obstacle[ix], dev_obs,
		m_node_size, hipMemcpyDeviceToHost) not_eq hipSuccess )
	{
		helper.GetCUDALastError( "hipMemcpy failed", __FILE__, __LINE__ );
		FreeResource();
		exit( 1 );
	}
};

void FluidSimProc::VelocitySolver( void )
{
	// diffuse the velocity field (per axis):
	hostDiffusion( dev_u0, dev_u, VISOCITY, dev_obs, MACRO_VELOCITY_U );
	hostDiffusion( dev_v0, dev_v, VISOCITY, dev_obs, MACRO_VELOCITY_V );
	hostDiffusion( dev_w0, dev_w, VISOCITY, dev_obs, MACRO_VELOCITY_W );
	
	std::swap( dev_u0, dev_u );
	std::swap( dev_v0, dev_v );
	std::swap( dev_w0, dev_w );

	// stabilize it: (vx0, vy0 are whatever, being used as temporaries to store gradient field)
	hostProject( dev_u, dev_v, dev_w, dev_div, dev_p, dev_obs );
	
	// advect the velocity field (per axis):
	hostAdvection( dev_u0, dev_u, dev_obs, MACRO_VELOCITY_U, dev_u, dev_v, dev_w );
	hostAdvection( dev_v0, dev_v, dev_obs, MACRO_VELOCITY_V, dev_u, dev_v, dev_w );
	hostAdvection( dev_w0, dev_w, dev_obs, MACRO_VELOCITY_W, dev_u, dev_v, dev_w );
	std::swap( dev_u0, dev_u );
	std::swap( dev_v0, dev_v );
	std::swap( dev_w0, dev_w );
	
	// stabilize it: (vx0, vy0 are whatever, being used as temporaries to store gradient field)
	hostProject( dev_u, dev_v, dev_w, dev_div, dev_p, dev_obs );
};

void FluidSimProc::DensitySolver( void )
{
	hostDiffusion( dev_den0, dev_den, DIFFUSION, dev_obs, MACRO_DENSITY );
	std::swap( dev_den0, dev_den );
	hostAdvection ( dev_den, dev_den0, dev_obs, MACRO_DENSITY, dev_u, dev_v, dev_w );
};

void FluidSimProc::ZeroBuffers( void )
{
	cudaDeviceDim3D();

	/* zero host buffer */
	for ( int i = 0; i < NODES_X * NODES_X * NODES_X; i++ )
	{
		hipMemcpy( dev_density[i],    host_density[i],    m_node_size, hipMemcpyHostToDevice );
		hipMemcpy( dev_velocity_u[i], host_velocity_u[i], m_node_size, hipMemcpyHostToDevice );
		hipMemcpy( dev_velocity_v[i], host_velocity_v[i], m_node_size, hipMemcpyHostToDevice );
		hipMemcpy( dev_velocity_w[i], host_velocity_w[i], m_node_size, hipMemcpyHostToDevice );
		hipMemcpy( dev_obstacle[i],   host_obstacle[i],   m_node_size, hipMemcpyHostToDevice );

		if ( helper.GetCUDALastError( "hipMemcpy failed when zero buffers", __FILE__, __LINE__ ) )
		{
			FreeResource();
			exit(1);
		}
	}
	
	/* zero GPU buffer */
	for ( int i = 0; i < dev_buffers_num; i++ ) 
		kernelZeroGrids <<<gridDim, blockDim>>> ( dev_buffers[i] );

	/* zero visual buffer */
	kernelZeroVolumetric <<< gridDim, blockDim>>> ( dev_visual );
	hipMemcpy( host_visual, dev_visual, m_volm_size, hipMemcpyDeviceToHost );
};

void FluidSimProc::TracingTheFlow( void )
{
#if TESTING_MODE_SWITCH
	int ix  = cudaIndex3D( 1, 1, 1, NODES_X );
	int nix = cudaIndex3D( nPos.x, nPos.y, nPos.z, NODES_X );
#endif

	cudaDeviceDim3D();

	/* flooding data */
	kernelFloodingBuffers <<<gridDim, blockDim>>> ( dens_L, dens_R, dens_U, dens_D, dens_F, dens_B, dens_C );
	kernelFloodingBuffers <<<gridDim, blockDim>>> ( velu_L, velu_R, velu_U, velu_D, velu_F, velu_B, velu_C );
	kernelFloodingBuffers <<<gridDim, blockDim>>> ( velv_L, velv_R, velv_U, velv_D, velv_F, velv_B, velv_C );
	kernelFloodingBuffers <<<gridDim, blockDim>>> ( velw_L, velw_R, velw_U, velw_D, velw_F, velw_B, velw_C );

	/* clear temporary buffers for next step */
#if TESTING_MODE_SWITCH
	if ( ix eqt nix )
#endif
	kernelZeroTemporaryBuffers <<<1, TPBUFFER_X>>> ( dev_tpbufs );

	/* clear halo to avoid data obstruction */
	kernelClearHalo <<<gridDim, blockDim>>> ( dens_L, dens_R, dens_U, dens_D, dens_F, dens_B, dens_C );
	kernelClearHalo <<<gridDim, blockDim>>> ( velu_L, velu_R, velu_U, velu_D, velu_F, velu_B, velu_C );
	kernelClearHalo <<<gridDim, blockDim>>> ( velv_L, velv_R, velv_U, velv_D, velv_F, velv_B, velv_C );
	kernelClearHalo <<<gridDim, blockDim>>> ( velw_L, velw_R, velw_U, velw_D, velw_F, velw_B, velw_C );

	/* zero buffers if they not exists */
	SimNode *ptr = host_node[cudaIndex3D( nPos.x, nPos.y, nPos.z, NODES_X )];

	if ( !ptr->ptrLeft )
	{
		kernelZeroGrids <<<gridDim, blockDim>>> ( dens_L );
		kernelZeroGrids <<<gridDim, blockDim>>> ( velu_L );
		kernelZeroGrids <<<gridDim, blockDim>>> ( velv_L );
		kernelZeroGrids <<<gridDim, blockDim>>> ( velw_L );
	}
	if ( !ptr->ptrRight )
	{
		kernelZeroGrids <<<gridDim, blockDim>>> ( dens_R );
		kernelZeroGrids <<<gridDim, blockDim>>> ( velu_R );
		kernelZeroGrids <<<gridDim, blockDim>>> ( velv_R );
		kernelZeroGrids <<<gridDim, blockDim>>> ( velw_R );
	}
	if ( !ptr->ptrUp )
	{
		kernelZeroGrids <<<gridDim, blockDim>>> ( dens_U );
		kernelZeroGrids <<<gridDim, blockDim>>> ( velu_U );
		kernelZeroGrids <<<gridDim, blockDim>>> ( velv_U );
		kernelZeroGrids <<<gridDim, blockDim>>> ( velw_U );
	}
	if ( !ptr->ptrDown )
	{
		kernelZeroGrids <<<gridDim, blockDim>>> ( dens_D );
		kernelZeroGrids <<<gridDim, blockDim>>> ( velu_D );
		kernelZeroGrids <<<gridDim, blockDim>>> ( velv_D );
		kernelZeroGrids <<<gridDim, blockDim>>> ( velw_D );	
	}
	if ( !ptr->ptrFront )
	{
		kernelZeroGrids <<<gridDim, blockDim>>> ( dens_F );
		kernelZeroGrids <<<gridDim, blockDim>>> ( velu_F );
		kernelZeroGrids <<<gridDim, blockDim>>> ( velv_F );
		kernelZeroGrids <<<gridDim, blockDim>>> ( velw_F );
	}
	if ( !ptr->ptrBack )
	{
		kernelZeroGrids <<<gridDim, blockDim>>> ( dens_B );
		kernelZeroGrids <<<gridDim, blockDim>>> ( velu_B );
		kernelZeroGrids <<<gridDim, blockDim>>> ( velv_B );
		kernelZeroGrids <<<gridDim, blockDim>>> ( velw_B );
	}

#if TESTING_MODE_SWITCH
	if ( ix eqt nix )
#endif
	/* sum the density of each node */
	kernelSumBufsDens <<<gridDim, blockDim>>>
		( dev_tpbufs, dens_L, dens_R, dens_U, dens_D, dens_F, dens_B, dens_C );

	/* retrieve temporary buffer back */
	if ( hipMemcpy(host_tpbufs, dev_tpbufs, 
		sizeof(double) * TPBUFFER_X, hipMemcpyDeviceToHost ) not_eq hipSuccess )
	{
		helper.GetCUDALastError( "hipMemcpy failed", __FILE__, __LINE__ );
		FreeResource();
		exit( 1 );
	}

#if TESTING_MODE_SWITCH
	system( "cls" );
	printf( "CENTER: %f\n", host_tpbufs[TEMP_BUF_CENTER] );
	printf( "LEFT:   %f\n", host_tpbufs[TEMP_BUF_LEFT] );
	printf( "RIGHT:  %f\n", host_tpbufs[TEMP_BUF_RIGHT] );
	printf( "UP:     %f\n", host_tpbufs[TEMP_BUF_UP] );
	printf( "DOWN:   %f\n", host_tpbufs[TEMP_BUF_DOWN] );
	printf( "FRONT:  %f\n", host_tpbufs[TEMP_BUF_FRONT] );
	printf( "BACK:   %f\n", host_tpbufs[TEMP_BUF_BACK] );
#endif

#if 0
	/* dead or live */
	if ( ptr->ptrLeft not_eq nullptr )
		if ( host_tpbufs[TEMP_BUF_LEFT] > 0.f )
			ptr->ptrLeft->active = true;
		else
			ptr->ptrLeft->active = false;

	if ( ptr->ptrRight not_eq nullptr )
		if ( host_tpbufs[TEMP_BUF_RIGHT] > 0.f )
			ptr->ptrRight->active = true;
		else
			ptr->ptrRight->active = false;

	if ( ptr->ptrUp not_eq nullptr )
		if ( host_tpbufs[TEMP_BUF_UP] > 0.f )
			ptr->ptrUp->active = true;
		else
			ptr->ptrUp->active = false;

	if ( ptr->ptrDown not_eq nullptr )
		if ( host_tpbufs[TEMP_BUF_DOWN] > 0.f )
			ptr->ptrDown->active = true;
		else
			ptr->ptrDown->active = false;

	if ( ptr->ptrFront not_eq nullptr )
		if ( host_tpbufs[TEMP_BUF_FRONT] > 0.f )
			ptr->ptrFront->active = true;
		else
			ptr->ptrFront->active = false;

	if ( ptr->ptrBack not_eq nullptr )
		if ( host_tpbufs[TEMP_BUF_BACK] > 0.f )
			ptr->ptrBack->active = true;
		else
			ptr->ptrBack->active = false;
#endif
};