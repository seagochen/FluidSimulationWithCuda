#include "hip/hip_runtime.h"
/**
* <Author>        Orlando Chen
* <Email>         seagochen@gmail.com
* <First Time>    Dec 15, 2013
* <Last Time>     Feb 18, 2014
* <File Name>     FluidSimDynamic.cu
*/

#include <iostream>
#include <hip/hip_runtime.h>
#include "MacroDefinition.h"
#include "FluidSimProc.h"

using namespace sge;

#define DENSITY_FIELD     0
#define VELOCITY_FIELD_U  1
#define VELOCITY_FIELD_V  2
#define VELOCITY_FIELD_W  3

size_t node_size = GRIDS_X * GRIDS_X * GRIDS_X * sizeof(double);
size_t visual_size = VOLUME_X * VOLUME_X * VOLUME_X * sizeof(SGUCHAR);

FluidSimProc::FluidSimProc ( FLUIDSPARAM *fluid )
{
	if ( AllocateResource ( fluid ) != SG_RUNTIME_OK )
	{
		FreeResource ();
		exit (1);
	}

	/* initialize FPS */
	InitFPS( fluid );

	/* build order */
	BuildOrder();

	/* select node */
	SelectNode(0, 0, 0);

	/* clear buffer */
	ZeroBuffers();

	/* finally, print message */
	printf( "fluid simulation ready, zero the data and preparing the stage now" );
};

void FluidSimProc::InitFPS( FLUIDSPARAM *fluid )
{
	fluid->fps.dwCurrentTime = 0;
	fluid->fps.dwElapsedTime = 0;
	fluid->fps.dwFrames = 0;
	fluid->fps.dwLastUpdateTime = 0;
	fluid->fps.uFPS = 0;
};

void FluidSimProc::BuildOrder( void )
{
	printf( "structure:\n" );

	for ( int i = 0; i < NODES_X; i++ )
	{
		for ( int j = 0; j < NODES_X; j++ )
		{
			for ( int k = 0; k < NODES_X; k++ )
			{
				int index = cudaIndex3D( i, j, k, NODES_X );

				if ( index >= host_node.size() or index < 0 )
				{
					printf ( "index out of range! %s, line: %d \n", __FILE__, __LINE__ );
					exit ( 1 );
				}

				/* left */
				if ( i >= 1 )
					host_node[index]->ptrLeft = host_node[index-1];
				/* right */
				if ( i <= NODES_X - 2 )
					host_node[index]->ptrRight = host_node[index+1];
				/* down */
				if ( j >= 1 )
					host_node[index]->ptrDown = host_node[index-NODES_X];
				/* up */
				if ( j <= NODES_X - 2 )
					host_node[index]->ptrUp = host_node[index+NODES_X];
				/* back */
				if ( k >= 1 )
					host_node[index]->ptrBack = host_node[index-NODES_X*NODES_X];
				/* front */
				if ( k <= NODES_X - 2 )
					host_node[index]->ptrFront = host_node[index+NODES_X*NODES_X];

				host_node[index]->nodeIX.x = i;
				host_node[index]->nodeIX.y = j;
				host_node[index]->nodeIX.z = k;

				printf ( "no: %d | offset: %d%d%d | L: %d | R: %d | U: %d | D: %d | F: %d | B: %d \n",
					index,
					host_node[index]->nodeIX.x, 
					host_node[index]->nodeIX.y, 
					host_node[index]->nodeIX.z,
					host_node[index]->ptrLeft not_eq nullptr,
					host_node[index]->ptrRight not_eq nullptr,
					host_node[index]->ptrUp not_eq nullptr,
					host_node[index]->ptrDown not_eq nullptr,
					host_node[index]->ptrFront not_eq nullptr,
					host_node[index]->ptrBack not_eq nullptr );
			}
		}
	}

	printf( "-----------------------------------------------\n" );
};

SGRUNTIMEMSG FluidSimProc::AllocateResource ( FLUIDSPARAM *fluid )
{
	/* choose which GPU to run on, change this on a multi-GPU system. */
	if ( hipSetDevice ( 0 ) != hipSuccess )
		helper.CheckRuntimeErrors ( "cudaSetDevices", __FILE__, __LINE__ );

	/* allocate memory on host */
	for ( int i = 0; i < NODES_X * NODES_X * NODES_X; i++ )
	{
		double *ptrDens, *ptrU, *ptrV, *ptrW, *ptrObs;

		if ( helper.CreateHostBuffers( node_size, 1, &ptrDens ) not_eq SG_RUNTIME_OK )
			return SG_RUNTIME_FALSE;
		if ( helper.CreateHostBuffers( node_size, 1, &ptrU ) not_eq SG_RUNTIME_OK )
			return SG_RUNTIME_FALSE;
		if ( helper.CreateHostBuffers( node_size, 1, &ptrV ) not_eq SG_RUNTIME_OK )
			return SG_RUNTIME_FALSE;
		if ( helper.CreateHostBuffers( node_size, 1, &ptrW ) not_eq SG_RUNTIME_OK )
			return SG_RUNTIME_FALSE;
		if ( helper.CreateHostBuffers( node_size, 1, &ptrObs ) not_eq SG_RUNTIME_OK )
			return SG_RUNTIME_FALSE;

		/* simulation nodes */
		SimNode *node = (SimNode*)malloc(sizeof(SimNode));
		node->ptrFront = node->ptrBack = nullptr;
		node->ptrLeft = node->ptrRight = nullptr;
		node->ptrDown = node->ptrUp = nullptr;
		host_node.push_back( node );

		host_density.push_back( ptrDens );
		host_velocity_u.push_back( ptrU );
		host_velocity_v.push_back( ptrV );
		host_velocity_w.push_back( ptrW );
		host_obstacle.push_back( ptrObs );
	}

	/* allocate memory on GPU devices */
	for ( int i = 0; i < dev_buffers_num; i++ )
	{
		double *ptr;
		if ( helper.CreateDeviceBuffers( node_size, 1, &ptr ) not_eq SG_RUNTIME_OK )
			return SG_RUNTIME_FALSE;

		dev_buffers.push_back(ptr);
	}

	/* allocate visual buffers */
	if ( helper.CreateDeviceBuffers( visual_size, 1, &dev_visual ) not_eq SG_RUNTIME_OK )
		return SG_RUNTIME_FALSE;
	if ( helper.CreateHostBuffers( visual_size, 1, &host_visual ) not_eq SG_RUNTIME_OK )
		return SG_RUNTIME_FALSE;

	/* finally */
	return SG_RUNTIME_OK;
}  

void FluidSimProc::FreeResource ( void )
{
	/* free host resource */
	for ( int i = 0; i < NODES_X * NODES_X * NODES_X; i++ )
	{
		helper.FreeHostBuffers( 1, &host_density[i] );
		helper.FreeHostBuffers( 1, &host_velocity_u[i] );
		helper.FreeHostBuffers( 1, &host_velocity_v[i] );
		helper.FreeHostBuffers( 1, &host_velocity_w[i] );
		helper.FreeHostBuffers( 1, &host_obstacle[i] );
	}

	/* free device resource */
	for ( int i = 0; i < dev_buffers_num; i++ )
	{
		helper.FreeDeviceBuffers( 1, &dev_buffers[i] );
	}
	SAFE_FREE_PTR( host_visual );
	hipFree( dev_visual );

	/* empty the vector */
	host_density.empty();
	host_velocity_u.empty();
	host_velocity_v.empty();
	host_velocity_w.empty();
	host_obstacle.empty();
	dev_buffers.empty( );
}

__global__ void kernelZeroBuffer ( double *grid )
{
	GetIndex ();
	grid [ Index(i,j,k) ] = 0.f;
};


void FluidSimProc::ZeroBuffers ( void )
{
	cudaDeviceDim3D();

	/* zero GPU buffer first */
	for ( int i = 0; i < dev_buffers_num; i++ )
		kernelZeroBuffer <<<gridDim, blockDim>>> ( dev_buffers[i] );

	/* zero host buffer */
	for ( int i = 0; i < NODES_X * NODES_X * NODES_X; i++ )
	{
		hipMemcpy( host_density[i], dev_den, node_size, hipMemcpyDeviceToHost);
		hipMemcpy( host_velocity_u[i], dev_u, node_size, hipMemcpyDeviceToHost );
		hipMemcpy( host_velocity_v[i], dev_v, node_size, hipMemcpyDeviceToHost );
		hipMemcpy( host_velocity_w[i], dev_w, node_size, hipMemcpyDeviceToHost );
	}
};

void FluidSimProc::NodetoDevice ( void )
{
	int ix = cudaIndex3D( nPos.x, nPos.y, nPos.z, NODES_X );

	hipMemcpy( dev_u, host_velocity_u[ix], node_size, hipMemcpyHostToDevice );
	hipMemcpy( dev_v, host_velocity_v[ix], node_size, hipMemcpyHostToDevice );
	hipMemcpy( dev_w, host_velocity_w[ix], node_size, hipMemcpyHostToDevice );
	hipMemcpy( dev_den, host_density[ix], node_size, hipMemcpyHostToDevice  );
	hipMemcpy( dev_obs, host_obstacle[ix], node_size, hipMemcpyHostToDevice );
};


void FluidSimProc::DevicetoNode ( void )
{
	int ix = cudaIndex3D( nPos.x, nPos.y, nPos.z, NODES_X );

	hipMemcpy( host_velocity_u[ix], dev_u, node_size, hipMemcpyDeviceToHost );
	hipMemcpy( host_velocity_v[ix], dev_v, node_size, hipMemcpyDeviceToHost );
	hipMemcpy( host_velocity_w[ix], dev_w, node_size, hipMemcpyDeviceToHost );
	hipMemcpy( host_density[ix], dev_den, node_size, hipMemcpyDeviceToHost  );
};

inline __host__ __device__ int atomicRand( int *seed )
{
	*seed = (69069 * *seed + 1);
	return *seed;
};

inline __host__ __device__ double atomicRandom( int *seed ) 
{
	return ( atomicRand( seed ) & 0xffff ) / (double)0x10000;
};

inline __host__ __device__  double sgcrandom( int *seed )
{
	return 2.0 * ( atomicRandom( seed ) - 0.5 );
};

inline __host__ __device__ double atomicInvsqrt( double x ) 
{
	double xhalf = 0.5f*x;
	int i = *(int*)&x;
	i = 0x5f3759df - (i>>1);
	x = *(double*)&i;
	x = x*(1.5f - xhalf*x*x);
	return x;
};

inline __host__ __device__ double atomicSqrt( double x )
{
	double xhalf = 0.5f*x;
	int i = *(int*)&x;
	i = 0x5f3759df - (i>>1);
	x = *(double*)&i;
	x = x*(1.5f - xhalf*x*x);
	return 1/x;
};

inline __host__ __device__ int atomicRound( double x)
{
     return (x >= 0) ? (int)(x + 0.5) : (int)(x - 0.5);
};

inline __host__ __device__ int atomicCeil( double x )
{
	int val = atomicRound(x);
	if (x > 0)
	{
		return (val >= (int)x) ? val : (int)x;
	}
	else
	{
		return ((int)x >= val) ? x : val;
	}
};

inline __host__ __device__  int atomicFloor(double x)
{
	int val = atomicRound(x);
	if (x > 0)
	{
		return (val < (int)x) ? val : x;
	}
	else
	{
		return ((int)x < val) ? x : val;
	}
};

inline __host__ __device__ int atomicFabs(int value)
{
	return (value >= 0) ? value : -value;
};

inline __host__ __device__ double atomicFabs(double value)
{
	return (value >= 0.f) ? value : -value;
};

inline __host__ __device__  double atomicGetValue
	( double const *grid, int const x, int const y, int const z )
{
	if ( x < gst_header ) return 0.f;
	if ( y < gst_header ) return 0.f;
	if ( z < gst_header ) return 0.f;
	if ( x > gst_tailer ) return 0.f;
	if ( y > gst_tailer ) return 0.f;
	if ( z > gst_tailer ) return 0.f;

	return grid[ Index(x,y,z) ];
};

inline __host__ __device__  void atomicVertices
	( double *c000, double *c001, double *c011, double *c010,
	double *c100, double *c101, double *c111, double *c110,
	double const *grid, double const x, double const y, double const z )
{
	int i = (int)x;
	int j = (int)y;
	int k = (int)z;

	*c000 = atomicGetValue ( grid, i, j, k );
	*c001 = atomicGetValue ( grid, i, j+1, k );
	*c011 = atomicGetValue ( grid, i, j+1, k+1 );
	*c010 = atomicGetValue ( grid, i, j, k+1 );
	*c100 = atomicGetValue ( grid, i+1, j, k );
	*c101 = atomicGetValue ( grid, i+1, j+1, k );
	*c111 = atomicGetValue ( grid, i+1, j+1, k+1 );
	*c110 = atomicGetValue ( grid, i+1, j, k+1 );
}

inline __host__ __device__  double atomicTrilinear
	( double const *grid, double const x, double const y, double const z )
{
	double v000, v001, v010, v011, v100, v101, v110, v111;
	atomicVertices ( &v000, &v001, &v011, &v010,
		&v100, &v101, &v111, &v110,
		grid, x, y, z );

	double dx = x - (int)(x);
	double dy = y - (int)(y);
	double dz = z - (int)(z);

	double c00 = v000 * ( 1 - dx ) + v001 * dx;
	double c10 = v010 * ( 1 - dx ) + v011 * dx;
	double c01 = v100 * ( 1 - dx ) + v101 * dx;
	double c11 = v110 * ( 1 - dx ) + v111 * dx;

	double c0 = c00 * ( 1 - dy ) + c10 * dy;
	double c1 = c01 * ( 1 - dy ) + c11 * dy;

	double c = c0 * ( 1 - dz ) + c1 * dz;

	return c;
};

__global__ void kernelPickData
( unsigned char *data, const double *bufs, int const offseti, int const offsetj, int const offsetk )
{
	GetIndex();

	int di = offseti + i;
	int dj = offsetj + j;
	int dk = offsetk + k;

	/* zero data first */
	data[ cudaIndex3D(di, dj, dk, VOLUME_X) ] = 0;

	/* retrieve data from grid */
	double value = bufs[ Index(i, j, k) ];

	/* append data to volume data */
	int temp = atomicRound( value );
	if ( temp > 0 and temp < 250 )
		data [ cudaIndex3D(di, dj, dk, VOLUME_X) ] = (unsigned char) temp;
};

/* �ɼ��������ݣ���ת��Ϊvolumetric data */
__host__ void hostPickData( SGUCHAR *data, const double *bufs, SGINT3 *nodeIX )
{
	cudaDeviceDim3D();

	nodeIX->x *= GRIDS_X;
	nodeIX->y *= GRIDS_X;
	nodeIX->z *= GRIDS_X;

	kernelPickData cudaDevice(gridDim, blockDim)
		( data, bufs, nodeIX->x, nodeIX->y, nodeIX->z );
};

__global__ void kernelCopyBuffer
	( double *grid_out, double const *grid_in )
{
	GetIndex ();

	grid_out [ Index(i,j,k) ] = grid_in [ Index(i, j, k) ];
};

__global__ void kernelSwapBuffer
	( double *grid1, double *grid2 )
{
	GetIndex ();

	double temp = grid1 [ Index(i,j,k) ];
	grid1 [ Index(i,j,k) ] = grid2 [ Index(i,j,k) ];
	grid2 [ Index(i,j,k) ] = temp;
};

__host__ void hostSwapBuffer
	( double *grid1, double *grid2 )
{
	cudaDeviceDim3D();
	kernelSwapBuffer cudaDevice(gridDim, blockDim) (grid1, grid2);
};


__global__ 	void kernelAddSource( double *grid, int const number )
{
	GetIndex();
	BeginSimArea();

	const int half = GRIDS_X / 2;

	switch ( number )
	{
	case 0: // density
		if ( j < 3 ) 
			if ( i >= half-2 and i <= half+2 ) if ( k >= half-2 and k <= half+2 )
				grid [ Index(i,j,k) ] = 100.f;
	case 1: // velocity v
		if ( j < 3 ) 
			if ( i >= half-2 and i <= half+2 ) if ( k >= half-2 and k <= half+2 )
				grid [ Index(i,j,k) ] = 100.f;

	default: // add external force if need
		break;
	}

	EndSimArea();
};

__host__ void hostAddSource( double *dens, double *vel_u, double *vel_v, double *vel_w  )
{
	cudaDeviceDim3D();

	if ( dens != NULL )
		kernelAddSource cudaDevice(gridDim, blockDim) ( dens, 0 );
	if ( vel_v != NULL )
		kernelAddSource cudaDevice(gridDim, blockDim) ( vel_v, 1 );
};

__global__ void kernelBoundary ( double *grid, int const cd )
{
	GetIndex();
	BeginSimArea();
	// ...
	EndSimArea();
};

__global__ void kernelJacobi
( double *grid_out, double const *grid_in, int const cd, double const diffusion, double const divisor )
{
	GetIndex();
	BeginSimArea();

	double div = 0.f;
	if ( divisor <= 0.f ) div = 1.f;
	else div = divisor;

	grid_out [ Index(i,j,k) ] = 
		( grid_in [ Index(i,j,k) ] + diffusion * 
			(
				grid_out [ Index(i-1, j, k) ] + grid_out [ Index(i+1, j, k) ] +
				grid_out [ Index(i, j-1, k) ] + grid_out [ Index(i, j+1, k) ] +
				grid_out [ Index(i, j, k-1) ] + grid_out [ Index(i, j, k+1) ]
			) 
		) / div;

	EndSimArea();
}

__host__ void hostJacobi
( double *grid_out, double const *grid_in, int const cd, double const diffusion, double const divisor )
{
	cudaDeviceDim3D();
	for ( int k=0; k<20; k++)
	{
		kernelJacobi cudaDevice(gridDim, blockDim) (grid_out, grid_in, cd, diffusion, divisor);
		kernelBoundary cudaDevice(gridDim, blockDim) (grid_out, cd);
	}
};

__global__ void kernelGridAdvection
( double *grid_out, double const *grid_in, double const *u_in, double const *v_in, double const *w_in )
{
	GetIndex();
	BeginSimArea();

	double u = i - u_in [ Index(i,j,k) ] * DELTATIME;
	double v = j - v_in [ Index(i,j,k) ] * DELTATIME;
	double w = k - w_in [ Index(i,j,k) ] * DELTATIME;
	grid_out [ Index(i,j,k) ] = atomicTrilinear ( grid_in, u, v, w );

	EndSimArea();
};

__host__ void hostAdvection
	( double *grid_out, double const *grid_in, int const cd, 
	double const *u_in, double const *v_in, double const *w_in )
{
	cudaDeviceDim3D();
	kernelGridAdvection cudaDevice(gridDim, blockDim) ( grid_out, grid_in, u_in, v_in, w_in );
	kernelBoundary cudaDevice(gridDim, blockDim) ( grid_out, cd );

};
#pragma endregion


__host__ void hostDiffusion
	( double *grid_out, double const *grid_in, int const cd, double const diffusion )
{
//	double rate = diffusion * GRIDS_X * GRIDS_X * GRIDS_X;
	double rate = diffusion;
	hostJacobi ( grid_out, grid_in, cd, rate, 1+6*rate );
};


__global__ void kernelGradient
	( double *div, double *p, double const *vel_u, double const *vel_v, double const *vel_w )
{
	GetIndex();
	BeginSimArea();
	
	const double h = 1.f / GRIDS_X;

	// previous instantaneous magnitude of velocity gradient 
	//		= (sum of velocity gradients per axis)/2N:
	div [ Index(i,j,k) ] = -0.5f * h * (
			vel_u [ Index(i+1, j, k) ] - vel_u [ Index(i-1, j, k) ] + // gradient of u
			vel_v [ Index(i, j+1, k) ] - vel_v [ Index(i, j-1, k) ] + // gradient of v
			vel_w [ Index(i, j, k+1) ] - vel_w [ Index(i, j, k-1) ]   // gradient of w
		);
	// zero out the present velocity gradient
	p [ Index(i,j,k) ] = 0.f;
	
	EndSimArea();
};

__global__ void kernelSubtract
	( double *vel_u, double *vel_v, double *vel_w, double const *p )
{
	GetIndex();
	BeginSimArea();

	// gradient calculated by neighbors

	vel_u [ Index(i, j, k) ] -= 0.5f * GRIDS_X * ( p [ Index(i+1, j, k) ] - p [ Index(i-1, j, k) ] );
	vel_v [ Index(i, j, k) ] -= 0.5f * GRIDS_X * ( p [ Index(i, j+1, k) ] - p [ Index(i, j-1, k) ] );
	vel_w [ Index(i, j, k) ] -= 0.5f * GRIDS_X * ( p [ Index(i, j, k+1) ] - p [ Index(i, j, k-1) ] );

	EndSimArea();
};

__host__ void hostProject
	( double *vel_u, double *vel_v, double *vel_w, double *div, double *p )
{
	cudaDeviceDim3D();

	// the velocity gradient
	kernelGradient cudaDevice(gridDim, blockDim) (div, p, vel_u, vel_v, vel_w);
	kernelBoundary cudaDevice(gridDim, blockDim) (div, DENSITY_FIELD);
	kernelBoundary cudaDevice(gridDim, blockDim) (p, DENSITY_FIELD);

	// reuse the Gauss-Seidel relaxation solver to safely diffuse the velocity gradients from p to div
	hostJacobi(p, div, DENSITY_FIELD, 1.f, 6.f);

	// now subtract this gradient from our current velocity field
	kernelSubtract cudaDevice(gridDim, blockDim) (vel_u, vel_v, vel_w, p);
	kernelBoundary cudaDevice(gridDim, blockDim) (vel_u, VELOCITY_FIELD_U);
	kernelBoundary cudaDevice(gridDim, blockDim) (vel_v, VELOCITY_FIELD_V);
	kernelBoundary cudaDevice(gridDim, blockDim) (vel_w, VELOCITY_FIELD_W);
};

void FluidSimProc::VelocitySolver( void )
{
	hostAddSource( NULL, NULL, dev_v, NULL );

	// diffuse the velocity field (per axis):
	hostDiffusion( dev_u0, dev_u, VELOCITY_FIELD_U, VISOCITY );
	hostDiffusion( dev_v0, dev_v, VELOCITY_FIELD_V, VISOCITY );
	hostDiffusion( dev_w0, dev_w, VELOCITY_FIELD_W, VISOCITY );
	hostSwapBuffer( dev_u0, dev_u );
	hostSwapBuffer( dev_v0, dev_v );
	hostSwapBuffer( dev_w0, dev_w );

	// stabilize it: (vx0, vy0 are whatever, being used as temporaries to store gradient field)
	hostProject( dev_u, dev_v, dev_w, dev_div, dev_p );
	
	// advect the velocity field (per axis):
	hostAdvection( dev_u0, dev_u, VELOCITY_FIELD_U, dev_u, dev_v, dev_w );
	hostAdvection( dev_v0, dev_v, VELOCITY_FIELD_V, dev_u, dev_v, dev_w );
	hostAdvection( dev_w0, dev_w, VELOCITY_FIELD_W, dev_u, dev_v, dev_w );
	hostSwapBuffer( dev_u0, dev_u );
	hostSwapBuffer( dev_v0, dev_v );
	hostSwapBuffer( dev_w0, dev_w );
	
	// stabilize it: (vx0, vy0 are whatever, being used as temporaries to store gradient field)
	hostProject( dev_u, dev_v, dev_w, dev_div, dev_p );
};

void FluidSimProc::DensitySolver( void )
{
	hostAddSource( dev_den, NULL, NULL, NULL );
	hostDiffusion( dev_den0, dev_den, DENSITY_FIELD, DIFFUSION );
	hostSwapBuffer( dev_den0, dev_den );
	hostAdvection ( dev_den, dev_den0, DENSITY_FIELD, dev_u, dev_v, dev_w );
};

void FluidSimProc::FluidSimSolver( FLUIDSPARAM *fluid )
{
	if ( !fluid->run ) return;

	for ( int i = 0; i < NODES_X; i++ )
	{
		for ( int j = 0; j < NODES_X; j++ )
		{
			for ( int k = 0; k < NODES_X; k++ )
			{
				/* select node */
				SelectNode( i, j, k );
				
				/* for fluid simulation, copy the data to device */
				NodetoDevice();
				
				/* Fluid process */
				VelocitySolver();
				DensitySolver();
				
				/* Synchronize the device */
				if ( hipDeviceSynchronize() not_eq hipSuccess ) 
				{
					helper.CheckRuntimeErrors("hipDeviceSynchronize failed", __FILE__, __LINE__);
					FreeResource();
					exit (1);
				}
				
				// After simulation process, retrieve data back to host, in order to 
				// avoid data flipping
				DevicetoNode();

				/* pick density */
				DensitytoVolumetric();
			}
		}
	}

	/* finally, generate volumetric image */
	GetVolumetric( fluid );
};

void FluidSimProc::DensitytoVolumetric( void )
{
	hostPickData( dev_visual, dev_den, &nPos );
}

void FluidSimProc::GetVolumetric( FLUIDSPARAM *fluid )
{
	hipMemcpy( host_visual, dev_visual, visual_size, hipMemcpyDeviceToHost );
	fluid->volume.ptrData = host_visual;
};

void FluidSimProc::SelectNode( int i, int j, int k )
{
	if ( i >= 0 and i < NODES_X and j >= 0 and j < NODES_X and k >= 0 and k < NODES_X )
	{
		nPos.x = i;
		nPos.y = j;
		nPos.z = k;
	}
};

bool FluidSimProc::ActiveNode( int i, int j, int k )
{
	int ix;
	if ( i >= 0 and i < NODES_X and j >= 0 and j < NODES_X and k >= 0 and k < NODES_X )
	{
		ix = cudaIndex3D( i, j, k, NODES_X );
		host_node[ix]->active = true;
	}

	return host_node[ix]->active == true;
};

bool FluidSimProc::DeactiveNode( int i, int j, int k )
{
	int ix;
	if ( i >= 0 and i < NODES_X and j >= 0 and j < NODES_X and k >= 0 and k < NODES_X )
	{
		ix = cudaIndex3D( i, j, k, NODES_X );
		host_node[ix]->active = false;
	}

	return host_node[ix]->active == false;
};