#include "hip/hip_runtime.h"
/**
* <Author>      Orlando Chen
* <First>       Dec 12, 2013
* <Last>		Jan 26, 2013
* <File>        BasicFluidKernels.cu
*/

#include <hip/hip_runtime_api.h>
#include <>
#include <SGE\config\ISO646.h>
#include "Parameters.h"
#include "CUDADef.h"
#include "DataStructures.h"
#include "FunctionHelper.h"
#include "BasicFluidKernels.h"

using namespace sge;

CUDAHelper m_cudahelper;

__host__
void hostPreBasicFluidKernel(double **dStores, int **nStores )
{
	if ( hipMalloc( (void**)dStores, sizeof(double) * TPBUFFER_X ) != hipSuccess )
	{
		m_cudahelper.CheckErrors( "malloc temporary stores failed!", __FILE__, __LINE__ );
		goto Error;
	}
	if ( hipMalloc( (void**)nStores, sizeof(int) * TPBUFFER_X ) != hipSuccess )
	{
		m_cudahelper.CheckErrors( "malloc temporary stores failed!", __FILE__, __LINE__ );
		goto Error;
	}

	goto Success;

Error:
	hipFree( *dStores );
	hipFree( *nStores );
	exit( 1 );

Success:
	;
};

__global__
void kernelAddSource( SGCUDAGRID *grid )
{
	GetIndex();
	int ix = Index(i,j,k);

	if ( grid[ix].obstacle eqt SG_BD_SOURCE )
	{
		grid[ix].v   = SOURCE * DELTATIME;
		grid[ix].den = SOURCE;
	}
};

__host__
void hostAddSource( SGCUDAGRID *grid )
{
	dim3 gridDim, blockDim;
	m_cudahelper.DeviceDim3D( &gridDim, &blockDim );

	kernelAddSource<<<gridDim, blockDim>>>( grid );
};

__device__
void atomicJacobi( double *dStores )
{
	jacobi_out = ( jacobi_in + jacobi_dif *
		( jacobi_x0 + jacobi_x1 + 
		jacobi_y0 + jacobi_y1 +
		jacobi_z0 + jacobi_z1 ) ) / jacobi_div;
};

__global__
void kernelJacobi( SGCUDAGRID *grid, double *dStores, 
	int const type, double const diffusion, double const divisor )
{
	GetIndex();
	BeginSimArea();

	jacobi_dif = diffusion;
	jacobi_div = 0.f;
	if ( divisor <= 0.f ) jacobi_div = 1.f;
	else jacobi_div = divisor;
	
	if ( type eqt SG_SOLVE_DENSITY )
	{
		jacobi_in = grid[ Index(i,j,k) ].den;
		jacobi_x0 = grid[ Index(i-1,j,k) ].den0;
		jacobi_x1 = grid[ Index(i+1,j,k) ].den0;
		jacobi_y0 = grid[ Index(i,j-1,k) ].den0;
		jacobi_y1 = grid[ Index(i,j+1,k) ].den0;
		jacobi_z0 = grid[ Index(i,j,k-1) ].den0;
		jacobi_z1 = grid[ Index(i,j,k+1) ].den0;
		atomicJacobi( dStores );
		grid[ Index(i,j,k) ].den0 = jacobi_out;
	}
	elif ( type eqt SG_SOLVE_VELOCITY )
	{
		/* velocity u */
		jacobi_in = grid[ Index(i,j,k) ].u;
		jacobi_x0 = grid[ Index(i-1,j,k) ].u0;
		jacobi_x1 = grid[ Index(i+1,j,k) ].u0;
		jacobi_y0 = grid[ Index(i,j-1,k) ].u0;
		jacobi_y1 = grid[ Index(i,j+1,k) ].u0;
		jacobi_z0 = grid[ Index(i,j,k-1) ].u0;
		jacobi_z1 = grid[ Index(i,j,k+1) ].u0;
		atomicJacobi( dStores );
		grid[ Index(i,j,k) ].u0 = jacobi_out;

		/* velocity v */
		jacobi_in = grid[ Index(i,j,k) ].v;
		jacobi_x0 = grid[ Index(i-1,j,k) ].v0;
		jacobi_x1 = grid[ Index(i+1,j,k) ].v0;
		jacobi_y0 = grid[ Index(i,j-1,k) ].v0;
		jacobi_y1 = grid[ Index(i,j+1,k) ].v0;
		jacobi_z0 = grid[ Index(i,j,k-1) ].v0;
		jacobi_z1 = grid[ Index(i,j,k+1) ].v0;
		atomicJacobi( dStores );
		grid[ Index(i,j,k) ].v0 = jacobi_out;

		/* velocity w */
		jacobi_in = grid[ Index(i,j,k) ].w;
		jacobi_x0 = grid[ Index(i-1,j,k) ].w0;
		jacobi_x1 = grid[ Index(i+1,j,k) ].w0;
		jacobi_y0 = grid[ Index(i,j-1,k) ].w0;
		jacobi_y1 = grid[ Index(i,j+1,k) ].w0;
		jacobi_z0 = grid[ Index(i,j,k-1) ].w0;
		jacobi_z1 = grid[ Index(i,j,k+1) ].w0;
		atomicJacobi( dStores );
		grid[ Index(i,j,k) ].w0 = jacobi_out;
	}

	EndSimArea();
}

__host__
void hostJacobi ( SGCUDAGRID *grid, double *dStores, 
	int const type, double const diffusion, double const divisor )
{
	dim3 gridDim, blockDim;
	m_cudahelper.DeviceDim3D( &gridDim, &blockDim );

	for ( int k = 0; k < 20; k++)
		kernelJacobi<<<gridDim, blockDim>>>( grid, dStores, type, diffusion, divisor );
};