#include "hip/hip_runtime.h"
/**
* <Author>      Orlando Chen
* <First>       Dec 12, 2013
* <Last>		Jan 29, 2013
* <File>        BasicFluidKernels.cu
*/

#include <hip/hip_runtime_api.h>
#include <>
#include <SGE\config\ISO646.h>
#include "Parameters.h"
#include "CUDADef.h"
#include "DataStructures.h"
#include "FunctionHelper.h"
#include "BasicFluidKernels.h"

using namespace sge;

CUDAHelper m_cudahelper;

__host__
void hostPreBasicFluidKernel( double **dStores, int **nStores, int nPtrs, ... )
{
	if ( hipMalloc( (void**)dStores, sizeof(double) * TPBUFFER_X ) != hipSuccess )
	{
		m_cudahelper.CheckErrors( "malloc temporary stores failed!", __FILE__, __LINE__ );
		goto Error;
	}
	if ( hipMalloc( (void**)nStores, sizeof(int) * TPBUFFER_X ) != hipSuccess )
	{
		m_cudahelper.CheckErrors( "malloc temporary stores failed!", __FILE__, __LINE__ );
		goto Error;
	}

	va_list ap; double **ptr;
	va_start( ap, nPtrs );
	for ( int i = 0; i < nPtrs; i++ )
	{
		ptr = va_arg( ap, double** );
		if ( hipMalloc( (void**)ptr, sizeof(double) * CUBESIZE_X ) != hipSuccess )
		{
			m_cudahelper.CheckErrors( "malloc temporary stores failed!", __FILE__, __LINE__ );
			goto Error;
		}
	}
	va_end( ap );

	goto Success;

Error:
	hipFree( *dStores );
	hipFree( *nStores );
	exit( 1 );

Success:
	;
};

__global__ 
void kernelCopyBuffer( double *buf, SGCUDAGRID *grids, const SGGRIDTYPE type )
{
	GetIndex();

	switch (type)
	{
	case SG_DENS_GRID0:
		buf[ Index(i,j,k) ] = grids[ Index(i,j,k) ].den0;
		break;
	case SG_VELU_GRID0:
		buf[ Index(i,j,k) ] = grids[ Index(i,j,k) ].u0;
		break;
	case SG_VELV_GRID0:
		break;
	case SG_VELW_GRID0:
		break;
	case SG_DENS_GRID:
		break;
	case SG_VELU_GRID:
		break;
	case SG_VELV_GRID:
		break;
	case SG_VELW_GRID:
		break;
	case SG_DIV_GRID:
		break;
	case SG_PRES_GRID:
		break;
	default:
		break;
	}
};

__global__
void kernelAddSource( SGCUDAGRID *grid )
{
	GetIndex();
	int ix = Index(i,j,k);

	if ( grid[ix].obstacle eqt SG_BD_SOURCE )
	{
		grid[ix].v   = SOURCE * DELTATIME;
		grid[ix].den = SOURCE;
	}
};

__host__
void hostAddSource( SGCUDAGRID *grid )
{
	dim3 gridDim, blockDim;
	m_cudahelper.DeviceDim3D( &gridDim, &blockDim );

	kernelAddSource<<<gridDim, blockDim>>>( grid );
};

#include "InterpKernels.h"

__device__
void atomicJacobi( double *dStores )
{
	jacobi_out = ( jacobi_in + jacobi_dif *
		( jacobi_x0 + jacobi_x1 + 
		jacobi_y0 + jacobi_y1 +
		jacobi_z0 + jacobi_z1 ) ) / jacobi_div;
};

__global__
void kernelJacobi( SGDEVBUFF *buff, double *dStores,
	const SGJACOBITYPE type, const double diffusion, const double divisor )
{
	GetIndex();

	jacobi_dif = diffusion;
	jacobi_div = 0.f;
	if ( divisor <= 0.f ) jacobi_div = 1.f;
	else jacobi_div = divisor;
	
	if ( type eqt SG_SOLVE_DENSITY )
	{
		jacobi_in = atomicGetDeviceBuffer( buff, SG_DENS_GRID, i, j, k );
		jacobi_x0 = atomicGetDeviceBuffer( buff, SG_DENS_GRID, i-1,j,k );
		jacobi_x1 = atomicGetDeviceBuffer( buff, SG_DENS_GRID, i+1,j,k );
		jacobi_y0 = atomicGetDeviceBuffer( buff, SG_DENS_GRID, i,j-1,k );
		jacobi_y1 = atomicGetDeviceBuffer( buff, SG_DENS_GRID, i,j+1,k );
		jacobi_z0 = atomicGetDeviceBuffer( buff, SG_DENS_GRID, i,j,k-1 );
		jacobi_z1 = atomicGetDeviceBuffer( buff, SG_DENS_GRID, i,j,k+1 );
		atomicJacobi( dStores );
		atomicSetDeviceBuffer( buff, SG_DENS_GRID
		grid[ Index(i,j,k) ].den0 = jacobi_out;
	}
	elif ( type eqt SG_SOLVE_VELOCITY )
	{
		/* velocity u */
		jacobi_in = grid[ Index(i,j,k) ].u;
		jacobi_x0 = grid[ Index(i-1,j,k) ].u0;
		jacobi_x1 = grid[ Index(i+1,j,k) ].u0;
		jacobi_y0 = grid[ Index(i,j-1,k) ].u0;
		jacobi_y1 = grid[ Index(i,j+1,k) ].u0;
		jacobi_z0 = grid[ Index(i,j,k-1) ].u0;
		jacobi_z1 = grid[ Index(i,j,k+1) ].u0;
		atomicJacobi( dStores );
		grid[ Index(i,j,k) ].u0 = jacobi_out;

		/* velocity v */
		jacobi_in = grid[ Index(i,j,k) ].v;
		jacobi_x0 = grid[ Index(i-1,j,k) ].v0;
		jacobi_x1 = grid[ Index(i+1,j,k) ].v0;
		jacobi_y0 = grid[ Index(i,j-1,k) ].v0;
		jacobi_y1 = grid[ Index(i,j+1,k) ].v0;
		jacobi_z0 = grid[ Index(i,j,k-1) ].v0;
		jacobi_z1 = grid[ Index(i,j,k+1) ].v0;
		atomicJacobi( dStores );
		grid[ Index(i,j,k) ].v0 = jacobi_out;

		/* velocity w */
		jacobi_in = grid[ Index(i,j,k) ].w;
		jacobi_x0 = grid[ Index(i-1,j,k) ].w0;
		jacobi_x1 = grid[ Index(i+1,j,k) ].w0;
		jacobi_y0 = grid[ Index(i,j-1,k) ].w0;
		jacobi_y1 = grid[ Index(i,j+1,k) ].w0;
		jacobi_z0 = grid[ Index(i,j,k-1) ].w0;
		jacobi_z1 = grid[ Index(i,j,k+1) ].w0;
		atomicJacobi( dStores );
		grid[ Index(i,j,k) ].w0 = jacobi_out;
	}

};

__host__
void hostJacobi ( SGCUDAGRID *grid, double *dStores, 
	int const type, double const diffusion, double const divisor )
{
	dim3 gridDim, blockDim;
	m_cudahelper.DeviceDim3D( &gridDim, &blockDim );

	for ( int k = 0; k < 20; k++)
		kernelJacobi<<<gridDim, blockDim>>>( grid, dStores, type, diffusion, divisor );
};

