#include "hip/hip_runtime.h"
/**
* <Author>        Orlando Chen
* <Email>         seagochen@gmail.com
* <First Time>    Dec 15, 2013
* <Last Time>     Feb 20, 2014
* <File Name>     FluidSimulation.cu
*/

#include <iostream>
#include <hip/hip_runtime.h>
#include "MacroDefinition.h"
#include "FluidSimProc.h"

using namespace sge;

__global__ void kernelZeroBuffer( double *grid )
{
	GetIndex ();
	grid [ Index(i,j,k) ] = 0.f;
};

__global__ void kernelZeroVisual( SGUCHAR *visual )
{
	GetIndex();

	for ( int ii = 0; ii < NODES_X; ii++ )
	{
		for ( int jj = 0; jj < NODES_X; jj++ )
		{
			for ( int kk = 0; kk < NODES_X; kk++ )
			{
				int di = ii * GRIDS_X + i;
				int dj = jj * GRIDS_X + j;
				int dk = kk * GRIDS_X + k;
				
				/* zero data */
				visual[ cudaIndex3D(di, dj, dk, VOLUME_X) ] = 0;
			}
		}
	}
};

inline __host__ __device__ int atomicRand( int *seed )
{
	*seed = (69069 * *seed + 1);
	return *seed;
};

inline __host__ __device__ double atomicRandom( int *seed ) 
{
	return ( atomicRand( seed ) & 0xffff ) / (double)0x10000;
};

inline __host__ __device__  double atomicCrandom( int *seed )
{
	return 2.0 * ( atomicRandom( seed ) - 0.5 );
};

inline __host__ __device__ double atomicInvsqrt( double x ) 
{
	double xhalf = 0.5f*x;
	int i = *(int*)&x;
	i = 0x5f3759df - (i>>1);
	x = *(double*)&i;
	x = x*(1.5f - xhalf*x*x);
	return x;
};

inline __host__ __device__ double atomicSqrt( double x )
{
	double xhalf = 0.5f*x;
	int i = *(int*)&x;
	i = 0x5f3759df - (i>>1);
	x = *(double*)&i;
	x = x*(1.5f - xhalf*x*x);
	return 1/x;
};

inline __host__ __device__ int atomicRound( double x)
{
     return (x >= 0) ? (int)(x + 0.5) : (int)(x - 0.5);
};

inline __host__ __device__ int atomicCeil( double x )
{
	int val = atomicRound(x);
	if (x > 0)
	{
		return (val >= (int)x) ? val : (int)x;
	}
	else
	{
		return ((int)x >= val) ? x : val;
	}
};

inline __host__ __device__  int atomicFloor(double x)
{
	int val = atomicRound(x);
	if (x > 0)
	{
		return (val < (int)x) ? val : x;
	}
	else
	{
		return ((int)x < val) ? x : val;
	}
};

inline __host__ __device__ int atomicFabs(int value)
{
	return (value >= 0) ? value : -value;
};

inline __host__ __device__ double atomicFabs(double value)
{
	return (value >= 0.f) ? value : -value;
};

inline __host__ __device__  double atomicGetValue
	( double const *grid, int const x, int const y, int const z )
{
	if ( x < gst_header ) return 0.f;
	if ( y < gst_header ) return 0.f;
	if ( z < gst_header ) return 0.f;
	if ( x > gst_tailer ) return 0.f;
	if ( y > gst_tailer ) return 0.f;
	if ( z > gst_tailer ) return 0.f;

	return grid[ Index(x,y,z) ];
};

inline __host__ __device__  void atomicVertices
	( double *c000, double *c001, double *c011, double *c010,
	double *c100, double *c101, double *c111, double *c110,
	double const *grid, double const x, double const y, double const z )
{
	int i = (int)x;
	int j = (int)y;
	int k = (int)z;

	*c000 = atomicGetValue ( grid, i, j, k );
	*c001 = atomicGetValue ( grid, i, j+1, k );
	*c011 = atomicGetValue ( grid, i, j+1, k+1 );
	*c010 = atomicGetValue ( grid, i, j, k+1 );
	*c100 = atomicGetValue ( grid, i+1, j, k );
	*c101 = atomicGetValue ( grid, i+1, j+1, k );
	*c111 = atomicGetValue ( grid, i+1, j+1, k+1 );
	*c110 = atomicGetValue ( grid, i+1, j, k+1 );
}

inline __host__ __device__  double atomicTrilinear
	( double const *grid, double const x, double const y, double const z )
{
	double v000, v001, v010, v011, v100, v101, v110, v111;
	atomicVertices ( &v000, &v001, &v011, &v010,
		&v100, &v101, &v111, &v110,
		grid, x, y, z );

	double dx = x - (int)(x);
	double dy = y - (int)(y);
	double dz = z - (int)(z);

	double c00 = v000 * ( 1 - dx ) + v001 * dx;
	double c10 = v010 * ( 1 - dx ) + v011 * dx;
	double c01 = v100 * ( 1 - dx ) + v101 * dx;
	double c11 = v110 * ( 1 - dx ) + v111 * dx;

	double c0 = c00 * ( 1 - dy ) + c10 * dy;
	double c1 = c01 * ( 1 - dy ) + c11 * dy;

	double c = c0 * ( 1 - dz ) + c1 * dz;

	return c;
};

__global__ void kernelPickData
( unsigned char *data, const double *bufs, int const offseti, int const offsetj, int const offsetk )
{
	GetIndex();

	int di = offseti + i;
	int dj = offsetj + j;
	int dk = offsetk + k;

	/* zero data first */
	data[ cudaIndex3D(di, dj, dk, VOLUME_X) ] = 0;

	/* retrieve data from grid */
	double value = bufs[ Index(i, j, k) ];

	/* append data to volume data */
	int temp = atomicRound( value );
	if ( temp > 0 and temp < 250 )
		data [ cudaIndex3D(di, dj, dk, VOLUME_X) ] = (unsigned char) temp;
};

__host__ void hostPickData( SGUCHAR *data, const double *bufs, SGINT3 *nodeIX )
{
	cudaDeviceDim3D();

	nodeIX->x *= GRIDS_X;
	nodeIX->y *= GRIDS_X;
	nodeIX->z *= GRIDS_X;

	kernelPickData cudaDevice(gridDim, blockDim)
		( data, bufs, nodeIX->x, nodeIX->y, nodeIX->z );
};

__global__ void kernelCopyBuffer( double *grid_out, double const *grid_in )
{
	GetIndex ();

	grid_out [ Index(i,j,k) ] = grid_in [ Index(i, j, k) ];
};

__global__ void kernelSwapBuffer( double *grid1, double *grid2 )
{
	GetIndex ();

	double temp = grid1 [ Index(i,j,k) ];
	grid1 [ Index(i,j,k) ] = grid2 [ Index(i,j,k) ];
	grid2 [ Index(i,j,k) ] = temp;
};

__host__ void hostSwapBuffer( double *grid1, double *grid2 )
{
	cudaDeviceDim3D();
	kernelSwapBuffer cudaDevice(gridDim, blockDim) (grid1, grid2);
};

__device__ void atomicDensityObs( double *grids, const double *obstacle )
{
	GetIndex();
	BeginSimArea();
	/* ��ǰ������ϰ�����ܶȴ���0 */
	if ( obstacle[Index(i,j,k)] eqt MACRO_BOUNDARY_OBSTACLE and grids[Index(i,j,k)] > 0.f )
	{
		int cells  = 0;
		double val = 0; 

		if ( obstacle[Index(i-1,j,k)] eqt MACRO_BOUNDARY_BLANK ) cells++;
		if ( obstacle[Index(i+1,j,k)] eqt MACRO_BOUNDARY_BLANK ) cells++;
		if ( obstacle[Index(i,j-1,k)] eqt MACRO_BOUNDARY_BLANK ) cells++;
		if ( obstacle[Index(i,j+1,k)] eqt MACRO_BOUNDARY_BLANK ) cells++;
		if ( obstacle[Index(i,j,k-1)] eqt MACRO_BOUNDARY_BLANK ) cells++;
		if ( obstacle[Index(i,j,k+1)] eqt MACRO_BOUNDARY_BLANK ) cells++;

		if ( cells > 0 ) val = grids[Index(i,j,k)] / cells;
		else val = 0.f;

		if ( obstacle[Index(i-1,j,k)] eqt MACRO_BOUNDARY_BLANK ) grids[Index(i-1,j,k)] += val;
		if ( obstacle[Index(i+1,j,k)] eqt MACRO_BOUNDARY_BLANK ) grids[Index(i+1,j,k)] += val;
		if ( obstacle[Index(i,j-1,k)] eqt MACRO_BOUNDARY_BLANK ) grids[Index(i,j-1,k)] += val;
		if ( obstacle[Index(i,j+1,k)] eqt MACRO_BOUNDARY_BLANK ) grids[Index(i,j+1,k)] += val;
		if ( obstacle[Index(i,j,k-1)] eqt MACRO_BOUNDARY_BLANK ) grids[Index(i,j,k-1)] += val;
		if ( obstacle[Index(i,j,k+1)] eqt MACRO_BOUNDARY_BLANK ) grids[Index(i,j,k+1)] += val;

		grids[Index(i,j,k)] = 0.f;
	}
	EndSimArea();
};

__device__ void atomicVelocityObs_U( double *grids, const double *obstacle )
{
	GetIndex();
	BeginSimArea();
	if ( obstacle[Index(i,j,k)] eqt MACRO_BOUNDARY_OBSTACLE )
	{
		if ( grids[Index(i,j,k)] > 0.f )
		{
			if ( obstacle[Index(i-1,j,k)] eqt MACRO_BOUNDARY_BLANK )
				grids[Index(i-1,j,k)] = grids[Index(i-1,j,k)] -  grids[Index(i,j,k)];
		}
		else
		{
			if ( obstacle[Index(i+1,j,k)] eqt MACRO_BOUNDARY_BLANK )
				grids[Index(i+1,j,k)] = grids[Index(i+1,j,k)] -  grids[Index(i,j,k)];
		}
		grids[Index(i,j,k)] = 0.f;
	}
	EndSimArea();
};

__device__ void atomicVelocityObs_V( double *grids, const double *obstacle )
{
	GetIndex();
	BeginSimArea();
	if ( obstacle[Index(i,j,k)] eqt MACRO_BOUNDARY_OBSTACLE )
	{
		if ( grids[Index(i,j,k)] > 0.f )
		{
			if ( obstacle[Index(i,j-1,k)] eqt MACRO_BOUNDARY_BLANK )
				grids[Index(i,j-1,k)] = grids[Index(i,j-1,k)] - grids[Index(i,j,k)];
		}
		else
		{
			if ( obstacle[Index(i,j+1,k)] eqt MACRO_BOUNDARY_BLANK )
				grids[Index(i,j+1,k)] = grids[Index(i,j+1,k)] - grids[Index(i,j,k)];
		}
		grids[Index(i,j,k)] = 0.f;
	}
	EndSimArea();
};

__device__ void atomicVelocityObs_W( double *grids, const double *obstacle )
{
	GetIndex();
	BeginSimArea();
	if ( obstacle[Index(i,j,k)] eqt MACRO_BOUNDARY_OBSTACLE )
	{
		if ( grids[Index(i,j,k)] > 0.f )
		{
			if ( obstacle[Index(i,j,k-1)] eqt MACRO_BOUNDARY_BLANK )
				grids[Index(i,j,k-1)] = grids[Index(i,j,k-1)] - grids[Index(i,j,k)];
		}
		else
		{
			if ( obstacle[Index(i,j,k+1)] eqt MACRO_BOUNDARY_BLANK )
				grids[Index(i,j,k+1)] = grids[Index(i,j,k+1)] - grids[Index(i,j,k)];
		}
		grids[Index(i,j,k)] = 0.f;
	}
	EndSimArea();
};

__global__ void kernelObstacle( double *grids, const double *obstacle, const int field )
{
	switch( field )
	{
	case MACRO_DENSITY:
		atomicDensityObs( grids, obstacle );
		break;

	case MACRO_VELOCITY_U:
		atomicVelocityObs_U( grids, obstacle );
		break;

	case MACRO_VELOCITY_V:
		atomicVelocityObs_V( grids, obstacle );
		break;

	case MACRO_VELOCITY_W:
		atomicVelocityObs_W( grids, obstacle );
		break;

	default:
		break;
	}
};

__global__ void kernelJacobi( double *grid_out, double const *grid_in, double const diffusion, double const divisor )
{
	GetIndex();
	BeginSimArea();

	double div = 0.f;
	if ( divisor <= 0.f ) div = 1.f;
	else div = divisor;

	grid_out [ Index(i,j,k) ] = 
		( grid_in [ Index(i,j,k) ] + diffusion * 
			(
				grid_out [ Index(i-1, j, k) ] + grid_out [ Index(i+1, j, k) ] +
				grid_out [ Index(i, j-1, k) ] + grid_out [ Index(i, j+1, k) ] +
				grid_out [ Index(i, j, k-1) ] + grid_out [ Index(i, j, k+1) ]
			) 
		) / div;

	EndSimArea();
}

__host__ void hostJacobi
	( double *grid_out, double const *grid_in,
	double const *obstacle, int const field, double const diffusion, double const divisor )
{
	cudaDeviceDim3D();
	for ( int k=0; k<20; k++)
	{
		kernelJacobi cudaDevice(gridDim, blockDim) (grid_out, grid_in, diffusion, divisor);
	}
	kernelObstacle cudaDevice(gridDim, blockDim) ( grid_out, obstacle, field );
};

__global__ void kernelGridAdvection( double *grid_out, double const *grid_in, double const *u_in, double const *v_in, double const *w_in )
{
	GetIndex();
	BeginSimArea();

	double u = i - u_in [ Index(i,j,k) ] * DELTATIME;
	double v = j - v_in [ Index(i,j,k) ] * DELTATIME;
	double w = k - w_in [ Index(i,j,k) ] * DELTATIME;
	
	grid_out [ Index(i,j,k) ] = atomicTrilinear ( grid_in, u, v, w );

	EndSimArea();
};

__host__ void hostAdvection
	( double *grid_out, double const *grid_in,
	double const *obstacle, int const field,
	double const *u_in, double const *v_in, double const *w_in )
{
	cudaDeviceDim3D();
	kernelGridAdvection cudaDevice(gridDim, blockDim) ( grid_out, grid_in, u_in, v_in, w_in );
	kernelObstacle cudaDevice(gridDim, blockDim) ( grid_out, obstacle, field );
};

__host__ void hostDiffusion
	( double *grid_out, double const *grid_in, double const diffusion,
	double const *obstacle, int const field )
{
//	double rate = diffusion * GRIDS_X * GRIDS_X * GRIDS_X;
	double rate = diffusion;
	hostJacobi ( grid_out, grid_in, obstacle, field, rate, 1+6*rate );
};


__global__ void kernelGradient( double *div, double *p, double const *vel_u, double const *vel_v, double const *vel_w )
{
	GetIndex();
	BeginSimArea();
	
	const double h = 1.f / GRIDS_X;

	// previous instantaneous magnitude of velocity gradient 
	//		= (sum of velocity gradients per axis)/2N:
	div [ Index(i,j,k) ] = -0.5f * h * (
			vel_u [ Index(i+1, j, k) ] - vel_u [ Index(i-1, j, k) ] + // gradient of u
			vel_v [ Index(i, j+1, k) ] - vel_v [ Index(i, j-1, k) ] + // gradient of v
			vel_w [ Index(i, j, k+1) ] - vel_w [ Index(i, j, k-1) ]   // gradient of w
		);
	// zero out the present velocity gradient
	p [ Index(i,j,k) ] = 0.f;
	
	EndSimArea();
};

__global__ void kernelSubtract( double *vel_u, double *vel_v, double *vel_w, double const *p )
{
	GetIndex();
	BeginSimArea();

	// gradient calculated by neighbors

	vel_u [ Index(i, j, k) ] -= 0.5f * GRIDS_X * ( p [ Index(i+1, j, k) ] - p [ Index(i-1, j, k) ] );
	vel_v [ Index(i, j, k) ] -= 0.5f * GRIDS_X * ( p [ Index(i, j+1, k) ] - p [ Index(i, j-1, k) ] );
	vel_w [ Index(i, j, k) ] -= 0.5f * GRIDS_X * ( p [ Index(i, j, k+1) ] - p [ Index(i, j, k-1) ] );

	EndSimArea();
};

__host__ void hostProject( double *vel_u, double *vel_v, double *vel_w, double *div, double *p, double const *obs )
{
	cudaDeviceDim3D();

	// the velocity gradient
	kernelGradient cudaDevice(gridDim, blockDim) ( div, p, vel_u, vel_v, vel_w );
	kernelObstacle cudaDevice(gridDim, blockDim) ( div, obs, MACRO_SIMPLE );
	kernelObstacle cudaDevice(gridDim, blockDim) ( p, obs, MACRO_SIMPLE );

	// reuse the Gauss-Seidel relaxation solver to safely diffuse the velocity gradients from p to div
	hostJacobi(p, div, obs, MACRO_SIMPLE, 1.f, 6.f);

	// now subtract this gradient from our current velocity field
	kernelSubtract cudaDevice(gridDim, blockDim) ( vel_u, vel_v, vel_w, p );
	kernelObstacle cudaDevice(gridDim, blockDim) ( vel_u, obs, MACRO_VELOCITY_U );
	kernelObstacle cudaDevice(gridDim, blockDim) ( vel_v, obs, MACRO_VELOCITY_V );
	kernelObstacle cudaDevice(gridDim, blockDim) ( vel_w, obs, MACRO_VELOCITY_W );
};

__global__ void kernelSetBoundary( double *grids )
{
	GetIndex();
	
	const int half = GRIDS_X / 2;
	
	if ( j < 3 and i >= half-2 and i <= half+2 and k >= half-2 and k <= half+2 )
		grids[ Index(i,j,k) ] = MACRO_BOUNDARY_SOURCE;
};

__global__ void kernelAddSource
	( double *density, double *vel_u, double *vel_v, double *vel_w, double *obs )
{
	GetIndex();

	const int half = GRIDS_X / 2;

	if ( obs[ Index(i,j,k) ] eqt MACRO_BOUNDARY_SOURCE )
	{
		/* add source to grids */
		density[Index(i,j,k)] = SOURCE_DENSITY;

		/* add velocity to grids */
		if ( i < half )
			vel_u[Index(i,j,k)] = -SOURCE_VELOCITY * DELTATIME * DELTATIME;
		elif( i >= half )
			vel_u[Index(i,j,k)] =  SOURCE_VELOCITY * DELTATIME * DELTATIME;

		vel_v[Index(i,j,k)] = SOURCE_VELOCITY;

		if ( k < half )
			vel_w[Index(i,j,k)] = -SOURCE_VELOCITY * DELTATIME * DELTATIME;
		elif ( k >= half )
			vel_w[Index(i,j,k)] =  SOURCE_VELOCITY * DELTATIME * DELTATIME;
	}
};

void FluidSimProc::AddSource( void )
{
	if ( decrease_times eqt 0 )
	{
		cudaDeviceDim3D();
		kernelAddSource<<<gridDim, blockDim>>> ( dev_den, dev_u, dev_v, dev_w, dev_obs );
		increase_times++;

		if ( increase_times eqt 200 )
		{
			decrease_times = increase_times;
			increase_times = 0;
		}
	}
	else
	{
		decrease_times--;
	}
};

void FluidSimProc::InitBoundary( int i, int j, int k )
{
	cudaDeviceDim3D();

	/* zero boundary buffers */
	kernelZeroBuffer<<<gridDim, blockDim>>>( dev_obs );

	for ( int i = 0; i < host_obstacle.size(); i++ )
	{
		if ( hipMemcpy( host_obstacle[i], dev_obs,
			m_node_size, hipMemcpyDeviceToHost ) not_eq hipSuccess )
		{
			helper.CheckRuntimeErrors( "hipMemcpy failed", __FILE__, __LINE__ );
			FreeResource();
			exit( 1 );
		}
	}

	/* select middle node */
	SelectNode( i, j, k );

	const int ix = cudaIndex3D( nPos.x, nPos.y, nPos.z, NODES_X );

	/* set boundary */
	kernelSetBoundary<<<gridDim, blockDim>>>( dev_obs );
	
	if ( hipMemcpy( host_obstacle[ix], dev_obs,
		m_node_size, hipMemcpyDeviceToHost) not_eq hipSuccess )
	{
		helper.CheckRuntimeErrors( "hipMemcpy failed", __FILE__, __LINE__ );
		FreeResource();
		exit( 1 );
	}
};

void FluidSimProc::VelocitySolver( void )
{
	// diffuse the velocity field (per axis):
	hostDiffusion( dev_u0, dev_u, VISOCITY, dev_obs, MACRO_VELOCITY_U );
	hostDiffusion( dev_v0, dev_v, VISOCITY, dev_obs, MACRO_VELOCITY_V );
	hostDiffusion( dev_w0, dev_w, VISOCITY, dev_obs, MACRO_VELOCITY_W );
	hostSwapBuffer( dev_u0, dev_u );
	hostSwapBuffer( dev_v0, dev_v );
	hostSwapBuffer( dev_w0, dev_w );

	// stabilize it: (vx0, vy0 are whatever, being used as temporaries to store gradient field)
	hostProject( dev_u, dev_v, dev_w, dev_div, dev_p, dev_obs );
	
	// advect the velocity field (per axis):
	hostAdvection( dev_u0, dev_u, dev_obs, MACRO_VELOCITY_U, dev_u, dev_v, dev_w );
	hostAdvection( dev_v0, dev_v, dev_obs, MACRO_VELOCITY_V, dev_u, dev_v, dev_w );
	hostAdvection( dev_w0, dev_w, dev_obs, MACRO_VELOCITY_W, dev_u, dev_v, dev_w );
	hostSwapBuffer( dev_u0, dev_u );
	hostSwapBuffer( dev_v0, dev_v );
	hostSwapBuffer( dev_w0, dev_w );
	
	// stabilize it: (vx0, vy0 are whatever, being used as temporaries to store gradient field)
	hostProject( dev_u, dev_v, dev_w, dev_div, dev_p, dev_obs );
};

void FluidSimProc::DensitySolver( void )
{
	hostDiffusion( dev_den0, dev_den, DIFFUSION, dev_obs, MACRO_DENSITY );
	hostSwapBuffer( dev_den0, dev_den );
	hostAdvection ( dev_den, dev_den0, dev_obs, MACRO_DENSITY, dev_u, dev_v, dev_w );
};

void FluidSimProc::DensitytoVolumetric( void )
{
	hostPickData( dev_visual, dev_den, &nPos );
}

void FluidSimProc::ZeroBuffers( void )
{
	cudaDeviceDim3D();

	/* zero GPU buffer */
	for ( int i = 0; i < dev_buffers_num; i++ )
		kernelZeroBuffer <<<gridDim, blockDim>>> ( dev_buffers[i] );

	/* zero host buffer */
	for ( int i = 0; i < NODES_X * NODES_X * NODES_X; i++ )
	{
		hipMemcpy( host_density[i], dev_den, m_node_size, hipMemcpyDeviceToHost);
		hipMemcpy( host_velocity_u[i], dev_u, m_node_size, hipMemcpyDeviceToHost );
		hipMemcpy( host_velocity_v[i], dev_v, m_node_size, hipMemcpyDeviceToHost );
		hipMemcpy( host_velocity_w[i], dev_w, m_node_size, hipMemcpyDeviceToHost );
	}

	/* zero visual buffer */
	kernelZeroVisual <<< gridDim, blockDim>>> ( dev_visual );

	hipMemcpy( host_visual, dev_visual, m_volm_size, hipMemcpyDeviceToHost );
};