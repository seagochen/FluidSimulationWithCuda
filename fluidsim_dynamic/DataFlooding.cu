#include "hip/hip_runtime.h"
/**
* <Author>        Orlando Chen
* <Email>         seagochen@gmail.com
* <First Time>    Feb 21, 2013
* <Last Time>     Feb 21, 2014
* <File Name>     DataFlooding.cu
*/

#include <iostream>
#include <hip/hip_runtime.h>
#include "MacroDefinition.h"
#include "FluidSimProc.h"

using namespace sge;

#define MACRO_DENSITY     0
#define MACRO_VELOCITY_U  1
#define MACRO_VELOCITY_V  2
#define MACRO_VELOCITY_W  3
#define MACRO_SIMPLE      4

#define MACRO_BOUNDARY_BLANK      0
#define MACRO_BOUNDARY_SOURCE     1
#define MACRO_BOUNDARY_OBSTACLE 100

#define MACRO_CENTER     0
#define MACRO_LEFT       1
#define MACRO_RIGHT      2
#define MACRO_UP         3
#define MACRO_DOWN       4
#define MACRO_FRONT      5
#define MACRO_BACK       6

__device__ void atomicHandleFaceLR( double *grids, double const *center, int header, int tailer )
{
	GetIndex();
	
	/* copy entire face from center node */
	grids[Index(header,j,k)] = center[Index(tailer,j,k)];

	/* wait, we have left something which may cause un-conservation of mass */
	grids[Index(header,sim_header,k)] = center[Index(tailer,sim_header,k)] / 2.f;
	grids[Index(header,sim_tailer,k)] = center[Index(tailer,sim_tailer,k)] / 2.f;
	grids[Index(header,j,sim_header)] = center[Index(tailer,j,sim_header)] / 2.f;
	grids[Index(header,j,sim_tailer)] = center[Index(tailer,j,sim_tailer)] / 2.f;

	grids[Index(header,sim_header,sim_header)] = center[Index(tailer,sim_header,sim_header)] / 3.f;
	grids[Index(header,sim_header,sim_tailer)] = center[Index(tailer,sim_header,sim_tailer)] / 3.f;
	grids[Index(header,sim_tailer,sim_header)] = center[Index(tailer,sim_tailer,sim_header)] / 3.f;
	grids[Index(header,sim_tailer,sim_tailer)] = center[Index(tailer,sim_tailer,sim_tailer)] / 3.f;
};
 
__device__ void atomicHandleFaceFB( double *grids, double const *center, int header, int tailer )
{
	GetIndex();

	grids[Index(i,header,k)] = center[Index(i,tailer,k)];

	grids[Index(sim_header,header,k)] = center[Index(sim_header,tailer,k)] / 2.f;
	grids[Index(sim_tailer,header,k)] = center[Index(sim_tailer,tailer,k)] / 2.f;
	grids[Index(i,header,sim_header)] = center[Index(i,tailer,sim_header)] / 2.f;
	grids[Index(i,header,sim_tailer)] = center[Index(i,tailer,sim_tailer)] / 2.f;

	grids[Index(sim_header,header,sim_header)] = center[Index(sim_header,tailer,sim_header)] / 2.f;
	grids[Index(sim_tailer,header,sim_header)] = center[Index(sim_tailer,tailer,sim_header)] / 2.f;
	grids[Index(sim_header,header,sim_tailer)] = center[Index(sim_header,tailer,sim_tailer)] / 2.f;
	grids[Index(sim_tailer,header,sim_tailer)] = center[Index(sim_tailer,tailer,sim_tailer)] / 2.f;
};

__device__ void atomicHandleFaceUD( double *grids, double const *center, int header, int tailer )
{
	GetIndex();
 
	grids[Index(i,j,header)] = center[Index(i,j,tailer)];
 
	grids[Index(sim_header,j,header)] = center[Index(sim_header,j,tailer)] / 2.f;
	grids[Index(i,sim_header,header)] = center[Index(i,sim_header,tailer)] / 2.f;
	grids[Index(sim_tailer,j,header)] = center[Index(sim_tailer,j,tailer)] / 2.f;
	grids[Index(i,sim_tailer,header)] = center[Index(i,sim_tailer,tailer)] / 2.f;

	grids[Index(sim_header,sim_header,header)] = center[Index(sim_header,sim_header,tailer)] / 3.f;
	grids[Index(sim_header,sim_tailer,header)] = center[Index(sim_header,sim_tailer,tailer)] / 3.f;
	grids[Index(sim_tailer,sim_header,header)] = center[Index(sim_tailer,sim_header,tailer)] / 3.f;
	grids[Index(sim_tailer,sim_tailer,header)] = center[Index(sim_tailer,sim_tailer,tailer)] / 3.f;
};

__global__ void kernelFloodBuffersBetweenNodes( double *grids, double *center, const int ops )
{
	switch ( ops )
 	{
	case MACRO_LEFT:
		atomicHandleFaceLR( grids, center, sim_tailer, sim_header );
		break;

	case MACRO_RIGHT:
		atomicHandleFaceLR( grids, center, sim_header, sim_tailer );
		break;

	case MACRO_UP:
		atomicHandleFaceUD( grids, center, sim_header, sim_tailer );
		break;

	case MACRO_DOWN:
		atomicHandleFaceUD( grids, center, sim_tailer, sim_header );
		break;

	case MACRO_FRONT:
		atomicHandleFaceFB( grids, center, sim_tailer, sim_header );
		break;

	case MACRO_BACK:
		atomicHandleFaceFB( grids, center, sim_header, sim_tailer );
		break;

	default:
		break;
	}

	GetIndex();

	center[Index(gst_header,j,k)] = grids[Index(gst_header,j,k)] = 0.f;
	center[Index(gst_tailer,j,k)] = grids[Index(gst_tailer,j,k)] = 0.f;
	center[Index(i,gst_header,k)] = grids[Index(i,gst_header,k)] = 0.f;
	center[Index(i,gst_tailer,k)] = grids[Index(i,gst_tailer,k)] = 0.f;
	center[Index(i,j,gst_header)] = grids[Index(i,j,gst_header)] = 0.f;
	center[Index(i,j,gst_tailer)] = grids[Index(i,j,gst_tailer)] = 0.f;
};

__global__ void kernelSumDensity( double *grids, double *buffs, const int ops )
{
	GetIndex();

#define num_dens_center  buffs[ 0 ]
#define num_dens_left    buffs[ 1 ]
#define num_dens_right   buffs[ 2 ]
#define num_dens_up      buffs[ 3 ]
#define num_dens_down    buffs[ 4 ]
#define num_dens_front   buffs[ 5 ]
#define num_dens_back    buffs[ 6 ]
 	
	if ( grids[Index(i,j,k)] > 0.f )
 	{
		switch ( ops )
		{
		case MACRO_LEFT:
			num_dens_left += grids[Index(i,j,k)];
			break;
		case MACRO_RIGHT:
			num_dens_right += grids[Index(i,j,k)];
			break;
		case MACRO_UP:
			num_dens_up += grids[Index(i,j,k)];
			break;
		case MACRO_DOWN:
			num_dens_down += grids[Index(i,j,k)];
			break;
		case MACRO_FRONT:
			num_dens_front += grids[Index(i,j,k)];
			break;
		case MACRO_BACK:
			num_dens_back += grids[Index(i,j,k)];
			break;
		case MACRO_CENTER:
			num_dens_center += grids[Index(i,j,k)];
			break;

		default:
			break;
		}
 	}
#undef num_dens_center
#undef num_dens_left  
#undef num_dens_right 
#undef num_dens_up    
#undef num_dens_down  
#undef num_dens_front 
#undef num_dens_back 

};

__global__ void kernelZeroBuffers
	( double *center, double *left, double *right, double *front, double *back, double *up, double *down )
{
	GetIndex();

	int ix = Index(i,j,k);
	center[ix] = left[ix] = right[ix] = up[ix] = down[ix] = front[ix] = back[ix] = 0.f;
};

__global__ void kernelFloodBoundary( double *grids )
{
	GetIndex();

	/* faces */
	grids[Index(gst_header,j,k)] = grids[Index(sim_header,j,k)];
	grids[Index(gst_tailer,j,k)] = grids[Index(sim_tailer,j,k)];
	grids[Index(i,gst_header,k)] = grids[Index(i,sim_header,k)];
	grids[Index(i,gst_tailer,k)] = grids[Index(i,sim_tailer,k)];
	grids[Index(i,j,gst_header)] = grids[Index(i,j,sim_header)];
	grids[Index(i,j,gst_tailer)] = grids[Index(i,j,sim_tailer)];

	/* edges */
	grids[Index(i,gst_header,gst_header)] = ( grids[Index(i,sim_header,gst_header)] + grids[Index(i,gst_header,sim_header)] ) / 2.f;
	grids[Index(i,gst_tailer,gst_header)] = ( grids[Index(i,sim_tailer,gst_header)] + grids[Index(i,gst_tailer,sim_header)] ) / 2.f;
	grids[Index(i,gst_header,gst_tailer)] = ( grids[Index(i,sim_header,gst_tailer)] + grids[Index(i,gst_header,sim_tailer)] ) / 2.f;
	grids[Index(i,gst_tailer,gst_tailer)] = ( grids[Index(i,sim_tailer,gst_tailer)] + grids[Index(i,gst_tailer,sim_tailer)] ) / 2.f;

	grids[Index(gst_header,j,gst_header)] = ( grids[Index(sim_header,j,gst_header)] + grids[Index(gst_header,j,sim_header)] ) / 2.f;
	grids[Index(gst_tailer,j,gst_header)] = ( grids[Index(sim_tailer,j,gst_header)] + grids[Index(gst_tailer,j,sim_header)] ) / 2.f;
	grids[Index(gst_header,j,gst_tailer)] = ( grids[Index(sim_header,j,gst_tailer)] + grids[Index(gst_header,j,sim_tailer)] ) / 2.f;
	grids[Index(gst_tailer,j,gst_tailer)] = ( grids[Index(sim_tailer,j,gst_tailer)] + grids[Index(gst_tailer,j,sim_tailer)] ) / 2.f;

	grids[Index(gst_header,gst_header,k)] = ( grids[Index(sim_header,gst_header,k)] + grids[Index(gst_header,sim_header,k)] ) / 2.f;
	grids[Index(gst_tailer,gst_header,k)] = ( grids[Index(sim_tailer,gst_header,k)] + grids[Index(gst_tailer,sim_header,k)] ) / 2.f;
	grids[Index(gst_header,gst_tailer,k)] = ( grids[Index(sim_header,gst_tailer,k)] + grids[Index(gst_header,sim_tailer,k)] ) / 2.f;
	grids[Index(gst_tailer,gst_tailer,k)] = ( grids[Index(sim_tailer,gst_tailer,k)] + grids[Index(gst_tailer,sim_tailer,k)] ) / 2.f;

	/* vetices */
	grids[Index(gst_header,gst_header,gst_header)] = ( grids[Index(sim_header,gst_header,gst_header)] + grids[Index(gst_header,sim_header,gst_header)] + grids[Index(gst_header,gst_header,sim_header)] ) / 3.f;
	grids[Index(gst_header,gst_header,gst_tailer)] = ( grids[Index(sim_header,gst_header,gst_tailer)] + grids[Index(gst_header,sim_header,gst_tailer)] + grids[Index(gst_header,gst_header,sim_tailer)] ) / 3.f;
	grids[Index(gst_header,gst_tailer,gst_header)] = ( grids[Index(sim_header,gst_tailer,gst_header)] + grids[Index(gst_header,sim_tailer,gst_header)] + grids[Index(gst_header,gst_tailer,sim_header)] ) / 3.f;
	grids[Index(gst_header,gst_tailer,gst_tailer)] = ( grids[Index(sim_header,gst_tailer,gst_tailer)] + grids[Index(gst_header,sim_tailer,gst_tailer)] + grids[Index(gst_header,gst_tailer,sim_tailer)] ) / 3.f;
	grids[Index(gst_tailer,gst_header,gst_header)] = ( grids[Index(sim_tailer,gst_header,gst_header)] + grids[Index(gst_tailer,sim_header,gst_header)] + grids[Index(gst_tailer,gst_header,sim_header)] ) / 3.f;
	grids[Index(gst_tailer,gst_header,gst_tailer)] = ( grids[Index(sim_tailer,gst_header,gst_tailer)] + grids[Index(gst_tailer,sim_header,gst_tailer)] + grids[Index(gst_tailer,gst_header,sim_tailer)] ) / 3.f;
	grids[Index(gst_tailer,gst_tailer,gst_header)] = ( grids[Index(sim_tailer,gst_tailer,gst_header)] + grids[Index(gst_tailer,sim_tailer,gst_header)] + grids[Index(gst_tailer,gst_tailer,sim_header)] ) / 3.f;
	grids[Index(gst_tailer,gst_tailer,gst_tailer)] = ( grids[Index(sim_tailer,gst_tailer,gst_tailer)] + grids[Index(gst_tailer,sim_tailer,gst_tailer)] + grids[Index(gst_tailer,gst_tailer,sim_tailer)]) / 3.f;
};

void FluidSimProc::TracingDensity( void )
{
//	for ( int i = 0; i < NODES_X; i++ )
//	{
//		for ( int j = 0; j < NODES_X; j++ )
//		{
//			for ( int k = 0; k < NODES_X; k++ )
//			{
//				if ( SelectNode( i, j, k ) )
//				{
//					DataFlooding( host_density, true );
//					DataFlooding( host_velocity_u, false );
//					DataFlooding( host_velocity_v, false );
//					DataFlooding( host_velocity_w, false );
//				}
//			}
//		}
//	}
};

void FluidSimProc::DataFlooding( vector<double*> container, bool bDens )
{
	cudaDeviceDim3D();
	
	kernelZeroBuffers cudaDevice(gridDim, blockDim)
		( dev_center, dev_left, dev_right, dev_up, dev_down, dev_front, dev_back );
 
 	int ix = cudaIndex3D( nPos.x, nPos.y, nPos.z, NODES_X );
	hipMemcpy( dev_center, container[ix], m_node_size, hipMemcpyHostToDevice );
 	
 	if ( nPos.x - 1 >= 0 ) // left
 	{
		ix = cudaIndex3D( nPos.x-1, nPos.y, nPos.z, NODES_X );
		hipMemcpy( dev_left, container[ix], m_node_size, hipMemcpyHostToDevice );
 	}
 	if ( nPos.x + 1 < NODES_X ) // right
 	{
 		ix = cudaIndex3D( nPos.x + 1, nPos.y, nPos.z, NODES_X );
		hipMemcpy( dev_right, container[ix], m_node_size, hipMemcpyHostToDevice );
 	}
 	if ( nPos.y - 1 >= 0 ) // down
 	{
 		ix = cudaIndex3D( nPos.x, nPos.y - 1, nPos.z, NODES_X );
		hipMemcpy( dev_down, container[ix], m_node_size, hipMemcpyHostToDevice );
 	}
 	if ( nPos.y + 1 < NODES_X ) // up
 	{
 		ix = cudaIndex3D( nPos.x, nPos.y + 1, nPos.z, NODES_X );
		hipMemcpy( dev_up, container[ix], m_node_size, hipMemcpyHostToDevice );
 	}
 	if ( nPos.z - 1 >= 0 ) // back
 	{
 		ix = cudaIndex3D( nPos.x, nPos.y, nPos.z - 1, NODES_X );
		hipMemcpy( dev_back, container[ix], m_node_size, hipMemcpyHostToDevice );
 	}
 	if ( nPos.z + 1 < NODES_X ) // front
 	{
 		ix = cudaIndex3D( nPos.x, nPos.y, nPos.z + 1, NODES_X );
		hipMemcpy( dev_front, container[ix], m_node_size, hipMemcpyHostToDevice );
 	}

	kernelFloodBuffersBetweenNodes <<<gridDim,blockDim>>> ( dev_left, dev_center, MACRO_LEFT );
	kernelFloodBuffersBetweenNodes <<<gridDim,blockDim>>> ( dev_right, dev_center, MACRO_RIGHT );
	kernelFloodBuffersBetweenNodes <<<gridDim,blockDim>>> ( dev_up, dev_center, MACRO_UP );
	kernelFloodBuffersBetweenNodes <<<gridDim,blockDim>>> ( dev_down, dev_center, MACRO_DOWN );
	kernelFloodBuffersBetweenNodes <<<gridDim,blockDim>>> ( dev_front, dev_center, MACRO_FRONT );
	kernelFloodBuffersBetweenNodes <<<gridDim,blockDim>>> ( dev_back, dev_center, MACRO_BACK );

	if ( bDens )
 	{
		for ( int i = 0; i < TPBUFFER_X; i++)
		{
			host_tpbufs[i] = 0.f;
		}
		if ( hipMemcpy( dev_tpbufs, host_tpbufs, 
			sizeof(double) * TPBUFFER_X, hipMemcpyHostToDevice ) not_eq hipSuccess )
		{
			helper.CheckRuntimeErrors( "hipMemcpy failed", __FILE__, __LINE__ );
			FreeResource();
			exit( 1 );
		}

		kernelSumDensity<<<gridDim, blockDim>>> ( dev_center, dev_tpbufs, MACRO_CENTER );
		kernelSumDensity<<<gridDim, blockDim>>> ( dev_left, dev_tpbufs, MACRO_LEFT );
		kernelSumDensity<<<gridDim, blockDim>>> ( dev_right, dev_tpbufs, MACRO_RIGHT );
		kernelSumDensity<<<gridDim, blockDim>>> ( dev_up, dev_tpbufs, MACRO_UP );
		kernelSumDensity<<<gridDim, blockDim>>> ( dev_down, dev_tpbufs, MACRO_DOWN );
		kernelSumDensity<<<gridDim, blockDim>>> ( dev_front, dev_tpbufs, MACRO_FRONT );
		kernelSumDensity<<<gridDim, blockDim>>> ( dev_back, dev_tpbufs, MACRO_BACK );
		
		if ( hipMemcpy( host_tpbufs, dev_tpbufs, sizeof(double) * TPBUFFER_X, hipMemcpyDeviceToHost ) not_eq hipSuccess )
		{
			helper.CheckRuntimeErrors( "hipMemcpy failed", __FILE__, __LINE__ );
			FreeResource();
			exit( 1 );
		}
		
#define num_dens_center  host_tpbufs[ 0 ]
#define num_dens_left    host_tpbufs[ 1 ]
#define num_dens_right   host_tpbufs[ 2 ]
#define num_dens_up      host_tpbufs[ 3 ]
#define num_dens_down    host_tpbufs[ 4 ]
#define num_dens_front   host_tpbufs[ 5 ]
#define num_dens_back    host_tpbufs[ 6 ]

#if 0
		system( "cls" );
		printf( "center:  %f\n", num_dens_center );
		printf( "left:    %f\n", num_dens_left );
		printf( "right:   %f\n", num_dens_right );
		printf( "up:      %f\n", num_dens_up );
		printf( "down:    %f\n", num_dens_down );
		printf( "front:   %f\n", num_dens_front );
		printf( "back:    %f\n", num_dens_back );
#endif
 	}

	/* retrieve data back to node if density filled */
	if ( nPos.x - 1 >= 0 and num_dens_left > 0.f ) // left
 	{
		ix = cudaIndex3D( nPos.x-1, nPos.y, nPos.z, NODES_X );
		host_node[ix]->active = true;
		hipMemcpy( container[ix], dev_left, m_node_size, hipMemcpyDeviceToHost );
 	}
	if ( nPos.x + 1 < NODES_X and num_dens_right > 0.f ) // right
 	{
 		ix = cudaIndex3D( nPos.x + 1, nPos.y, nPos.z, NODES_X );
		host_node[ix]->active = true;
		hipMemcpy( container[ix], dev_right, m_node_size, hipMemcpyDeviceToHost );
 	}
	if ( nPos.y - 1 >= 0 and num_dens_down > 0.f ) // down
 	{
 		ix = cudaIndex3D( nPos.x, nPos.y - 1, nPos.z, NODES_X );
		host_node[ix]->active = true;
		hipMemcpy( container[ix], dev_down, m_node_size, hipMemcpyDeviceToHost );
 	}
	if ( nPos.y + 1 < NODES_X and num_dens_up > 0.f ) // up
 	{
 		ix = cudaIndex3D( nPos.x, nPos.y + 1, nPos.z, NODES_X );
		host_node[ix]->active = true;
		hipMemcpy( container[ix], dev_up, m_node_size, hipMemcpyDeviceToHost );
 	}
	if ( nPos.z - 1 >= 0 and num_dens_back > 0.f ) // back
 	{
 		ix = cudaIndex3D( nPos.x, nPos.y, nPos.z - 1, NODES_X );
		host_node[ix]->active = true;
		hipMemcpy( container[ix], dev_back, m_node_size, hipMemcpyDeviceToHost );
 	}
	if ( nPos.z + 1 < NODES_X and num_dens_front > 0.f ) // front
 	{
 		ix = cudaIndex3D( nPos.x, nPos.y, nPos.z + 1, NODES_X );
		host_node[ix]->active = true;
		hipMemcpy( container[ix], dev_front, m_node_size, hipMemcpyDeviceToHost );
 	}

#undef num_dens_center
#undef num_dens_left  
#undef num_dens_right 
#undef num_dens_up    
#undef num_dens_down  
#undef num_dens_front 
#undef num_dens_back
};