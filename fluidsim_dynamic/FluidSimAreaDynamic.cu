#include "hip/hip_runtime.h"
/**
* <Author>      Orlando Chen
* <First>       Nov 15, 2013
* <Last>		Jan 13, 2014
* <File>        FluidSimAreaDynamic.cpp
*/

#include <iostream>
#include <hip/hip_runtime_api.h>
#include <>
#include "FluidSimAreaDynamic.h"
#include "FunctionHelperDynamic.h"


#pragma region helper kernel functions

__global__ void kernelZeroNode ( double *u, double *v, double *w, double *dens )
{
	GetIndex();

	int ind = Index(i,j,k);
	u [ ind ] = 0.f;
	v [ ind ] = 0.f;
	w [ ind ] = 0.f;
};

#pragma endregion

sge::FluidSimProc::FluidSimProc ( fluidsim *fluid )
{
	if ( AllocateResourcePtrs ( fluid ) != SG_RUNTIME_OK )
	{
		FreeResourcePtrs ();
		exit (1);
	}

	fluid->fps.dwCurrentTime    = 0;
	fluid->fps.dwElapsedTime    = 0;
	fluid->fps.dwFrames         = 0;
	fluid->fps.dwLastUpdateTime = 0;
	fluid->fps.uFPS             = 0;

	std::cout << "fluid simulation ready, zero the data and preparing the stage now" << std::endl;
	SelectNode (0, 0, 0);
	ZeroData ();
};

sge::SGRUNTIMEMSG sge::FluidSimProc::AllocateResourcePtrs ( fluidsim *fluid )
{
	/* choose which GPU to run on, change this on a multi-GPU system. */
	if ( hipSetDevice ( 0 ) != hipSuccess )
		cudaCheckErrors ( "cudaSetDevices", __FILE__, __LINE__ );

#pragma region allocate memory on both host, device and volume

	/* allocate memory on host */
	size_t size = NODES_X * NODES_X * NODES_X;
	for ( int i = 0; i < size; i++ )
	{
		static node node;
		node.ptrDens = (double*) malloc ( SIMSIZE_X * sizeof(double) );
		node.ptrVelU = (double*) malloc ( SIMSIZE_X * sizeof(double) );
		node.ptrVelV = (double*) malloc ( SIMSIZE_X * sizeof(double) );
		node.ptrVelW = (double*) malloc ( SIMSIZE_X * sizeof(double) );
		node_list.push_back ( node );
	}

	/* allocate memory on GPU devices */
	for ( int i = 0; i < DevListNum; i++ )
	{
		/* alarm if hipMalloc failed */
		static double *ptr;
		if ( hipMalloc( (void **) &ptr, SIMSIZE_X * sizeof(double) ) != hipSuccess )
		{
			cudaCheckErrors ( "hipMalloc failed!", __FILE__, __LINE__ );
			return SG_RUNTIME_FALSE;
		}
		dev_list.push_back(ptr);
	}

	size = fluid->volume.uWidth * fluid->volume.uHeight * fluid->volume.uDepth;	
	host_visual = (uchar*) malloc ( sizeof(uchar) * size );
	if ( hipMalloc ((void**)&dev_visual, sizeof(uchar) * size ) != hipSuccess )
	{
		cudaCheckErrors ( "hipMalloc failed!", __FILE__, __LINE__ );
		return SG_RUNTIME_FALSE;
	}
	
#pragma endregion


#pragma region assign node position

	for ( int i = 0; i < NODES_X; i++ )
	{
		for ( int j = 0; j < NODES_X; j++ )
		{
			for ( int k = 0; k < NODES_X; k++ )
			{
				int index = i + j * NODES_X + k * NODES_X * NODES_X;

				if ( index >= node_list.size() or index < 0 )
				{
					printf ( "index out of range! %s, line: %d \n", __FILE__, __LINE__ );
					exit ( 1 );
				}

				/* left */
				if ( i >= 1 )
					node_list[index].ptrLeft = &node_list[index-1];
				/* right */
				if ( i <= NODES_X - 2 )
					node_list[index].ptrRight = &node_list[index+1];
				/* down */
				if ( j >= 1 )
					node_list[index].ptrDown = &node_list[index-NODES_X];
				/* up */
				if ( j <= NODES_X - 2 )
					node_list[index].ptrUp = &node_list[index+NODES_X];
				/* back */
				if ( k >= 1 )
					node_list[index].ptrBack = &node_list[index-NODES_X*NODES_X];
				/* front */
				if ( k <= NODES_X - 2 )
					node_list[index].ptrFront = &node_list[index+NODES_X*NODES_X];

				node_list[index].i = i;
				node_list[index].j = j;
				node_list[index].k = k;

				printf ( "no: %d | offset: %d%d%d "
					"| L: %d | R: %d | U: %d | D: %d | F: %d | B: %d \n",
					index,
					node_list[index].i, 
					node_list[index].j, 
					node_list[index].k,
					node_list[index].ptrLeft != NULL,
					node_list[index].ptrRight != NULL,
					node_list[index].ptrUp != NULL,
					node_list[index].ptrDown != NULL,
					node_list[index].ptrFront != NULL,
					node_list[index].ptrBack != NULL );
			}
		}
	}

#pragma endregion

	/* finally */
	return SG_RUNTIME_OK;
}  

void sge::FluidSimProc::FreeResourcePtrs ( void )
{
	size_t size = NODES_X * NODES_X * NODES_X;

	for ( int i = 0; i < size; i++ )
	{
		SAFE_FREE_PTR ( node_list[ i ].ptrDens );
		SAFE_FREE_PTR ( node_list[ i ].ptrVelU );
		SAFE_FREE_PTR ( node_list[ i ].ptrVelV );
		SAFE_FREE_PTR ( node_list[ i ].ptrVelW );
	}
	node_list.empty ( );

	for ( int i = 0; i < DevListNum; i++ )
	{
		hipFree ( dev_list [ i ] );
	}
	dev_list.empty ( );

	SAFE_FREE_PTR( host_visual );
	hipFree ( dev_visual );
}

void sge::FluidSimProc::ZeroData ( void )
{
	cudaDeviceDim3D();
	kernelZeroNode cudaDevice(gridDim, blockDim) ( dev_u, dev_v, dev_w, dev_den );

	/* zero each node one by one */
	for ( int i = 0; i < node_list.size(); i++ )
	{
		if ( hipMemcpy (node_list[ i ].ptrDens, dev_den,
			sizeof(double) * SIMSIZE_X, hipMemcpyDeviceToHost) != hipSuccess )
			goto Error;
		if ( hipMemcpy (node_list[ i ].ptrVelU, dev_u,
			sizeof(double) * SIMSIZE_X, hipMemcpyDeviceToHost) != hipSuccess )
			goto Error;
		if ( hipMemcpy (node_list[ i ].ptrVelV, dev_v,
			sizeof(double) * SIMSIZE_X, hipMemcpyDeviceToHost) != hipSuccess )
			goto Error;
		if ( hipMemcpy (node_list[ i ].ptrVelW, dev_w,
			sizeof(double) * SIMSIZE_X, hipMemcpyDeviceToHost) != hipSuccess )
			goto Error;
	}

	goto Success;

Error:
	cudaCheckErrors ( "hipMemcpy failed", __FILE__, __LINE__ );
	FreeResourcePtrs ();
	exit(1);

Success:
	;
}

void sge::FluidSimProc::CopyDataToDevice ( void )
{
	if ( hipMemcpy (dev_den, node_list[ IX ].ptrDens, 
		sizeof(double) * SIMSIZE_X, hipMemcpyHostToDevice) != hipSuccess )
		goto Error;
	if ( hipMemcpy (dev_u, node_list[ IX ].ptrVelU, 
		sizeof(double) * SIMSIZE_X, hipMemcpyHostToDevice) != hipSuccess )
		goto Error;
	if ( hipMemcpy (dev_v, node_list[ IX ].ptrVelV, 
		sizeof(double) * SIMSIZE_X, hipMemcpyHostToDevice) != hipSuccess )
		goto Error;
	if ( hipMemcpy (dev_w, node_list[ IX ].ptrVelW, 
		sizeof(double) * SIMSIZE_X, hipMemcpyHostToDevice) != hipSuccess )
		goto Error;

	goto Success;

Error:
	cudaCheckErrors ( "hipMemcpy failed", __FILE__, __LINE__ );
	FreeResourcePtrs ();
	exit(1);

Success:
	;	
};

void sge::FluidSimProc::CopyDataToHost ( void )
{
	if ( hipMemcpy (node_list[ IX ].ptrDens, dev_den,
		sizeof(double) * SIMSIZE_X, hipMemcpyDeviceToHost) != hipSuccess )
		goto Error;
	if ( hipMemcpy (node_list[ IX ].ptrVelU, dev_u,
		sizeof(double) * SIMSIZE_X, hipMemcpyDeviceToHost) != hipSuccess )
		goto Error;
	if ( hipMemcpy (node_list[ IX ].ptrVelV, dev_v,
		sizeof(double) * SIMSIZE_X, hipMemcpyDeviceToHost) != hipSuccess )
		goto Error;
	if ( hipMemcpy (node_list[ IX ].ptrVelW, dev_w,
		sizeof(double) * SIMSIZE_X, hipMemcpyDeviceToHost) != hipSuccess )
		goto Error;

	goto Success;

Error:
	cudaCheckErrors ( "hipMemcpy failed", __FILE__, __LINE__ );
	FreeResourcePtrs ();
	exit(1);

Success:
	;	
};

void sge::FluidSimProc::SelectNode ( int i, int j, int k )
{
	if ( i >= 0 and i < NODES_X ) 
	if ( i >= 0 and i < NODES_X )
	if ( i >= 0 and i < NODES_X )
	{
		offi = i;
		offj = j;
		offk = k;
		IX = offi + offj * NODES_X + offk * NODES_X * NODES_X;
	}	
};

void sge::FluidSimProc::SelectNode ( int index )
{
	size_t size = NODES_X * NODES_X * NODES_X;
	if ( index >= 0 and index < size )
		IX = index;
};