#include "hip/hip_runtime.h"
/**
* <Author>        Orlando Chen
* <Email>         seagochen@gmail.com
* <First Time>    Dec 15, 2013
* <Last Time>     Feb 23, 2014
* <File Name>     FluidSimProcKernels.cu
*/

#include <iostream>
#include <utility>
#include "FluidSimulationKernels.h"
#include "CUDATracingKernels.h"

using namespace sge;

__global__ void kernelZeroGrids( double *grid )
{
	GetIndex ();
	grid [ Index(i,j,k) ] = 0.f;
};

__global__ void kernelZeroVolumetric( SGUCHAR *visual )
{
	GetIndex();

	for ( int ii = 0; ii < NODES_X; ii++ )
	{
		for ( int jj = 0; jj < NODES_X; jj++ )
		{
			for ( int kk = 0; kk < NODES_X; kk++ )
			{
				int di = ii * GRIDS_X + i;
				int dj = jj * GRIDS_X + j;
				int dk = kk * GRIDS_X + k;
				
				/* zero data */
				visual[ cudaIndex3D(di, dj, dk, VOLUME_X) ] = 0;
			}
		}
	}
};

__global__ void kernelZeroTemporaryBuffers( double *bufs )
{
	GetIndex();

	bufs[threadIdx.x] = 0.f;
};

__global__ void kernelZeroTemporaryBuffers( int *bufs )
{
	bufs[threadIdx.x] = 0;
};

__global__ void kernelPickData
( unsigned char *data, const double *bufs, int const offseti, int const offsetj, int const offsetk )
{
	GetIndex();

	int di = offseti + i;
	int dj = offsetj + j;
	int dk = offsetk + k;

	/* zero data first */
	data[ cudaIndex3D(di, dj, dk, VOLUME_X) ] = 0;

	/* retrieve data from grid */
	double value = bufs[ Index(i, j, k) ];

	/* append data to volume data */
	int temp = atomicRound( value );
	if ( temp > 0 and temp < 250 )
		data [ cudaIndex3D(di, dj, dk, VOLUME_X) ] = (unsigned char) temp;
};


FluidSimProc::FluidSimProc ( FLUIDSPARAM *fluid )
{
	/* initialize FPS */
	InitParams( fluid );

	/* allocate resources */
	if ( !AllocateResource ( fluid ) ) { FreeResource (); exit (1); }

	/* build order */
	BuildOrder();

#if !TESTING_MODE_SWITCH
	/* select node */
	ActiveTheNode( 1, 0, 1 );

	/* clear buffer */
	ZeroBuffers();

	/* set boundary */
	InitBoundary( 1, 0, 1 );
#else
	/* select node */
	ActiveTheNode( 1, 1, 1 );

	/* clear buffer */
	ZeroBuffers();

	/* set boundary */
	InitBoundary( 1, 1, 1 );
#endif

	/* finally, print message */
	printf( "fluid simulation ready...\n" );
};

void FluidSimProc::InitParams( FLUIDSPARAM *fluid )
{
	fluid->fps.dwCurrentTime = 0;
	fluid->fps.dwElapsedTime = 0;
	fluid->fps.dwFrames = 0;
	fluid->fps.dwLastUpdateTime = 0;
	fluid->fps.uFPS = 0;

	m_node_size = GRIDS_X * GRIDS_X * GRIDS_X * sizeof(double);
	m_volm_size = VOLUME_X * VOLUME_X * VOLUME_X * sizeof(SGUCHAR);

	increase_times = decrease_times = 0;
};

void FluidSimProc::BuildOrder( void )
{
	for ( int k = 0; k < NODES_X; k++ )
	{
		for ( int j = 0; j < NODES_X; j++ )
		{
			for ( int i = 0; i < NODES_X; i++ )
			{
				/* left */
				if ( i >= 1 )
					host_node[cudaIndex3D( i, j, k, NODES_X )]->ptrLeft  = host_node[cudaIndex3D( i-1, j, k, NODES_X )];
				/* right */
				if ( i <= NODES_X - 2 )
					host_node[cudaIndex3D( i, j, k, NODES_X )]->ptrRight = host_node[cudaIndex3D( i+1, j, k, NODES_X )];
				/* down */
				if ( j >= 1 )
					host_node[cudaIndex3D( i, j, k, NODES_X )]->ptrDown  = host_node[cudaIndex3D( i, j-1, k, NODES_X )];
				/* up */
				if ( j <= NODES_X - 2 )
					host_node[cudaIndex3D( i, j, k, NODES_X )]->ptrUp    = host_node[cudaIndex3D( i, j+1, k, NODES_X )];
				/* back */
				if ( k >= 1 )
					host_node[cudaIndex3D( i, j, k, NODES_X )]->ptrBack  = host_node[cudaIndex3D( i, j, k-1, NODES_X )];
				/* front */
				if ( k <= NODES_X - 2 )
					host_node[cudaIndex3D( i, j, k, NODES_X )]->ptrFront = host_node[cudaIndex3D( i, j, k+1, NODES_X )];

				host_node[cudaIndex3D( i, j, k, NODES_X )]->nodeIX.x = i;
				host_node[cudaIndex3D( i, j, k, NODES_X )]->nodeIX.y = j;
				host_node[cudaIndex3D( i, j, k, NODES_X )]->nodeIX.z = k;
			}
		}
	}

	for ( int i = 0; i < NODES_X; i++ )
	{
		for ( int j = 0; j < NODES_X; j++ )
		{
			for ( int k = 0; k < NODES_X; k++ )
			{
				printf ( "offset: %d %d %d | L: %d | R: %d | U: %d | D: %d | F: %d | B: %d \n",
					host_node[cudaIndex3D( i, j, k, NODES_X )]->nodeIX.x, 
					host_node[cudaIndex3D( i, j, k, NODES_X )]->nodeIX.y, 
					host_node[cudaIndex3D( i, j, k, NODES_X )]->nodeIX.z,
					host_node[cudaIndex3D( i, j, k, NODES_X )]->ptrLeft not_eq nullptr,
					host_node[cudaIndex3D( i, j, k, NODES_X )]->ptrRight not_eq nullptr,
					host_node[cudaIndex3D( i, j, k, NODES_X )]->ptrUp not_eq nullptr,
					host_node[cudaIndex3D( i, j, k, NODES_X )]->ptrDown not_eq nullptr,
					host_node[cudaIndex3D( i, j, k, NODES_X )]->ptrFront not_eq nullptr,
					host_node[cudaIndex3D( i, j, k, NODES_X )]->ptrBack not_eq nullptr );
			}
		}
	}
};

bool FluidSimProc::AllocateResource ( FLUIDSPARAM *fluid )
{
	/* choose which GPU to run on, change this on a multi-GPU system. */
	if ( hipSetDevice ( 0 ) != hipSuccess )
	{
		helper.GetCUDALastError ( "cudaSetDevices", __FILE__, __LINE__ );
		return false;
	}

	/* allocate memory on host */
	for ( int i = 0; i < NODES_X * NODES_X * NODES_X; i++ )
	{
		double *ptrDens, *ptrU, *ptrV, *ptrW, *ptrObs;

		if ( helper.CreateHostBuffers( m_node_size, 1, &ptrDens ) not_eq SG_RUNTIME_OK )
			return false;
		if ( helper.CreateHostBuffers( m_node_size, 1, &ptrU ) not_eq SG_RUNTIME_OK )
			return false;
		if ( helper.CreateHostBuffers( m_node_size, 1, &ptrV ) not_eq SG_RUNTIME_OK )
			return false;
		if ( helper.CreateHostBuffers( m_node_size, 1, &ptrW ) not_eq SG_RUNTIME_OK )
			return false;
		if ( helper.CreateHostBuffers( m_node_size, 1, &ptrObs ) not_eq SG_RUNTIME_OK )
			return false;

		/* simulation nodes */
		SimNode *node = (SimNode*)malloc(sizeof(SimNode));
		node->ptrFront = node->ptrBack = nullptr;
		node->ptrLeft = node->ptrRight = nullptr;
		node->ptrDown = node->ptrUp = nullptr;
		host_node.push_back( node );

		node->active = false;

		host_density.push_back( ptrDens );
		host_velocity_u.push_back( ptrU );
		host_velocity_v.push_back( ptrV );
		host_velocity_w.push_back( ptrW );
		host_obstacle.push_back( ptrObs );
	}

	/* allocate memory on GPU devices */
	for ( int i = 0; i < dev_buffers_num; i++ )
	{
		double *ptr;
		if ( helper.CreateDeviceBuffers( m_node_size, 1, &ptr ) not_eq SG_RUNTIME_OK )
			return false;

		dev_buffers.push_back(ptr);
	}

	/* allocate visual buffers */
	if ( helper.CreateDeviceBuffers( m_volm_size, 1, &dev_visual ) not_eq SG_RUNTIME_OK )
		return false;
	if ( helper.CreateHostBuffers( m_volm_size, 1, &host_visual ) not_eq SG_RUNTIME_OK )
		return false;

	/* allocate temporary buffers */
	if ( helper.CreateDeviceBuffers( sizeof(double)*TPBUFFER_X, 1, &dev_tpbufs ) not_eq SG_RUNTIME_OK )
		return false;
	if ( helper.CreateHostBuffers(sizeof(double)*TPBUFFER_X, 1, &host_tpbufs ) not_eq SG_RUNTIME_OK )
		return false;

	/* finally */
	return true;
}  

void FluidSimProc::FreeResource ( void )
{
	/* free host resource */
	for ( int i = 0; i < NODES_X * NODES_X * NODES_X; i++ )
	{
		helper.FreeHostBuffers( 1, &host_density[i] );
		helper.FreeHostBuffers( 1, &host_velocity_u[i] );
		helper.FreeHostBuffers( 1, &host_velocity_v[i] );
		helper.FreeHostBuffers( 1, &host_velocity_w[i] );
		helper.FreeHostBuffers( 1, &host_obstacle[i] );
	}

	/* free device resource */
	for ( int i = 0; i < dev_buffers_num; i++ )
	{
		helper.FreeDeviceBuffers( 1, &dev_buffers[i] );
	}

	/* free... */
	helper.FreeHostBuffers( 2, &host_visual, &host_tpbufs );
	helper.FreeDeviceBuffers( 2, &dev_visual, &dev_tpbufs );

	/* empty the vector */
	host_density.empty();
	host_velocity_u.empty();
	host_velocity_v.empty();
	host_velocity_w.empty();
	host_obstacle.empty();
	dev_buffers.empty( );
}

bool FluidSimProc::SelectTheNode( int i, int j, int k )
{
	if ( i >= 0 and i < NODES_X and j >= 0 and j < NODES_X and k >= 0 and k < NODES_X )
	{
		nPos.x = i;
		nPos.y = j;
		nPos.z = k;
		
#if !TESTING_MODE_SWITCH
		int ix = cudaIndex3D( i, j, k, NODES_X );
		return host_node[ix]->active;
#else
		int ix    = cudaIndex3D( i, j, k, NODES_X );
		int left  = cudaIndex3D( 0, 1, 1, NODES_X );
		int right = cudaIndex3D( 2, 1, 1, NODES_X );
		int up    = cudaIndex3D( 1, 2, 1, NODES_X );
		int down  = cudaIndex3D( 1, 0, 1, NODES_X );
		int front = cudaIndex3D( 1, 1, 2, NODES_X );
		int back  = cudaIndex3D( 1, 1, 0, NODES_X );
		int center= cudaIndex3D( 1, 1, 1, NODES_X );

		return 
			ix eqt center or
			ix eqt left  or ix eqt right or
			ix eqt up 	 or ix eqt down  or
			ix eqt front or ix eqt back;
#endif
	}

	return false;
};

bool FluidSimProc::ActiveTheNode( int i, int j, int k )
{
	int ix;
	if ( i >= 0 and i < NODES_X and j >= 0 and j < NODES_X and k >= 0 and k < NODES_X )
	{
		ix = cudaIndex3D( i, j, k, NODES_X );
		host_node[ix]->active = true;
	}

	return host_node[ix]->active == true;
};

bool FluidSimProc::DeactiveTheNode( int i, int j, int k )
{
	int ix;
	if ( i >= 0 and i < NODES_X and j >= 0 and j < NODES_X and k >= 0 and k < NODES_X )
	{
		ix = cudaIndex3D( i, j, k, NODES_X );
		host_node[ix]->active = false;
	}

	return host_node[ix]->active == false;
};

void FluidSimProc::FluidSimSolver( FLUIDSPARAM *fluid )
{
	if ( !fluid->run ) return;

	for ( int i = 0; i < NODES_X; i++ )
	{
		for ( int j = 0; j < NODES_X; j++ )
		{
			for ( int k = 0; k < NODES_X; k++ )
			{
				//if ( SelectTheNode( i, j, k ) )
				{
					/* select node */
					SelectTheNode( i, j, k );

					/* for fluid simulation, copy the data to device */
					NodeToDevice();
					
					/* Fluid process */
					AddSource();
					VelocitySolver();
					DensitySolver();
					
					/* tracing */
					TracingTheFlow();

					/* retrieve data back to host */
					DeviceToNode();

					if ( hipDeviceSynchronize() not_eq hipSuccess )
					{
						printf( "hipDeviceSynchronize failed\n" );
						FreeResource();
						exit( 1 );
					}
				}
			}
		}
	}
	/* finally, generate volumetric image */
	GetVolumetric( fluid );
};

void FluidSimProc::GetVolumetric( FLUIDSPARAM *fluid )
{
	hipMemcpy( host_visual, dev_visual, m_volm_size, hipMemcpyDeviceToHost );
	fluid->volume.ptrData = host_visual;
};

void FluidSimProc::NodeToDevice ( void )
{
	/* navigate the node's position */
	int i = nPos.x;
	int j = nPos.y;
	int k = nPos.z;

	SimNode *ptr = host_node[cudaIndex3D( i, j, k, NODES_X )];

	/* upload center node to GPU device */
	hipMemcpy( dev_u, host_velocity_u[cudaIndex3D( i, j, k, NODES_X )], m_node_size, hipMemcpyHostToDevice );
	hipMemcpy( dev_v, host_velocity_v[cudaIndex3D( i, j, k, NODES_X )], m_node_size, hipMemcpyHostToDevice );
	hipMemcpy( dev_w, host_velocity_w[cudaIndex3D( i, j, k, NODES_X )], m_node_size, hipMemcpyHostToDevice );
	hipMemcpy( dev_den,  host_density[cudaIndex3D( i, j, k, NODES_X )], m_node_size, hipMemcpyHostToDevice );
	hipMemcpy( dev_obs, host_obstacle[cudaIndex3D( i, j, k, NODES_X )], m_node_size, hipMemcpyHostToDevice );

	if ( helper.GetCUDALastError( "hipMemcpy failed", __FILE__, __LINE__ ) )
	{
		FreeResource();
		exit( 1 );
	}

	/* upload neighbouring buffers to GPU device */
	if ( ptr->ptrLeft not_eq nullptr )
	{
		hipMemcpy( velu_L, host_velocity_u[cudaIndex3D( i-1, j, k, NODES_X )], m_node_size, hipMemcpyHostToDevice );
		hipMemcpy( velv_L, host_velocity_v[cudaIndex3D( i-1, j, k, NODES_X )], m_node_size, hipMemcpyHostToDevice );
		hipMemcpy( velw_L, host_velocity_w[cudaIndex3D( i-1, j, k, NODES_X )], m_node_size, hipMemcpyHostToDevice );
		hipMemcpy( dens_L,    host_density[cudaIndex3D( i-1, j, k, NODES_X )], m_node_size, hipMemcpyHostToDevice );

		if ( helper.GetCUDALastError( "hipMemcpy failed", __FILE__, __LINE__ ) )
		{
			FreeResource();
			exit( 1 );
		}
	}

	if ( ptr->ptrRight not_eq nullptr )
	{
		hipMemcpy( velu_R, host_velocity_u[cudaIndex3D( i+1, j, k, NODES_X )], m_node_size, hipMemcpyHostToDevice );
		hipMemcpy( velv_R, host_velocity_v[cudaIndex3D( i+1, j, k, NODES_X )], m_node_size, hipMemcpyHostToDevice );
		hipMemcpy( velw_R, host_velocity_w[cudaIndex3D( i+1, j, k, NODES_X )], m_node_size, hipMemcpyHostToDevice );
		hipMemcpy( dens_R,    host_density[cudaIndex3D( i+1, j, k, NODES_X )], m_node_size, hipMemcpyHostToDevice );

		if ( helper.GetCUDALastError( "hipMemcpy failed", __FILE__, __LINE__ ) )
		{
			FreeResource();
			exit( 1 );
		}
	}

	if ( ptr->ptrUp not_eq nullptr )
	{
		hipMemcpy( velu_U, host_velocity_u[cudaIndex3D( i, j+1, k, NODES_X )], m_node_size, hipMemcpyHostToDevice );
		hipMemcpy( velv_U, host_velocity_v[cudaIndex3D( i, j+1, k, NODES_X )], m_node_size, hipMemcpyHostToDevice );
		hipMemcpy( velw_U, host_velocity_w[cudaIndex3D( i, j+1, k, NODES_X )], m_node_size, hipMemcpyHostToDevice );
		hipMemcpy( dens_U,    host_density[cudaIndex3D( i, j+1, k, NODES_X )], m_node_size, hipMemcpyHostToDevice );

		if ( helper.GetCUDALastError( "hipMemcpy failed", __FILE__, __LINE__ ) )
		{
			FreeResource();
			exit( 1 );
		}
	}

	if ( ptr->ptrDown not_eq nullptr )
	{
		hipMemcpy( velu_D, host_velocity_u[cudaIndex3D( i, j-1, k, NODES_X )], m_node_size, hipMemcpyHostToDevice );
		hipMemcpy( velv_D, host_velocity_v[cudaIndex3D( i, j-1, k, NODES_X )], m_node_size, hipMemcpyHostToDevice );
		hipMemcpy( velw_D, host_velocity_w[cudaIndex3D( i, j-1, k, NODES_X )], m_node_size, hipMemcpyHostToDevice );
		hipMemcpy( dens_D,    host_density[cudaIndex3D( i, j-1, k, NODES_X )], m_node_size, hipMemcpyHostToDevice );

		if ( helper.GetCUDALastError( "hipMemcpy failed", __FILE__, __LINE__ ) )
		{
			FreeResource();
			exit( 1 );
		}
	}

	if ( ptr->ptrFront not_eq nullptr )
	{
		hipMemcpy( velu_F, host_velocity_u[cudaIndex3D( i, j, k+1, NODES_X )], m_node_size, hipMemcpyHostToDevice );
		hipMemcpy( velv_F, host_velocity_v[cudaIndex3D( i, j, k+1, NODES_X )], m_node_size, hipMemcpyHostToDevice );
		hipMemcpy( velw_F, host_velocity_w[cudaIndex3D( i, j, k+1, NODES_X )], m_node_size, hipMemcpyHostToDevice );
		hipMemcpy( dens_F,    host_density[cudaIndex3D( i, j, k+1, NODES_X )], m_node_size, hipMemcpyHostToDevice );

		if ( helper.GetCUDALastError( "hipMemcpy failed", __FILE__, __LINE__ ) )
		{
			FreeResource();
			exit( 1 );
		}
	}

	if ( ptr->ptrBack not_eq nullptr )
	{
		hipMemcpy( velu_B, host_velocity_u[cudaIndex3D( i, j, k-1, NODES_X )], m_node_size, hipMemcpyHostToDevice );
		hipMemcpy( velv_B, host_velocity_v[cudaIndex3D( i, j, k-1, NODES_X )], m_node_size, hipMemcpyHostToDevice );
		hipMemcpy( velw_B, host_velocity_w[cudaIndex3D( i, j, k-1, NODES_X )], m_node_size, hipMemcpyHostToDevice );
		hipMemcpy( dens_B,    host_density[cudaIndex3D( i, j, k-1, NODES_X )], m_node_size, hipMemcpyHostToDevice );

		if ( helper.GetCUDALastError( "hipMemcpy failed", __FILE__, __LINE__ ) )
		{
			FreeResource();
			exit( 1 );
		}
	}
};

void FluidSimProc::DeviceToNode ( void )
{
	/* navigate the node's position */
	int i = nPos.x;
	int j = nPos.y;
	int k = nPos.z;
	SimNode *ptr = host_node[cudaIndex3D( i, j, k, NODES_X )];

	/* draw data back */
	hipMemcpy( host_velocity_u[cudaIndex3D( i, j, k, NODES_X )], dev_u, m_node_size, hipMemcpyDeviceToHost );
	hipMemcpy( host_velocity_v[cudaIndex3D( i, j, k, NODES_X )], dev_v, m_node_size, hipMemcpyDeviceToHost );
	hipMemcpy( host_velocity_w[cudaIndex3D( i, j, k, NODES_X )], dev_w, m_node_size, hipMemcpyDeviceToHost );
	hipMemcpy( host_density[cudaIndex3D( i, j, k, NODES_X )],  dev_den, m_node_size, hipMemcpyDeviceToHost );

	if ( helper.GetCUDALastError( "hipMemcpy failed", __FILE__, __LINE__ ) )
	{
		FreeResource();
		exit( 1 );
	}

	/* draw neighbouring buffers back */
	if ( ptr->ptrLeft not_eq nullptr )
	{
		hipMemcpy( host_velocity_u[cudaIndex3D( i-1, j, k, NODES_X )], velu_L, m_node_size, hipMemcpyDeviceToHost );
		hipMemcpy( host_velocity_v[cudaIndex3D( i-1, j, k, NODES_X )], velv_L, m_node_size, hipMemcpyDeviceToHost );
		hipMemcpy( host_velocity_w[cudaIndex3D( i-1, j, k, NODES_X )], velw_L, m_node_size, hipMemcpyDeviceToHost );
		hipMemcpy(    host_density[cudaIndex3D( i-1, j, k, NODES_X )], dens_L, m_node_size, hipMemcpyDeviceToHost );

		if ( helper.GetCUDALastError( "hipMemcpy failed", __FILE__, __LINE__ ) )
		{
			FreeResource();
			exit( 1 );
		}
	}

	if ( ptr->ptrRight not_eq nullptr )
	{
		hipMemcpy( host_velocity_u[cudaIndex3D( i+1, j, k, NODES_X )], velu_R, m_node_size, hipMemcpyDeviceToHost );
		hipMemcpy( host_velocity_v[cudaIndex3D( i+1, j, k, NODES_X )], velv_R, m_node_size, hipMemcpyDeviceToHost );
		hipMemcpy( host_velocity_w[cudaIndex3D( i+1, j, k, NODES_X )], velw_R, m_node_size, hipMemcpyDeviceToHost );
		hipMemcpy(    host_density[cudaIndex3D( i+1, j, k, NODES_X )], dens_R, m_node_size, hipMemcpyDeviceToHost );

		if ( helper.GetCUDALastError( "hipMemcpy failed", __FILE__, __LINE__ ) )
		{
			FreeResource();
			exit( 1 );
		}
	}

	if ( ptr->ptrUp not_eq nullptr )
	{
		hipMemcpy( host_velocity_u[cudaIndex3D( i, j+1, k, NODES_X )], velu_U, m_node_size, hipMemcpyDeviceToHost );
		hipMemcpy( host_velocity_v[cudaIndex3D( i, j+1, k, NODES_X )], velv_U, m_node_size, hipMemcpyDeviceToHost );
		hipMemcpy( host_velocity_w[cudaIndex3D( i, j+1, k, NODES_X )], velw_U, m_node_size, hipMemcpyDeviceToHost );
		hipMemcpy(    host_density[cudaIndex3D( i, j+1, k, NODES_X )], dens_U, m_node_size, hipMemcpyDeviceToHost );

		if ( helper.GetCUDALastError( "hipMemcpy failed", __FILE__, __LINE__ ) )
		{
			FreeResource();
			exit( 1 );
		}
	}

	if ( ptr->ptrDown not_eq nullptr )
	{
		hipMemcpy( host_velocity_u[cudaIndex3D( i, j-1, k, NODES_X )], velu_D, m_node_size, hipMemcpyDeviceToHost );
		hipMemcpy( host_velocity_v[cudaIndex3D( i, j-1, k, NODES_X )], velv_D, m_node_size, hipMemcpyDeviceToHost );
		hipMemcpy( host_velocity_w[cudaIndex3D( i, j-1, k, NODES_X )], velw_D, m_node_size, hipMemcpyDeviceToHost );
		hipMemcpy(    host_density[cudaIndex3D( i, j-1, k, NODES_X )], dens_D, m_node_size, hipMemcpyDeviceToHost );

		if ( helper.GetCUDALastError( "hipMemcpy failed", __FILE__, __LINE__ ) )
		{
			FreeResource();
			exit( 1 );
		}
	}

	if ( ptr->ptrFront not_eq nullptr )
	{
		hipMemcpy( host_velocity_u[cudaIndex3D( i, j, k+1, NODES_X )], velu_F, m_node_size, hipMemcpyDeviceToHost );
		hipMemcpy( host_velocity_v[cudaIndex3D( i, j, k+1, NODES_X )], velv_F, m_node_size, hipMemcpyDeviceToHost );
		hipMemcpy( host_velocity_w[cudaIndex3D( i, j, k+1, NODES_X )], velw_F, m_node_size, hipMemcpyDeviceToHost );
		hipMemcpy(    host_density[cudaIndex3D( i, j, k+1, NODES_X )], dens_F, m_node_size, hipMemcpyDeviceToHost );

		if ( helper.GetCUDALastError( "hipMemcpy failed", __FILE__, __LINE__ ) )
		{
			FreeResource();
			exit( 1 );
		}
	}

	if ( ptr->ptrBack not_eq nullptr )
	{
		hipMemcpy( host_velocity_u[cudaIndex3D( i, j, k-1, NODES_X )], velu_B, m_node_size, hipMemcpyDeviceToHost );
		hipMemcpy( host_velocity_v[cudaIndex3D( i, j, k-1, NODES_X )], velv_B, m_node_size, hipMemcpyDeviceToHost );
		hipMemcpy( host_velocity_w[cudaIndex3D( i, j, k-1, NODES_X )], velw_B, m_node_size, hipMemcpyDeviceToHost );
		hipMemcpy(    host_density[cudaIndex3D( i, j, k-1, NODES_X )], dens_B, m_node_size, hipMemcpyDeviceToHost );

		if ( helper.GetCUDALastError( "hipMemcpy failed", __FILE__, __LINE__ ) )
		{
			FreeResource();
			exit( 1 );
		}
	}

	/* draw volumetric data back */
	cudaDeviceDim3D();
	kernelPickData <<<gridDim, blockDim>>>
		( dev_visual, dev_den, nPos.x * GRIDS_X, nPos.y * GRIDS_X, nPos.z * GRIDS_X );

};

void FluidSimProc::AddSource( void )
{
#if TESTING_MODE_SWITCH
	if ( decrease_times eqt 0 )
	{
		cudaDeviceDim3D();
		kernelAddSource<<<gridDim, blockDim>>> ( dev_den, dev_u, dev_v, dev_w, dev_obs );
		increase_times++;

		if ( increase_times eqt 200 )
		{
			decrease_times = increase_times;
			increase_times = 0;
		}
	}
	else
	{
		decrease_times--;
	}
#else
	cudaDeviceDim3D();
	kernelAddSource<<<gridDim, blockDim>>> ( dev_den, dev_u, dev_v, dev_w, dev_obs );
#endif
};

void FluidSimProc::InitBoundary( int i, int j, int k )
{
	cudaDeviceDim3D();

	/* zero boundary buffers */
	kernelZeroGrids<<<gridDim, blockDim>>>( dev_obs );

	for ( int i = 0; i < host_obstacle.size(); i++ )
	{
		if ( hipMemcpy( host_obstacle[i], dev_obs,
			m_node_size, hipMemcpyDeviceToHost ) not_eq hipSuccess )
		{
			helper.GetCUDALastError( "hipMemcpy failed", __FILE__, __LINE__ );
			FreeResource();
			exit( 1 );
		}
	}

	/* select middle node */
	SelectTheNode( i, j, k );

	const int ix = cudaIndex3D( nPos.x, nPos.y, nPos.z, NODES_X );

	/* set boundary */
	kernelSetBoundary<<<gridDim, blockDim>>>( dev_obs );
	
	if ( hipMemcpy( host_obstacle[ix], dev_obs,
		m_node_size, hipMemcpyDeviceToHost) not_eq hipSuccess )
	{
		helper.GetCUDALastError( "hipMemcpy failed", __FILE__, __LINE__ );
		FreeResource();
		exit( 1 );
	}
};

void FluidSimProc::VelocitySolver( void )
{
	// diffuse the velocity field (per axis):
	hostDiffusion( dev_u0, dev_u, VISOCITY, dev_obs, MACRO_VELOCITY_U );
	hostDiffusion( dev_v0, dev_v, VISOCITY, dev_obs, MACRO_VELOCITY_V );
	hostDiffusion( dev_w0, dev_w, VISOCITY, dev_obs, MACRO_VELOCITY_W );
	
	std::swap( dev_u0, dev_u );
	std::swap( dev_v0, dev_v );
	std::swap( dev_w0, dev_w );

	// stabilize it: (vx0, vy0 are whatever, being used as temporaries to store gradient field)
	hostProject( dev_u, dev_v, dev_w, dev_div, dev_p, dev_obs );
	
	// advect the velocity field (per axis):
	hostAdvection( dev_u0, dev_u, dev_obs, MACRO_VELOCITY_U, dev_u, dev_v, dev_w );
	hostAdvection( dev_v0, dev_v, dev_obs, MACRO_VELOCITY_V, dev_u, dev_v, dev_w );
	hostAdvection( dev_w0, dev_w, dev_obs, MACRO_VELOCITY_W, dev_u, dev_v, dev_w );
	std::swap( dev_u0, dev_u );
	std::swap( dev_v0, dev_v );
	std::swap( dev_w0, dev_w );
	
	// stabilize it: (vx0, vy0 are whatever, being used as temporaries to store gradient field)
	hostProject( dev_u, dev_v, dev_w, dev_div, dev_p, dev_obs );
};

void FluidSimProc::DensitySolver( void )
{
	hostDiffusion( dev_den0, dev_den, DIFFUSION, dev_obs, MACRO_DENSITY );
	std::swap( dev_den0, dev_den );
	hostAdvection ( dev_den, dev_den0, dev_obs, MACRO_DENSITY, dev_u, dev_v, dev_w );
};

void FluidSimProc::ZeroBuffers( void )
{
	cudaDeviceDim3D();

	/* zero GPU buffer */
	for ( int i = 0; i < dev_buffers_num; i++ )
		kernelZeroGrids <<<gridDim, blockDim>>> ( dev_buffers[i] );

	/* zero host buffer */
	for ( int i = 0; i < NODES_X * NODES_X * NODES_X; i++ )
	{
		hipMemcpy( host_density[i],  dev_den, m_node_size, hipMemcpyDeviceToHost);
		hipMemcpy( host_velocity_u[i], dev_u, m_node_size, hipMemcpyDeviceToHost );
		hipMemcpy( host_velocity_v[i], dev_v, m_node_size, hipMemcpyDeviceToHost );
		hipMemcpy( host_velocity_w[i], dev_w, m_node_size, hipMemcpyDeviceToHost );
	}

	/* zero visual buffer */
	kernelZeroVolumetric <<< gridDim, blockDim>>> ( dev_visual );
	hipMemcpy( host_visual, dev_visual, m_volm_size, hipMemcpyDeviceToHost );
};

void FluidSimProc::TracingTheFlow( void )
{
#if TESTING_MODE_SWITCH
	int ix  = cudaIndex3D( 1, 1, 1, NODES_X );
	int nix = cudaIndex3D( nPos.x, nPos.y, nPos.z, NODES_X );
#endif

	cudaDeviceDim3D();

	/* flooding data */
	kernelFloodingBuffers <<<gridDim, blockDim>>> ( dens_L, dens_R, dens_U, dens_D, dens_F, dens_B, dens_C );
	kernelFloodingBuffers <<<gridDim, blockDim>>> ( velu_L, velu_R, velu_U, velu_D, velu_F, velu_B, velu_C );
	kernelFloodingBuffers <<<gridDim, blockDim>>> ( velv_L, velv_R, velv_U, velv_D, velv_F, velv_B, velv_C );
	kernelFloodingBuffers <<<gridDim, blockDim>>> ( velw_L, velw_R, velw_U, velw_D, velw_F, velw_B, velw_C );

	/* clear temporary buffers for next step */
#if TESTING_MODE_SWITCH
	if ( ix eqt nix )
#endif
	kernelZeroTemporaryBuffers <<<1, TPBUFFER_X>>> ( dev_tpbufs );

	/* clear halo to avoid data obstruction */
	kernelClearHalo <<<gridDim, blockDim>>> ( dens_L, dens_R, dens_U, dens_D, dens_F, dens_B, dens_C );
	kernelClearHalo <<<gridDim, blockDim>>> ( velu_L, velu_R, velu_U, velu_D, velu_F, velu_B, velu_C );
	kernelClearHalo <<<gridDim, blockDim>>> ( velv_L, velv_R, velv_U, velv_D, velv_F, velv_B, velv_C );
	kernelClearHalo <<<gridDim, blockDim>>> ( velw_L, velw_R, velw_U, velw_D, velw_F, velw_B, velw_C );

	/* zero buffers if they not exists */
	SimNode *ptr = host_node[cudaIndex3D( nPos.x, nPos.y, nPos.z, NODES_X )];

	if ( !ptr->ptrLeft )
	{
		kernelZeroGrids <<<gridDim, blockDim>>> ( dens_L );
		kernelZeroGrids <<<gridDim, blockDim>>> ( velu_L );
		kernelZeroGrids <<<gridDim, blockDim>>> ( velv_L );
		kernelZeroGrids <<<gridDim, blockDim>>> ( velw_L );
	}
	if ( !ptr->ptrRight )
	{
		kernelZeroGrids <<<gridDim, blockDim>>> ( dens_R );
		kernelZeroGrids <<<gridDim, blockDim>>> ( velu_R );
		kernelZeroGrids <<<gridDim, blockDim>>> ( velv_R );
		kernelZeroGrids <<<gridDim, blockDim>>> ( velw_R );
	}
	if ( !ptr->ptrUp )
	{
		kernelZeroGrids <<<gridDim, blockDim>>> ( dens_U );
		kernelZeroGrids <<<gridDim, blockDim>>> ( velu_U );
		kernelZeroGrids <<<gridDim, blockDim>>> ( velv_U );
		kernelZeroGrids <<<gridDim, blockDim>>> ( velw_U );
	}
	if ( !ptr->ptrDown )
	{
		kernelZeroGrids <<<gridDim, blockDim>>> ( dens_D );
		kernelZeroGrids <<<gridDim, blockDim>>> ( velu_D );
		kernelZeroGrids <<<gridDim, blockDim>>> ( velv_D );
		kernelZeroGrids <<<gridDim, blockDim>>> ( velw_D );	
	}
	if ( !ptr->ptrFront )
	{
		kernelZeroGrids <<<gridDim, blockDim>>> ( dens_F );
		kernelZeroGrids <<<gridDim, blockDim>>> ( velu_F );
		kernelZeroGrids <<<gridDim, blockDim>>> ( velv_F );
		kernelZeroGrids <<<gridDim, blockDim>>> ( velw_F );
	}
	if ( !ptr->ptrBack )
	{
		kernelZeroGrids <<<gridDim, blockDim>>> ( dens_B );
		kernelZeroGrids <<<gridDim, blockDim>>> ( velu_B );
		kernelZeroGrids <<<gridDim, blockDim>>> ( velv_B );
		kernelZeroGrids <<<gridDim, blockDim>>> ( velw_B );
	}

#if TESTING_MODE_SWITCH
	if ( ix eqt nix )
#endif
	/* sum the density of each node */
	kernelSumBufsDens <<<gridDim, blockDim>>>
		( dev_tpbufs, dens_L, dens_R, dens_U, dens_D, dens_F, dens_B, dens_C );

	/* retrieve temporary buffer back */
	if ( hipMemcpy(host_tpbufs, dev_tpbufs, 
		sizeof(double) * TPBUFFER_X, hipMemcpyDeviceToHost ) not_eq hipSuccess )
	{
		helper.GetCUDALastError( "hipMemcpy failed", __FILE__, __LINE__ );
		FreeResource();
		exit( 1 );
	}

#if TESTING_MODE_SWITCH
	system( "cls" );
	printf( "CENTER: %f\n", host_tpbufs[TEMP_BUF_CENTER] );
	printf( "LEFT:   %f\n", host_tpbufs[TEMP_BUF_LEFT] );
	printf( "RIGHT:  %f\n", host_tpbufs[TEMP_BUF_RIGHT] );
	printf( "UP:     %f\n", host_tpbufs[TEMP_BUF_UP] );
	printf( "DOWN:   %f\n", host_tpbufs[TEMP_BUF_DOWN] );
	printf( "FRONT:  %f\n", host_tpbufs[TEMP_BUF_FRONT] );
	printf( "BACK:   %f\n", host_tpbufs[TEMP_BUF_BACK] );
#endif

#if 0
	/* dead or live */
	if ( ptr->ptrLeft not_eq nullptr )
		if ( host_tpbufs[TEMP_BUF_LEFT] > 0.f )
			ptr->ptrLeft->active = true;
		else
			ptr->ptrLeft->active = false;

	if ( ptr->ptrRight not_eq nullptr )
		if ( host_tpbufs[TEMP_BUF_RIGHT] > 0.f )
			ptr->ptrRight->active = true;
		else
			ptr->ptrRight->active = false;

	if ( ptr->ptrUp not_eq nullptr )
		if ( host_tpbufs[TEMP_BUF_UP] > 0.f )
			ptr->ptrUp->active = true;
		else
			ptr->ptrUp->active = false;

	if ( ptr->ptrDown not_eq nullptr )
		if ( host_tpbufs[TEMP_BUF_DOWN] > 0.f )
			ptr->ptrDown->active = true;
		else
			ptr->ptrDown->active = false;

	if ( ptr->ptrFront not_eq nullptr )
		if ( host_tpbufs[TEMP_BUF_FRONT] > 0.f )
			ptr->ptrFront->active = true;
		else
			ptr->ptrFront->active = false;

	if ( ptr->ptrBack not_eq nullptr )
		if ( host_tpbufs[TEMP_BUF_BACK] > 0.f )
			ptr->ptrBack->active = true;
		else
			ptr->ptrBack->active = false;
#endif
};