#include "hip/hip_runtime.h"
/**
* <Author>        Orlando Chen
* <Email>         seagochen@gmail.com
* <First Time>    Dec 15, 2013
* <Last Time>     Mar 25, 2014
* <File Name>     FluidSimProc.cu
*/

#include <time.h>
#include <iostream>
#include <utility>
#include <hip/hip_runtime_api.h>
#include <>
#include "MacroDefinition.h"
#include "FluidSimProc.h"
#include "MacroDefinition.h"
#include "Kernels.h"

using namespace sge;
using std::cout;
using std::endl;


FluidSimProc::FluidSimProc( FLUIDSPARAM *fluid )
{
	/* choose which GPU to run on, change this on a multi-GPU system. */
	if ( hipSetDevice ( 0 ) != hipSuccess )
	{
		m_scHelper.GetCUDALastError( "cannot set device", __FILE__, __LINE__ );
		exit(1);
	}

	/* initialize FPS */
	InitParams( fluid );

	/* allocate resources */
	AllocateResource();
	
	/* clear buffer */
	ClearBuffers();

	/* create boundary condition */
	InitBoundary();

	/* finally, print message */
	printf( "fluid simulation ready...\n" );
};

void FluidSimProc::InitParams( FLUIDSPARAM *fluid )
{
	fluid->fps.dwCurrentTime = 0;
	fluid->fps.dwElapsedTime = 0;
	fluid->fps.dwFrames = 0;
	fluid->fps.dwLastUpdateTime = 0;
	fluid->fps.uFPS = 0;

	srand(time(NULL));

	m_szTitle = APP_TITLE;
};

void FluidSimProc::AllocateResource( void )
{
	if ( not m_scHelper.CreateCompNodesForDevice( &m_vectCompBufs, 
		GRIDS_X * GRIDS_Y * GRIDS_Z * sizeof(double), COMP_BUFS ) ) goto Error;

	if ( not m_scHelper.CreateCompNodesForDevice( &m_vectBulletBufs, 
		BULLET_X * BULLET_Y * BULLET_Z * sizeof(double), BUL_BUFS ) ) goto Error;

	m_scHelper.CreateDeviceBuffers( VOLUME_X * VOLUME_Y * VOLUME_Z * sizeof(SGUCHAR),
		1, &m_ptrDeviceVisual );
	m_scHelper.CreateHostBuffers( VOLUME_X * VOLUME_Y * VOLUME_Z * sizeof(SGUCHAR),
		1, &m_ptrHostVisual );

	if ( not m_scHelper.CreateCompNodesForDevice( &m_vectBigBufs,
		VOLUME_X * VOLUME_Y * VOLUME_Z * sizeof(double), BIG_BUFS ) ) goto Error;

	goto Success;

Error:
		cout << "create computation buffers device failed" << endl;
		FreeResource();
		exit(1);

Success:
		cout << "size of m_vectBulletBufs: " << m_vectBulletBufs.size() << endl
			<< "size of m_vectCompBufs: " << m_vectCompBufs.size() << endl;
};

void FluidSimProc::FreeResource( void )
{
	for ( int i = 0; i < m_vectBulletBufs.size(); i++ )
		m_scHelper.FreeDeviceBuffers( 1, &m_vectBulletBufs[i] );

	for ( int i = 0; i < m_vectCompBufs.size(); i++ )
		m_scHelper.FreeDeviceBuffers( 1, &m_vectCompBufs[i] );

	for ( int i = 0; i < m_vectBigBufs.size(); i++ )
		m_scHelper.FreeDeviceBuffers( 1, &m_vectBigBufs[i] );

	m_scHelper.FreeDeviceBuffers( 1, &m_ptrDeviceVisual );
	m_scHelper.FreeHostBuffers( 1, &m_ptrHostVisual );
};

void FluidSimProc::RefreshStatus( FLUIDSPARAM *fluid )
{
	/* waiting for all kernels end */
	if ( hipDeviceSynchronize() not_eq hipSuccess )
	{
		printf( "hipDeviceSynchronize failed\n" );
		FreeResource();
		exit( 1 );
	}

	/* counting FPS */
	fluid->fps.dwFrames ++;
	fluid->fps.dwCurrentTime = GetTickCount();
	fluid->fps.dwElapsedTime = fluid->fps.dwCurrentTime - fluid->fps.dwLastUpdateTime;

	/* 1 second */
	if ( fluid->fps.dwElapsedTime >= 1000 )
	{
		fluid->fps.uFPS     = fluid->fps.dwFrames * 1000 / fluid->fps.dwElapsedTime;
		fluid->fps.dwFrames = 0;
		fluid->fps.dwLastUpdateTime = fluid->fps.dwCurrentTime;
	}

	/* updating image */
	if ( hipMemcpy( m_ptrHostVisual, m_ptrDeviceVisual, 
		VOLUME_X * VOLUME_Y * VOLUME_Z * sizeof(SGUCHAR), hipMemcpyDeviceToHost ) not_eq hipSuccess )
	{
		m_scHelper.GetCUDALastError( "host function: hipMemcpy failed", __FILE__, __LINE__ );
		FreeResource();
		exit( 1 );
	}
	fluid->volume.ptrData = m_ptrHostVisual;
};

void FluidSimProc::FluidSimSolver( FLUIDSPARAM *fluid )
{
	if ( not fluid->run ) return;
	
	SolveNavierStokesEquation( DELTATIME, true, true, true );

	GenerVolumeImg();

	RefreshStatus( fluid );
};

void FluidSimProc::GenerVolumeImg( void )
{
	m_scHelper.DeviceParamDim( &gridDim, &blockDim, THREADS_S, TILE_X, TILE_Y, GRIDS_X, GRIDS_X, GRIDS_X );
	kernelExitBullet __device_func__
		( comp_den, dev_den, GRIDS_X, GRIDS_Y, GRIDS_Z, BULLET_X, BULLET_Y, BULLET_Z );

//	m_scHelper.DeviceParamDim( &gridDim, &blockDim, THREADS_S, TILE_X, TILE_Y, VOLUME_X, VOLUME_Y, VOLUME_Z );
//	kernelUpScalingInterpolation __device_func__ ( big_den, comp_den, 
//		GRIDS_X, GRIDS_Y, GRIDS_Z, 
//		VOLUME_X, VOLUME_Y, VOLUME_Z,
//		2, 2, 2 );
//
//	kernelPickData __device_func__ ( m_ptrDeviceVisual, VOLUME_X, VOLUME_Y, VOLUME_Z,
//		big_den, VOLUME_X, VOLUME_Y, VOLUME_Z, 0, 0, 0, 1.f, 1.f, 1.f );
	kernelPickData __device_func__ ( m_ptrDeviceVisual, comp_den, VOLUME_X, VOLUME_Y, VOLUME_Z );
};

void FluidSimProc::ClearBuffers( void )
{
	m_scHelper.DeviceParamDim( &gridDim, &blockDim, THREADS_S, 26, 26, BULLET_X, BULLET_Y, BULLET_Z );
	for ( int i = 0; i < m_vectBulletBufs.size(); i++ )
		kernelZeroBuffers __device_func__ ( m_vectBulletBufs[i], BULLET_X, BULLET_Y, BULLET_Z );

	if ( m_scHelper.GetCUDALastError( "host function failed: ZeroBuffers", __FILE__, __LINE__ ) )
	{
		FreeResource();
		exit( 1 );
	}
};

inline __device__ void _thread( int *i )
{
	*i = blockIdx.x * blockDim.x + threadIdx.x;
};

inline __device__ void _thread( int *i, int *j )
{
	*i = blockIdx.x * blockDim.x + threadIdx.x;
	*j = blockIdx.y * blockDim.y + threadIdx.y;
};

inline __device__ void _thread
	( int *i, int *j, int *k, cint tilex, cint tiley, cint tilez )
{
	_thread( i, j );
	
	*k = *i + *j * tilex;
	*k = *k / ( tilex * tiley );
	*i = *i % tilex;
	*j = *j % tiley;
};

inline __device__ __host__ int ix( cint x, cint tilex)
{
	if ( x >= tilex or x < 0 ) return -1;
	return x;
};

inline __device__ __host__ int ix( cint i, cint j, cint tilex, cint tiley )
{
	if ( i < 0 or j < 0 ) return -1;

	int x; if ( i >= tilex ) x = tilex - 1;
	int y; if ( j >= tiley ) y = tiley - 1;

	x = i; y = j;
	return x + y * tilex;
};

inline __host__ __device__ int ix
	( cint i, cint j, cint k, cint tilex, cint tiley, cint tilez )
{
	if ( i < 0 or i >= tilex ) return -1;
	if ( j < 0 or j >= tiley ) return -1;
	if ( k < 0 or k >= tilez ) return -1;

	return i + j * tilex + k * tilex * tiley;
};

__global__ void kernelSetBound( double *dst, cint tilex, cint tiley, cint tilez )
{
	int i, j, k;
	_thread( &i, &j, &k, tilex, tiley, tilez );

	cint halfx = tilex / 2;
	cint halfz = tilez / 2;

	if ( j < 6 and 
		i >= halfx - 4 and i < halfx + 4 and 
		k >= halfz - 4 and k < halfz + 4 )
	{
		dst[ix(i,j,k,tilex,tiley,tilez)] = MACRO_BOUNDARY_SOURCE;
	}
	else
	{
		dst[ix(i,j,k,tilex,tiley,tilez)] = MACRO_BOUNDARY_BLANK;
	}
};

void FluidSimProc::InitBoundary( void )
{
	m_scHelper.DeviceParamDim( &gridDim, &blockDim, THREADS_S, TILE_X, TILE_Y, GRIDS_X, GRIDS_Y, GRIDS_Z );

	kernelSetBound __device_func__ ( comp_obst, GRIDS_X, GRIDS_Y, GRIDS_Z );

	if ( m_scHelper.GetCUDALastError( "call member function InitBound failed", __FILE__, __LINE__ ) )
	{
		FreeResource();
		exit(1);
	}
};

void FluidSimProc::SaveCurStage( void )
{
};

void FluidSimProc::LoadPreStage( void )
{
};

void FluidSimProc::SolveNavierStokesEquation( cdouble dt, bool add, bool vel, bool dens )
{
	m_scHelper.DeviceParamDim( &gridDim, &blockDim, THREADS_S, TILE_X, TILE_Y, GRIDS_X, GRIDS_Y, GRIDS_Z );

	kernelLoadBullet __device_func__
		( dev_obs, comp_obst,  BULLET_X, BULLET_Y, BULLET_Z, GRIDS_X, GRIDS_Y, GRIDS_Z );

	SolveGlobal( dt, add, vel, dens );
};

void FluidSimProc::SolveGlobal( cdouble dt, bool add, bool vel, bool dens )
{
	if ( add ) SourceSolverGlobal( dt );
	if ( vel ) VelocitySolverGlobal( dt );
	if ( dens ) DensitySolverGlobal( dt );
};